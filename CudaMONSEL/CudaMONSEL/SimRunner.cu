#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"

#include "gov/nist/microanalysis/NISTMonte/MonteCarloSS.cu"
//#include "gov/nist/microanalysis/NISTMonte/Electron.cu"
//#include "gov/nist/microanalysis/Utility/CSVReader.h"

#include "Amphibian\String.cuh"
#include "Amphibian\LinkedList.cuh"

#include "CudaUtil.h"
#include "ImageUtil.h"
#include "TimeUtil.h"

__global__ void spawnElectron(unsigned int *d_arr, int idx_x, int idx_y, size_t size_x, size_t size_y)
{
   int idx = idx_y * size_x + idx_x;
   //MonteCarloSS::RegionBase e(idx);
   //d_arr[idx] = e.GetId();

   d_arr[idx] = idx;
}

__global__ void spawnElectrons(unsigned int *d_arr, size_t size_x, size_t size_y)
{
   int idx_x = threadIdx.x + blockDim.x * blockIdx.x;
   int idx_y = threadIdx.y + blockDim.y * blockIdx.y;

   printf("%d, %d", idx_x, idx_y);

   spawnElectron << <1, 1 >> >(d_arr, idx_x, idx_y, size_x, size_y);
}

void PrintArray2D(unsigned int *h_arr, size_t img_x, size_t img_y)
{
   for (int k = 0; k < img_y; ++k) {
      for (int l = 0; l < img_x; ++l) {
         std::cout << h_arr[k*img_x + l] << " ";
      }
      std::cout << std::endl;
   }
   std::cout << std::endl;
}

//int main()
//{
//   checkCudaErrors(hipSetDevice(0));
//
//   const size_t img_x = 256;
//   const size_t img_y = 256;
//   const dim3 blockSize(2, 2, 1);
//   const size_t grid_x = (img_x / blockSize.x) + ((img_x % blockSize.x > 0) ? 1 : 0);
//   const size_t grid_y = (img_y / blockSize.y) + ((img_y % blockSize.y > 0) ? 1 : 0);
//   const dim3 gridSize(grid_x, grid_y, 1);
//
//   unsigned int *d_arr;
//   checkCudaErrors(hipMalloc((void **)&d_arr, sizeof(int) * img_x * img_y));
//
//   spawnElectrons << <gridSize, blockSize >> >(d_arr, img_x, img_y);
//   checkCudaErrors(hipDeviceSynchronize());
//   checkCudaErrors(hipGetLastError());
//
//   unsigned int *h_arr = new unsigned int[img_x * img_y];
//   checkCudaErrors(hipMemcpy(h_arr, d_arr, sizeof(int) * img_x * img_y, hipMemcpyDeviceToHost));
//   checkCudaErrors(hipFree(d_arr));
//
//   saveResults("a.bmp", h_arr, img_x, img_y);
//   delete[] h_arr;
//
//   return 0;
//}

__host__ __device__ void BuildListTest(Node<String, float>** head)
{
   String a("a");
   String b("b");
   String c("c");
   Node<String, float>::InsertHead(head, a, 0.0f);
   Node<String, float>::InsertHead(head, b, 1.0f);
   Node<String, float>::InsertHead(head, c, 2.0f);
}

__host__ __device__ void PrintListInOrder(Node<String, float>* head)
{
   if (head != NULL) {
      printf("%s: %f\n", head->GetKey().Get(), head->GetValue());
      PrintListInOrder(head->GetNext());
   }
}

__global__ void Test1()
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
   __syncthreads();
#endif

   Node<String, float>* head = NULL;
   printf("A\n");
   BuildListTest(&head);
   printf("B\n");
   PrintListInOrder(head);
   printf("C\n");
   Node<String, float>::RemoveAll(&head);

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
   __syncthreads();
#endif
}

__global__ void kernel(int n)
{
   char n_a[16] = "\0";
   String::IToA(n_a, n);
   printf("%s\n", n_a);
}

int main()
{
   kernel << <1, 1 >> >(12345678);
   //Test1 << < 1, 1 >> > ();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   return 0;
}
