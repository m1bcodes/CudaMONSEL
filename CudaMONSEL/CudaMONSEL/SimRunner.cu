#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>

#include "gov/nist/microanalysis/NISTMonte/MonteCarloSS.cu"
//#include "gov/nist/microanalysis/NISTMonte/Electron.cu"
//#include "gov/nist/microanalysis/Utility/CSVReader.h"
#include "gov\nist\microanalysis\Utility\UncertainValue2.cuh"

#include "Amphibian\String.cuh"
#include "Amphibian\LinkedList.cuh"

#include "CudaUtil.h"
#include "ImageUtil.h"
#include "TimeUtil.h"

__global__ void spawnElectron(unsigned int *d_arr, int idx_x, int idx_y, size_t size_x, size_t size_y)
{
   int idx = idx_y * size_x + idx_x;
   //MonteCarloSS::RegionBase e(idx);
   //d_arr[idx] = e.GetId();

   d_arr[idx] = idx;
}

__global__ void spawnElectrons(unsigned int *d_arr, size_t size_x, size_t size_y)
{
   int idx_x = threadIdx.x + blockDim.x * blockIdx.x;
   int idx_y = threadIdx.y + blockDim.y * blockIdx.y;

   printf("%d, %d", idx_x, idx_y);

   spawnElectron << <1, 1 >> >(d_arr, idx_x, idx_y, size_x, size_y);
}

void PrintArray2D(unsigned int *h_arr, size_t img_x, size_t img_y)
{
   for (int k = 0; k < img_y; ++k) {
      for (int l = 0; l < img_x; ++l) {
         std::cout << h_arr[k*img_x + l] << " ";
      }
      std::cout << std::endl;
   }
   std::cout << std::endl;
}

//int main()
//{
//   checkCudaErrors(hipSetDevice(0));
//
//   const size_t img_x = 256;
//   const size_t img_y = 256;
//   const dim3 blockSize(2, 2, 1);
//   const size_t grid_x = (img_x / blockSize.x) + ((img_x % blockSize.x > 0) ? 1 : 0);
//   const size_t grid_y = (img_y / blockSize.y) + ((img_y % blockSize.y > 0) ? 1 : 0);
//   const dim3 gridSize(grid_x, grid_y, 1);
//
//   unsigned int *d_arr;
//   checkCudaErrors(hipMalloc((void **)&d_arr, sizeof(int) * img_x * img_y));
//
//   spawnElectrons << <gridSize, blockSize >> >(d_arr, img_x, img_y);
//   checkCudaErrors(hipDeviceSynchronize());
//   checkCudaErrors(hipGetLastError());
//
//   unsigned int *h_arr = new unsigned int[img_x * img_y];
//   checkCudaErrors(hipMemcpy(h_arr, d_arr, sizeof(int) * img_x * img_y, hipMemcpyDeviceToHost));
//   checkCudaErrors(hipFree(d_arr));
//
//   saveResults("a.bmp", h_arr, img_x, img_y);
//   delete[] h_arr;
//
//   return 0;
//}

__global__ void kernel()
{
   UncertainValue2::UncertainValue2 v0(0, "abc", 5);
   UncertainValue2::UncertainValue2 v1(1);
   UncertainValue2::UncertainValue2 v2(2, 10);
   UncertainValue2::UncertainValue2 v3(2, 10);

   printf("%d\n", v1.equals(&v2));
}

int main()
{
   kernel << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   return 0;
}
