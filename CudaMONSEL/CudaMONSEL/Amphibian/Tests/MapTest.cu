
#include <hip/hip_runtime.h>
//#include "Amphibian/Tests/MapTest.cuh"
//
//#include "Amphibian/String.cuh"
//
//#include <math.h>
//
//namespace MapTest
//{
//   __device__ void AssertEqual(int a, int b)
//   {
//      if (a != b) {
//         printf("not equal: (%d, %d)\n", a, b);
//      }
//   }
//
//   __device__ MapTest::MapTest()
//   {
//   }
//
//   typedef Map::Map<int, int, Comparator::IntCompareFcn, Comparator::IntCompareFcn, Hasher::IntHashFcn, Hasher::IntHashFcn> IntTestType;
//   typedef Map::Iterator<int, int, Comparator::IntCompareFcn, Comparator::IntCompareFcn, Hasher::IntHashFcn, Hasher::IntHashFcn> IntTestTypeItr;
//
//   __device__ void MapTest::TestInteger()
//   {
//      int k = 0, v = 1;
//      IntTestType m1 =
//         CreateMapA<int, int, Comparator::IntCompareFcn, Comparator::IntCompareFcn, Hasher::IntHashFcn, Hasher::IntHashFcn>(k, v);
//      int k0 = 1, v0 = 1;
//      m1.Put(k0, v0);
//
//      AssertEqual(m1.Size(), 2);
//
//      IntTestType m2(m1);
//      AssertEqual(m2.Size(), 2);
//      int k1 = 2, v1 = 3;
//      m2.Put(k1, v1);
//      AssertEqual(m2.Size(), 3);
//      AssertEqual(m1.Size(), 2);
//
//      int c1 = 0;
//      IntTestTypeItr itr1(m1);
//      while (itr1.HasNext()) {
//         //printf("(%d, %d) ", itr1.GetKey(), itr1.GetValue());
//         c1++;
//         itr1.Next();
//      }
//      AssertEqual(c1, 2);
//
//      int c2 = 0;
//      IntTestTypeItr itr2(m2);
//      while (itr2.HasNext()) {
//         //printf("(%d, %d) ", itr2.GetKey(), itr2.GetValue());
//         c2++;
//         itr2.Next();
//      }
//      AssertEqual(c2, 3);
//
//      IntTestType m3 = m2;
//      AssertEqual(m3.Size(), 3);
//      int k2 = 2, v2 = 3;
//      m3.Put(k2, v2);
//      AssertEqual(m3.Size(), 3);
//      unsigned int h2 = m2.HashCode();
//      unsigned int h3 = m3.HashCode();
//      if (h2 != h3) {
//         printf("HashCodes are different: %d, %d\n", h2, h3);
//      }
//
//      printf("MapTest::TestInteger() completed\n");
//   }
//
//   typedef Map::Map<String::String, double, String::CompareFcn, Comparator::DoubleCompareFcn, String::HashFcn, Hasher::DoubleHashFcn> StringTestType;
//   typedef Map::Iterator<String::String, double, String::CompareFcn, Comparator::DoubleCompareFcn, String::HashFcn, Hasher::DoubleHashFcn> StringTestTypeItr;
//
//   __device__ StringTestType makeMap()
//   {
//      StringTestType map;
//      double v0 = 1, v1 = 2, v2 = 3, v3 = 4, v4 = 5;
//      String::String A("A"), B("B"), C("C"), D("D"), E("E");
//      map.Put(A, v0);
//      map.Put(B, v1);
//      map.Put(C, v2);
//      map.Put(D, v3);
//      map.Put(E, v4);
//      return map;
//   }
//
//   __device__ void MapTest::TestString()
//   {
//      StringTestType m1;
//      double v0 = 1, v1 = 2, v2 = 3, v3 = 4;
//      String::String A("A"), B("B"), C("C"), D("D"), E("E");
//      m1.Put(A, v0);
//      m1.Put(B, v1);
//
//      AssertEqual(m1.Size(), 2);
//
//      StringTestType m2(m1);
//      AssertEqual(m2.Size(), 2);
//      m2.Put(C, v2);
//      AssertEqual(m2.Size(), 3);
//      AssertEqual(m1.Size(), 2);
//
//      int c1 = 0;
//      StringTestTypeItr itr1(m1);
//      while (itr1.HasNext()) {
//         //printf("(%s, %lf) ", itr1.GetKey().Get(), itr1.GetValue());
//         c1++;
//         itr1.Next();
//      }
//      AssertEqual(c1, 2);
//      //printf("\n");
//
//      int c2 = 0;
//      StringTestTypeItr itr2(m2);
//      while (itr2.HasNext()) {
//         //printf("(%s, %lf) ", itr2.GetKey().Get(), itr2.GetValue());
//         c2++;
//         itr2.Next();
//      }
//      AssertEqual(c2, 3);
//      //printf("\n");
//
//      StringTestType m3 = m2;
//      AssertEqual(m3.Size(), 3);
//      m3.Put(D, v3);
//      AssertEqual(m3.Size(), 4);
//
//      int c3 = 0;
//      StringTestTypeItr itr3(m3);
//      while (itr3.HasNext()) {
//         //printf("(%s, %lf) ", itr3.GetKey().Get(), itr3.GetValue());
//         c3++;
//         itr3.Next();
//      }
//      AssertEqual(m3.Size(), 4);
//      //printf("\n");
//
//      StringTestType map4;
//      double v5 = ::sqrt(0.05), v6 = ::sqrt(0.04);
//      String::String V1("V1"), V2("V2");
//      map4.Put(V1, v5);
//      map4.Put(V2, v6);
//
//      int c4 = 0;
//      StringTestTypeItr itr4(map4);
//      while (itr4.HasNext()) {
//         //printf("(%s, %lf) ", itr4.GetKey().Get(), itr4.GetValue());
//         c4++;
//         itr4.Next();
//      }
//      AssertEqual(map4.Size(), 2);
//      AssertEqual(c4, 2);
//
//      StringTestType m5 = map4;
//      if (!(m5 == map4)) {
//         printf("maps are different\n");
//      }
//      if (!(m5.Size() == map4.Size())) {
//         printf("maps sizes are different: %d, %d\n", map4.Size(), m5.Size());
//      }
//      if (m5.HashCode() != map4.HashCode()) {
//         printf("maps hashcodes are different\n");
//      }
//
//      StringTestTypeItr itr5(m5);
//      while (itr5.HasNext()) {
//         //printf("(%s, %lf) ", itr5.GetKey().Get(), itr5.GetValue());
//         itr5.Next();
//      }
//
//      StringTestType m6 = makeMap();
//      StringTestTypeItr itr6(m6);
//      while (itr6.HasNext()) {
//         //printf("(%s, %lf) ", itr6.GetKey().Get(), itr6.GetValue());
//         itr6.Next();
//      }
//      //printf("\n");
//
//      printf("MapTest::TestString() completed\n");
//   }
//
//   class TestClass
//   {
//   public:
//      __device__ TestClass() {}
//      __device__ StringTestType& GetMap() { return m; }
//
//   private:
//      StringTestType m;
//   };
//
//   struct TestClassCompare
//   {
//      __device__ inline bool operator() (TestClass& lhs, TestClass& rhs)
//      {
//         return lhs.GetMap() == rhs.GetMap();
//      }
//   };
//
//   struct TestClassHashFcn
//   {
//      __device__ inline unsigned int operator() (TestClass& t)
//      {
//         return t.GetMap().HashCode();
//      }
//   };
//
//   typedef Map::Map<String::String, TestClass, String::CompareFcn, TestClassCompare, String::HashFcn, TestClassHashFcn> MapTestT;
//   typedef Map::Iterator<String::String, TestClass, String::CompareFcn, TestClassCompare, String::HashFcn, TestClassHashFcn> MapTestTItr;
//
//   __device__ void MapTest::TestMapOfMap()
//   {
//      MapTestT m1;
//      TestClass a;
//      TestClass b;
//
//      double v1 = 1.2, v2 = 1.2, v3 = 3.3;
//      String::String s0("A1"), s1("A2"), s2("A3"), s3("B1");
//      String::String s4("A"), s5("B");
//      a.GetMap().Put(s0, v1);
//      a.GetMap().Put(s1, v2);
//      a.GetMap().Put(s2, v3);
//      b.GetMap().Put(s3, v1);
//      m1.Put(s4, a);
//      m1.Put(s5, b);
//
//      printf("MapTest::TestMapOfMap() completed\n");
//   }
//
//   __device__ void MapTest::TestAggregate()
//   {
//      StringTestType t;
//      double v0 = 0.1, v1 = 0.5, v2 = 100;
//      String::String s0("A"), s1("B"), s2("C");
//      t.Put(s0, v0);
//      t.Put(s1, v1);
//      t.Put(s2, v2);
//
//      double ret1 = t.Aggregate([](double a) {return a*a; });
//      double ret2 = t.Aggregate([](double a) {return a*a; });
//      double ret3 = t.Aggregate([](double a) {return a*a; });
//      double ans = 0.1*0.1 + 0.5*0.5 + 100 * 100;
//      if (ret1 != ans) {
//         printf("wrong : %lf, %lf", ret1, ans);
//      }
//      if (ret2 != ans) {
//         printf("wrong : %lf, %lf", ret2, ans);
//      }
//      if (ret3 != ans) {
//         printf("wrong : %lf, %lf", ret3, ans);
//      }
//      printf("MapTest::TestAggregate() completed\n");
//   }
//}
