#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include "Tests\HasherTest.cuh"
#include "Tests\LinkedListTest.cuh"
#include "Tests\MapTest.cuh"
#include "Tests\SetTest.cuh"

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
   if (err != hipSuccess) {
      std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
      std::cerr << hipGetErrorString(err) << " " << func << std::endl;
      exit(1);
   }
}

__global__ void TestKernel()
{
   printf("INT_MIN %d\n", INT_MIN);
   printf("INT_MAX %d\n", INT_MAX);

   HasherTest::TestOne();

   LinkedListTest::LinkedListTest lltest;
   lltest.TestAddAllAsSet();

   SetTest::SetTest setTest;
   setTest.TestIntBasic();
   setTest.TestInt();
   setTest.TestInt2();
   setTest.TestString();
   setTest.TestSetOfSetOfString();

   MapTest::MapTest mapTest;
   mapTest.TestInteger();
   mapTest.TestString();
   mapTest.TestMapOfMap();
}

struct CompareDouble
{
   inline bool operator() (double& a, double& b) {
      return a < b;
   }
};

int main()
{
   //printf("%d\n", sizeof(CompareDouble));
   TestKernel<<<1, 1>>> ();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

    return 0;
}
