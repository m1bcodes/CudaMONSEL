#include "hip/hip_runtime.h"
/*
* - without length, the array parameter (eg const double a[]) is always 3 dimensional
* - use "auto" keyword for pointers ONLY
*/

#include <stdio.h>

#include <hip/hip_runtime.h>

#include "Amphibian\Tests\HasherTest.cuh"
#include "Amphibian\Tests\StringTest.cuh"
#include "Amphibian\Tests\LinkedListTest.cuh"
#include "Amphibian\Tests\SetTest.cuh"
#include "Amphibian\Tests\MapTest.cuh"
#include "Amphibian\Tests\VectorTest.cuh"
#include "Amphibian\Tests\StackTest.cuh"

#include "CudaUtil.h"
#include "ImageUtil.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "gov\nist\microanalysis\Utility\Math2.cuh"

#include "gov\nist\microanalysis\Utility\UncertainValue2.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Material.cuh"

#include "gov\nist\microanalysis\EPQLibrary\EdgeEnergy.cuh"
#include "gov\nist\microanalysis\EPQLibrary\MaterialFactory.cuh"
#include "gov\nist\microanalysis\EPQLibrary\BrowningEmpiricalCrossSection.cuh"
#include "gov\nist\microanalysis\EPQLibrary\ScreenedRutherfordScatteringAngle.cuh"
#include "gov\nist\microanalysis\EPQLibrary\CzyzewskiMottScatteringAngle.cuh"
#include "gov\nist\microanalysis\EPQLibrary\GasScatteringCrossSection.cuh"
#include "gov\nist\microanalysis\EPQLibrary\NISTMottScatteringAngle.cuh"
#include "gov\nist\microanalysis\EPQLibrary\MeanIonizationPotential.cuh"

#include "gov\nist\nanoscalemetrology\JMONSEL\NISTMottRS.cuh"

#include "gov\nist\microanalysis\EPQTests\UncertainValue2Test.cuh"
#include "gov\nist\microanalysis\EPQTests\ElementTest.cuh"
#include "gov\nist\microanalysis\EPQTests\MaterialTest.cuh"
#include "gov\nist\microanalysis\EPQTests\AtomicShellTest.cuh"
#include "gov\nist\microanalysis\EPQTests\EdgeEnergyTest.cuh"
#include "gov\nist\microanalysis\EPQTests\MeanIonizationPotentialTest.cuh"
#include "gov\nist\microanalysis\EPQTests\SphereTest.cuh"
#include "gov\nist\microanalysis\EPQTests\Math2Test.cuh"
#include "gov\nist\microanalysis\EPQTests\CylindricalShapeTest.cuh"
#include "gov\nist\microanalysis\EPQTests\SumShapeTest.cuh"
#include "gov\nist\microanalysis\EPQTests\BetheElectronEnergyLossTest.cuh"
#include "gov\nist\microanalysis\EPQTests\MonteCarloSSTest.cuh"

#include "gov\nist\nanoscalemetrology\JMONSELTests\LinesOnLayers.cuh"

//__device__ __host__ float function(float x)
//{
//   #if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
//      return 10.0f * __sinf(x);
//   #else // host code here
//   #endif
//}

__global__ void testKernel()
{
   printf("%d, %d, %d, %d, %d, %d\n", threadIdx.x, blockIdx.x, threadIdx.y, blockIdx.y, threadIdx.z, blockIdx.z);
   printf("%d, %d, %d\n", gridDim.x, gridDim.y, gridDim.z);

   //int i = threadIdx.x + blockDim.x * blockIdx.x;
   Math2Test::testRandom1Cuda();
   Math2Test::testRandom2Cuda();

   HasherTest::TestOne();

   StringTest::EmptyTest();
   StringTest::TestOne();
   StringTest::AtoITest();
   StringTest::AtoFTest();
   StringTest::ItoATest();
   StringTest::findTest();
   StringTest::addTest();

   LinkedListTest::LinkedListTest lltest;
   lltest.InsertionTest();
   lltest.TestAddAllAsSet();
   LinkedListTest::TestListKV();
   LinkedListTest::testDLinkedList();

   SetTest::SetTest setTest;
   setTest.testIntBasic();
   setTest.testInt();
   setTest.testInt2();
   setTest.testString();
   //setTest.TestSetOfSetOfString();

   MapTest::MapTest mapTest;
   mapTest.testInteger();
   mapTest.testString();
   mapTest.testMapOfMap();

   VectorTest::testOne();
   VectorTest::testTwo();
   VectorTest::testThree();

   StackTest::testOne();
}

const unsigned int NUM_ROWS = 1;
const unsigned int NUM_COLS = 16;

__global__ void printRand()
{
   const unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
   printf("thread id %d: %.10e\n", i, Math2::random());
}

void testGPU()
{
   printf("-----------------GPU-----------------------------\n");
   Math2::initCudaStates << <1, 1 >> >(NUM_ROWS * NUM_COLS);
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   printRand << <1, 16 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   testKernel << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
}

void testsCPU()
{
   printf("-----------------CPU-----------------------------\n");
   HasherTest::TestOne();

   StringTest::EmptyTest();
   StringTest::TestOne();
   StringTest::AtoITest();
   StringTest::AtoFTest();
   StringTest::ItoATest();
   StringTest::findTest();
   StringTest::addTest();

   LinkedListTest::LinkedListTest lltest;
   lltest.InsertionTest();
   lltest.TestAddAllAsSet();
   LinkedListTest::TestListKV();
   LinkedListTest::testDLinkedList();

   SetTest::SetTest setTest;
   setTest.testIntBasic();
   setTest.testInt();
   setTest.testInt2();
   setTest.testString();
   //setTest.TestSetOfSetOfString();

   MapTest::MapTest mapTest;
   mapTest.testInteger();
   mapTest.testString();
   mapTest.testMapOfMap();

   VectorTest::testOne();
   VectorTest::testTwo();
   VectorTest::testThree();
   VectorTest::testFour();

   StackTest::testOne();

   EdgeEnergy::DiracHartreeSlaterIonizationEnergies::loadxionUis();
   EdgeEnergy::NISTEdgeEnergy::loadNISTxrtdb();
   EdgeEnergy::ChantlerEdgeEnergy::loadFFastEdgeDB();
   EdgeEnergy::DTSAEdgeEnergy::loadEdgeEnergies();

   Element::init();
   MaterialFactory::init();

   BrowningEmpiricalCrossSection::init();
   ScreenedRutherfordScatteringAngle::init();
   CzyzewskiMottScatteringAngle::init();
   NISTMottScatteringAngle::init();
   GasScatteringCrossSection::init();
   NISTMottRS::init();
   MeanIonizationPotential::Berger64.readTabulatedValues();
   MeanIonizationPotential::Berger83.readTabulatedValues();
   //MeanIonizationPotential::Berger64MeanIonizationPotential::readTabulatedValues();
   //MeanIonizationPotential::Berger83MeanIonizationPotential::readTabulatedValues();

   Math2Test::testRandom1();
   Math2Test::testRandom2();

   UncertainValue2::UncertainValue2 v0(0, "abc", 5);
   UncertainValue2::UncertainValue2 v1(1);
   UncertainValue2::UncertainValue2 v2(2, 10);
   UncertainValue2::UncertainValue2 v3(2, 10);
   printf("%d\n", v1.equals(v2));
   printf("%d\n", v1.equals(v3));
   printf("%d\n", v2.equals(v3));

   UncertainValue2Test::UncertainValue2Test uvTest;
   uvTest.testSpecialValues();
   uvTest.testA();
   uvTest.testB();
   uvTest.testC();
   uvTest.testAB();
   uvTest.testAdd1();
   uvTest.testAdd2();
   uvTest.testAdd3();
   uvTest.testMultiply();
   uvTest.testDivide();
   uvTest.testFunctions();

   ElementTest::ElementTest elementTest;
   elementTest.testZero();
   elementTest.testOne();

   MaterialTest::MaterialTest mat;
   mat.testOne();

   AtomicShellTest::testOne();

   EdgeEnergyTest::testOne();

   MeanIonizationPotentialTest::testOne();

   SphereTest::testContains();
   SphereTest::testGetFirstIntersection();

   CylindricalShapeTest::CylindricalShapeTest cylindricalShapeTest;
   cylindricalShapeTest.testZero();
   cylindricalShapeTest.testOne();
   cylindricalShapeTest.testTwo();
   cylindricalShapeTest.testThree();
   cylindricalShapeTest.testFour();
   cylindricalShapeTest.testFive();
   cylindricalShapeTest.testSix();
   cylindricalShapeTest.testSeven();
   cylindricalShapeTest.testEight();
   cylindricalShapeTest.testNine();
   cylindricalShapeTest.testTen();
   cylindricalShapeTest.testEleven();
   cylindricalShapeTest.testTwelve();

   BetheElectronEnergyLossTest::testOne();

   MonteCarloSSTest::testOne();

   SumShapeTest::SumShapeTest sumShapeTest;
   sumShapeTest.testGetFirstIntersection();
   sumShapeTest.testAll();
}

__global__ void printNullReference()
{
   printf("%s\n", Reference::dNullReference->getLongForm().c_str());
}

__device__ float *d_arr;
__device__ float d_arr1[8];

__global__ void initArr(float *d_tmp)
{
   d_arr = new float[8];
   memcpy(d_arr, d_tmp, sizeof(float) * 8);
}

__global__ void cpyker()
{
   memcpy(d_arr1, d_arr, sizeof(float) * 8);
   for (int i = 0; i < 8; ++i) {
      printf("%lf ", d_arr1[i]);
   }
   printf("\n");
}

__global__ void printSpwem()
{
   printf("GPU: %d\n", NISTMottScatteringAngle::getNISTMSA(59).getSpwem().size());
   for (auto a : NISTMottScatteringAngle::getNISTMSA(59).getSpwem()) {
      printf("%.10e ", a);
   }
   printf("GPU end\n");
}

__global__ void printMeanIonizationPotential()
{
   printf("GPU:\n");
   printf("Berger64: %d\n", MeanIonizationPotential::dBerger64->getData().size());
   for (auto a : MeanIonizationPotential::dBerger64->getData()) {
      printf("%.10e ", a);
   }
   printf("Berger64 end\n");
   printf("Berger83: %d\n", MeanIonizationPotential::dBerger83->getData().size());
   for (auto a : MeanIonizationPotential::dBerger83->getData()) {
      printf("%.10e ", a);
   }
   printf("Berger83 end\n");
   printf("GPU end\n");
}

void initCuda()
{
   printf("-----------------initCuda-----------------------------\n");

   char *d_data = nullptr;
   checkCudaErrors(hipMalloc((void **)&d_data, sizeof(char) * 128));
   checkCudaErrors(hipMemcpy(d_data, Reference::NullReference.getReference().c_str(), sizeof(char) * 128, hipMemcpyHostToDevice));
   Reference::initCuda << <1, 1 >> >(d_data);
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   checkCudaErrors(hipFree(d_data));
   printNullReference << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   Element::copyDataToDevice();
   Element::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   BrowningEmpiricalCrossSection::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   ScreenedRutherfordScatteringAngle::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   
   NISTMottScatteringAngle::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   NISTMottScatteringAngle::copyDataToCuda();
   printSpwem << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   printf("CPU: %d\n", NISTMottScatteringAngle::getNISTMSA(59).getSpwem().size());
   for (auto a : NISTMottScatteringAngle::getNISTMSA(59).getSpwem()) {
      printf("%.10e ", a);
   }
   printf("CPU end\n");
   NISTMottRS::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   MeanIonizationPotential::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   MeanIonizationPotential::copyDataToCuda();
   printMeanIonizationPotential << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   printf("CPU:\n");
   printf("Berger64: %d\n", MeanIonizationPotential::Berger64.getData().size());
   for (auto a : MeanIonizationPotential::Berger64.getData()) {
      printf("%.10e ", a);
   }
   printf("Berger64 end\n");
   printf("Berger83: %d\n", MeanIonizationPotential::Berger83.getData().size());
   for (auto a : MeanIonizationPotential::Berger83.getData()) {
      printf("%.10e ", a);
   }
   printf("Berger83 end\n");
   printf("CPU end\n");

   float tmp[] = { 0, 1, 2, 3, 4, 5, 6, 7 };
   float *d_tmp;
   checkCudaErrors(hipMalloc((void**)&d_tmp, sizeof(float) * 8));
   checkCudaErrors(hipMemcpy(d_tmp, tmp, sizeof(float) * 8, hipMemcpyHostToDevice));
   initArr << <1, 1 >> >(d_tmp);
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   cpyker << <1, 1 >> >();
   checkCudaErrors(hipFree(d_tmp));

   //char *d_data;
   //checkCudaErrors(hipMalloc((void **)&d_data, sizeof(char) * 128));
   //checkCudaErrors(hipMemcpy(d_data, Reference::NullReference.getReference().c_str(), sizeof(char) * 128, hipMemcpyHostToDevice));
   //Reference::initCuda << <1, 1 >> >(d_data);
   //checkCudaErrors(hipDeviceSynchronize());
   //checkCudaErrors(hipGetLastError());
   //checkCudaErrors(hipFree(d_data));
   //print << <1, 1 >> >();
   //checkCudaErrors(hipDeviceSynchronize());
   //checkCudaErrors(hipGetLastError());
}

__device__ hiprandState state[16];

__device__ double generateRandomNumber()
{
   const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
   const double res = hiprand_uniform_double(&state[id]);
   printf("thread no. %d: %.10e\n", id, res);
   return res;
}

__global__ void printRandomNumbers()
{
   const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
   hiprand_init(0, id, id, &state[id]);
   printf("thread no. %d: %.10e\n", id, generateRandomNumber());
}

int main()
{
   //CudaClass c;
   //CudaClass *d_c = nullptr;
   //checkCudaErrors(hipMalloc((void **)&d_c, sizeof(CudaClass)));
   //checkCudaErrors(hipMemcpy(d_c, &c, sizeof(CudaClass), hipMemcpyHostToDevice));
   ////int *hdata;
   ////checkCudaErrors(hipMalloc((void **)&hdata, sizeof(int)));
   ////checkCudaErrors(hipMemcpy(hdata, c.get_data(), sizeof(int), hipMemcpyHostToDevice));
   ////checkCudaErrors(hipMemcpy(d_c->get_data(), hdata, sizeof(int), hipMemcpyDeviceToDevice));
   ////checkCudaErrors(hipMemcpy(d_c->get_data(), c.get_data(), sizeof(int) * 8, hipMemcpyHostToDevice));
   //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_ptr), &d_c, sizeof(CudaClass*)));
   //useClass<<<1, 1>>>();
   //checkCudaErrors(hipDeviceSynchronize());
   //checkCudaErrors(hipFree(d_c));

   printRandomNumbers << <1, 16 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   testsCPU();
   testGPU();

   initCuda();

   LinesOnLayers::run();

   return 0;
}

//static const int N = 2;
//
//class vecarray
//{
//public:
//   int *vecptr[N]; //array of pointers pointing to array
//   int dim[N]; //store length of each array pointed to
//
//   __device__ __host__ vecarray(); //constructor
//   __device__ __host__ int sum();  //sum up all the elements in the array being pointed to
//};
//
//vecarray::vecarray()
//{
//   for (int i = 0; i<N; i++)
//   {
//      vecptr[i] = NULL;
//      dim[i] = 0;
//   }
//}
//
//__device__ __host__ int vecarray::sum()
//{
//   int i = 0, j = 0, s = 0;
//   for (i = 0; i<N; i++)
//      for (j = 0; j < dim[i]; j++)
//         s += vecptr[i][j];
//   return s;
//}
//
//__global__ void addvecarray(vecarray * v, int *s)
//{
//   *s = v->sum();
//}
//
//int main()
//{
//   //copy *V to device, do sum() and pass back
//   vecarray *v, *dev_v; // the result by dev_v
//   v = new vecarray;
//   int a[3] = { 1, 2, 3 }; //initialize v manually
//   int b[4] = { 4, 5, 6, 7 };
//   int result = 0;
//   int *dev_result;
//   v->vecptr[0] = a;
//   v->vecptr[1] = b;
//   v->dim[0] = 3; v->dim[1] = 4;
//   int *vptr[N];
//
//   checkCudaErrors(hipMalloc((void**)&dev_v, sizeof(vecarray)));
//   checkCudaErrors(hipMemcpy(dev_v, v, sizeof(vecarray), hipMemcpyHostToDevice)); //copy class object
//   checkCudaErrors("cudaMemcpy1 fail");
//
//   for (int i = 0; i < N; i++){
//      checkCudaErrors(hipMalloc((void**)&(vptr[i]), v->dim[i] * sizeof(int)));
//      checkCudaErrors(hipMemcpy(&(dev_v->vecptr[i]), &vptr[i], sizeof(int*), hipMemcpyHostToDevice));
//   }
//
//   for (int i = 0; i<N; i++) { // copy arrays
//      checkCudaErrors(hipMemcpy(vptr[i], v->vecptr[i], v->dim[i] * sizeof(int), hipMemcpyHostToDevice));
//   }
//   checkCudaErrors(hipMalloc((void **)&dev_result, sizeof(int)));
//   addvecarray<<<1, 1>>>(dev_v, dev_result);
//
//   checkCudaErrors(hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost));
//   printf("the result is %d\n", result);
//
//   return 0;
//}
