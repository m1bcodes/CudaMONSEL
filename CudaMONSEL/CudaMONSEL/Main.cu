#include "hip/hip_runtime.h"
/*
* - without length, the array parameter (eg const double a[]) is always 3 dimensional
* - use "auto" keyword for pointers ONLY
*/

#include <stdio.h>

#include <hip/hip_runtime.h>

#include "Amphibian\Tests\HasherTest.cuh"
#include "Amphibian\Tests\StringTest.cuh"
#include "Amphibian\Tests\LinkedListTest.cuh"
#include "Amphibian\Tests\SetTest.cuh"
#include "Amphibian\Tests\MapTest.cuh"

#include "CudaUtil.h"

#include "gov\nist\microanalysis\Utility\UncertainValue2.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Material.cuh"

#include "gov\nist\microanalysis\EPQLibrary\EdgeEnergy.cuh"
#include "gov\nist\microanalysis\EPQLibrary\MaterialFactory.cuh"
#include "gov\nist\microanalysis\EPQLibrary\CzyzewskiMottScatteringAngle.cuh"
#include "gov\nist\microanalysis\EPQLibrary\GasScatteringCrossSection.cuh"
#include "gov\nist\microanalysis\EPQLibrary\NISTMottScatteringAngle.cuh"
#include "gov\nist\microanalysis\EPQLibrary\MeanIonizationPotential.cuh"

#include "gov\nist\nanoscalemetrology\JMONSEL\NISTMottRS.cuh"

#include "gov\nist\microanalysis\EPQTests\UncertainValue2Test.cuh"
#include "gov\nist\microanalysis\EPQTests\ElementTest.cuh"
#include "gov\nist\microanalysis\EPQTests\MaterialTest.cuh"
#include "gov\nist\microanalysis\EPQTests\AtomicShellTest.cuh"
#include "gov\nist\microanalysis\EPQTests\EdgeEnergyTest.cuh"
#include "gov\nist\microanalysis\EPQTests\MeanIonizationPotentialTest.cuh"
#include "gov\nist\microanalysis\EPQTests\SphereTest.cuh"
#include "gov\nist\microanalysis\EPQTests\Math2Test.cuh"
#include "gov\nist\microanalysis\EPQTests\CylindricalShapeTest.cuh"
#include "gov\nist\microanalysis\EPQTests\SumShapeTest.cuh"
#include "gov\nist\microanalysis\EPQTests\BetheElectronEnergyLossTest.cuh"
#include "gov\nist\microanalysis\EPQTests\MonteCarloSSTest.cuh"

#include "gov\nist\nanoscalemetrology\JMONSELTests\LinesOnLayers.cuh"

#include "ImageUtil.h"

__global__ void TestKernel()
{
   HasherTest::TestOne();

   LinkedListTest::LinkedListTest lltest;
   lltest.TestAddAllAsSet();

   SetTest::SetTest setTest;
   setTest.TestIntBasic();
   setTest.TestInt();
   setTest.TestInt2();
   setTest.TestString();
   //setTest.TestSetOfSetOfString();

   MapTest::MapTest mapTest;
   mapTest.TestInteger();
   mapTest.TestString();
   mapTest.TestMapOfMap();
}

int main()
{
   TestKernel<<<1, 1>>>();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   EdgeEnergy::DiracHartreeSlaterIonizationEnergies::loadxionUis();
   EdgeEnergy::NISTEdgeEnergy::loadNISTxrtdb();
   EdgeEnergy::ChantlerEdgeEnergy::loadFFastEdgeDB();
   EdgeEnergy::DTSAEdgeEnergy::loadEdgeEnergies();

   Element::Init();
   MaterialFactory::Init();

   CzyzewskiMottScatteringAngle::init();
   NISTMottScatteringAngle::init();
   GasScatteringCrossSection::init();
   NISTMottRS::init();
   MeanIonizationPotential::Berger64MeanIonizationPotential::readTabulatedValues();
   MeanIonizationPotential::Berger83MeanIonizationPotential::readTabulatedValues();

   Math2Test::testRandom1();
   Math2Test::testRandom2();

   UncertainValue2::UncertainValue2 v0(0, "abc", 5);
   UncertainValue2::UncertainValue2 v1(1);
   UncertainValue2::UncertainValue2 v2(2, 10);
   UncertainValue2::UncertainValue2 v3(2, 10);
   printf("%d\n", v1.equals(v2));
   printf("%d\n", v1.equals(v3));
   printf("%d\n", v2.equals(v3));
   
   UncertainValue2Test::UncertainValue2Test uvTest;
   uvTest.testSpecialValues();
   uvTest.testA();
   uvTest.testB();
   uvTest.testC();
   uvTest.testAB();
   uvTest.testAdd1();
   uvTest.testAdd2();
   uvTest.testAdd3();
   uvTest.testMultiply();
   uvTest.testDivide();
   uvTest.testFunctions();
   
   ElementTest::ElementTest elementTest;
   elementTest.testZero();
   elementTest.testOne();

   MaterialTest::MaterialTest mat;
   mat.testOne();

   AtomicShellTest::testOne();

   EdgeEnergyTest::testOne();

   MeanIonizationPotentialTest::testOne();
   
   SphereTest::testContains();
   SphereTest::testGetFirstIntersection();

   CylindricalShapeTest::CylindricalShapeTest cylindricalShapeTest;
   cylindricalShapeTest.testZero();
   cylindricalShapeTest.testOne();
   cylindricalShapeTest.testTwo();
   cylindricalShapeTest.testThree();
   cylindricalShapeTest.testFour();
   cylindricalShapeTest.testFive();
   cylindricalShapeTest.testSix();
   cylindricalShapeTest.testSeven();
   cylindricalShapeTest.testEight();
   cylindricalShapeTest.testNine();
   cylindricalShapeTest.testTen();
   cylindricalShapeTest.testEleven();
   cylindricalShapeTest.testTwelve();

   BetheElectronEnergyLossTest::testOne();

   MonteCarloSSTest::testOne();

   SumShapeTest::testGetFirstIntersection();
   SumShapeTest::testAll();

   LinesOnLayers::run();

   return 0;
}
