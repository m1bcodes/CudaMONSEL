#include "hip/hip_runtime.h"
/*
* - without length, the array parameter (eg const double a[]) is always 3 dimensional
* - use "auto" keyword for pointers ONLY
*/

#include <stdio.h>

#include <hip/hip_runtime.h>

#include "Amphibian\Tests\HasherTest.cuh"
#include "Amphibian\Tests\StringTest.cuh"
#include "Amphibian\Tests\LinkedListTest.cuh"
#include "Amphibian\Tests\SetTest.cuh"
#include "Amphibian\Tests\MapTest.cuh"
#include "Amphibian\Tests\VectorTest.cuh"
#include "Amphibian\Tests\StackTest.cuh"
#include "Amphibian\random.cuh"
#include "Amphibian\Algorithm.cuh"

#include "CudaUtil.h"
#include "ImageUtil.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "gov\nist\microanalysis\Utility\Math2.cuh"

#include "gov\nist\microanalysis\Utility\UncertainValue2.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Material.cuh"

#include "gov\nist\microanalysis\EPQLibrary\EdgeEnergy.cuh"
#include "gov\nist\microanalysis\EPQLibrary\MaterialFactory.cuh"
#include "gov\nist\microanalysis\EPQLibrary\BrowningEmpiricalCrossSection.cuh"
#include "gov\nist\microanalysis\EPQLibrary\ScreenedRutherfordScatteringAngle.cuh"
#include "gov\nist\microanalysis\EPQLibrary\CzyzewskiMottScatteringAngle.cuh"
#include "gov\nist\microanalysis\EPQLibrary\GasScatteringCrossSection.cuh"
#include "gov\nist\microanalysis\EPQLibrary\NISTMottScatteringAngle.cuh"
#include "gov\nist\microanalysis\EPQLibrary\MeanIonizationPotential.cuh"

#include "gov\nist\nanoscalemetrology\JMONSEL\NISTMottRS.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NUTableInterpolation.cuh"

#include "gov\nist\microanalysis\EPQTests\UncertainValue2Test.cuh"
#include "gov\nist\microanalysis\EPQTests\ElementTest.cuh"
#include "gov\nist\microanalysis\EPQTests\MaterialTest.cuh"
#include "gov\nist\microanalysis\EPQTests\AtomicShellTest.cuh"
#include "gov\nist\microanalysis\EPQTests\EdgeEnergyTest.cuh"
#include "gov\nist\microanalysis\EPQTests\MeanIonizationPotentialTest.cuh"
#include "gov\nist\microanalysis\EPQTests\SphereTest.cuh"
#include "gov\nist\microanalysis\EPQTests\CylindricalShapeTest.cuh"
#include "gov\nist\microanalysis\EPQTests\SumShapeTest.cuh"
#include "gov\nist\microanalysis\EPQTests\BetheElectronEnergyLossTest.cuh"
#include "gov\nist\microanalysis\EPQTests\MonteCarloSSTest.cuh"

#include "gov\nist\nanoscalemetrology\JMONSELTests\LinesOnLayers0.cuh"

#include <chrono>

//__device__ __host__ float function(float x)
//{
//   #if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
//      return 10.0f * __sinf(x);
//   #else // host code here
//   #endif
//}

__host__ __device__ void testRandom1()
{
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());

   printf("Math2Test::testRandom1() completed.\n");
}

__host__ __device__ void testRandom2()
{
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());
   printf("%.10e\n", Random::random());

   printf("Math2Test::testRandom2() completed.\n");
}

__global__ void testLibraryCuda()
{
   printf("%d, %d, %d, %d, %d, %d\n", threadIdx.x, blockIdx.x, threadIdx.y, blockIdx.y, threadIdx.z, blockIdx.z);
   printf("%d, %d, %d\n", gridDim.x, gridDim.y, gridDim.z);

   testRandom1();
   testRandom2();

   HasherTest::TestOne();

   StringTest::EmptyTest();
   StringTest::TestOne();
   StringTest::AtoITest();
   StringTest::AtoFTest();
   StringTest::ItoATest();
   StringTest::findTest();
   StringTest::addTest();

   LinkedListTest::LinkedListTest lltest;
   lltest.InsertionTest();
   lltest.TestAddAllAsSet();
   LinkedListTest::TestListKV();
   LinkedListTest::testDLinkedList();

   SetTest::SetTest setTest;
   setTest.testIntBasic();
   setTest.testInt();
   setTest.testInt2();
   setTest.testString();
   //setTest.TestSetOfSetOfString();

   MapTest::MapTest mapTest;
   mapTest.testInteger();
   mapTest.testString();
   mapTest.testMapOfMap();

   VectorTest::testOne();
   VectorTest::testTwo();
   VectorTest::testThree();

   StackTest::testOne();
}

//const unsigned int H = 128, W = 80;
//const unsigned int TX = 16, TY = 16;
const unsigned int H = 4, W = 4;
const unsigned int TX = 2, TY = 2;
dim3 blockSize(TX, TY); // Equivalent to dim3 blockSize(TX, TY, 1);
int bx = (W + blockSize.x - 1) / blockSize.x;
int by = (H + blockSize.y - 1) / blockSize.y;
dim3 gridSize = dim3(bx, by);

__global__ void printRand()
{
   const unsigned int c = threadIdx.x + blockDim.x * blockIdx.x;
   const unsigned int r = blockIdx.y*blockDim.y + threadIdx.y;

   int blockId = blockIdx.x + blockIdx.y * gridDim.x;
   int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
   printf("%d, %d (%d): %.5e | ", r, c, threadId, Random::random());
}

void testGPU()
{
   printf("-----------------GPU-----------------------------\n");
   Random::initCudaStates << <1, 1 >> >(H * W);
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   printRand << <gridSize, blockSize >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   testLibraryCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
}

void testLibrary()
{
   printf("-----------------CPU-----------------------------\n");
   HasherTest::TestOne();

   StringTest::EmptyTest();
   StringTest::TestOne();
   StringTest::AtoITest();
   StringTest::AtoFTest();
   StringTest::ItoATest();
   StringTest::findTest();
   StringTest::addTest();

   LinkedListTest::LinkedListTest lltest;
   lltest.InsertionTest();
   lltest.TestAddAllAsSet();
   LinkedListTest::TestListKV();
   LinkedListTest::testDLinkedList();

   SetTest::SetTest setTest;
   setTest.testIntBasic();
   setTest.testInt();
   setTest.testInt2();
   setTest.testString();
   //setTest.TestSetOfSetOfString();

   MapTest::MapTest mapTest;
   mapTest.testInteger();
   mapTest.testString();
   mapTest.testMapOfMap();

   VectorTest::testOne();
   VectorTest::testTwo();
   VectorTest::testThree();
   VectorTest::testFour();

   StackTest::testOne();

   testRandom1();
   testRandom2();
}

void initSim()
{
   EdgeEnergy::DiracHartreeSlaterIonizationEnergies::loadxionUis();
   EdgeEnergy::NISTEdgeEnergy::loadNISTxrtdb();
   EdgeEnergy::ChantlerEdgeEnergy::loadFFastEdgeDB();
   EdgeEnergy::DTSAEdgeEnergy::loadEdgeEnergies();

   Element::init();
   MaterialFactory::init();

   BrowningEmpiricalCrossSection::init();
   ScreenedRutherfordScatteringAngle::init();
   CzyzewskiMottScatteringAngle::init();
   NISTMottScatteringAngle::init();
   GasScatteringCrossSection::init();
   NISTMottRS::init();
   MeanIonizationPotential::Berger64.readTabulatedValues();
   MeanIonizationPotential::Berger83.readTabulatedValues();
}

void testSim()
{
   UncertainValue2::UncertainValue2 v0(0, "abc", 5);
   UncertainValue2::UncertainValue2 v1(1);
   UncertainValue2::UncertainValue2 v2(2, 10);
   UncertainValue2::UncertainValue2 v3(2, 10);
   printf("%d\n", v1.equals(v2));
   printf("%d\n", v1.equals(v3));
   printf("%d\n", v2.equals(v3));

   UncertainValue2Test::UncertainValue2Test uvTest;
   uvTest.testSpecialValues();
   uvTest.testA();
   uvTest.testB();
   uvTest.testC();
   uvTest.testAB();
   uvTest.testAdd1();
   uvTest.testAdd2();
   uvTest.testAdd3();
   uvTest.testMultiply();
   uvTest.testDivide();
   uvTest.testFunctions();

   ElementTest::ElementTest elementTest;
   elementTest.testZero();
   elementTest.testOne();

   MaterialTest::MaterialTest mat;
   mat.testOne();

   AtomicShellTest::testOne();

   EdgeEnergyTest::testOne();

   MeanIonizationPotentialTest::testOne();

   SphereTest::testContains();
   SphereTest::testGetFirstIntersection();

   CylindricalShapeTest::CylindricalShapeTest cylindricalShapeTest;
   cylindricalShapeTest.testZero();
   cylindricalShapeTest.testOne();
   cylindricalShapeTest.testTwo();
   cylindricalShapeTest.testThree();
   cylindricalShapeTest.testFour();
   cylindricalShapeTest.testFive();
   cylindricalShapeTest.testSix();
   cylindricalShapeTest.testSeven();
   cylindricalShapeTest.testEight();
   cylindricalShapeTest.testNine();
   cylindricalShapeTest.testTen();
   cylindricalShapeTest.testEleven();
   cylindricalShapeTest.testTwelve();

   BetheElectronEnergyLossTest::testOne();

   MonteCarloSSTest::testOne();

   //SumShapeTest::SumShapeTest sumShapeTest;
   //sumShapeTest.testGetFirstIntersection();
   //sumShapeTest.testAll();
}

__global__ void printNullReference()
{
   printf("%s\n", Reference::d_NullReference->getLongForm().c_str());
}

__global__ void printSpwem()
{
   printf("GPU: %d\n", NISTMottScatteringAngle::getNISTMSA(59).getSpwem().size());
   for (auto a : NISTMottScatteringAngle::getNISTMSA(59).getSpwem()) {
      printf("%.10e ", a);
   }
   printf("GPU end\n");
}

__global__ void printMeanIonizationPotential()
{
   printf("GPU:\n");
   printf("Berger64: %d\n", MeanIonizationPotential::d_Berger64->getData().size());
   for (auto a : MeanIonizationPotential::d_Berger64->getData()) {
      printf("%.10e ", a);
   }
   printf("Berger64 end\n");
   printf("Berger83: %d\n", MeanIonizationPotential::d_Berger83->getData().size());
   for (auto a : MeanIonizationPotential::d_Berger83->getData()) {
      printf("%.10e ", a);
   }
   printf("Berger83 end\n");
   printf("GPU end\n");
}

void initCuda()
{
   printf("-----------------initCuda-----------------------------\n");

   Material::initCuda << <1, 1 >> >();
   AlgorithmUser::initCuda << <1, 1 >> >();
   NISTMottRS::initFactory << <1, 1 >> >();
   NUTableInterpolation::initFactory << <1, 1 >> >();

   char *d_data = nullptr;
   checkCudaErrors(hipMalloc((void **)&d_data, sizeof(char) * 128));
   checkCudaErrors(hipMemcpy(d_data, Reference::NullReference.getReference().c_str(), sizeof(char) * 128, hipMemcpyHostToDevice));
   Reference::initCuda << <1, 1 >> >(d_data);
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   checkCudaErrors(hipFree(d_data));
   printNullReference << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   Element::copyDataToCuda();
   Element::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   BrowningEmpiricalCrossSection::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   ScreenedRutherfordScatteringAngle::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   
   NISTMottScatteringAngle::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   NISTMottScatteringAngle::initFactory << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   NISTMottScatteringAngle::transferDataToCuda();
   printSpwem << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   printf("CPU: %d\n", NISTMottScatteringAngle::getNISTMSA(59).getSpwem().size());
   for (auto a : NISTMottScatteringAngle::getNISTMSA(59).getSpwem()) {
      printf("%.10e ", a);
   }
   printf("CPU end\n");
   NISTMottRS::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   MeanIonizationPotential::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   MeanIonizationPotential::transferDataToCuda();
   printMeanIonizationPotential << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   printf("CPU:\n");
   printf("Berger64: %d\n", MeanIonizationPotential::Berger64.getData().size());
   for (auto a : MeanIonizationPotential::Berger64.getData()) {
      printf("%.10e ", a);
   }
   printf("Berger64 end\n");
   printf("Berger83: %d\n", MeanIonizationPotential::Berger83.getData().size());
   for (auto a : MeanIonizationPotential::Berger83.getData()) {
      printf("%.10e ", a);
   }
   printf("Berger83 end\n");
   printf("CPU end\n");
}
// causes stack overflow on GPU since it is sorting on worst case (ie already sorted array)
// use case: Histogram::Histogram
//__global__ void AlgorithmTest()
//{
//   VectorXd bins(360 + 1, 0);
//   bins[0] = 0;
//   const double delta = (Math2::PI * 2) / (360 + 1);
//   for (int i = 1; i < bins.size(); ++i)
//      bins[i] = i * delta;
//   Algorithm::quicksort(bins.data(), 0, bins.size() - 1);
//}

__global__ void printTotalCrossSection()
{
   printf("%.5e\n", NISTMottScatteringAngle::d_Factory->get(*Element::dC).totalCrossSection(8.01088e-17));
   //printf("%.5e\n", NISTMottScatteringAngle::d_Factory->get(*Element::dO));
   //printf("%.5e\n", NISTMottScatteringAngle::d_Factory->get(*Element::dH));

   //printf("%.5e\n", NISTMottScatteringAngle::d_Factory->get(*Element::dC).totalCrossSection(8.01088e-17));
   //printf("%.5e\n", NISTMottScatteringAngle::d_Factory->get(*Element::dO).totalCrossSection(8.01088e-17));
   //printf("%.5e\n", NISTMottScatteringAngle::d_Factory->get(*Element::dH).totalCrossSection(8.01088e-17));
}

int main()
{
   hipDeviceSetLimit(hipLimitMallocHeapSize, 1e9);
   hipDeviceSetLimit(hipLimitStackSize, 131072);
   size_t pValue;
   hipDeviceGetLimit(&pValue, hipLimitMallocHeapSize);
   printf("hipLimitMallocHeapSize: %d\n", pValue);
   hipDeviceGetLimit(&pValue, hipLimitStackSize);
   printf("hipLimitStackSize: %d\n", pValue);

   testLibrary();
   initSim();
   testSim();
   testGPU();

   initCuda();

   LinesOnLayers::transferDataToCuda();

   size_t a, t;
   checkCudaErrors(hipMemGetInfo(&a, &t));
   printf("free/total: %lu/%lu\n", a, t);

   LinesOnLayers::initCuda << <1, 1 >> >();
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());

   //LinesOnLayers::runCuda << <1, 1 >> >();
   //checkCudaErrors(hipDeviceSynchronize());
   //checkCudaErrors(hipGetLastError());

   //hipStream_t streams[2];
   //hipStreamCreate(&streams[0]);
   //hipStreamCreate(&streams[1]);

   auto start = std::chrono::system_clock::now();
   //LinesOnLayers::runCudaSinglePixel << <gridSize, blockSize >> >();
   //LinesOnLayers::runCudaSinglePixel << <1, 1, 0, streams[0]>> >(0, 0);
   //LinesOnLayers::runCudaSinglePixel << <1, 1, 0, streams[1] >> >(0, 1);
   //LinesOnLayers::runCudaSinglePixel << <8, 1 >> >();
   LinesOnLayers::runCudaSinglePixel << <1, 1 >> >(0, 4);
   checkCudaErrors(hipDeviceSynchronize());
   checkCudaErrors(hipGetLastError());
   auto end = std::chrono::system_clock::now();
   std::chrono::duration<double> elapsed_seconds = end - start;
   std::time_t end_time = std::chrono::system_clock::to_time_t(end);
   std::cout << std::endl << "finished computation at " << std::ctime(&end_time) << "elapsed time: " << elapsed_seconds.count() << "s" << std::endl;

   printf("done\n");

   //testsCPU();
   //LinesOnLayers::initCuda();
   //LinesOnLayers::runCuda();

   return 0;
}
