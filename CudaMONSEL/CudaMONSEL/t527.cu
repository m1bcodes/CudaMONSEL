
#include <hip/hip_runtime.h>
//#include <stdio.h>
//#include <curand.h>
//#include <curand_kernel.h>
//#include <math.h>
//#include <assert.h>
//
//#define MIN 2
//#define MAX 7
//#define ITER 10000000
//
//__global__ void setup_kernel(curandState *state)
//{
//   int idx = threadIdx.x + blockDim.x * blockIdx.x;
//   printf("%d\n", rand());
//   curand_init(1234, idx, 0, &state[idx]);
//}
//
//__global__ void generate_kernel(curandState *my_curandstate, const unsigned int n, const unsigned *max_rand_int, const unsigned *min_rand_int, unsigned int *result)
//{
//   int idx = threadIdx.x + blockDim.x * blockIdx.x;
//
//   int count = 0;
//   while (count < n) {
//      float myrandf = curand_uniform(my_curandstate + idx);
//      myrandf *= (max_rand_int[idx] - min_rand_int[idx] + 0.999999);
//      myrandf += min_rand_int[idx];
//      int myrand = (int)truncf(myrandf);
//
//      assert(myrand <= max_rand_int[idx]);
//      assert(myrand >= min_rand_int[idx]);
//      result[myrand - min_rand_int[idx]]++;
//      count++;
//   }
//}
//
//int main()
//{
//   curandState *d_state;
//   cudaMalloc(&d_state, sizeof(curandState));
//   unsigned *d_result, *h_result;
//   unsigned *d_max_rand_int, *h_max_rand_int, *d_min_rand_int, *h_min_rand_int;
//   cudaMalloc(&d_result, (MAX - MIN + 1) * sizeof(unsigned));
//   h_result = (unsigned *)malloc((MAX - MIN + 1) * sizeof(unsigned));
//   cudaMalloc(&d_max_rand_int, sizeof(unsigned));
//   h_max_rand_int = (unsigned *)malloc(sizeof(unsigned));
//   cudaMalloc(&d_min_rand_int, sizeof(unsigned));
//   h_min_rand_int = (unsigned *)malloc(sizeof(unsigned));
//   cudaMemset(d_result, 0, (MAX - MIN + 1) * sizeof(unsigned));
//   setup_kernel<<<1, 1>>>(d_state);
//
//   *h_max_rand_int = MAX;
//   *h_min_rand_int = MIN;
//   cudaMemcpy(d_max_rand_int, h_max_rand_int, sizeof(unsigned), cudaMemcpyHostToDevice);
//   cudaMemcpy(d_min_rand_int, h_min_rand_int, sizeof(unsigned), cudaMemcpyHostToDevice);
//   generate_kernel<<<1, 1>>>(d_state, ITER, d_max_rand_int, d_min_rand_int, d_result);
//   cudaMemcpy(h_result, d_result, (MAX - MIN + 1) * sizeof(unsigned), cudaMemcpyDeviceToHost);
//   printf("Bin:    Count: \n");
//   for (int i = MIN; i <= MAX; i++)
//      printf("%d    %d\n", i, h_result[i - MIN]);
//
//   return 0;
//}