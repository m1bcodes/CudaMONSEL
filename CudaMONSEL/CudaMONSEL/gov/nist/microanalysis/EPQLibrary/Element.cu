#include "hip/hip_runtime.h"
#include "Element.cuh"

namespace Element
{
   const long long serialVersionUID = 0x987360133793L;

   float mIonizationEnergy[104];
   float mAtomicWeight[112];

   const Element None(0);
   const Element H(1);
   const Element He(2);
   const Element Li(3);
   const Element Be(4);
   const Element B(5);
   const Element C(6);
   const Element N(7);
   const Element O(8);
   const Element F(9);
   const Element Ne(10);
   const Element Na(11);
   const Element Mg(12);
   const Element Al(13);
   const Element Si(14);
   const Element P(15);
   const Element S(16);
   const Element Cl(17);
   const Element Ar(18);
   const Element K(19);
   const Element Ca(20);
   const Element Sc(21);
   const Element Ti(22);
   const Element V(23);
   const Element Cr(24);
   const Element Mn(25);
   const Element Fe(26);
   const Element Co(27);
   const Element Ni(28);
   const Element Cu(29);
   const Element Zn(30);
   const Element Ga(31);
   const Element Ge(32);
   const Element As(33);
   const Element Se(34);
   const Element Br(35);
   const Element Kr(36);
   const Element Rb(37);
   const Element Sr(38);
   const Element Y(39);
   const Element Zr(40);
   const Element Nb(41);
   const Element Mo(42);
   const Element Tc(43);
   const Element Ru(44);
   const Element Rh(45);
   const Element Pd(46);
   const Element Ag(47);
   const Element Cd(48);
   const Element In(49);
   const Element Sn(50);
   const Element Sb(51);
   const Element Te(52);
   const Element I(53);
   const Element Xe(54);
   const Element Cs(55);
   const Element Ba(56);
   const Element La(57);
   const Element Ce(58);
   const Element Pr(59);
   const Element Nd(60);
   const Element Pm(61);
   const Element Sm(62);
   const Element Eu(63);
   const Element Gd(64);
   const Element Tb(65);
   const Element Dy(66);
   const Element Ho(67);
   const Element Er(68);
   const Element Tm(69);
   const Element Yb(70);
   const Element Lu(71);
   const Element Hf(72);
   const Element Ta(73);
   const Element W(74);
   const Element Re(75);
   const Element Os(76);
   const Element Ir(77);
   const Element Pt(78);
   const Element Au(79);
   const Element Hg(80);
   const Element Tl(81);
   const Element Pb(82);
   const Element Bi(83);
   const Element Po(84);
   const Element At(85);
   const Element Rn(86);
   const Element Fr(87);
   const Element Ra(88);
   const Element Ac(89);
   const Element Th(90);
   const Element Pa(91);
   const Element U(92);
   const Element Np(93);
   const Element Pu(94);
   const Element Am(95);
   const Element Cm(96);
   const Element Bk(97);
   const Element Cf(98);
   const Element Es(99);
   const Element Fm(100);
   const Element Md(101);
   const Element No(102);
   const Element Lr(103);
   const Element Rf(104);
   const Element Db(105);
   const Element Sg(106);
   const Element Bh(107);
   const Element Hs(108);
   const Element Mt(109);
   const Element Uun(110);
   const Element Uuu(111);
   const Element Uub(112);

   const Element mAllElements[] = {
      H,
      He,
      Li,
      Be,
      B,
      C,
      N,
      O,
      F,
      Ne,
      Na,
      Mg,
      Al,
      Si,
      P,
      S,
      Cl,
      Ar,
      K,
      Ca,
      Sc,
      Ti,
      V,
      Cr,
      Mn,
      Fe,
      Co,
      Ni,
      Cu,
      Zn,
      Ga,
      Ge,
      As,
      Se,
      Br,
      Kr,
      Rb,
      Sr,
      Y,
      Zr,
      Nb,
      Mo,
      Tc,
      Ru,
      Rh,
      Pd,
      Ag,
      Cd,
      In,
      Sn,
      Sb,
      Te,
      I,
      Xe,
      Cs,
      Ba,
      La,
      Ce,
      Pr,
      Nd,
      Pm,
      Sm,
      Eu,
      Gd,
      Tb,
      Dy,
      Ho,
      Er,
      Tm,
      Yb,
      Lu,
      Hf,
      Ta,
      W,
      Re,
      Os,
      Ir,
      Pt,
      Au,
      Hg,
      Tl,
      Pb,
      Bi,
      Po,
      At,
      Rn,
      Fr,
      Ra,
      Ac,
      Th,
      Pa,
      U,
      Np,
      Pu,
      Am,
      Cm,
      Bk,
      Cf,
      Es,
      Fm,
      Md,
      No,
      Lr,
      Rf,
      Db,
      Sg,
      Bh,
      Hs,
      Mt,
      Uun,
      Uuu,
      Uub
   };

   char const * const mElementNames[] = {
      "None",
      "Hydrogen",
      "Helium",
      "Lithium",
      "Beryllium",
      "Boron",
      "Carbon",
      "Nitrogen",
      "Oxygen",
      "Fluorine",
      "Neon",
      "Sodium",
      "Magnesium",
      "Aluminum",
      "Silicon",
      "Phosphorus",
      "Sulfur",
      "Chlorine",
      "Argon",
      "Potassium",
      "Calcium",
      "Scandium",
      "Titanium",
      "Vanadium",
      "Chromium",
      "Manganese",
      "Iron",
      "Cobalt",
      "Nickel",
      "Copper",
      "Zinc",
      "Gallium",
      "Germanium",
      "Arsenic",
      "Selenium",
      "Bromine",
      "Krypton",
      "Rubidium",
      "Strontium",
      "Yttrium",
      "Zirconium",
      "Niobium",
      "Molybdenum",
      "Technetium",
      "Ruthenium",
      "Rhodium",
      "Palladium",
      "Silver",
      "Cadmium",
      "Indium",
      "Tin",
      "Antimony",
      "Tellurium",
      "Iodine",
      "Xenon",
      "Cesium",
      "Barium",
      "Lanthanum",
      "Cerium",
      "Praseodymium",
      "Neodymium",
      "Promethium",
      "Samarium",
      "Europium",
      "Gadolinium",
      "Terbium",
      "Dysprosium",
      "Holmium",
      "Erbium",
      "Thulium",
      "Ytterbium",
      "Lutetium",
      "Hafnium",
      "Tantalum",
      "Tungsten",
      "Rhenium",
      "Osmium",
      "Iridium",
      "Platinum",
      "Gold",
      "Mercury",
      "Thallium",
      "Lead",
      "Bismuth",
      "Polonium",
      "Astatine",
      "Radon",
      "Francium",
      "Radium",
      "Actinium",
      "Thorium",
      "Protactinium",
      "Uranium",
      "Neptunium",
      "Plutonium",
      "Americium",
      "Curium",
      "Berkelium",
      "Californium",
      "Einsteinium",
      "Fermium",
      "Mendelevium",
      "Nobelium",
      "Lawrencium",
      "Rutherfordium",
      "Dubnium",
      "Seaborgium",
      "Bohrium",
      "Hassium",
      "Meitnerium",
      "Ununnilium",
      "Unununium",
      "Ununbium",
      "End-of-elements"
   };

   char const * const mAbbreviations[] = {
      "",
      "H",
      "He",
      "Li",
      "Be",
      "B",
      "C",
      "N",
      "O",
      "F",
      "Ne",
      "Na",
      "Mg",
      "Al",
      "Si",
      "P",
      "S",
      "Cl",
      "Ar",
      "K",
      "Ca",
      "Sc",
      "Ti",
      "V",
      "Cr",
      "Mn",
      "Fe",
      "Co",
      "Ni",
      "Cu",
      "Zn",
      "Ga",
      "Ge",
      "As",
      "Se",
      "Br",
      "Kr",
      "Rb",
      "Sr",
      "Y",
      "Zr",
      "Nb",
      "Mo",
      "Tc",
      "Ru",
      "Rh",
      "Pd",
      "Ag",
      "Cd",
      "In",
      "Sn",
      "Sb",
      "Te",
      "I",
      "Xe",
      "Cs",
      "Ba",
      "La",
      "Ce",
      "Pr",
      "Nd",
      "Pm",
      "Sm",
      "Eu",
      "Gd",
      "Tb",
      "Dy",
      "Ho",
      "Er",
      "Tm",
      "Yb",
      "Lu",
      "Hf",
      "Ta",
      "W",
      "Re",
      "Os",
      "Ir",
      "Pt",
      "Au",
      "Hg",
      "Tl",
      "Pb",
      "Bi",
      "Po",
      "At",
      "Rn",
      "Fr",
      "Ra",
      "Ac",
      "Th",
      "Pa",
      "U",
      "Np",
      "Pu",
      "Am",
      "Cm",
      "Bk",
      "Cf",
      "Es",
      "Fm",
      "Md",
      "No",
      "Lr",
      "Rf",
      "Db",
      "Sg",
      "Bh",
      "Hs",
      "Mt",
      "Uun",
      "Uuu",
      "Uub",
      "EOE"
   };

   Element::Element(int atomicNo)
   {
      if ((atomicNo >= elmNone) && (atomicNo < elmEndOfElements)) {
         mAtomicNumber = atomicNo;
      }
      else {
         printf("Wrong atomic number %d\n", atomicNo);
      }
   }

   Element::Element()
   {
      mAtomicNumber = elmNone;
   }

   void Element::readAtomicWeights()
   {
      try {
         std::ifstream file("AtomicWeights.csv");

         int idx = 0;
         for (CSVIterator loop(file); loop != CSVIterator(); ++loop) { // TODO: check if the first line should be removed
            mAtomicWeight[idx] = std::stof((*loop)[0]);
         }
         ++idx;
      }
      catch (std::exception&) {
         throw 0; //throw new EPQFatalException("Fatal error while attempting to load the atomic weights data file.");
      }
   }

   int Element::atomicNumberForName(char* name)
   {
      for (int i = 0; i < elmEndOfElements + 1; ++i) {
         if ((strcmp(mElementNames[i], name) == 0) || (strcmp(mAbbreviations[i], name) == 0)) { // TODO: make it case insensitive
            return i;
         }
      }
      try {
         return std::stoi(name);
      }
      catch (std::exception&) {
         return elmNone;
      }
   }

   Element Element::byName(char* name)
   {
      int z = atomicNumberForName(name);
      return z == 0 ? None : mAllElements[z - 1];
   }

   Element Element::byAtomicNumber(int an)
   {
      return (an >= 1) && (an < elmEndOfElements - 1) ? mAllElements[an - 1] : None;
   }

   double Element::getAtomicWeight(int atomicNo)
   {
      if (mAtomicWeight == NULL) {
         readAtomicWeights();
      }
      return mAtomicWeight[atomicNo - 1];
   }

   Element const * Element::allElements()
   {
      return mAllElements;
   }

   //Element* Element::range(Element min, Element max)
   //{
   //   if (min.getAtomicNumber() <= max.getAtomicNumber()) {
   //      throw 0;
   //   }
   //   return Arrays.copyOfRange(mAllElements, min.getAtomicNumber() - 1, max.getAtomicNumber() - 1);
   //}
   //
   //double Element::meanIonizationPotential(int atomicNo) {
   //   try {
   //      return MeanIonizationPotential.Berger64.compute(Element.byAtomicNumber(atomicNo));
   //   }
   //   catch (std::exception& ex) {
   //      return MeanIonizationPotential.Sternheimer64.compute(Element.byAtomicNumber(atomicNo));
   //   }
   //}

   int Element::getAtomicNumber() {
      return mAtomicNumber;
   }

   double Element::getAtomicWeight()
   {
      return getAtomicWeight(mAtomicNumber);
   }

   double Element::getMass()
   {
      return ToSI::AMU(getAtomicWeight(mAtomicNumber));
   }

   char const * Element::toAbbrev()
   {
      return mAbbreviations[mAtomicNumber];
   }

   char const * Element::toAbbrev(int atomicNo)
   {
      return mAbbreviations[atomicNo];
   }

   char const * Element::toString(int el)
   {
      return mElementNames[el];
   }

   //double meanIonizationPotential()
   //{
   //   try {
   //      return MeanIonizationPotential.Berger64.compute(this);
   //   }
   //   catch (final Exception ex) {
   //      return MeanIonizationPotential.Sternheimer64.compute(this);
   //   }
   //}

   //public double energyLoss(double eK) {
   //   return BetheElectronEnergyLoss.JoyLuo1989.compute(this, eK);
   //}
   //

   //public double massAbsorptionCoefficient(double energy) {
   //   return AlgorithmUser.getDefaultMAC().compute(this, energy);
   //}
   //

   //public double massAbsorptionCoefficient(XRayTransition xrt)
   //throws EPQException{
   //   return AlgorithmUser.getDefaultMAC().compute(this, xrt);
   //}
   //

   bool Element::isValid(int atomicNo)
   {
      return (atomicNo >= elmH) && (atomicNo < elmEndOfElements);
   }

   bool Element::isValid()
   {
      return (mAtomicNumber >= elmH) && (mAtomicNumber < elmEndOfElements);
   }

   int Element::compareTo(Element e)
   {
      if (mAtomicNumber < e.mAtomicNumber) {
         return -1;
      }
      else {
         return mAtomicNumber == e.mAtomicNumber ? 0 : 1;
      }
   }

   int Element::hashCode()
   {
      // mAtomicNumber is always less than 128 (1<<7). Int has 31 + 1 bits. 31-7
      // = 24
      return mAtomicNumber << 24;
   }

   bool Element::equals(Element el)
   {
      return el.mAtomicNumber == mAtomicNumber;
   }

   char const * Element::toString()
   {
      return mElementNames[mAtomicNumber];
   }

   double Element::getIonizationEnergy()
   {
      int idx = 0;
      try {
         std::ifstream file("IonizationEnergies.csv");

         for (CSVIterator loop(file); loop != CSVIterator(); ++loop) { // TODO: check if the first line should be removed
            if ((*loop)[0][0] == '/' && (*loop)[0][1] == '/') {
               continue;
            }
            if (CSVIterator::IsNaN((*loop)[0])) {
               mIonizationEnergy[idx] = -1.0;
            }
            else {
               mIonizationEnergy[idx] = std::stof((*loop)[0]);
            }
         }
         ++idx;
      }
      catch (std::exception&) {
         throw 0; // throw new EPQFatalException("Fatal error while attempting to load the atomic weights data file.");
      }

      double res = (mAtomicNumber - 1 <= 104) ? mIonizationEnergy[mAtomicNumber - 1] : -1.0;
      if (res == -1.0) {
         throw 0; // new EPQFatalException("The ionization energy is not available for " + toAbbrev());
      }
      return res;
   }

   Element Element::readResolve()
   {
      return Element::byAtomicNumber(mAtomicNumber);
   }

   //char * const Element::getListOfAbbreviations(Element minEl, Element maxEl)
   //{
   //   int numEl = maxEl.getAtomicNumber() - minEl.getAtomicNumber() + 1;
   //   char *res[] = new char*[numEl];
   //   for (int z = minEl.getAtomicNumber(); z <= maxEl.getAtomicNumber(); ++z)
   //      res.add(toAbbrev(z));
   //   return res;
   //}

   //static public final ArrayList<String> getListOfElements(Element minEl, Element maxEl)
   //{
   //   final ArrayList<String> res = new ArrayList<String>();
   //   for (int z = minEl.getAtomicNumber(); z <= maxEl.getAtomicNumber(); ++z)
   //      res.add(toString(z));
   //   return res;
   //}

}
