#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\EPQLibrary\Composition.cuh"

#include "Amphibian\random.cuh"

#include "gov\nist\microanalysis\Utility\Math2.cuh"

#include <algorithm>

namespace Composition
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
   __device__ static const long long serialVersionUID = 0x42;
   __device__ static const data_type OUT_OF_THIS_MANY_ATOMS = 1.0;
#else
   static const long long serialVersionUID = 0x42;
   static const data_type OUT_OF_THIS_MANY_ATOMS = 1.0;
#endif

   //static std::pair<const Element::Element*, UncertainValue2::UncertainValue2> make_pair(const Element::Element* first, const UncertainValue2::UncertainValue2& second)
   //{
   //   return std::make_pair(first, second);
   //}

   __host__ __device__ static LinkedListKV::Node<const Element::Element*, UncertainValue2::UncertainValue2>* make_pair(const Element::Element* first, const UncertainValue2::UncertainValue2& second)
   {
      return amp::make_pair(first, second);
   }

   __host__ __device__ void Composition::renormalize()
   {
      if (!mConstituents.empty()) {
         mNormalization = UncertainValue2::ZERO();
         for (auto &e : mConstituents) {
            auto const &uv = e.second;
            if (((UncertainValue2::UncertainValue2&)uv).doubleValue() > 0.0) {
               mNormalization = UncertainValue2::add(mNormalization, uv);
            }
         }

         mAtomicNormalization = UncertainValue2::ZERO();
         for (auto e : mConstituentsAtomic) {
            auto const &uv = e.second;
            if (((UncertainValue2::UncertainValue2&)uv).doubleValue() > 0.0) {
               mAtomicNormalization = UncertainValue2::add(mAtomicNormalization, uv);
            }
         }
      }
      else {
         mNormalization = UncertainValue2::ONE();
         mAtomicNormalization = UncertainValue2::ONE();
      }
      mMoleNorm = UncertainValue2::NaN();
   }

   __host__ __device__ Composition::Composition() :
      mNormalization(UncertainValue2::UncertainValue2(1)),
      mAtomicNormalization(UncertainValue2::UncertainValue2(1)),
      mName(""),
      mOptimalRepresentation(Representation::UNDETERMINED),
      mMoleNorm(UncertainValue2::NaN())
   {
      renormalize();
   }

   __host__ __device__ Composition::Composition(const Composition& comp)
   {
      replicate(comp);
   }

   Composition::Composition(const Element::Element* elms[], int elmsLen, const data_type massFracs[], int massFracsLen)
   {
      if (elmsLen != massFracsLen) {
         printf("Composition::Composition: elmsLen != massFracsLen, (%d, %d)", elmsLen, massFracsLen);
      }
      for (int i = 0; i < elmsLen; ++i) {
         mConstituents.insert(make_pair(elms[i], UncertainValue2::UncertainValue2(massFracs[i])));
      }
      recomputeStoiciometry();
      renormalize();
   }

   Composition::Composition(const Element::Element& elm)
   {
      mConstituents.insert(make_pair(&elm, UncertainValue2::ONE()));
      recomputeStoiciometry();
      renormalize();
   }

   __host__ __device__ Composition::Composition(const Element::Element* elms[], int elmsLen, const data_type massFracs[], int massFracsLen, char const* name)
   {
      data_type* wf = new data_type[elmsLen];
      if (!wf) printf("Composition::Composition: failed creating array.\n");
      if (elmsLen == massFracsLen - 1) {
         data_type sum = 0.0;
         for (int i = 0; i < massFracsLen; ++i) {
            sum += massFracs[i];
            wf[i] = massFracs[i];
         }
         if (sum > 1.0) {
            printf("Composition::Composition: sum is greater than 1 (%lf)", sum);
         }
         wf[elmsLen - 1] = 1.0 - sum;
         elmsLen = massFracsLen;
      }
      if (elmsLen != massFracsLen) {
         printf("Composition::Composition: elmsLen != massFracsLen, (%d, %d)", elmsLen, massFracsLen);
      }
      for (int i = 0; i < elmsLen; ++i) {
         if (massFracs[i] < 0.0) {
            printf("A mass fraction was less than zero while defining the material %s", name);
         }
         UncertainValue2::UncertainValue2 tmp(massFracs[i]);
         mConstituents.insert(make_pair(elms[i], tmp));
      }
      delete[] wf;
      mName = name;
      recomputeStoiciometry();
      renormalize();
   }

   bool Composition::operator==(const Composition& obj) const
   {
      if (this == &obj) {
         return true;
      }

      if (!sameConstituents(obj.mConstituents)) {
         return false;
      }

      if (!sameConstituentsAtomic(obj.mConstituentsAtomic)) {
         return false;
      }
      if (!(mName == obj.mName)) {
         return false;
      }

      if (!(mNormalization == obj.mNormalization)) {
         return false;
      }
      if (!(mOptimalRepresentation == obj.mOptimalRepresentation)) {
         return false;
      }
      return true;
   }

   void Composition::operator=(const Composition& comp)
   {
      replicate(comp);
   }

   Composition Composition::readResolve()
   {
      renormalize();
      return *this;
   }

   __host__ __device__ void Composition::replicate(const Composition& comp)
   {
      if (&comp == this) return;

      mConstituents.clear();
      mConstituentsAtomic.clear();

      for (auto &itr : comp.mConstituents) {
         mConstituents.insert(make_pair((const Element::Element*)itr.first, (UncertainValue2::UncertainValue2&)itr.second));
      }

      for (auto &ca : comp.mConstituentsAtomic) {
         mConstituentsAtomic.insert(make_pair((const Element::Element*)ca.first, (UncertainValue2::UncertainValue2&)ca.second));
      }
      mNormalization = comp.mNormalization;
      mAtomicNormalization = comp.mAtomicNormalization;
      mMoleNorm = comp.mMoleNorm;
      mName = comp.mName;
      mOptimalRepresentation = comp.mOptimalRepresentation;
   }

   __host__ __device__ Element::UnorderedSetT Composition::getElementSet() const
   {
      Element::UnorderedSetT elmset;
      for (auto &c : mConstituents) {
         elmset.insert(c.first);
      }
      //std::transform(mConstituents.begin(), mConstituents.end(), std::inserter(elmset, elmset.end()), [](std::pair<const Element::Element*, UncertainValue2::UncertainValue2> p) { return p.first; });
      return elmset;
   }

   Element::OrderedSetT Composition::getSortedElements() const
   {
      Element::OrderedSetT elmset;
      for (auto c : mConstituents) {
         elmset.insert(c.first);
      }
      return elmset;
   }

   __host__ __device__ int Composition::getElementCount() const
   {
      return mConstituents.size();
   }

   void Composition::addElement(int atomicNo, data_type massFrac)
   {
      addElement(Element::byAtomicNumber(atomicNo), massFrac);
   }

   void Composition::addElement(int atomicNo, const UncertainValue2::UncertainValue2 massFrac)
   {
      addElement(Element::byAtomicNumber(atomicNo), massFrac);
   }

   void Composition::addElement(const Element::Element& elm, data_type massFrac)
   {
      auto uv = UncertainValue2::UncertainValue2(massFrac);
      addElement(elm, uv);
   }

   __host__ __device__ data_type Composition::weightFraction(const Element::Element& elm, const bool normalized) const
   {
      auto itr = mConstituents.find(&elm);
      if (itr != mConstituents.end()) {
         auto &d = itr->second;
         return normalized ? normalize(d, mNormalization, true).doubleValue() : ((UncertainValue2::UncertainValue2&)d).doubleValue();
      }
      return 0.0;
   }

   __host__ __device__ void Composition::recomputeStoiciometry()
   {
      mMoleNorm = UncertainValue2::ZERO();
      for (auto &itr : mConstituents) {
         mMoleNorm = UncertainValue2::add(mMoleNorm, UncertainValue2::multiply(1.0 / ((const Element::Element*)itr.first)->getAtomicWeight(), itr.second));
      }

      mConstituentsAtomic.clear();
      Element::UnorderedSetT constituentsAtomicKeys;
      for (auto &ca : mConstituentsAtomic) {
         constituentsAtomicKeys.insert(ca.first);
      }

      for (const auto &elm : constituentsAtomicKeys) {
         UncertainValue2::UncertainValue2 uv0(UncertainValue2::ZERO());
         UncertainValue2::UncertainValue2 uv1(mConstituents.find(elm)->second);
         UncertainValue2::UncertainValue2 uv2(UncertainValue2::multiply(elm->getAtomicWeight() / OUT_OF_THIS_MANY_ATOMS, mMoleNorm));
         UncertainValue2::UncertainValue2 uv3(UncertainValue2::divide(uv1, uv2));
         UncertainValue2::UncertainValue2 moleFrac((mMoleNorm.doubleValue() > 0.0 ? uv3 : uv0));

         mConstituentsAtomic.insert(make_pair(elm, moleFrac));
      }
      mOptimalRepresentation = Representation::WEIGHT_PCT;
   }

   void Composition::addElement(const Element::Element& elm, const UncertainValue2::UncertainValue2& massFrac)
   {
      mConstituents.insert(make_pair(&elm, massFrac));
      recomputeStoiciometry();
      renormalize();
   }

   UncertainValue2::UncertainValue2 Composition::weightFractionU(const Element::Element& elm, bool normalized) const
   {
      return weightFractionU(elm, normalized, true);
   }

   UncertainValue2::UncertainValue2 Composition::weightFractionU(const Element::Element& elm, bool normalized, bool positiveOnly) const
   {
      auto itr = mConstituents.find(&elm);
      if (itr != mConstituents.end()) {
         auto d = itr->second;
         return normalized ? normalize(d, mNormalization, positiveOnly) : d;
      }
      return UncertainValue2::ZERO();
   }

   Composition positiveDefinite(const Composition& comp)
   {
      Composition res;
      const Element::UnorderedSetT& elemSet = comp.getElementSet();
      for (auto elm : elemSet) {
         if (comp.weightFraction(*elm, false) > 0.0) {
            res.addElement(*elm, comp.weightFractionU(*elm, false));
         }
      }
      return res;
   }

   void Composition::addElementByStoiciometry(const Element::Element& elm, const UncertainValue2::UncertainValue2& moleFrac)
   {
      mConstituentsAtomic.insert(make_pair(&elm, moleFrac));
      recomputeWeightFractions();
      renormalize();
   }

   void Composition::addElementByStoiciometry(const Element::Element& elm, data_type moleFrac)
   {
      addElementByStoiciometry(elm, UncertainValue2::UncertainValue2(moleFrac));
   }

   __host__ __device__ void Composition::clear()
   {
      mConstituents.clear();
      mConstituentsAtomic.clear();
      mNormalization = UncertainValue2::ONE();
      mAtomicNormalization = UncertainValue2::ONE();
      mMoleNorm = UncertainValue2::NaN();
   }

   void Composition::defineByWeightFraction(const Element::Element* elms[], int elmsLen, data_type wgtFracs[], int wgtFracsLen)
   {
      clear();
      if (elmsLen != wgtFracsLen) {
         printf("Composition::defineByWeightFraction1: elmsLen != wgtFracsLen (%d, %d)", elmsLen, wgtFracsLen);
      }
      for (int i = 0; i < elmsLen; ++i) {
         mConstituents.insert(make_pair(elms[i], UncertainValue2::UncertainValue2(wgtFracs[i])));
      }
      recomputeStoiciometry();
      renormalize();
   }

   void Composition::defineByWeightFraction(const Element::Element* elms[], int elmsLen, const UncertainValue2::UncertainValue2 wgtFracs[], int wgtFracsLen)
   {
      clear();
      if (elmsLen != wgtFracsLen) {
         printf("Composition::defineByWeightFraction2: elmsLen != wgtFracsLen (%d, %d)", elmsLen, wgtFracsLen);
      }
      for (int i = 0; i < elmsLen; ++i) {
         mConstituents.insert(make_pair(elms[i], wgtFracs[i]));
      }
      recomputeStoiciometry();
      renormalize();
   }

   __host__ __device__ void Composition::defineByMoleFraction(const Element::Element* elms[], int elmsLen, const data_type moleFracs[], int moleFracsLen)
   {
      clear();
      if (elmsLen != moleFracsLen) {
         printf("Composition::defineByWeightFraction: elmsLen != moleFracsLen (%d, %d)", elmsLen, moleFracsLen);
      }
      data_type mfSum = 0;
      for (int k = 0; k < moleFracsLen; ++k) {
         mfSum += moleFracs[k];
      }
      for (int i = 0; i < moleFracsLen; ++i) {
         mConstituentsAtomic.insert(make_pair(elms[i], UncertainValue2::UncertainValue2(moleFracs[i] / mfSum)));
      }
      recomputeWeightFractions();
      renormalize();
   }

   __host__ __device__ UncertainValue2::UncertainValue2 Composition::atomicPercentU(const Element::Element& elm) const
   {
      return atomicPercentU(elm, true);
   }

   __host__ __device__ UncertainValue2::UncertainValue2 Composition::atomicPercentU(const Element::Element& elm, const bool positiveOnly) const
   {
      ConstituentsMapT::const_iterator itr = mConstituentsAtomic.find(&elm);
      return itr == mConstituentsAtomic.cend() ? UncertainValue2::ZERO() : normalize(itr->second, mAtomicNormalization, positiveOnly);
   }

   __host__ __device__ void Composition::recomputeWeightFractions()
   {
      UncertainValue2::UncertainValue2 totalWgt(UncertainValue2::ZERO());
      Element::UnorderedSetT constituentsAtomicKeys;
      for (const auto &ca : mConstituentsAtomic) {
         constituentsAtomicKeys.insert(ca.first);
      }
      for (const auto &elm : constituentsAtomicKeys) {
         UncertainValue2::UncertainValue2 uv0(atomicPercentU(*elm));
         UncertainValue2::UncertainValue2 uv1(UncertainValue2::multiply(elm->getAtomicWeight(), uv0));
         totalWgt = UncertainValue2::add(totalWgt, uv1);
      }

      mConstituents.clear();
      for (const auto &elm : constituentsAtomicKeys) {
         UncertainValue2::UncertainValue2 wgtFrac(UncertainValue2::multiply(elm->getAtomicWeight(), UncertainValue2::divide(atomicPercentU(*elm), totalWgt)));
         mConstituents.insert(make_pair(elm, wgtFrac));
      }
      mOptimalRepresentation = Representation::STOICIOMETRY;
   }

   __host__ __device__ UncertainValue2::UncertainValue2 normalize(const UncertainValue2::UncertainValue2& val, const UncertainValue2::UncertainValue2& norm, const bool positive)
   {
      if (norm.doubleValue() > 0.0) {
         //UncertainValue2::UncertainValue2& quotient = UncertainValue2::divide(val, norm);
         UncertainValue2::UncertainValue2 quotient(val.doubleValue() / norm.doubleValue());
         UncertainValue2::divide(val, norm, quotient);
         return positive ? UncertainValue2::positiveDefinite(quotient) : quotient;
      }
      else {
         return positive ? UncertainValue2::positiveDefinite(val) : val;
      }
   }

   void Composition::setOptimalRepresentation(const Representation opt)
   {
      switch (opt)
      {
      case UNDETERMINED:
         break;
      case WEIGHT_PCT:
         recomputeStoiciometry();
         break;
      case STOICIOMETRY:
         recomputeWeightFractions();
         break;
      }
   }

   Element::UnorderedSetT elementSet(const Composition* compositions[], int len)
   {
      Element::UnorderedSetT elms;
      for (int i = 0; i < len; ++i) {
         const Element::UnorderedSetT& elmset = compositions[i]->getElementSet();
         for (auto elm : elmset) {
            elms.insert(elm);
         }
      }
      return elms;
   }

   void Composition::defineByMaterialFraction(const Composition* compositions[], int compLen, const data_type matFracs[], int matFracsLen)
   {
      if (compLen != matFracsLen) {
         printf("Composition::defineByMaterialFraction: lengths are different (%d, %d)", compLen, matFracsLen);
      }
      clear();
      const Element::UnorderedSetT& elms = elementSet(compositions, compLen);
      const int len = elms.size();
      //std::vector<const Element::Element*> newElms(len);
      //std::vector<UncertainValue2::UncertainValue2> frac(len);
      const Element::Element** newElms = new const Element::Element*[len];
      UncertainValue2::UncertainValue2* frac = new UncertainValue2::UncertainValue2[len];

      int ji = 0;
      for (auto el : elms) {
         UncertainValue2::UncertainValue2 sum = UncertainValue2::ZERO();
         for (int i = 0; i < compLen; ++i) {
            auto uv = UncertainValue2::multiply(matFracs[i], compositions[i]->weightFractionU(*el, true));
            sum = UncertainValue2::add(sum, uv);
         }
         frac[ji] = sum;
         newElms[ji] = el;
         ++ji;
      }
      //defineByWeightFraction(newElms.data(), len, frac.data(), len);
      defineByWeightFraction(newElms, len, frac, len);

      delete[] newElms;
      delete[] frac;
   }

   void Composition::removeElement(const Element::Element& el)
   {
      if (mConstituents.find(&el) != mConstituents.end()) {
         mConstituents.erase(&el);
         mConstituentsAtomic.erase(&el);
         // Don't recomputeStoiciometry or recomputeWeightFractions
         renormalize();
      }
   }

   bool Composition::containsElement(const Element::Element& el) const
   {
      return (mConstituents.find(&el) != mConstituents.end() && ((UncertainValue2::UncertainValue2&)(mConstituents.find(&el)->second)).doubleValue() > 0.0);
   }

   bool Composition::containsAll(const Element::UnorderedSetT& elms) const
   {
      for (auto elm : elms) {
         if (!containsElement(*elm)) {
            return false;
         }
      }
      return true;
   }

   data_type Composition::atomicPercent(const Element::Element& elm) const
   {
      return atomicPercentU(elm).doubleValue();
   }

   UncertainValue2::UncertainValue2 Composition::stoichiometryU(const Element::Element& elm) const
   {
      ConstituentsMapT::const_iterator itr = mConstituentsAtomic.find(&elm);
      if (itr != mConstituentsAtomic.cend()) {
         return itr->second;
      }
      return UncertainValue2::ZERO();
   }

   data_type Composition::stoichiometry(const Element::Element& elm) const
   {
      return stoichiometryU(elm).doubleValue();
   }

   data_type Composition::atomsPerKg(Element::Element& elm, bool normalized)
   {
      return weightFraction(elm, normalized) / elm.getMass();
   }

   UncertainValue2::UncertainValue2 Composition::atomsPerKgU(const Element::Element& elm, bool normalized) const
   {
      return UncertainValue2::multiply(1.0 / elm.getMass(), weightFractionU(elm, normalized));
   }

   UncertainValue2::UncertainValue2 Composition::weightAvgAtomicNumberU() const
   {
      UncertainValue2::UncertainValue2 res = UncertainValue2::ZERO();
      for (auto itr : mConstituents) {
         auto elm = itr.first;
         auto uv0 = weightFractionU(*elm, true);
         auto uv = UncertainValue2::multiply(((const Element::Element*)elm)->getAtomicNumber(), uv0);
         res = UncertainValue2::add(res, uv);
      }
      return res;
   }

   data_type Composition::weightAvgAtomicNumber() const
   {
      return weightAvgAtomicNumberU().doubleValue();
   }

   data_type Composition::sumWeightFraction() const
   {
      data_type sum = 0.0;
      for (auto itr : mConstituents) {
         const UncertainValue2::UncertainValue2& uv = itr.second;
         if (uv.doubleValue() > 0.0) {
            sum += uv.doubleValue();
         }
      }
      return sum;
   }

   UncertainValue2::UncertainValue2 Composition::sumWeightFractionU() const
   {
      UncertainValue2::UncertainValue2 res = UncertainValue2::ZERO();
      for (auto itr : mConstituents) {
         const UncertainValue2::UncertainValue2& val = itr.second;
         if (val.doubleValue() > 0.0) {
            res = UncertainValue2::add(res, val);
         }
      }
      return res;
   }

   __host__ __device__ const char* Composition::toString() const
   {
      //if (mName.size() == 0) {
      //   return descriptiveString(false);
      //}
      return mName.c_str();
   }

   //String::String Composition::stoichiometryString()
   //{
   //   final StringBuffer sb = new StringBuffer();
   //   final NumberFormat nf = new HalfUpFormat("0.####");
   //   for (final Element elm : getElementSet()) {
   //      final UncertainValue2 d0 = atomicPercentU(elm);
   //      if (sb.length() > 1)
   //         sb.append(",");
   //      sb.append(elm.toAbbrev());
   //      sb.append("(");
   //      sb.append(d0.format(nf));
   //      sb.append(" atoms)");
   //   }
   //   return sb.toString();
   //}

   //String::String weightPercentString(bool normalize)
   //{
   //   final StringBuffer sb = new StringBuffer();
   //   final NumberFormat nf = new HalfUpFormat("0.0000");
   //   for (final Element elm : getElementSet()) {
   //      final UncertainValue2 d0 = weightFractionU(elm, normalize);
   //      if (sb.length() > 1)
   //         sb.append(",");
   //      sb.append(elm.toAbbrev());
   //      sb.append("(");
   //      sb.append(d0.format(nf));
   //      sb.append(" mass frac)");
   //   }
   //   if (!normalize) {
   //      sb.append(",\u03A3=");
   //      sb.append(sumWeightPercentU().format(nf));
   //   }
   //   return sb.toString();
   //}

   //String::String descriptiveString(bool normalize)
   //{
   //   StringBuffer sb = new StringBuffer();
   //   if ((mName != null) && (mName.length() > 0))
   //      sb.append(mName + " = ");
   //   sb.append("[");
   //   if (mOptimalRepresentation == Representation.STOICIOMETRY)
   //      sb.append(stoichiometryString());
   //   else
   //      sb.append(weightPercentString(normalize));
   //   sb.append("]");
   //   return sb.toString();
   //}

   //Element::Element Composition::getNthElementByWeight(int n)
   //{
   //   LinkedListKV::Node<UncertainValue2::UncertainValue2, Element::Element>* tm = NULL;
   //   auto constituentsItr = mConstituents;
   //   while (constituentsItr != NULL) {
   //      auto el = constituentsItr->GetKey();
   //      UncertainValue2::UncertainValue2 wf = weightFractionU(el, true);
   //      // Add hoc mechanism to handle the case in which multiple elements are
   //      // present in the same weightPct.
   //      hiprandState state;
   //      hiprand_init(1, 0, 0, &state);
   //      while (LinkedListKV::ContainsKey(tm, wf, UncertainValue2::AreEqual)) {
   //         wf = UncertainValue2::add(1.0e-10 * hiprand_uniform(&state), wf);
   //      }
   //      LinkedListKV::InsertHead(&tm, wf, el);
   //      constituentsItr = constituentsItr->GetNext();
   //   }
   //   int j = 0;
   //   auto tmItr = tm;
   //   while (tmItr != NULL) {
   //      ++j;
   //      if (j == LinkedListKV::Size(mConstituents) - n) {
   //         return tmItr->GetValue();
   //      }
   //      tmItr = tmItr->GetNext();
   //   }
   //   return Element::None;
   //}

   //Element::Element Composition::getNthElementByAtomicFraction(int n)
   //{
   //   LinkedListKV::Node<UncertainValue2::UncertainValue2, Element::Element>* tm = NULL;
   //   auto constituentsItr = mConstituents;
   //   while (constituentsItr != NULL) {
   //      auto el = constituentsItr->GetKey();
   //      auto mf = atomicPercentU(el);
   //      hiprandState state;
   //      hiprand_init(1, 0, 0, &state);
   //      while (LinkedListKV::ContainsKey(tm, mf, UncertainValue2::AreEqual)) {
   //         mf = UncertainValue2::add(1.0e-10 * hiprand_uniform(&state), mf);
   //      }
   //      LinkedListKV::InsertHead(&tm, mf, el);
   //      constituentsItr = constituentsItr->GetNext();
   //   }
   //   int j = 0;
   //   auto tmItr = tm;
   //   while (tmItr != NULL) {
   //      ++j;
   //      if (j == n) {
   //         return tmItr->GetValue();
   //      }
   //      tmItr = tmItr->GetNext();
   //   }
   //   return Element::None;
   //}

   __host__ __device__ void Composition::setName(const char* name)
   {
      mName = name;
   }

   __host__ __device__ const char* Composition::getName() const
   {
      return mName.c_str();
   }

   int Composition::compareTo(const Composition& comp) const
   {
      if (this == &comp) {
         return 0;
      }
      // hashers have to be the same also
      auto i = mConstituents.begin();
      auto j = comp.mConstituents.begin();
      while (i != mConstituents.end() && j != comp.mConstituents.end()) {
         int zi = ((const Element::Element*)(i->first))->getAtomicNumber();
         int zj = ((const Element::Element*)(j->first))->getAtomicNumber();
         if (zi < zj) {
            return +1;
         }
         else if (zi > zj) {
            return -1;
         }
         else {
            UncertainValue2::UncertainValue2 ci = i->second;
            UncertainValue2::UncertainValue2 cj = j->second;
            if (ci.lessThan(cj)) {
               return -1;
            }
            else if (ci.greaterThan(cj)) {
               return +1;
            }
         }
         ++i;
         ++j;
      }
      if (!(i == mConstituents.end())) {
         return +1;
      }
      if (!(j == comp.mConstituents.end())) {
         return -1;
      }
      return 0;
   }

   Composition Composition::asComposition() const
   {
      Composition res;
      res.replicate(*this);
      return res;
   }

   Composition Composition::clone() const
   {
      return asComposition();
   }

   UncertainValue2::UncertainValue2 Composition::differenceU(const Composition& comp) const
   {
      // assert (comp.getElementCount() == this.getElementCount());
      UncertainValue2::UncertainValue2 delta = UncertainValue2::ZERO();
      Element::UnorderedSetT allElms;
      const Element::UnorderedSetT& s0 = getElementSet();
      const Element::UnorderedSetT& s1 = comp.getElementSet();
      allElms.insert(s0.begin(), s0.end());
      allElms.insert(s1.begin(), s1.end());

      for (auto el : allElms) {
         auto uv0 = comp.weightFractionU(*el, false);
         auto uv1 = weightFractionU(*el, false);
         auto uv2 = UncertainValue2::subtract(uv0, uv1);
         delta = UncertainValue2::add(delta, UncertainValue2::sqr(uv2));
      }
      return UncertainValue2::multiply(1.0 / allElms.size(), delta).sqrt();
   }

   data_type Composition::difference(const Composition& comp) const
   {
      return differenceU(comp).doubleValue();
   }

   Representation Composition::getOptimalRepresentation() const
   {
      return mOptimalRepresentation;
   }

   unsigned int Composition::hashCode() const
   {
      unsigned int result = 1;
      static const unsigned int PRIME = 31;
      result = PRIME * result + mConstituents.hashCode();
      //for (auto c : mConstituents) {
      //   result = PRIME * result + c.first->hashCode() ^ c.second.hashCode();
      //}
      result = PRIME * result + mConstituentsAtomic.hashCode();
      //for (auto ca : mConstituentsAtomic) {
      //   result = PRIME * result + ca.first->hashCode() ^ ca.second.hashCode();
      //}
      //result = PRIME * result + std::hash<std::string>()(mName);
      result = PRIME * result + mName.hashCode();
      //result = PRIME * result + mName.hashCode();
      long temp;
      temp = mNormalization.hashCode();
      result = PRIME * result + (int)(temp ^ (temp >> 32));
      result = PRIME * result + mOptimalRepresentation;
      if (result == INT_MAX)
         result = INT_MIN;
      return result;
   }

   Composition::ConstituentsMapT& Composition::GetConstituents()
   {
      return mConstituents;
   }

   bool Composition::sameConstituents(const ConstituentsMapT& constituents) const
   {
      for (auto c : mConstituents) {
         auto itr = constituents.find(c.first);
         if (itr == constituents.end()) {
            return false;
         }
         if (!((UncertainValue2::UncertainValue2&)(c.second) == (UncertainValue2::UncertainValue2&)(itr->second))) {
            return false;
         }
      }

      return true;
   }

   bool Composition::sameConstituentsAtomic(const Composition::ConstituentsMapT& constituentsAtomic) const
   {
      for (auto c : mConstituentsAtomic) {
         auto itr = constituentsAtomic.find(c.first);
         if (itr == constituentsAtomic.end()) return false;
         if ((UncertainValue2::UncertainValue2&)(c.second) == (UncertainValue2::UncertainValue2&)(itr->second)) return false;
      }

      return true;
   }

   bool Composition::equals(const Composition& obj) const
   {
      return this == &obj || *this == obj;
   }

   bool Composition::almostEquals(const Composition& other, data_type tol) const
   {
      if (this == &other) {
         return true;
      }
      //if (*((int*)&other) == NULL) {
      //   return false;
      //}
      if (abs(mNormalization.doubleValue() - other.mNormalization.doubleValue()) > tol) {
         return false;
      }
      Element::UnorderedSetT allElms;
      const Element::UnorderedSetT& elms0 = other.getElementSet();
      for (auto e : elms0) {
         allElms.insert(e);
      }
      const Element::UnorderedSetT& elms1 = getElementSet();
      for (auto e : elms1) {
         allElms.insert(e);
      }
      for (auto elm : allElms) {
         {
            UncertainValue2::UncertainValue2 uv1 = weightFractionU(*elm, false);
            UncertainValue2::UncertainValue2 uv2 = other.weightFractionU(*elm, false);
            if ((*((int*)&uv1) == NULL) || (*((int*)&uv2) == NULL)) {
               return false;
            }
            if ((abs(uv1.doubleValue() - uv2.doubleValue()) > tol)
               || (abs(uv1.uncertainty() - uv2.uncertainty()) > tol)) {
               return false;
            }
         }
         {
            UncertainValue2::UncertainValue2 uv1 = this->atomicPercentU(*elm);
            UncertainValue2::UncertainValue2 uv2 = other.atomicPercentU(*elm);
            //if ((*((int*)&uv1) == NULL) || (*((int*)&uv2) == NULL)) {
            //   return false;
            //}
            if ((abs(uv1.doubleValue() - uv2.doubleValue()) > tol)
               || (abs(uv1.uncertainty() - uv2.uncertainty()) > tol)) {
               return false;
            }
         }
      }
      return true;
   }

   //Composition::ErrorMapT Composition::absoluteError(const Composition& std, bool normalize) const
   //{
   //   Element::UnorderedSetT elms;
   //   const Element::UnorderedSetT& elms0 = std.getElementSet();
   //   for (auto e : elms0) {
   //      elms.insert(e);
   //   }
   //   const Element::UnorderedSetT& elms1 = getElementSet();
   //   for (auto e : elms1) {
   //      elms.insert(e);
   //   }
   //   ErrorMapT res;
   //   for (auto elm : elms) {
   //      data_type u = weightFractionU(*elm, normalize).doubleValue();
   //      data_type s = std.weightFractionU(*elm, normalize).doubleValue();
   //      res.insert(std::make_pair(elm, s != 0.0 ? (u - s) / s : (u == 0.0 ? 0.0 : 1.0)));
   //   }
   //   return res;
   //}

   //Composition::ErrorMapT Composition::relativeError(const Composition& std, bool normalize) const
   //{
   //   Element::UnorderedSetT elms;
   //   const Element::UnorderedSetT& elms0 = std.getElementSet();
   //   for (auto e : elms0) {
   //      elms.insert(e);
   //   }
   //   const Element::UnorderedSetT& elms1 = getElementSet();
   //   for (auto e : elms1) {
   //      elms.insert(e);
   //   }
   //   ErrorMapT res;
   //   for (auto elm : elms) {
   //      data_type u = weightFractionU(*elm, normalize).doubleValue();
   //      data_type s = std.weightFractionU(*elm, normalize).doubleValue();
   //      res.insert(std::make_pair(elm, u - s));
   //   }
   //   return res;
   //}

   bool Composition::isUncertain()
   {
      switch (mOptimalRepresentation) {
      case WEIGHT_PCT:
      case UNDETERMINED:
         for (auto i : mConstituents) {
            const UncertainValue2::UncertainValue2& v = i.second;
            if (v.isUncertain()) {
               return true;
            }
         }
         break;
      case STOICIOMETRY:
         auto j = mConstituentsAtomic;
         for (auto j : mConstituentsAtomic) {
            const UncertainValue2::UncertainValue2& v = j.second;
            if (v.isUncertain()) {
               return true;
            }
         }
         break;
      }
      return false;
   }

   UncertainValue2::UncertainValue2 Composition::meanAtomicNumberU() const
   {
      UncertainValue2::UncertainValue2 res = UncertainValue2::ZERO();
      const Element::UnorderedSetT& elms = getElementSet();
      for (auto elm : elms) {
         const UncertainValue2::UncertainValue2& uv = UncertainValue2::multiply(elm->getAtomicNumber(), atomicPercentU(*elm));
         res = UncertainValue2::add(res, uv);
      }
      return res;
   }

   data_type Composition::meanAtomicNumber() const
   {
      data_type res = 0.0;
      const Element::UnorderedSetT& elms = getElementSet();
      for (auto elm : elms) {
         res += elm->getAtomicNumber() * atomicPercent(*elm);
      }
      return res;
   }

   void Composition::forceNormalization()
   {
      UncertainValue2::UncertainValue2 norm = sumWeightFractionU();
      ConstituentsMapT newConst;
      for (auto itr : mConstituents) {
         newConst.insert(make_pair((const Element::Element*)(itr.first), norm.doubleValue() > 0.0 ? UncertainValue2::divide((UncertainValue2::UncertainValue2&)(itr.second), norm) : UncertainValue2::ZERO()));
      }
      mConstituents.clear();
      mConstituents = newConst;
      mOptimalRepresentation = Representation::WEIGHT_PCT;
      renormalize();
   }

   Composition parseGlass(char str[], int numlines)
   {
      Composition result;
      int pos = 0;
      int c = 0;
      for (int n = 0; n < numlines; ++n) {
         char line[256];
         int k = 0;
         while (str[c] != '\n') {
            line[k] = str[c];
            ++c;
            ++k;
         }
         ++c;
         Composition::StringT strline(line);
         if (pos == 0) {
            if (!strline.find("NBS GLASS K ")) {
               result.setName("K" "NBS GLASS K");
            }
            else if (!strline.find("CATIO")) {
               pos = 1;
            }
         }
         else if (pos == 1) {
            if (!strline.find("AVERAGE ATOMIC NUMBER")) {
               pos = 2;
            }
            else {
               char elmName[4];
               char elmWgtPct[16];
               int r = 0, l = 0;
               int tabCount = 0;
               while (true) {
                  if (line[r] == '\t') {
                     l = 0;
                     ++tabCount;
                  }
                  if (tabCount > 6) {
                     break;
                  }
                  if (tabCount == 0) {
                     elmName[l] = line[r];
                     ++l;
                  }
                  if (tabCount == 5) {
                     elmWgtPct[l] = line[r];
                     ++l;
                  }
                  ++r;
               }

               Element::Element elm = Element::byName(elmName);
               data_type wgtPct = std::atof(elmWgtPct);
               result.addElement(elm, wgtPct / 100.0);
            }
         }
         else if (pos == 2) {
            if (!strline.find("WEIGHT PERCENT OXYGEN")) {
               char oWgtPctStr[256];
               int r = 0, l = 0;
               int tabCount = 0;
               while (true) {
                  if (line[r] == '\t') {
                     ++tabCount;
                  }
                  if (tabCount > 1) {
                     break;
                  }
                  if (tabCount == 1) {
                     oWgtPctStr[l] = line[r];
                     ++l;
                  }
                  ++r;
               }

               data_type oWgtPct = std::atof(oWgtPctStr);
               result.addElement(Element::O, oWgtPct / 100.0);
               break;
            }
         }
      }
      return result;
   }

   Composition Composition::randomize(data_type offset, data_type proportional) const
   {
      srand(0);
      Composition res;
      const Element::UnorderedSetT& elms = getElementSet();
      for (auto elm : elms) {
         data_type w = weightFraction(*elm, false);
         data_type v = w + w * Random::generateGaussianNoise(0, 1) * proportional + offset * Random::generateGaussianNoise(0, 1);
         v = v > 0.0 ? v : 0.0;
         v = v < 1.1 ? v : 1.1;
         res.addElement(*elm, v);
      }
      return res;
   }

   int DIM = 9;
   long PROJECTORS[100]; // = createProjectors(2762689630628022905L);

   long mIndexHashS = INT_MAX;
   long mIndexHashL = INT_MAX;

   void createProjectors(long seed)
   {
      srand(NULL);

      std::unordered_set<int> eval;
      for (int j = 0; j < 100; ++j) {
         long tmp;
         do {
            long mult = 1;
            tmp = 0;
            for (int i = 0; i < DIM; ++i, mult *= 10) {
               data_type r = (data_type)rand() / (data_type)RAND_MAX;
               tmp += r * 2 * mult;
            }
         } while (eval.find(tmp) != eval.end());
         PROJECTORS[j] = tmp;
      }
   }

   long Composition::indexHashCodeS() const
   {
      if (mIndexHashS == INT_MAX) {
         long res = 0;
         const Element::UnorderedSetT& elms = getElementSet();
         for (auto elm : elms) {
            int v = (int)sqrt(100.0 * weightFraction(*elm, false));
            v = v > 0 ? v : 0;
            v = v < 10 ? v : 0;
            res += v * PROJECTORS[elm->getAtomicNumber()];
         }
         mIndexHashS = res;
      }
      return mIndexHashS;
   }

   long Composition::indexHashCodeL() const
   {
      if (mIndexHashL == INT_MAX) {
         long res = 0;
         const Element::UnorderedSetT& elms = getElementSet();
         for (auto elm : elms) {
            int v = (int)(10.0 * weightFraction(*elm, false));
            v = v > 0 ? v : 0;
            v = v < 10 ? v : 0;
            res += v * PROJECTORS[elm->getAtomicNumber()];
         }
         mIndexHashL = res;
      }
      return mIndexHashL;
   }
}
