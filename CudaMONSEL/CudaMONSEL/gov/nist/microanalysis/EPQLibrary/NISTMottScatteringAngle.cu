#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\EPQLibrary\NISTMottScatteringAngle.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Reference.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\EPQLibrary\PhysicalConstants.cuh"
#include "gov\nist\microanalysis\EPQLibrary\FromSI.cuh"
#include "gov\nist\microanalysis\EPQLibrary\ScreenedRutherfordScatteringAngle.cuh"

#include "CudaUtil.h"

#include "Amphibian\random.cuh"

namespace NISTMottScatteringAngle
{
   static const Reference::Author* auRef[] = { &Reference::CPowell, &Reference::FSalvat, &Reference::AJablonski };
   static const Reference::WebSite REFERENCE("http://www.nist.gov/srd/nist64.htm", "NIST Electron Elastic-Scattering Cross-Section Database version 3.1", "2007 AUGUST 24", auRef, 3);


#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
   __constant__ const int SPWEM_LEN = 61;
   __constant__ const int X1_LEN = 201;
   __constant__ const double DL50 = 1.69897000434;
   __constant__ const double PARAM = 0.04336766652;

   __constant__ static const double MAX_NISTMOTT = 3.2043531e-15;
#else
   const int SPWEM_LEN = 61;
   const int X1_LEN = 201;
   const double DL50 = ::log(50.0);
   const double PARAM = (::log(2.0e4) - DL50) / 60.0;

   static const double MAX_NISTMOTT = ToSI::keV(20.0);
#endif

   __host__ __device__ static double value(double a, double b, double c, double y0, double y1, double y2, double x)
   {
      return (x - b) * (x - c) * y0 / ((a - b) * (a - c)) + (x - a) * (x - c) * y1 / ((b - a) * (b - c)) + (x - a) * (x - b) * y2 / ((c - a) * (c - b));
   }

   // https://www.oreilly.com/library/view/c-cookbook/0596007612/ch03s06.html
   static double sciToDub(const std::string& str)
   {
      std::string tmp = str.substr(str.find_first_not_of(" "));
      std::stringstream ss(tmp);
      double d = 0;
      ss >> d;

      if (ss.fail()) {
         std::string s = "Unable to format ";
         s += tmp;
         s += " as a number!";
         throw (s);
      }

      return (d);
   }

   void NISTMottScatteringAngle::loadData(int an)
   {
      std::string name(an < 10 ? ".\\gov\\nist\\microanalysis\\EPQLibrary\\NistXSec/E0" + std::to_string(an) + ".D64" : ".\\gov\\nist\\microanalysis\\EPQLibrary\\NistXSec/E" + std::to_string(an) + ".D64");
      printf("Reading: %s\n", name.c_str());
      try {
         std::ifstream t(name);
         if (!t.good()) throw 0;
         std::string line;
         std::getline(t, line);
         for (int j = 0; j < SPWEM_LEN; ++j) {
            std::getline(t, line);
            mSpwem[j] = sciToDub(line);
            for (int i = 0; i < X1_LEN; ++i) {
               std::getline(t, line);
               mX1[j][i] = sciToDub(line);
            }
         }
         t.close();
      }
      catch (std::exception&) {
         printf("Unable to construct NISTMottScatteringAngle: %s\n", name.c_str());
      }
   }

   __host__ __device__ NISTMottScatteringAngle::NISTMottScatteringAngle(const ElementT& elm) :
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      RandomizedScatterT("NIST Elastic cross-section", *Reference::dNullReference),
#else
      RandomizedScatterT("NIST Elastic cross-section", REFERENCE),
#endif
      mElement(elm),
      mSpwem(SPWEM_LEN, 0),
      mX1(SPWEM_LEN, VectorXf(X1_LEN, 0)),
      mRutherford(ScreenedRutherfordScatteringAngle::getSRSA(elm.getAtomicNumber()))
   {
#if (!(defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
      loadData(elm.getAtomicNumber());
#endif
   }

   StringT NISTMottScatteringAngle::toString() const
   {
      return "CrossSection[NIST-Mott," + StringT(mElement.toAbbrev()) + "]";
   }

   __host__ __device__ const ElementT& NISTMottScatteringAngle::getElement() const
   {
      return mElement;
   }
   
   __host__ __device__ double NISTMottScatteringAngle::totalCrossSection(const double energy) const
   {
      if (energy < MAX_NISTMOTT) {
         const double scale = PhysicalConstants::BohrRadius * PhysicalConstants::BohrRadius;
         const double logE = ::logf(FromSI::eV(energy));
         int j = 1 + (int)((logE - DL50) / PARAM);
         if (j == 1)
            return value(DL50, DL50 + PARAM, DL50 + 2.0 * PARAM, mSpwem[0], mSpwem[1], mSpwem[2], logE) * scale;
         else if (j == SPWEM_LEN)
            return value(DL50 + 58.0 * PARAM, DL50 + 59.0 * PARAM, DL50 + 60.0 * PARAM, mSpwem[SPWEM_LEN - 3], mSpwem[SPWEM_LEN - 2], mSpwem[SPWEM_LEN - 1], logE) * scale;
         else {
            double e0 = DL50 + (j - 2) * PARAM;
            return value(e0, e0 + PARAM, e0 + 2.0 * PARAM, mSpwem[j - 2], mSpwem[j - 1], mSpwem[j], logE) * scale;
         }
      }
      else {
         return mRutherford.totalCrossSection(energy);
      }
   }

   __host__ __device__ double NISTMottScatteringAngle::randomScatteringAngle(const double energy) const
   {
      if (energy < MAX_NISTMOTT) {
         const double logE = ::log(FromSI::eV(energy));
         const int j = (int)((logE - DL50) / PARAM); // offset to zero-based
         const double e2 = DL50 + (j + 1) * PARAM;
         const double e1 = e2 - PARAM;
         const int i = (logE - e1 < e2 - logE ? j : j + 1); // offset to zero-based
         if (!((i >= 0) && (i < SPWEM_LEN))) printf("%d %s %lf %s %lf %s %lf\n", i, "\t", FromSI::eV(energy), "\t", e1, "\t", e2);
         // via j
         const int k = Random::randomInt(200); // offset to zero-based
         const double x = (mX1[i][k + 1] - mX1[i][k]) * Random::random();
         const double q = mX1[i][k] + x;
         const double com = 1.0 - 2.0 * q * q;
         return com > -1.0 ? (com < 1.0 ? ::acos(com) : 0.0) : PhysicalConstants::PI;
      }
      else {
         return mRutherford.randomScatteringAngle(energy);
      }
   }

   __host__ __device__ const VectorXf& NISTMottScatteringAngle::getSpwem() const
   {
      return mSpwem;
   }

   __host__ __device__ const MatrixXf& NISTMottScatteringAngle::getX1() const
   {
      return mX1;
   }

   const NISTMottScatteringAngle* mScatter[113];

   void init()
   {
      mScatter[1] = new NISTMottScatteringAngle(Element::H);
      mScatter[2] = new NISTMottScatteringAngle(Element::He);
      mScatter[3] = new NISTMottScatteringAngle(Element::Li);
      mScatter[4] = new NISTMottScatteringAngle(Element::Be);
      mScatter[5] = new NISTMottScatteringAngle(Element::B);
      mScatter[6] = new NISTMottScatteringAngle(Element::C);
      mScatter[7] = new NISTMottScatteringAngle(Element::N);
      mScatter[8] = new NISTMottScatteringAngle(Element::O);
      mScatter[9] = new NISTMottScatteringAngle(Element::F);
      mScatter[10] = new NISTMottScatteringAngle(Element::Ne);
      mScatter[11] = new NISTMottScatteringAngle(Element::Na);
      mScatter[12] = new NISTMottScatteringAngle(Element::Mg);
      mScatter[13] = new NISTMottScatteringAngle(Element::Al);
      mScatter[14] = new NISTMottScatteringAngle(Element::Si);
      mScatter[15] = new NISTMottScatteringAngle(Element::P);
      mScatter[16] = new NISTMottScatteringAngle(Element::S);
      mScatter[17] = new NISTMottScatteringAngle(Element::Cl);
      mScatter[18] = new NISTMottScatteringAngle(Element::Ar);
      mScatter[19] = new NISTMottScatteringAngle(Element::K);
      mScatter[20] = new NISTMottScatteringAngle(Element::Ca);
      mScatter[21] = new NISTMottScatteringAngle(Element::Sc);
      mScatter[22] = new NISTMottScatteringAngle(Element::Ti);
      mScatter[23] = new NISTMottScatteringAngle(Element::V);
      mScatter[24] = new NISTMottScatteringAngle(Element::Cr);
      mScatter[25] = new NISTMottScatteringAngle(Element::Mn);
      mScatter[26] = new NISTMottScatteringAngle(Element::Fe);
      mScatter[27] = new NISTMottScatteringAngle(Element::Co);
      mScatter[28] = new NISTMottScatteringAngle(Element::Ni);
      mScatter[29] = new NISTMottScatteringAngle(Element::Cu);
      mScatter[30] = new NISTMottScatteringAngle(Element::Zn);
      mScatter[31] = new NISTMottScatteringAngle(Element::Ga);
      mScatter[32] = new NISTMottScatteringAngle(Element::Ge);
      mScatter[33] = new NISTMottScatteringAngle(Element::As);
      mScatter[34] = new NISTMottScatteringAngle(Element::Se);
      mScatter[35] = new NISTMottScatteringAngle(Element::Br);
      mScatter[36] = new NISTMottScatteringAngle(Element::Kr);
      mScatter[37] = new NISTMottScatteringAngle(Element::Rb);
      mScatter[38] = new NISTMottScatteringAngle(Element::Sr);
      mScatter[39] = new NISTMottScatteringAngle(Element::Y);
      mScatter[40] = new NISTMottScatteringAngle(Element::Zr);
      mScatter[41] = new NISTMottScatteringAngle(Element::Nb);
      mScatter[42] = new NISTMottScatteringAngle(Element::Mo);
      mScatter[43] = new NISTMottScatteringAngle(Element::Tc);
      mScatter[44] = new NISTMottScatteringAngle(Element::Ru);
      mScatter[45] = new NISTMottScatteringAngle(Element::Rh);
      mScatter[46] = new NISTMottScatteringAngle(Element::Pd);
      mScatter[47] = new NISTMottScatteringAngle(Element::Ag);
      mScatter[48] = new NISTMottScatteringAngle(Element::Cd);
      mScatter[49] = new NISTMottScatteringAngle(Element::In);
      mScatter[50] = new NISTMottScatteringAngle(Element::Sn);
      mScatter[51] = new NISTMottScatteringAngle(Element::Sb);
      mScatter[52] = new NISTMottScatteringAngle(Element::Te);
      mScatter[53] = new NISTMottScatteringAngle(Element::I);
      mScatter[54] = new NISTMottScatteringAngle(Element::Xe);
      mScatter[55] = new NISTMottScatteringAngle(Element::Cs);
      mScatter[56] = new NISTMottScatteringAngle(Element::Ba);
      mScatter[57] = new NISTMottScatteringAngle(Element::La);
      mScatter[58] = new NISTMottScatteringAngle(Element::Ce);
      mScatter[59] = new NISTMottScatteringAngle(Element::Pr);
      mScatter[60] = new NISTMottScatteringAngle(Element::Nd);
      mScatter[61] = new NISTMottScatteringAngle(Element::Pm);
      mScatter[62] = new NISTMottScatteringAngle(Element::Sm);
      mScatter[63] = new NISTMottScatteringAngle(Element::Eu);
      mScatter[64] = new NISTMottScatteringAngle(Element::Gd);
      mScatter[65] = new NISTMottScatteringAngle(Element::Tb);
      mScatter[66] = new NISTMottScatteringAngle(Element::Dy);
      mScatter[67] = new NISTMottScatteringAngle(Element::Ho);
      mScatter[68] = new NISTMottScatteringAngle(Element::Er);
      mScatter[69] = new NISTMottScatteringAngle(Element::Tm);
      mScatter[70] = new NISTMottScatteringAngle(Element::Yb);
      mScatter[71] = new NISTMottScatteringAngle(Element::Lu);
      mScatter[72] = new NISTMottScatteringAngle(Element::Hf);
      mScatter[73] = new NISTMottScatteringAngle(Element::Ta);
      mScatter[74] = new NISTMottScatteringAngle(Element::W);
      mScatter[75] = new NISTMottScatteringAngle(Element::Re);
      mScatter[76] = new NISTMottScatteringAngle(Element::Os);
      mScatter[77] = new NISTMottScatteringAngle(Element::Ir);
      mScatter[78] = new NISTMottScatteringAngle(Element::Pt);
      mScatter[79] = new NISTMottScatteringAngle(Element::Au);
      mScatter[80] = new NISTMottScatteringAngle(Element::Hg);
      mScatter[81] = new NISTMottScatteringAngle(Element::Tl);
      mScatter[82] = new NISTMottScatteringAngle(Element::Pb);
      mScatter[83] = new NISTMottScatteringAngle(Element::Bi);
      mScatter[84] = new NISTMottScatteringAngle(Element::Po);
      mScatter[85] = new NISTMottScatteringAngle(Element::At);
      mScatter[86] = new NISTMottScatteringAngle(Element::Rn);
      mScatter[87] = new NISTMottScatteringAngle(Element::Fr);
      mScatter[88] = new NISTMottScatteringAngle(Element::Ra);
      mScatter[89] = new NISTMottScatteringAngle(Element::Ac);
      mScatter[90] = new NISTMottScatteringAngle(Element::Th);
      mScatter[91] = new NISTMottScatteringAngle(Element::Pa);
      mScatter[92] = new NISTMottScatteringAngle(Element::U);
      mScatter[93] = new NISTMottScatteringAngle(Element::Np);
      mScatter[94] = new NISTMottScatteringAngle(Element::Pu);
      mScatter[95] = new NISTMottScatteringAngle(Element::Am);
      mScatter[96] = new NISTMottScatteringAngle(Element::Cm);
   }

   __device__ NISTMottScatteringAngle *dScatter[113];

   __global__ void initCuda()
   {
      dScatter[1] = new NISTMottScatteringAngle(*Element::dH);
      dScatter[2] = new NISTMottScatteringAngle(*Element::dHe);
      dScatter[3] = new NISTMottScatteringAngle(*Element::dLi);
      dScatter[4] = new NISTMottScatteringAngle(*Element::dBe);
      dScatter[5] = new NISTMottScatteringAngle(*Element::dB);
      dScatter[6] = new NISTMottScatteringAngle(*Element::dC);
      dScatter[7] = new NISTMottScatteringAngle(*Element::dN);
      dScatter[8] = new NISTMottScatteringAngle(*Element::dO);
      dScatter[9] = new NISTMottScatteringAngle(*Element::dF);
      dScatter[10] = new NISTMottScatteringAngle(*Element::dNe);
      dScatter[11] = new NISTMottScatteringAngle(*Element::dNa);
      dScatter[12] = new NISTMottScatteringAngle(*Element::dMg);
      dScatter[13] = new NISTMottScatteringAngle(*Element::dAl);
      dScatter[14] = new NISTMottScatteringAngle(*Element::dSi);
      dScatter[15] = new NISTMottScatteringAngle(*Element::dP);
      dScatter[16] = new NISTMottScatteringAngle(*Element::dS);
      dScatter[17] = new NISTMottScatteringAngle(*Element::dCl);
      dScatter[18] = new NISTMottScatteringAngle(*Element::dAr);
      dScatter[19] = new NISTMottScatteringAngle(*Element::dK);
      dScatter[20] = new NISTMottScatteringAngle(*Element::dCa);
      dScatter[21] = new NISTMottScatteringAngle(*Element::dSc);
      dScatter[22] = new NISTMottScatteringAngle(*Element::dTi);
      dScatter[23] = new NISTMottScatteringAngle(*Element::dV);
      dScatter[24] = new NISTMottScatteringAngle(*Element::dCr);
      dScatter[25] = new NISTMottScatteringAngle(*Element::dMn);
      dScatter[26] = new NISTMottScatteringAngle(*Element::dFe);
      dScatter[27] = new NISTMottScatteringAngle(*Element::dCo);
      dScatter[28] = new NISTMottScatteringAngle(*Element::dNi);
      dScatter[29] = new NISTMottScatteringAngle(*Element::dCu);
      dScatter[30] = new NISTMottScatteringAngle(*Element::dZn);
      dScatter[31] = new NISTMottScatteringAngle(*Element::dGa);
      dScatter[32] = new NISTMottScatteringAngle(*Element::dGe);
      dScatter[33] = new NISTMottScatteringAngle(*Element::dAs);
      dScatter[34] = new NISTMottScatteringAngle(*Element::dSe);
      dScatter[35] = new NISTMottScatteringAngle(*Element::dBr);
      dScatter[36] = new NISTMottScatteringAngle(*Element::dKr);
      dScatter[37] = new NISTMottScatteringAngle(*Element::dRb);
      dScatter[38] = new NISTMottScatteringAngle(*Element::dSr);
      dScatter[39] = new NISTMottScatteringAngle(*Element::dY);
      dScatter[40] = new NISTMottScatteringAngle(*Element::dZr);
      dScatter[41] = new NISTMottScatteringAngle(*Element::dNb);
      dScatter[42] = new NISTMottScatteringAngle(*Element::dMo);
      dScatter[43] = new NISTMottScatteringAngle(*Element::dTc);
      dScatter[44] = new NISTMottScatteringAngle(*Element::dRu);
      dScatter[45] = new NISTMottScatteringAngle(*Element::dRh);
      dScatter[46] = new NISTMottScatteringAngle(*Element::dPd);
      dScatter[47] = new NISTMottScatteringAngle(*Element::dAg);
      dScatter[48] = new NISTMottScatteringAngle(*Element::dCd);
      dScatter[49] = new NISTMottScatteringAngle(*Element::dIn);
      dScatter[50] = new NISTMottScatteringAngle(*Element::dSn);
      dScatter[51] = new NISTMottScatteringAngle(*Element::dSb);
      dScatter[52] = new NISTMottScatteringAngle(*Element::dTe);
      dScatter[53] = new NISTMottScatteringAngle(*Element::dI);
      dScatter[54] = new NISTMottScatteringAngle(*Element::dXe);
      dScatter[55] = new NISTMottScatteringAngle(*Element::dCs);
      dScatter[56] = new NISTMottScatteringAngle(*Element::dBa);
      dScatter[57] = new NISTMottScatteringAngle(*Element::dLa);
      dScatter[58] = new NISTMottScatteringAngle(*Element::dCe);
      dScatter[59] = new NISTMottScatteringAngle(*Element::dPr);
      dScatter[60] = new NISTMottScatteringAngle(*Element::dNd);
      dScatter[61] = new NISTMottScatteringAngle(*Element::dPm);
      dScatter[62] = new NISTMottScatteringAngle(*Element::dSm);
      dScatter[63] = new NISTMottScatteringAngle(*Element::dEu);
      dScatter[64] = new NISTMottScatteringAngle(*Element::dGd);
      dScatter[65] = new NISTMottScatteringAngle(*Element::dTb);
      dScatter[66] = new NISTMottScatteringAngle(*Element::dDy);
      dScatter[67] = new NISTMottScatteringAngle(*Element::dHo);
      dScatter[68] = new NISTMottScatteringAngle(*Element::dEr);
      dScatter[69] = new NISTMottScatteringAngle(*Element::dTm);
      dScatter[70] = new NISTMottScatteringAngle(*Element::dYb);
      dScatter[71] = new NISTMottScatteringAngle(*Element::dLu);
      dScatter[72] = new NISTMottScatteringAngle(*Element::dHf);
      dScatter[73] = new NISTMottScatteringAngle(*Element::dTa);
      dScatter[74] = new NISTMottScatteringAngle(*Element::dW);
      dScatter[75] = new NISTMottScatteringAngle(*Element::dRe);
      dScatter[76] = new NISTMottScatteringAngle(*Element::dOs);
      dScatter[77] = new NISTMottScatteringAngle(*Element::dIr);
      dScatter[78] = new NISTMottScatteringAngle(*Element::dPt);
      dScatter[79] = new NISTMottScatteringAngle(*Element::dAu);
      dScatter[80] = new NISTMottScatteringAngle(*Element::dHg);
      dScatter[81] = new NISTMottScatteringAngle(*Element::dTl);
      dScatter[82] = new NISTMottScatteringAngle(*Element::dPb);
      dScatter[83] = new NISTMottScatteringAngle(*Element::dBi);
      dScatter[84] = new NISTMottScatteringAngle(*Element::dPo);
      dScatter[85] = new NISTMottScatteringAngle(*Element::dAt);
      dScatter[86] = new NISTMottScatteringAngle(*Element::dRn);
      dScatter[87] = new NISTMottScatteringAngle(*Element::dFr);
      dScatter[88] = new NISTMottScatteringAngle(*Element::dRa);
      dScatter[89] = new NISTMottScatteringAngle(*Element::dAc);
      dScatter[90] = new NISTMottScatteringAngle(*Element::dTh);
      dScatter[91] = new NISTMottScatteringAngle(*Element::dPa);
      dScatter[92] = new NISTMottScatteringAngle(*Element::dU);
      dScatter[93] = new NISTMottScatteringAngle(*Element::dNp);
      dScatter[94] = new NISTMottScatteringAngle(*Element::dPu);
      dScatter[95] = new NISTMottScatteringAngle(*Element::dAm);
      dScatter[96] = new NISTMottScatteringAngle(*Element::dCm);
   }

   //__device__ void NISTMottScatteringAngle::copySpwem(float *dSpwem, unsigned int size)
   //{
   //   mSpwem.assign(dSpwem, dSpwem + size);
   //}

   //__device__ void NISTMottScatteringAngle::copyX1j(unsigned int r, float *dSpwem, unsigned int size)
   //{
   //   mX1[r].assign(dSpwem, dSpwem + size);
   //}

   __global__ void copySpwem(unsigned int i, float *dSpwem, unsigned int size)
   {
      dScatter[i]->copySpwem<float>(dSpwem, size);
   }

   __global__ void copyX1Row(unsigned int i, unsigned int r, float *dSX1r, unsigned int size)
   {
      dScatter[i]->copyX1Row<float>(r, dSX1r, size);
   }

   void copyDataToCuda()
   {
      for (int i = 1; i <= 96; ++i) {
         float *dSpwem = nullptr;
         const VectorXf& spwem = mScatter[i]->getSpwem();
         checkCudaErrors(hipMalloc((void **)&dSpwem, sizeof(float) * spwem.size()));
         checkCudaErrors(hipMemcpy(dSpwem, spwem.data(), sizeof(float) * spwem.size(), hipMemcpyHostToDevice));
         copySpwem << <1, 1 >> >(i, dSpwem, spwem.size());
         checkCudaErrors(hipDeviceSynchronize());
         checkCudaErrors(hipGetLastError());
         checkCudaErrors(hipFree(dSpwem));

         const MatrixXf& x1 = mScatter[i]->getX1();
         for (int r = 0; r < x1.size(); ++r) {
            float *dX1r = nullptr;
            checkCudaErrors(hipMalloc((void **)&dX1r, x1[r].size() * sizeof(float)));
            checkCudaErrors(hipMemcpy(dX1r, x1[r].data(), x1[r].size() * sizeof(float), hipMemcpyHostToDevice));
            copyX1Row << <1, 1 >> >(i, r, dX1r, x1[r].size());
            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipFree(dX1r));
         }
      }
   }

   __host__ __device__ const NISTMottScatteringAngle& getNISTMSA(int an)
   {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      return *dScatter[an];
#else
      return *mScatter[an];
#endif
   }

   __host__ __device__ NISTMottRandomizedScatterFactory::NISTMottRandomizedScatterFactory() :
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      RandomizedScatterFactoryT("NIST Mott Inelastic Cross-Section", *Reference::dNullReference)
#else
      RandomizedScatterFactoryT("NIST Mott Inelastic Cross-Section", REFERENCE)
#endif
   {
   }

   __host__ __device__ const RandomizedScatterT& NISTMottRandomizedScatterFactory::get(const ElementT& elm) const
   {
      return getNISTMSA(elm.getAtomicNumber());
   }

   const NISTMottRandomizedScatterFactory NISTMottRandomizedFactory;
   const RandomizedScatterFactoryT& Factory = NISTMottRandomizedFactory;
   __device__ const RandomizedScatterFactoryT* d_Factory = nullptr;

   __global__ void initFactory()
   {
      d_Factory = new NISTMottRandomizedScatterFactory();
   }
}
