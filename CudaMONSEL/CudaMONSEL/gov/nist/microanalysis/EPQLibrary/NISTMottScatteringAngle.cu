#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\EPQLibrary\NISTMottScatteringAngle.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Reference.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\EPQLibrary\PhysicalConstants.cuh"
#include "gov\nist\microanalysis\EPQLibrary\FromSI.cuh"
#include "gov\nist\microanalysis\EPQLibrary\ScreenedRutherfordScatteringAngle.cuh"

namespace NISTMottScatteringAngle
{
   static const Reference::Author* auRef[] = { &Reference::CPowell, &Reference::FSalvat, &Reference::AJablonski };
   static const Reference::WebSite REFERENCE("http://www.nist.gov/srd/nist64.htm", "NIST Electron Elastic-Scattering Cross-Section Database version 3.1", "2007 AUGUST 24", auRef, 3);

   const int SPWEM_LEN = 61;
   const int X1_LEN = 201;
   const double DL50 = ::log(50.0);
   const double PARAM = (::log(2.0e4) - DL50) / 60.0;

   const double MAX_NISTMOTT = ToSI::keV(20.0);

   static double value(double a, double b, double c, double y0, double y1, double y2, double x)
   {
      return (x - b) * (x - c) * y0 / ((a - b) * (a - c)) + (x - a) * (x - c) * y1 / ((b - a) * (b - c)) + (x - a) * (x - b) * y2 / ((c - a) * (c - b));
   }

   // https://www.oreilly.com/library/view/c-cookbook/0596007612/ch03s06.html
   static double sciToDub(const std::string& str)
   {
      std::string tmp = str.substr(str.find_first_not_of(" "));
      std::stringstream ss(tmp);
      double d = 0;
      ss >> d;

      if (ss.fail()) {
         std::string s = "Unable to format ";
         s += tmp;
         s += " as a number!";
         throw (s);
      }

      return (d);
   }

   void NISTMottScatteringAngle::loadData(int an)
   {
      std::string name(an < 10 ? ".\\gov\\nist\\microanalysis\\EPQLibrary\\NistXSec/E0" + std::to_string(an) + ".D64" : ".\\gov\\nist\\microanalysis\\EPQLibrary\\NistXSec/E" + std::to_string(an) + ".D64");
      printf("Reading: %s\n", name.c_str());
      try {
         std::ifstream t(name);
         if (!t.good()) throw 0;
         std::string line;
         std::getline(t, line);
         for (int j = 0; j < SPWEM_LEN; ++j) {
            std::getline(t, line);
            mSpwem[j] = sciToDub(line);
            for (int i = 0; i < X1_LEN; ++i) {
               std::getline(t, line);
               mX1[j][i] = sciToDub(line);
            }
         }
         t.close();
      }
      catch (std::exception&) {
         printf("Unable to construct NISTMottScatteringAngle: %s\n", name.c_str());
      }
   }

   NISTMottScatteringAngle::NISTMottScatteringAngle(const ElementT& elm) : RandomizedScatterT("NIST Elastic cross-section", REFERENCE), mElement(elm), mSpwem(SPWEM_LEN, 0), mX1(SPWEM_LEN, VectorXd(X1_LEN, 0)), mRutherford(ScreenedRutherfordScatteringAngle::getSRSA(elm.getAtomicNumber()))
   {
      loadData(elm.getAtomicNumber());
   }

   StringT NISTMottScatteringAngle::toString() const
   {
      return "CrossSection[NIST-Mott," + StringT(mElement.toAbbrev()) + "]";
   }

   const ElementT& NISTMottScatteringAngle::getElement() const
   {
      return mElement;
   }
   
   double NISTMottScatteringAngle::totalCrossSection(double energy) const
   {
      if (energy < MAX_NISTMOTT) {
         double scale = PhysicalConstants::BohrRadius * PhysicalConstants::BohrRadius;
         double logE = ::log(FromSI::eV(energy));
         int j = 1 + (int)((logE - DL50) / PARAM);
         if (j == 1)
            return value(DL50, DL50 + PARAM, DL50 + 2.0 * PARAM, mSpwem[0], mSpwem[1], mSpwem[2], logE) * scale;
         else if (j == SPWEM_LEN)
            return value(DL50 + 58.0 * PARAM, DL50 + 59.0 * PARAM, DL50 + 60.0 * PARAM, mSpwem[SPWEM_LEN - 3], mSpwem[SPWEM_LEN - 2], mSpwem[SPWEM_LEN - 1], logE)
            * scale;
         else {
            double e0 = DL50 + (j - 2) * PARAM;
            return value(e0, e0 + PARAM, e0 + 2.0 * PARAM, mSpwem[j - 2], mSpwem[j - 1], mSpwem[j], logE) * scale;
         }
      }
      else {
         return mRutherford.totalCrossSection(energy);
      }
   }

   double NISTMottScatteringAngle::randomScatteringAngle(double energy) const
   {
      if (energy < MAX_NISTMOTT) {
         double logE = ::log(FromSI::eV(energy));
         int j = (int)((logE - DL50) / PARAM); // offset to zero-based
         double e2 = DL50 + (j + 1) * PARAM;
         double e1 = e2 - PARAM;
         int i = (logE - e1 < e2 - logE ? j : j + 1); // offset to
         // zero-based
         if (!((i >= 0) && (i < SPWEM_LEN))) printf("%s\n", StringT(std::to_string(i) + "\t" + std::to_string(FromSI::eV(energy)) + "\t" + std::to_string(e1) + "\t" + std::to_string(e2)).c_str());
         // via j
         int k = Math2::randomInt(200); // offset to
         // zero-based
         double x = (mX1[i][k + 1] - mX1[i][k]) * Math2::random();
         double q = mX1[i][k] + x;
         double com = 1.0 - 2.0 * q * q;
         return com > -1.0 ? (com < 1.0 ? ::acos(com) : 0.0) : PhysicalConstants::PI;
      }
      else {
         return mRutherford.randomScatteringAngle(energy);
      }
   }

   const VectorXd& NISTMottScatteringAngle::getSpwem() const
   {
      return mSpwem;
   }

   const MatrixXd& NISTMottScatteringAngle::getX1() const
   {
      return mX1;
   }

   //const NISTMottScatteringAngle NISTMSA1(Element::H);
   //const NISTMottScatteringAngle NISTMSA2(Element::He);
   //const NISTMottScatteringAngle NISTMSA3(Element::Li);
   //const NISTMottScatteringAngle NISTMSA4(Element::Be);
   //const NISTMottScatteringAngle NISTMSA5(Element::B);
   //const NISTMottScatteringAngle NISTMSA6(Element::C);
   //const NISTMottScatteringAngle NISTMSA7(Element::N);
   //const NISTMottScatteringAngle NISTMSA8(Element::O);
   //const NISTMottScatteringAngle NISTMSA9(Element::F);
   //const NISTMottScatteringAngle NISTMSA10(Element::Ne);
   //const NISTMottScatteringAngle NISTMSA11(Element::Na);
   //const NISTMottScatteringAngle NISTMSA12(Element::Mg);
   //const NISTMottScatteringAngle NISTMSA13(Element::Al);
   //const NISTMottScatteringAngle NISTMSA14(Element::Si);
   //const NISTMottScatteringAngle NISTMSA15(Element::P);
   //const NISTMottScatteringAngle NISTMSA16(Element::S);
   //const NISTMottScatteringAngle NISTMSA17(Element::Cl);
   //const NISTMottScatteringAngle NISTMSA18(Element::Ar);
   //const NISTMottScatteringAngle NISTMSA19(Element::K);
   //const NISTMottScatteringAngle NISTMSA20(Element::Ca);
   //const NISTMottScatteringAngle NISTMSA21(Element::Sc);
   //const NISTMottScatteringAngle NISTMSA22(Element::Ti);
   //const NISTMottScatteringAngle NISTMSA23(Element::V);
   //const NISTMottScatteringAngle NISTMSA24(Element::Cr);
   //const NISTMottScatteringAngle NISTMSA25(Element::Mn);
   //const NISTMottScatteringAngle NISTMSA26(Element::Fe);
   //const NISTMottScatteringAngle NISTMSA27(Element::Co);
   //const NISTMottScatteringAngle NISTMSA28(Element::Ni);
   //const NISTMottScatteringAngle NISTMSA29(Element::Cu);
   //const NISTMottScatteringAngle NISTMSA30(Element::Zn);
   //const NISTMottScatteringAngle NISTMSA31(Element::Ga);
   //const NISTMottScatteringAngle NISTMSA32(Element::Ge);
   //const NISTMottScatteringAngle NISTMSA33(Element::As);
   //const NISTMottScatteringAngle NISTMSA34(Element::Se);
   //const NISTMottScatteringAngle NISTMSA35(Element::Br);
   //const NISTMottScatteringAngle NISTMSA36(Element::Kr);
   //const NISTMottScatteringAngle NISTMSA37(Element::Rb);
   //const NISTMottScatteringAngle NISTMSA38(Element::Sr);
   //const NISTMottScatteringAngle NISTMSA39(Element::Y);
   //const NISTMottScatteringAngle NISTMSA40(Element::Zr);
   //const NISTMottScatteringAngle NISTMSA41(Element::Nb);
   //const NISTMottScatteringAngle NISTMSA42(Element::Mo);
   //const NISTMottScatteringAngle NISTMSA43(Element::Tc);
   //const NISTMottScatteringAngle NISTMSA44(Element::Ru);
   //const NISTMottScatteringAngle NISTMSA45(Element::Rh);
   //const NISTMottScatteringAngle NISTMSA46(Element::Pd);
   //const NISTMottScatteringAngle NISTMSA47(Element::Ag);
   //const NISTMottScatteringAngle NISTMSA48(Element::Cd);
   //const NISTMottScatteringAngle NISTMSA49(Element::In);
   //const NISTMottScatteringAngle NISTMSA50(Element::Sn);
   //const NISTMottScatteringAngle NISTMSA51(Element::Sb);
   //const NISTMottScatteringAngle NISTMSA52(Element::Te);
   //const NISTMottScatteringAngle NISTMSA53(Element::I);
   //const NISTMottScatteringAngle NISTMSA54(Element::Xe);
   //const NISTMottScatteringAngle NISTMSA55(Element::Cs);
   //const NISTMottScatteringAngle NISTMSA56(Element::Ba);
   //const NISTMottScatteringAngle NISTMSA57(Element::La);
   //const NISTMottScatteringAngle NISTMSA58(Element::Ce);
   //const NISTMottScatteringAngle NISTMSA59(Element::Pr);
   //const NISTMottScatteringAngle NISTMSA60(Element::Nd);
   //const NISTMottScatteringAngle NISTMSA61(Element::Pm);
   //const NISTMottScatteringAngle NISTMSA62(Element::Sm);
   //const NISTMottScatteringAngle NISTMSA63(Element::Eu);
   //const NISTMottScatteringAngle NISTMSA64(Element::Gd);
   //const NISTMottScatteringAngle NISTMSA65(Element::Tb);
   //const NISTMottScatteringAngle NISTMSA66(Element::Dy);
   //const NISTMottScatteringAngle NISTMSA67(Element::Ho);
   //const NISTMottScatteringAngle NISTMSA68(Element::Er);
   //const NISTMottScatteringAngle NISTMSA69(Element::Tm);
   //const NISTMottScatteringAngle NISTMSA70(Element::Yb);
   //const NISTMottScatteringAngle NISTMSA71(Element::Lu);
   //const NISTMottScatteringAngle NISTMSA72(Element::Hf);
   //const NISTMottScatteringAngle NISTMSA73(Element::Ta);
   //const NISTMottScatteringAngle NISTMSA74(Element::W);
   //const NISTMottScatteringAngle NISTMSA75(Element::Re);
   //const NISTMottScatteringAngle NISTMSA76(Element::Os);
   //const NISTMottScatteringAngle NISTMSA77(Element::Ir);
   //const NISTMottScatteringAngle NISTMSA78(Element::Pt);
   //const NISTMottScatteringAngle NISTMSA79(Element::Au);
   //const NISTMottScatteringAngle NISTMSA80(Element::Hg);
   //const NISTMottScatteringAngle NISTMSA81(Element::Tl);
   //const NISTMottScatteringAngle NISTMSA82(Element::Pb);
   //const NISTMottScatteringAngle NISTMSA83(Element::Bi);
   //const NISTMottScatteringAngle NISTMSA84(Element::Po);
   //const NISTMottScatteringAngle NISTMSA85(Element::At);
   //const NISTMottScatteringAngle NISTMSA86(Element::Rn);
   //const NISTMottScatteringAngle NISTMSA87(Element::Fr);
   //const NISTMottScatteringAngle NISTMSA88(Element::Ra);
   //const NISTMottScatteringAngle NISTMSA89(Element::Ac);
   //const NISTMottScatteringAngle NISTMSA90(Element::Th);
   //const NISTMottScatteringAngle NISTMSA91(Element::Pa);
   //const NISTMottScatteringAngle NISTMSA92(Element::U);
   //const NISTMottScatteringAngle NISTMSA93(Element::Np);
   //const NISTMottScatteringAngle NISTMSA94(Element::Pu);

   const NISTMottScatteringAngle* mScatter[113];

   void init()
   {
      mScatter[1] = new NISTMottScatteringAngle(Element::H);
      mScatter[2] = new NISTMottScatteringAngle(Element::He);
      mScatter[3] = new NISTMottScatteringAngle(Element::Li);
      mScatter[4] = new NISTMottScatteringAngle(Element::Be);
      mScatter[5] = new NISTMottScatteringAngle(Element::B);
      mScatter[6] = new NISTMottScatteringAngle(Element::C);
      mScatter[7] = new NISTMottScatteringAngle(Element::N);
      mScatter[8] = new NISTMottScatteringAngle(Element::O);
      mScatter[9] = new NISTMottScatteringAngle(Element::F);
      mScatter[10] = new NISTMottScatteringAngle(Element::Ne);
      mScatter[11] = new NISTMottScatteringAngle(Element::Na);
      mScatter[12] = new NISTMottScatteringAngle(Element::Mg);
      mScatter[13] = new NISTMottScatteringAngle(Element::Al);
      mScatter[14] = new NISTMottScatteringAngle(Element::Si);
      mScatter[15] = new NISTMottScatteringAngle(Element::P);
      mScatter[16] = new NISTMottScatteringAngle(Element::S);
      mScatter[17] = new NISTMottScatteringAngle(Element::Cl);
      mScatter[18] = new NISTMottScatteringAngle(Element::Ar);
      mScatter[19] = new NISTMottScatteringAngle(Element::K);
      mScatter[20] = new NISTMottScatteringAngle(Element::Ca);
      mScatter[21] = new NISTMottScatteringAngle(Element::Sc);
      mScatter[22] = new NISTMottScatteringAngle(Element::Ti);
      mScatter[23] = new NISTMottScatteringAngle(Element::V);
      mScatter[24] = new NISTMottScatteringAngle(Element::Cr);
      mScatter[25] = new NISTMottScatteringAngle(Element::Mn);
      mScatter[26] = new NISTMottScatteringAngle(Element::Fe);
      mScatter[27] = new NISTMottScatteringAngle(Element::Co);
      mScatter[28] = new NISTMottScatteringAngle(Element::Ni);
      mScatter[29] = new NISTMottScatteringAngle(Element::Cu);
      mScatter[30] = new NISTMottScatteringAngle(Element::Zn);
      mScatter[31] = new NISTMottScatteringAngle(Element::Ga);
      mScatter[32] = new NISTMottScatteringAngle(Element::Ge);
      mScatter[33] = new NISTMottScatteringAngle(Element::As);
      mScatter[34] = new NISTMottScatteringAngle(Element::Se);
      mScatter[35] = new NISTMottScatteringAngle(Element::Br);
      mScatter[36] = new NISTMottScatteringAngle(Element::Kr);
      mScatter[37] = new NISTMottScatteringAngle(Element::Rb);
      mScatter[38] = new NISTMottScatteringAngle(Element::Sr);
      mScatter[39] = new NISTMottScatteringAngle(Element::Y);
      mScatter[40] = new NISTMottScatteringAngle(Element::Zr);
      mScatter[41] = new NISTMottScatteringAngle(Element::Nb);
      mScatter[42] = new NISTMottScatteringAngle(Element::Mo);
      mScatter[43] = new NISTMottScatteringAngle(Element::Tc);
      mScatter[44] = new NISTMottScatteringAngle(Element::Ru);
      mScatter[45] = new NISTMottScatteringAngle(Element::Rh);
      mScatter[46] = new NISTMottScatteringAngle(Element::Pd);
      mScatter[47] = new NISTMottScatteringAngle(Element::Ag);
      mScatter[48] = new NISTMottScatteringAngle(Element::Cd);
      mScatter[49] = new NISTMottScatteringAngle(Element::In);
      mScatter[50] = new NISTMottScatteringAngle(Element::Sn);
      mScatter[51] = new NISTMottScatteringAngle(Element::Sb);
      mScatter[52] = new NISTMottScatteringAngle(Element::Te);
      mScatter[53] = new NISTMottScatteringAngle(Element::I);
      mScatter[54] = new NISTMottScatteringAngle(Element::Xe);
      mScatter[55] = new NISTMottScatteringAngle(Element::Cs);
      mScatter[56] = new NISTMottScatteringAngle(Element::Ba);
      mScatter[57] = new NISTMottScatteringAngle(Element::La);
      mScatter[58] = new NISTMottScatteringAngle(Element::Ce);
      mScatter[59] = new NISTMottScatteringAngle(Element::Pr);
      mScatter[60] = new NISTMottScatteringAngle(Element::Nd);
      mScatter[61] = new NISTMottScatteringAngle(Element::Pm);
      mScatter[62] = new NISTMottScatteringAngle(Element::Sm);
      mScatter[63] = new NISTMottScatteringAngle(Element::Eu);
      mScatter[64] = new NISTMottScatteringAngle(Element::Gd);
      mScatter[65] = new NISTMottScatteringAngle(Element::Tb);
      mScatter[66] = new NISTMottScatteringAngle(Element::Dy);
      mScatter[67] = new NISTMottScatteringAngle(Element::Ho);
      mScatter[68] = new NISTMottScatteringAngle(Element::Er);
      mScatter[69] = new NISTMottScatteringAngle(Element::Tm);
      mScatter[70] = new NISTMottScatteringAngle(Element::Yb);
      mScatter[71] = new NISTMottScatteringAngle(Element::Lu);
      mScatter[72] = new NISTMottScatteringAngle(Element::Hf);
      mScatter[73] = new NISTMottScatteringAngle(Element::Ta);
      mScatter[74] = new NISTMottScatteringAngle(Element::W);
      mScatter[75] = new NISTMottScatteringAngle(Element::Re);
      mScatter[76] = new NISTMottScatteringAngle(Element::Os);
      mScatter[77] = new NISTMottScatteringAngle(Element::Ir);
      mScatter[78] = new NISTMottScatteringAngle(Element::Pt);
      mScatter[79] = new NISTMottScatteringAngle(Element::Au);
      mScatter[80] = new NISTMottScatteringAngle(Element::Hg);
      mScatter[81] = new NISTMottScatteringAngle(Element::Tl);
      mScatter[82] = new NISTMottScatteringAngle(Element::Pb);
      mScatter[83] = new NISTMottScatteringAngle(Element::Bi);
      mScatter[84] = new NISTMottScatteringAngle(Element::Po);
      mScatter[85] = new NISTMottScatteringAngle(Element::At);
      mScatter[86] = new NISTMottScatteringAngle(Element::Rn);
      mScatter[87] = new NISTMottScatteringAngle(Element::Fr);
      mScatter[88] = new NISTMottScatteringAngle(Element::Ra);
      mScatter[89] = new NISTMottScatteringAngle(Element::Ac);
      mScatter[90] = new NISTMottScatteringAngle(Element::Th);
      mScatter[91] = new NISTMottScatteringAngle(Element::Pa);
      mScatter[92] = new NISTMottScatteringAngle(Element::U);
      mScatter[93] = new NISTMottScatteringAngle(Element::Np);
      mScatter[94] = new NISTMottScatteringAngle(Element::Pu);
      mScatter[95] = new NISTMottScatteringAngle(Element::Am);
      mScatter[96] = new NISTMottScatteringAngle(Element::Cm);
   }

   const NISTMottScatteringAngle& getNISTMSA(int an)
   {
      return *mScatter[an];
   }

   NISTMottRandomizedScatterFactory::NISTMottRandomizedScatterFactory() : RandomizedScatterFactoryT("NIST Mott Inelastic Cross-Section", REFERENCE)
   {
   }

   void NISTMottRandomizedScatterFactory::initializeDefaultStrategy()
   {
   }

   const RandomizedScatterT& NISTMottRandomizedScatterFactory::get(const ElementT& elm) const
   {
      return getNISTMSA(elm.getAtomicNumber());
   }

   const NISTMottRandomizedScatterFactory NISTMottRandomizedFactory;
   const RandomizedScatterFactoryT& Factory = NISTMottRandomizedFactory;
}
