#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\EPQLibrary\NISTMottScatteringAngle.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Reference.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\EPQLibrary\PhysicalConstants.cuh"
#include "gov\nist\microanalysis\EPQLibrary\FromSI.cuh"
#include "gov\nist\microanalysis\EPQLibrary\ScreenedRutherfordScatteringAngle.cuh"

namespace NISTMottScatteringAngle
{
   static const Reference::Author* auRef[] = { &Reference::CPowell, &Reference::FSalvat, &Reference::AJablonski };
   static const Reference::WebSite REFERENCE("http://www.nist.gov/srd/nist64.htm", "NIST Electron Elastic-Scattering Cross-Section Database version 3.1", "2007 AUGUST 24", auRef, 3);

   const int SPWEM_LEN = 61;
   const int X1_LEN = 201;
   const double DL50 = ::log(50.0);
   const double PARAM = (::log(2.0e4) - DL50) / 60.0;

   const double MAX_NISTMOTT = ToSI::keV(20.0);

   static double value(double a, double b, double c, double y0, double y1, double y2, double x)
   {
      return (x - b) * (x - c) * y0 / ((a - b) * (a - c)) + (x - a) * (x - c) * y1 / ((b - a) * (b - c)) + (x - a) * (x - b) * y2 / ((c - a) * (c - b));
   }

   // https://www.oreilly.com/library/view/c-cookbook/0596007612/ch03s06.html
   static double sciToDub(const std::string& str)
   {
      std::string tmp = str.substr(str.find_first_not_of(" "));
      std::stringstream ss(tmp);
      double d = 0;
      ss >> d;

      if (ss.fail()) {
         std::string s = "Unable to format ";
         s += tmp;
         s += " as a number!";
         throw (s);
      }

      return (d);
   }

   void NISTMottScatteringAngle::loadData(int an)
   {
      std::string name(an < 10 ? ".\\gov\\nist\\microanalysis\\EPQLibrary\\NistXSec/E0" + std::to_string(an) + ".D64" : ".\\gov\\nist\\microanalysis\\EPQLibrary\\NistXSec/E" + std::to_string(an) + ".D64");
      printf("Reading: %s\n", name.c_str());
      try {
         std::ifstream t(name);
         if (!t.good()) throw 0;
         std::string line;
         std::getline(t, line);
         for (int j = 0; j < SPWEM_LEN; ++j) {
            std::getline(t, line);
            mSpwem[j] = sciToDub(line);
            for (int i = 0; i < X1_LEN; ++i) {
               std::getline(t, line);
               mX1[j][i] = sciToDub(line);
            }
         }
      }
      catch (std::exception ex) {
         printf("Unable to construct NISTMottScatteringAngle: %s\n", name.c_str());
      }
   }

   NISTMottScatteringAngle::NISTMottScatteringAngle(const ElementT& elm) : RandomizedScatterT("NIST Elastic cross-section", REFERENCE), mElement(elm), mSpwem(SPWEM_LEN, 0), mX1(SPWEM_LEN, VectorXd(X1_LEN, 0)), mRutherford(*ScreenedRutherfordScatteringAngle::mScatter[elm.getAtomicNumber() - 1])
   {
      loadData(elm.getAtomicNumber());
   }

   NISTMottScatteringAngle::NISTMottScatteringAngle(int an) : RandomizedScatterT("NIST Elastic cross-section", REFERENCE), mElement(Element::byAtomicNumber(an)), mSpwem(SPWEM_LEN, 0), mX1(SPWEM_LEN, VectorXd(X1_LEN, 0)), mRutherford(*ScreenedRutherfordScatteringAngle::mScatter[an - 1])
   {
      loadData(an);
   }

   NISTMottScatteringAngle::NISTMottScatteringAngle(const NISTMottScatteringAngle& other) : RandomizedScatterT("NIST Elastic cross-section", REFERENCE), mElement(other.mElement), mSpwem(other.mSpwem), mX1(other.mX1), mRutherford(other.mRutherford)
   {
   }

   StringT NISTMottScatteringAngle::toString() const
   {
      return "CrossSection[NIST-Mott," + StringT(mElement.toAbbrev()) + "]";
   }

   const ElementT& NISTMottScatteringAngle::getElement() const
   {
      return mElement;
   }
   
   double NISTMottScatteringAngle::totalCrossSection(double energy) const
   {
      if (energy < MAX_NISTMOTT) {
         double scale = PhysicalConstants::BohrRadius * PhysicalConstants::BohrRadius;
         double logE = ::log(FromSI::eV(energy));
         int j = 1 + (int)((logE - DL50) / PARAM);
         if (j == 1)
            return value(DL50, DL50 + PARAM, DL50 + 2.0 * PARAM, mSpwem[0], mSpwem[1], mSpwem[2], logE) * scale;
         else if (j == SPWEM_LEN)
            return value(DL50 + 58.0 * PARAM, DL50 + 59.0 * PARAM, DL50 + 60.0 * PARAM, mSpwem[SPWEM_LEN - 3], mSpwem[SPWEM_LEN - 2], mSpwem[SPWEM_LEN - 1], logE)
            * scale;
         else {
            double e0 = DL50 + (j - 2) * PARAM;
            return value(e0, e0 + PARAM, e0 + 2.0 * PARAM, mSpwem[j - 2], mSpwem[j - 1], mSpwem[j], logE) * scale;
         }
      }
      else {
         return mRutherford.totalCrossSection(energy);
      }
   }

   double NISTMottScatteringAngle::randomScatteringAngle(double energy) const
   {
      if (energy < MAX_NISTMOTT) {
         double logE = ::log(FromSI::eV(energy));
         int j = (int)((logE - DL50) / PARAM); // offset to zero-based
         double e2 = DL50 + (j + 1) * PARAM;
         double e1 = e2 - PARAM;
         int i = (logE - e1 < e2 - logE ? j : j + 1); // offset to
         // zero-based
         if (!((i >= 0) && (i < SPWEM_LEN))) printf("%s\n", StringT(std::to_string(i) + "\t" + std::to_string(FromSI::eV(energy)) + "\t" + std::to_string(e1) + "\t" + std::to_string(e2)).c_str());
         // via j
         int k = (int)(200.0 * Math2::random()); // offset to
         // zero-based
         double x = (mX1[i][k + 1] - mX1[i][k]) * Math2::random();
         double q = mX1[i][k] + x;
         double com = 1.0 - 2.0 * q * q;
         return com > -1.0 ? (com < 1.0 ? ::acos(com) : 0.0) : PhysicalConstants::PI;
      }
      else {
         return mRutherford.randomScatteringAngle(energy);
      }
   }

   //const NISTMottScatteringAngle NISTMSA1(Element::H);
   //const NISTMottScatteringAngle NISTMSA2(Element::He);
   //const NISTMottScatteringAngle NISTMSA3(Element::Li);
   //const NISTMottScatteringAngle NISTMSA4(Element::Be);
   //const NISTMottScatteringAngle NISTMSA5(Element::B);
   //const NISTMottScatteringAngle NISTMSA6(Element::C);
   //const NISTMottScatteringAngle NISTMSA7(Element::N);
   //const NISTMottScatteringAngle NISTMSA8(Element::O);
   //const NISTMottScatteringAngle NISTMSA9(Element::F);
   //const NISTMottScatteringAngle NISTMSA10(Element::Ne);
   //const NISTMottScatteringAngle NISTMSA11(Element::Na);
   //const NISTMottScatteringAngle NISTMSA12(Element::Mg);
   //const NISTMottScatteringAngle NISTMSA13(Element::Al);
   //const NISTMottScatteringAngle NISTMSA14(Element::Si);
   //const NISTMottScatteringAngle NISTMSA15(Element::P);
   //const NISTMottScatteringAngle NISTMSA16(Element::S);
   //const NISTMottScatteringAngle NISTMSA17(Element::Cl);
   //const NISTMottScatteringAngle NISTMSA18(Element::Ar);
   //const NISTMottScatteringAngle NISTMSA19(Element::K);
   //const NISTMottScatteringAngle NISTMSA20(Element::Ca);
   //const NISTMottScatteringAngle NISTMSA21(Element::Sc);
   //const NISTMottScatteringAngle NISTMSA22(Element::Ti);
   //const NISTMottScatteringAngle NISTMSA23(Element::V);
   //const NISTMottScatteringAngle NISTMSA24(Element::Cr);
   //const NISTMottScatteringAngle NISTMSA25(Element::Mn);
   //const NISTMottScatteringAngle NISTMSA26(Element::Fe);
   //const NISTMottScatteringAngle NISTMSA27(Element::Co);
   //const NISTMottScatteringAngle NISTMSA28(Element::Ni);
   //const NISTMottScatteringAngle NISTMSA29(Element::Cu);
   //const NISTMottScatteringAngle NISTMSA30(Element::Zn);
   //const NISTMottScatteringAngle NISTMSA31(Element::Ga);
   //const NISTMottScatteringAngle NISTMSA32(Element::Ge);
   //const NISTMottScatteringAngle NISTMSA33(Element::As);
   //const NISTMottScatteringAngle NISTMSA34(Element::Se);
   //const NISTMottScatteringAngle NISTMSA35(Element::Br);
   //const NISTMottScatteringAngle NISTMSA36(Element::Kr);
   //const NISTMottScatteringAngle NISTMSA37(Element::Rb);
   //const NISTMottScatteringAngle NISTMSA38(Element::Sr);
   //const NISTMottScatteringAngle NISTMSA39(Element::Y);
   //const NISTMottScatteringAngle NISTMSA40(Element::Zr);
   //const NISTMottScatteringAngle NISTMSA41(Element::Nb);
   //const NISTMottScatteringAngle NISTMSA42(Element::Mo);
   //const NISTMottScatteringAngle NISTMSA43(Element::Tc);
   //const NISTMottScatteringAngle NISTMSA44(Element::Ru);
   //const NISTMottScatteringAngle NISTMSA45(Element::Rh);
   //const NISTMottScatteringAngle NISTMSA46(Element::Pd);
   //const NISTMottScatteringAngle NISTMSA47(Element::Ag);
   //const NISTMottScatteringAngle NISTMSA48(Element::Cd);
   //const NISTMottScatteringAngle NISTMSA49(Element::In);
   //const NISTMottScatteringAngle NISTMSA50(Element::Sn);
   //const NISTMottScatteringAngle NISTMSA51(Element::Sb);
   //const NISTMottScatteringAngle NISTMSA52(Element::Te);
   //const NISTMottScatteringAngle NISTMSA53(Element::I);
   //const NISTMottScatteringAngle NISTMSA54(Element::Xe);
   //const NISTMottScatteringAngle NISTMSA55(Element::Cs);
   //const NISTMottScatteringAngle NISTMSA56(Element::Ba);
   //const NISTMottScatteringAngle NISTMSA57(Element::La);
   //const NISTMottScatteringAngle NISTMSA58(Element::Ce);
   //const NISTMottScatteringAngle NISTMSA59(Element::Pr);
   //const NISTMottScatteringAngle NISTMSA60(Element::Nd);
   //const NISTMottScatteringAngle NISTMSA61(Element::Pm);
   //const NISTMottScatteringAngle NISTMSA62(Element::Sm);
   //const NISTMottScatteringAngle NISTMSA63(Element::Eu);
   //const NISTMottScatteringAngle NISTMSA64(Element::Gd);
   //const NISTMottScatteringAngle NISTMSA65(Element::Tb);
   //const NISTMottScatteringAngle NISTMSA66(Element::Dy);
   //const NISTMottScatteringAngle NISTMSA67(Element::Ho);
   //const NISTMottScatteringAngle NISTMSA68(Element::Er);
   //const NISTMottScatteringAngle NISTMSA69(Element::Tm);
   //const NISTMottScatteringAngle NISTMSA70(Element::Yb);
   //const NISTMottScatteringAngle NISTMSA71(Element::Lu);
   //const NISTMottScatteringAngle NISTMSA72(Element::Hf);
   //const NISTMottScatteringAngle NISTMSA73(Element::Ta);
   //const NISTMottScatteringAngle NISTMSA74(Element::W);
   //const NISTMottScatteringAngle NISTMSA75(Element::Re);
   //const NISTMottScatteringAngle NISTMSA76(Element::Os);
   //const NISTMottScatteringAngle NISTMSA77(Element::Ir);
   //const NISTMottScatteringAngle NISTMSA78(Element::Pt);
   //const NISTMottScatteringAngle NISTMSA79(Element::Au);
   //const NISTMottScatteringAngle NISTMSA80(Element::Hg);
   //const NISTMottScatteringAngle NISTMSA81(Element::Tl);
   //const NISTMottScatteringAngle NISTMSA82(Element::Pb);
   //const NISTMottScatteringAngle NISTMSA83(Element::Bi);
   //const NISTMottScatteringAngle NISTMSA84(Element::Po);
   //const NISTMottScatteringAngle NISTMSA85(Element::At);
   //const NISTMottScatteringAngle NISTMSA86(Element::Rn);
   //const NISTMottScatteringAngle NISTMSA87(Element::Fr);
   //const NISTMottScatteringAngle NISTMSA88(Element::Ra);
   //const NISTMottScatteringAngle NISTMSA89(Element::Ac);
   //const NISTMottScatteringAngle NISTMSA90(Element::Th);
   //const NISTMottScatteringAngle NISTMSA91(Element::Pa);
   //const NISTMottScatteringAngle NISTMSA92(Element::U);
   //const NISTMottScatteringAngle NISTMSA93(Element::Np);
   //const NISTMottScatteringAngle NISTMSA94(Element::Pu);

   const NISTMottScatteringAngle NISTMSA1(1);
   const NISTMottScatteringAngle NISTMSA2(2);
   const NISTMottScatteringAngle NISTMSA3(3);
   const NISTMottScatteringAngle NISTMSA4(4);
   const NISTMottScatteringAngle NISTMSA5(5);
   const NISTMottScatteringAngle NISTMSA6(6);
   const NISTMottScatteringAngle NISTMSA7(7);
   const NISTMottScatteringAngle NISTMSA8(8);
   const NISTMottScatteringAngle NISTMSA9(9);
   const NISTMottScatteringAngle NISTMSA10(10);
   const NISTMottScatteringAngle NISTMSA11(11);
   const NISTMottScatteringAngle NISTMSA12(12);
   const NISTMottScatteringAngle NISTMSA13(13);
   const NISTMottScatteringAngle NISTMSA14(14);
   const NISTMottScatteringAngle NISTMSA15(15);
   const NISTMottScatteringAngle NISTMSA16(16);
   const NISTMottScatteringAngle NISTMSA17(17);
   const NISTMottScatteringAngle NISTMSA18(18);
   const NISTMottScatteringAngle NISTMSA19(19);
   const NISTMottScatteringAngle NISTMSA20(20);
   const NISTMottScatteringAngle NISTMSA21(21);
   const NISTMottScatteringAngle NISTMSA22(22);
   const NISTMottScatteringAngle NISTMSA23(23);
   const NISTMottScatteringAngle NISTMSA24(24);
   const NISTMottScatteringAngle NISTMSA25(25);
   const NISTMottScatteringAngle NISTMSA26(26);
   const NISTMottScatteringAngle NISTMSA27(27);
   const NISTMottScatteringAngle NISTMSA28(28);
   const NISTMottScatteringAngle NISTMSA29(29);
   const NISTMottScatteringAngle NISTMSA30(30);
   const NISTMottScatteringAngle NISTMSA31(31);
   const NISTMottScatteringAngle NISTMSA32(32);
   const NISTMottScatteringAngle NISTMSA33(33);
   const NISTMottScatteringAngle NISTMSA34(34);
   const NISTMottScatteringAngle NISTMSA35(35);
   const NISTMottScatteringAngle NISTMSA36(36);
   const NISTMottScatteringAngle NISTMSA37(37);
   const NISTMottScatteringAngle NISTMSA38(38);
   const NISTMottScatteringAngle NISTMSA39(39);
   const NISTMottScatteringAngle NISTMSA40(40);
   const NISTMottScatteringAngle NISTMSA41(41);
   const NISTMottScatteringAngle NISTMSA42(42);
   const NISTMottScatteringAngle NISTMSA43(43);
   const NISTMottScatteringAngle NISTMSA44(44);
   const NISTMottScatteringAngle NISTMSA45(45);
   const NISTMottScatteringAngle NISTMSA46(46);
   const NISTMottScatteringAngle NISTMSA47(47);
   const NISTMottScatteringAngle NISTMSA48(48);
   const NISTMottScatteringAngle NISTMSA49(49);
   const NISTMottScatteringAngle NISTMSA50(50);
   const NISTMottScatteringAngle NISTMSA51(51);
   const NISTMottScatteringAngle NISTMSA52(52);
   const NISTMottScatteringAngle NISTMSA53(53);
   const NISTMottScatteringAngle NISTMSA54(54);
   const NISTMottScatteringAngle NISTMSA55(55);
   const NISTMottScatteringAngle NISTMSA56(56);
   const NISTMottScatteringAngle NISTMSA57(57);
   const NISTMottScatteringAngle NISTMSA58(58);
   const NISTMottScatteringAngle NISTMSA59(59);
   const NISTMottScatteringAngle NISTMSA60(60);
   const NISTMottScatteringAngle NISTMSA61(61);
   const NISTMottScatteringAngle NISTMSA62(62);
   const NISTMottScatteringAngle NISTMSA63(63);
   const NISTMottScatteringAngle NISTMSA64(64);
   const NISTMottScatteringAngle NISTMSA65(65);
   const NISTMottScatteringAngle NISTMSA66(66);
   const NISTMottScatteringAngle NISTMSA67(67);
   const NISTMottScatteringAngle NISTMSA68(68);
   const NISTMottScatteringAngle NISTMSA69(69);
   const NISTMottScatteringAngle NISTMSA70(70);
   const NISTMottScatteringAngle NISTMSA71(71);
   const NISTMottScatteringAngle NISTMSA72(72);
   const NISTMottScatteringAngle NISTMSA73(73);
   const NISTMottScatteringAngle NISTMSA74(74);
   const NISTMottScatteringAngle NISTMSA75(75);
   const NISTMottScatteringAngle NISTMSA76(76);
   const NISTMottScatteringAngle NISTMSA77(77);
   const NISTMottScatteringAngle NISTMSA78(78);
   const NISTMottScatteringAngle NISTMSA79(79);
   const NISTMottScatteringAngle NISTMSA80(80);
   const NISTMottScatteringAngle NISTMSA81(81);
   const NISTMottScatteringAngle NISTMSA82(82);
   const NISTMottScatteringAngle NISTMSA83(83);
   const NISTMottScatteringAngle NISTMSA84(84);
   const NISTMottScatteringAngle NISTMSA85(85);
   const NISTMottScatteringAngle NISTMSA86(86);
   const NISTMottScatteringAngle NISTMSA87(87);
   const NISTMottScatteringAngle NISTMSA88(88);
   const NISTMottScatteringAngle NISTMSA89(89);
   const NISTMottScatteringAngle NISTMSA90(90);
   const NISTMottScatteringAngle NISTMSA91(91);
   const NISTMottScatteringAngle NISTMSA92(92);
   const NISTMottScatteringAngle NISTMSA93(93);
   const NISTMottScatteringAngle NISTMSA94(94);

   const NISTMottScatteringAngle* mScatter[113] = {
      &NISTMSA1,
      &NISTMSA2,
      &NISTMSA3,
      &NISTMSA4,
      &NISTMSA5,
      &NISTMSA6,
      &NISTMSA7,
      &NISTMSA8,
      &NISTMSA9,
      &NISTMSA10,
      &NISTMSA11,
      &NISTMSA12,
      &NISTMSA13,
      &NISTMSA14,
      &NISTMSA15,
      &NISTMSA16,
      &NISTMSA17,
      &NISTMSA18,
      &NISTMSA19,
      &NISTMSA20,
      &NISTMSA21,
      &NISTMSA22,
      &NISTMSA23,
      &NISTMSA24,
      &NISTMSA25,
      &NISTMSA26,
      &NISTMSA27,
      &NISTMSA28,
      &NISTMSA29,
      &NISTMSA30,
      &NISTMSA31,
      &NISTMSA32,
      &NISTMSA33,
      &NISTMSA34,
      &NISTMSA35,
      &NISTMSA36,
      &NISTMSA37,
      &NISTMSA38,
      &NISTMSA39,
      &NISTMSA40,
      &NISTMSA41,
      &NISTMSA42,
      &NISTMSA43,
      &NISTMSA44,
      &NISTMSA45,
      &NISTMSA46,
      &NISTMSA47,
      &NISTMSA48,
      &NISTMSA49,
      &NISTMSA50,
      &NISTMSA51,
      &NISTMSA52,
      &NISTMSA53,
      &NISTMSA54,
      &NISTMSA55,
      &NISTMSA56,
      &NISTMSA57,
      &NISTMSA58,
      &NISTMSA59,
      &NISTMSA60,
      &NISTMSA61,
      &NISTMSA62,
      &NISTMSA63,
      &NISTMSA64,
      &NISTMSA65,
      &NISTMSA66,
      &NISTMSA67,
      &NISTMSA68,
      &NISTMSA69,
      &NISTMSA70,
      &NISTMSA71,
      &NISTMSA72,
      &NISTMSA73,
      &NISTMSA74,
      &NISTMSA75,
      &NISTMSA76,
      &NISTMSA77,
      &NISTMSA78,
      &NISTMSA79,
      &NISTMSA80,
      &NISTMSA81,
      &NISTMSA82,
      &NISTMSA83,
      &NISTMSA84,
      &NISTMSA85,
      &NISTMSA86,
      &NISTMSA87,
      &NISTMSA88,
      &NISTMSA89,
      &NISTMSA90,
      &NISTMSA91,
      &NISTMSA92,
      &NISTMSA93,
      &NISTMSA94
   };

   NISTMottRandomizedScatterFactory::NISTMottRandomizedScatterFactory() : RandomizedScatterFactoryT("NIST Mott Inelastic Cross-Section", REFERENCE)
   {
   }

   void NISTMottRandomizedScatterFactory::initializeDefaultStrategy()
   {
   }

   const RandomizedScatterT& NISTMottRandomizedScatterFactory::get(const ElementT& elm) const
   {
      return *mScatter[elm.getAtomicNumber()];
   }

   const NISTMottRandomizedScatterFactory NISTMottRandomizedFactory;
   const RandomizedScatterFactoryT& Factory = NISTMottRandomizedFactory;
}
