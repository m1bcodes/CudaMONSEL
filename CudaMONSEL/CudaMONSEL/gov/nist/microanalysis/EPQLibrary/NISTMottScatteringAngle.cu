#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\EPQLibrary\NISTMottScatteringAngle.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Reference.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\EPQLibrary\PhysicalConstants.cuh"
#include "gov\nist\microanalysis\EPQLibrary\FromSI.cuh"
#include "gov\nist\microanalysis\EPQLibrary\ScreenedRutherfordScatteringAngle.cuh"

namespace NISTMottScatteringAngle
{
   static const Reference::Author auRef[] = { Reference::CPowell, Reference::FSalvat, Reference::AJablonski };
   static const Reference::WebSite REFERENCE("http://www.nist.gov/srd/nist64.htm", "NIST Electron Elastic-Scattering Cross-Section Database version 3.1", "2007 AUGUST 24", auRef, 3);

   const int SPWEM_LEN = 61;
   const int X1_LEN = 201;
   const double DL50 = ::log(50.0);
   const double PARAM = (::log(2.0e4) - DL50) / 60.0;

   const double MAX_NISTMOTT = ToSI::keV(20.0);

   static double value(double a, double b, double c, double y0, double y1, double y2, double x)
   {
      return (x - b) * (x - c) * y0 / ((a - b) * (a - c)) + (x - a) * (x - c) * y1 / ((b - a) * (b - c)) + (x - a) * (x - b)
         * y2 / ((c - a) * (c - b));
   }


   // https://www.oreilly.com/library/view/c-cookbook/0596007612/ch03s06.html
   static double sciToDub(const std::string& str)
   {
      std::string tmp = str.substr(str.find_first_not_of(" "));
      std::stringstream ss(tmp);
      double d = 0;
      ss >> d;

      if (ss.fail()) {
         std::string s = "Unable to format ";
         s += tmp;
         s += " as a number!";
         throw (s);
      }

      return (d);
   }

   NISTMottScatteringAngle::NISTMottScatteringAngle(const ElementT& elm) : RandomizedScatterT("NIST Elastic cross-section", REFERENCE), mElement(elm), mSpwem(SPWEM_LEN, 0), mX1(SPWEM_LEN, VectorXd(X1_LEN, 0)), mRutherford(*ScreenedRutherfordScatteringAngle::mScatter[elm.getAtomicNumber()])
   {
      try {
         std::string name = elm.getAtomicNumber() < 10 ? "NistXSec/E0" + std::to_string(elm.getAtomicNumber()) + ".D64" : "NistXSec/E" + std::to_string(elm.getAtomicNumber()) + ".D64";
         std::ifstream t(name);
         std::string line;
         std::getline(t, line);
         for (int j = 0; j < SPWEM_LEN; ++j) {
            std::getline(t, line);
            mSpwem[j] = sciToDub(line);
            for (int i = 0; i < X1_LEN; ++i) {
               std::getline(t, line);
               mX1[j][i] = sciToDub(line);
            }
         }
      }
      catch (std::exception ex) {
         printf("Unable to construct NISTMottScatteringAngle: ");
      }
   }

   StringT NISTMottScatteringAngle::toString() const
   {
      return "CrossSection[NIST-Mott," + StringT(mElement.toAbbrev()) + "]";
   }

   const ElementT& NISTMottScatteringAngle::getElement() const
   {
      return mElement;
   }

   
   double NISTMottScatteringAngle::totalCrossSection(double energy) const
   {
      if (energy < MAX_NISTMOTT) {
         double scale = PhysicalConstants::BohrRadius * PhysicalConstants::BohrRadius;
         double logE = ::log(FromSI::eV(energy));
         int j = 1 + (int)((logE - DL50) / PARAM);
         if (j == 1)
            return value(DL50, DL50 + PARAM, DL50 + 2.0 * PARAM, mSpwem[0], mSpwem[1], mSpwem[2], logE) * scale;
         else if (j == SPWEM_LEN)
            return value(DL50 + 58.0 * PARAM, DL50 + 59.0 * PARAM, DL50 + 60.0 * PARAM, mSpwem[SPWEM_LEN - 3], mSpwem[SPWEM_LEN - 2], mSpwem[SPWEM_LEN - 1], logE)
            * scale;
         else {
            double e0 = DL50 + (j - 2) * PARAM;
            return value(e0, e0 + PARAM, e0 + 2.0 * PARAM, mSpwem[j - 2], mSpwem[j - 1], mSpwem[j], logE) * scale;
         }
      }
      else {
         return mRutherford.totalCrossSection(energy);
      }
   }

   double NISTMottScatteringAngle::randomScatteringAngle(double energy) const
   {
      if (energy < MAX_NISTMOTT) {
         double logE = ::log(FromSI::eV(energy));
         int j = (int)((logE - DL50) / PARAM); // offset to zero-based
         double e2 = DL50 + (j + 1) * PARAM;
         double e1 = e2 - PARAM;
         int i = (logE - e1 < e2 - logE ? j : j + 1); // offset to
         // zero-based
         if (!((i >= 0) && (i < SPWEM_LEN))) printf("%s\n", std::to_string(i) + "\t" + std::to_string(FromSI::eV(energy)) + "\t" + std::to_string(e1) + "\t" + std::to_string(e2));
         // via j
         int k = (int)(200.0 * Math2::random()); // offset to
         // zero-based
         double x = (mX1[i][k + 1] - mX1[i][k]) * Math2::random();
         double q = mX1[i][k] + x;
         double com = 1.0 - 2.0 * q * q;
         return com > -1.0 ? (com < 1.0 ? ::acos(com) : 0.0) : PhysicalConstants::PI;
      }
      else {
         return mRutherford.randomScatteringAngle(energy);
      }
   }

   const NISTMottScatteringAngle NISTMSA1(Element::H);
   const NISTMottScatteringAngle NISTMSA2(Element::He);
   const NISTMottScatteringAngle NISTMSA3(Element::Li);
   const NISTMottScatteringAngle NISTMSA4(Element::Be);
   const NISTMottScatteringAngle NISTMSA5(Element::B);
   const NISTMottScatteringAngle NISTMSA6(Element::C);
   const NISTMottScatteringAngle NISTMSA7(Element::N);
   const NISTMottScatteringAngle NISTMSA8(Element::O);
   const NISTMottScatteringAngle NISTMSA9(Element::F);
   const NISTMottScatteringAngle NISTMSA10(Element::Ne);
   const NISTMottScatteringAngle NISTMSA11(Element::Na);
   const NISTMottScatteringAngle NISTMSA12(Element::Mg);
   const NISTMottScatteringAngle NISTMSA13(Element::Al);
   const NISTMottScatteringAngle NISTMSA14(Element::Si);
   const NISTMottScatteringAngle NISTMSA15(Element::P);
   const NISTMottScatteringAngle NISTMSA16(Element::S);
   const NISTMottScatteringAngle NISTMSA17(Element::Cl);
   const NISTMottScatteringAngle NISTMSA18(Element::Ar);
   const NISTMottScatteringAngle NISTMSA19(Element::K);
   const NISTMottScatteringAngle NISTMSA20(Element::Ca);
   const NISTMottScatteringAngle NISTMSA21(Element::Sc);
   const NISTMottScatteringAngle NISTMSA22(Element::Ti);
   const NISTMottScatteringAngle NISTMSA23(Element::V);
   const NISTMottScatteringAngle NISTMSA24(Element::Cr);
   const NISTMottScatteringAngle NISTMSA25(Element::Mn);
   const NISTMottScatteringAngle NISTMSA26(Element::Fe);
   const NISTMottScatteringAngle NISTMSA27(Element::Co);
   const NISTMottScatteringAngle NISTMSA28(Element::Ni);
   const NISTMottScatteringAngle NISTMSA29(Element::Cu);
   const NISTMottScatteringAngle NISTMSA30(Element::Zn);
   const NISTMottScatteringAngle NISTMSA31(Element::Ga);
   const NISTMottScatteringAngle NISTMSA32(Element::Ge);
   const NISTMottScatteringAngle NISTMSA33(Element::As);
   const NISTMottScatteringAngle NISTMSA34(Element::Se);
   const NISTMottScatteringAngle NISTMSA35(Element::Br);
   const NISTMottScatteringAngle NISTMSA36(Element::Kr);
   const NISTMottScatteringAngle NISTMSA37(Element::Rb);
   const NISTMottScatteringAngle NISTMSA38(Element::Sr);
   const NISTMottScatteringAngle NISTMSA39(Element::Y);
   const NISTMottScatteringAngle NISTMSA40(Element::Zr);
   const NISTMottScatteringAngle NISTMSA41(Element::Nb);
   const NISTMottScatteringAngle NISTMSA42(Element::Mo);
   const NISTMottScatteringAngle NISTMSA43(Element::Tc);
   const NISTMottScatteringAngle NISTMSA44(Element::Ru);
   const NISTMottScatteringAngle NISTMSA45(Element::Rh);
   const NISTMottScatteringAngle NISTMSA46(Element::Pd);
   const NISTMottScatteringAngle NISTMSA47(Element::Ag);
   const NISTMottScatteringAngle NISTMSA48(Element::Cd);
   const NISTMottScatteringAngle NISTMSA49(Element::In);
   const NISTMottScatteringAngle NISTMSA50(Element::Sn);
   const NISTMottScatteringAngle NISTMSA51(Element::Sb);
   const NISTMottScatteringAngle NISTMSA52(Element::Te);
   const NISTMottScatteringAngle NISTMSA53(Element::I);
   const NISTMottScatteringAngle NISTMSA54(Element::Xe);
   const NISTMottScatteringAngle NISTMSA55(Element::Cs);
   const NISTMottScatteringAngle NISTMSA56(Element::Ba);
   const NISTMottScatteringAngle NISTMSA57(Element::La);
   const NISTMottScatteringAngle NISTMSA58(Element::Ce);
   const NISTMottScatteringAngle NISTMSA59(Element::Pr);
   const NISTMottScatteringAngle NISTMSA60(Element::Nd);
   const NISTMottScatteringAngle NISTMSA61(Element::Pm);
   const NISTMottScatteringAngle NISTMSA62(Element::Sm);
   const NISTMottScatteringAngle NISTMSA63(Element::Eu);
   const NISTMottScatteringAngle NISTMSA64(Element::Gd);
   const NISTMottScatteringAngle NISTMSA65(Element::Tb);
   const NISTMottScatteringAngle NISTMSA66(Element::Dy);
   const NISTMottScatteringAngle NISTMSA67(Element::Ho);
   const NISTMottScatteringAngle NISTMSA68(Element::Er);
   const NISTMottScatteringAngle NISTMSA69(Element::Tm);
   const NISTMottScatteringAngle NISTMSA70(Element::Yb);
   const NISTMottScatteringAngle NISTMSA71(Element::Lu);
   const NISTMottScatteringAngle NISTMSA72(Element::Hf);
   const NISTMottScatteringAngle NISTMSA73(Element::Ta);
   const NISTMottScatteringAngle NISTMSA74(Element::W);
   const NISTMottScatteringAngle NISTMSA75(Element::Re);
   const NISTMottScatteringAngle NISTMSA76(Element::Os);
   const NISTMottScatteringAngle NISTMSA77(Element::Ir);
   const NISTMottScatteringAngle NISTMSA78(Element::Pt);
   const NISTMottScatteringAngle NISTMSA79(Element::Au);
   const NISTMottScatteringAngle NISTMSA80(Element::Hg);
   const NISTMottScatteringAngle NISTMSA81(Element::Tl);
   const NISTMottScatteringAngle NISTMSA82(Element::Pb);
   const NISTMottScatteringAngle NISTMSA83(Element::Bi);
   const NISTMottScatteringAngle NISTMSA84(Element::Po);
   const NISTMottScatteringAngle NISTMSA85(Element::At);
   const NISTMottScatteringAngle NISTMSA86(Element::Rn);
   const NISTMottScatteringAngle NISTMSA87(Element::Fr);
   const NISTMottScatteringAngle NISTMSA88(Element::Ra);
   const NISTMottScatteringAngle NISTMSA89(Element::Ac);
   const NISTMottScatteringAngle NISTMSA90(Element::Th);
   const NISTMottScatteringAngle NISTMSA91(Element::Pa);
   const NISTMottScatteringAngle NISTMSA92(Element::U);
   const NISTMottScatteringAngle NISTMSA93(Element::Np);
   const NISTMottScatteringAngle NISTMSA94(Element::Pu);

   const NISTMottScatteringAngle* mScatter[113] = {
      &NISTMSA1,
      &NISTMSA2,
      &NISTMSA3,
      &NISTMSA4,
      &NISTMSA5,
      &NISTMSA6,
      &NISTMSA7,
      &NISTMSA8,
      &NISTMSA9,
      &NISTMSA10,
      &NISTMSA11,
      &NISTMSA12,
      &NISTMSA13,
      &NISTMSA14,
      &NISTMSA15,
      &NISTMSA16,
      &NISTMSA17,
      &NISTMSA18,
      &NISTMSA19,
      &NISTMSA20,
      &NISTMSA21,
      &NISTMSA22,
      &NISTMSA23,
      &NISTMSA24,
      &NISTMSA25,
      &NISTMSA26,
      &NISTMSA27,
      &NISTMSA28,
      &NISTMSA29,
      &NISTMSA30,
      &NISTMSA31,
      &NISTMSA32,
      &NISTMSA33,
      &NISTMSA34,
      &NISTMSA35,
      &NISTMSA36,
      &NISTMSA37,
      &NISTMSA38,
      &NISTMSA39,
      &NISTMSA40,
      &NISTMSA41,
      &NISTMSA42,
      &NISTMSA43,
      &NISTMSA44,
      &NISTMSA45,
      &NISTMSA46,
      &NISTMSA47,
      &NISTMSA48,
      &NISTMSA49,
      &NISTMSA50,
      &NISTMSA51,
      &NISTMSA52,
      &NISTMSA53,
      &NISTMSA54,
      &NISTMSA55,
      &NISTMSA56,
      &NISTMSA57,
      &NISTMSA58,
      &NISTMSA59,
      &NISTMSA60,
      &NISTMSA61,
      &NISTMSA62,
      &NISTMSA63,
      &NISTMSA64,
      &NISTMSA65,
      &NISTMSA66,
      &NISTMSA67,
      &NISTMSA68,
      &NISTMSA69,
      &NISTMSA70,
      &NISTMSA71,
      &NISTMSA72,
      &NISTMSA73,
      &NISTMSA74,
      &NISTMSA75,
      &NISTMSA76,
      &NISTMSA77,
      &NISTMSA78,
      &NISTMSA79,
      &NISTMSA80,
      &NISTMSA81,
      &NISTMSA82,
      &NISTMSA83,
      &NISTMSA84,
      &NISTMSA85,
      &NISTMSA86,
      &NISTMSA87,
      &NISTMSA88,
      &NISTMSA89,
      &NISTMSA90,
      &NISTMSA91,
      &NISTMSA92,
      &NISTMSA93,
      &NISTMSA94
   };

   NISTMottRandomizedScatterFactory::NISTMottRandomizedScatterFactory() : RandomizedScatterFactoryT("NIST Mott Inelastic Cross-Section", REFERENCE)
   {
   }

   void NISTMottRandomizedScatterFactory::initializeDefaultStrategy()
   {
   }

   const RandomizedScatterT& NISTMottRandomizedScatterFactory::get(const ElementT& elm) const
   {
      return *mScatter[elm.getAtomicNumber()];
   }

   const NISTMottRandomizedScatterFactory NISTMottRandomizedFactory;
   const RandomizedScatterFactoryT& Factory = NISTMottRandomizedFactory;
}
