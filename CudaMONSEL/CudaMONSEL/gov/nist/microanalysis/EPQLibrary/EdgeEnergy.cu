
#include <hip/hip_runtime.h>
//#include "gov\nist\microanalysis\Utility\CSVReader.h"
//#include "gov\nist\microanalysis\Utility\Math2.cuh"
//
///**
//* <p>
//* Various different implementations of classes that return the edge energy for
//* a specified AtomicShell.
//* </p>
//* <p>
//* Copyright: Pursuant to title 17 Section 105 of the United States Code this
//* software is not subject to copyright protection and is in the public domain
//* </p>
//* <p>
//* Company: National Institute of Standards and Technology
//* </p>
//*
//* @author Nicholas W. M. Ritchie
//* @version 1.0
//*/
//
//namespace EdgeEnergy
//{
//
//   @Override
//      protected void initializeDefaultStrategy() {
//      // Don't do anything...
//   }
//
//   protected EdgeEnergy(String name, String ref) {
//      super("Edge Energy", name, ref);
//   }
//
//   protected EdgeEnergy(String name, Reference ref) {
//      super("Edge Energy", name, ref);
//   }
//
//   /**
//   * getAllImplementations - Returns a full list of all available algorithms.
//   * Each item is an implements the EdgeEnergy class.
//   *
//   * @return List
//   */
//   @Override
//      public List<AlgorithmClass> getAllImplementations() {
//      return Arrays.asList(mAllImplementations);
//   }
//
//   /**
//   * Returns the edge energy associated with the specified atomic shell (in
//   * Joules). Returns zero for shells not associated with edges.
//   *
//   * @param shell AtomicShell
//   * @return double
//   */
//   abstract public double compute(AtomicShell shell);
//
//   /**
//   * Returns the edge energy (in Joules) associated with the specified
//   * tranition.
//   *
//   * @param xrt XRayTransition
//   * @return double
//   */
//   public double compute(XRayTransition xrt) {
//      return compute(xrt.getDestination());
//   }
//
//   /**
//   * supports - Does this particular implementation provide a non-zero edge
//   * energy for the specified transition?
//   *
//   * @param shell AtomicShell
//   * @return boolean
//   */
//   abstract public boolean isSupported(AtomicShell shell);
//
//   public static class DiracHartreeSlaterIonizationEnergies
//      extends EdgeEnergy {
//
//      static double[][] mUis; // nominally [100][9]
//
//      public DiracHartreeSlaterIonizationEnergies() {
//         super("Bote-Salvat 2008", new Reference.JournalArticle(Reference.PhysRevA, "77", "042701-1 to 24", 2008, new Reference.Author[] {
//            new Reference.Author("David", "Bote", "Facultat de Física (ECM), Universitat de Barcelona, Diagonal 647, 08028 Barcelona, Spain"),
//               Reference.FSalvat
//         }));
//      }
//
//      private void initialize() {
//         synchronized(DiracHartreeSlaterIonizationEnergies.class) {
//            if (mUis == null) {
//               mUis = new double[100][];
//               final double[][] uisTmp = (new CSVReader.ResourceReader("SalvatXion/xionUis.csv", false)).getResource(EdgeEnergy.class);
//               assert uisTmp.length == 99;
//               for (int r = 0; r < uisTmp.length; ++r) {
//                  assert Math.round(uisTmp[r][0]) == r + 1;
//                  mUis[r + 1] = Math2.slice(uisTmp[r], 1, uisTmp[r].length - 1);
//               }
//            }
//         }
//      }
//
//      /**
//      * @see gov.nist.microanalysis.EPQLibrary.EdgeEnergy#compute(gov.nist.microanalysis.EPQLibrary.AtomicShell)
//      */
//      @Override
//         public double compute(AtomicShell shell) {
//         if (mUis == null)
//            initialize();
//         return ToSI.eV(mUis[shell.getElement().getAtomicNumber()][shell.getShell()]);
//      }
//
//      /**
//      * @see gov.nist.microanalysis.EPQLibrary.EdgeEnergy#isSupported(gov.nist.microanalysis.EPQLibrary.AtomicShell)
//      */
//      @Override
//         public boolean isSupported(AtomicShell shell) {
//         if (mUis == null)
//            initialize();
//         final int z = shell.getElement().getAtomicNumber();
//         final int sh = shell.getShell();
//         return (z >= 1) && (z <= 99) && (sh < mUis[z].length);
//      }
//   };
//
//   public static final EdgeEnergy DHSIonizationEnergy = new DiracHartreeSlaterIonizationEnergies();
//
//   /**
//   * NISTxrtdb - The NIST x-ray transition database provides edge energies for
//   * the K and L shells for atomic numbers from 10 to 100.
//   */
//   public static class NISTEdgeEnergy
//      extends EdgeEnergy {
//      NISTEdgeEnergy() {
//         super("NIST X-ray transition database", "http://physics.nist.gov/PhysRefData/XrayTrans/");
//      }
//
//      private double[][] mEnergies;
//
//      @Override
//         public double compute(AtomicShell shell) {
//         if (mEnergies == null)
//            mEnergies = (new CSVReader.ResourceReader("NISTxrtdb.csv", false)).getResource(EdgeEnergy.class);
//         final int an = shell.getElement().getAtomicNumber();
//         if ((an < Element.elmNe) || (an > Element.elmFm))
//            throw new IllegalArgumentException(toString() + " only supports elements Ne (10) to Fm (100)");
//         final int sh = shell.getShell();
//         if ((sh < AtomicShell.K) || (sh > AtomicShell.LIII))
//            throw new IllegalArgumentException(toString() + " only supports shells K, L1, L2 and L3");
//         return ToSI.eV(mEnergies[an - Element.elmNe][sh - AtomicShell.K]);
//      }
//
//      @Override
//         public boolean isSupported(AtomicShell shell) {
//         if (mEnergies == null)
//            mEnergies = (new CSVReader.ResourceReader("NISTxrtdb.csv", false)).getResource(EdgeEnergy.class);
//         final int an = shell.getElement().getAtomicNumber();
//         final int sh = shell.getShell();
//         return (an >= Element.elmNe) && (an <= Element.elmFm) && (sh >= AtomicShell.K) && (sh <= AtomicShell.LIII)
//            && (mEnergies[an - Element.elmNe][sh - AtomicShell.K] > 0.0);
//      }
//   };
//
//   public static final EdgeEnergy NISTxrtdb = new NISTEdgeEnergy();
//
//   /**
//   * Chantler2005 - A set of edge energies from "Chantler, C.T., Olsen, K.,
//   * Dragoset, R.A., Kishore, A.R., Kotochigova, S.A., and Zucker, D.S. (2005),
//   * X-Ray Form Factor, Attenuation and Scattering Tables (version 2.1).
//   * [Online] Available: http://physics.nist.gov/ffast 10-Mar-2005. National
//   * Institute of Standards and Technology, Gaithersburg, MD. Originally
//   * published as Chantler, C.T., J. Phys. Chem. Ref. Data 29(4), 597-1048
//   * (2000); and Chantler, C.T., J. Phys. Chem. Ref. Data 24, 71-643 (1995)."
//   * Supports elements H to U and shells K to O5, P1 to P3.
//   */
//
//   public static class ChantlerEdgeEnergy
//      extends EdgeEnergy {
//      ChantlerEdgeEnergy() {
//         super("NIST-Chantler 2005", "http://physics.nist.gov/ffast");
//      }
//
//      private double[][] mEnergies;
//
//      private final int index(int sh) {
//         if ((sh < AtomicShell.K) || (sh > AtomicShell.PIII))
//            return -1;
//         if (sh <= AtomicShell.OV)
//            return sh;
//         if (sh >= AtomicShell.PI)
//            return sh - 4; // (AtomicShell.PI-AtomicShell.OV + 1);
//         return -1;
//      }
//
//      private void load() {
//         synchronized(this) {
//            if (mEnergies == null) {
//               mEnergies = (new CSVReader.ResourceReader("FFastEdgeDB.csv", true)).getResource(EdgeEnergy.class);
//               // Convert from eV to Joules
//               for (int r = 0; r < mEnergies.length; ++r)
//                  if (mEnergies[r].length > 0)
//                     for (int c = 0; c < mEnergies[r].length; ++c)
//                        mEnergies[r][c] = ToSI.eV(mEnergies[r][c]);
//            }
//         }
//      }
//
//      @Override
//         public double compute(AtomicShell shell) {
//         if (mEnergies == null)
//            load();
//         final int an = shell.getElement().getAtomicNumber();
//         if ((an < Element.elmH) || (an > Element.elmU))
//            throw new IllegalArgumentException(toString() + " only supports elements H (1) to U (92)");
//         final int sh = shell.getShell();
//         final int i = index(sh);
//         if (i == -1)
//            throw new IllegalArgumentException(toString() + " only supports shells K to O5, P1 to P3");
//         assert mEnergies.length > an - Element.elmH : "Too few elements in EdgeEnergy database.";
//         return mEnergies[an - Element.elmH].length > i ? mEnergies[an - Element.elmH][i] : 0.0;
//      }
//
//      @Override
//         public boolean isSupported(AtomicShell shell) {
//         if (mEnergies == null)
//            load();
//         final int an = shell.getElement().getAtomicNumber();
//         final int i = index(shell.getShell());
//         return (an >= Element.elmLi) && (an <= Element.elmU) && (i >= 0) && (mEnergies[an - Element.elmH].length > i)
//            && (mEnergies[an - Element.elmH][i] > 0.0);
//      }
//   };
//
//   public static final EdgeEnergy Chantler2005 = new ChantlerEdgeEnergy();
//
//   /**
//   * Wernish84 - Wernisch et al., 1984 - Taken from Markowitz in the Handbook
//   * of X-ray Spectroscopy
//   */
//   public static class WernishEdgeEnergy
//      extends EdgeEnergy {
//      WernishEdgeEnergy() {
//         super("Wernisch et al., 1984", "Wernisch et al., 1984 - Taken from Markowitz in the Handbook of X-ray Spectroscopy");
//      }
//
//      @Override
//         public double compute(AtomicShell shell) {
//         final double z = shell.getElement().getAtomicNumber();
//         if (!isSupported(shell))
//            throw new IllegalArgumentException("Unsupported shell " + shell.toString() + " in " + toString());
//         switch (shell.getShell()) {
//         case AtomicShell.K:
//            return ToSI.keV(-1.304e-1 + z * (-2.633e-3 + z * (9.718e-3 + z * 4.144e-5)));
//         case AtomicShell.LI:
//            return ToSI.keV(-4.506e-1 + z * (1.566e-2 + z * (7.599e-4 + z * 1.792e-5)));
//         case AtomicShell.LII:
//            return ToSI.keV(-6.018e-1 + z * (1.964e-2 + z * (5.935e-4 + z * 1.843e-5)));
//         case AtomicShell.LIII:
//            return ToSI.keV(3.390e-1 + z * (-4.931e-2 + z * (2.336e-3 + z * 1.836e-6)));
//         case AtomicShell.MI:
//            return ToSI.keV(-8.645 + z * (3.977e-1 + z * (-5.963e-3 + z * 3.624e-5)));
//         case AtomicShell.MII:
//            return ToSI.keV(-7.499 + z * (3.459e-1 + z * (-5.250e-3 + z * 3.263e-5)));
//         case AtomicShell.MIII:
//            return ToSI.keV(-6.280 + z * (2.831e-1 + z * (-4.117e-3 + z * 2.505e-5)));
//         case AtomicShell.MIV:
//            return ToSI.keV(-4.778 + z * (2.184e-1 + z * (-3.303e-3 + z * 2.115e-5)));
//         case AtomicShell.MV:
//            return ToSI.keV(-2.421 + z * (1.172e-1 + z * (-1.845e-3 + z * 1.397e-5)));
//         default:
//            throw new IllegalArgumentException("Unsupported shell in " + toString());
//         }
//      }
//
//      @Override
//         public boolean isSupported(AtomicShell shell) {
//         final int z = shell.getElement().getAtomicNumber();
//         switch (shell.getShell()) {
//         case AtomicShell.K:
//            return ((z >= 11) && (z <= 63));
//         case AtomicShell.LI:
//            return ((z >= 28) && (z <= 83));
//         case AtomicShell.LII:
//            return ((z >= 30) && (z <= 83));
//         case AtomicShell.LIII:
//            return ((z >= 30) && (z <= 83));
//         case AtomicShell.MI:
//            return ((z >= 52) && (z <= 83));
//         case AtomicShell.MII:
//            return ((z >= 55) && (z <= 83));
//         case AtomicShell.MIII:
//            return ((z >= 55) && (z <= 83));
//         case AtomicShell.MIV:
//            return ((z >= 60) && (z <= 83));
//         case AtomicShell.MV:
//            return ((z >= 61) && (z <= 83));
//         default:
//            return false;
//         }
//      }
//   }
//
//   public static final EdgeEnergy Wernish84 = new WernishEdgeEnergy();
//
//   /**
//   * DTSA - From DTSA at
//   * http://www.cstl.nist.gov/div837/Division/outputs/DTSA/DTSA.htm
//   */
//   public static class DTSAEdgeEnergy
//      extends EdgeEnergy {
//      DTSAEdgeEnergy() {
//         super("DTSA", "From DTSA at http://www.cstl.nist.gov/div837/Division/outputs/DTSA/DTSA.htm");
//      }
//
//      private double[][] mEdgeEnergy;
//
//      @Override
//         public double compute(AtomicShell shell) {
//         if (mEdgeEnergy == null)
//            mEdgeEnergy = (new CSVReader.ResourceReader("EdgeEnergies.csv", true)).getResource(EdgeEnergy.class);
//         final int sh = shell.getShell();
//         if ((sh < AtomicShell.K) || (sh > AtomicShell.NI))
//            throw new EPQFatalException("Unsupported shell " + shell.toString() + " in " + toString());
//         final int i = shell.getElement().getAtomicNumber() - 1;
//         if ((i < 0) || (i >= mEdgeEnergy.length))
//            throw new EPQFatalException("Unsupported element " + shell.toString() + " in " + toString());
//         return (mEdgeEnergy[i] != null) && (mEdgeEnergy[i].length > sh) ? ToSI.eV(mEdgeEnergy[i][sh]) : 0.0;
//      }
//
//      @Override
//         public boolean isSupported(AtomicShell shell) {
//         if (mEdgeEnergy == null)
//            mEdgeEnergy = (new CSVReader.ResourceReader("EdgeEnergies.csv", true)).getResource(EdgeEnergy.class);
//         final int sh = shell.getShell();
//         final int zp = shell.getElement().getAtomicNumber() - 1;
//         return (zp < mEdgeEnergy.length) && (mEdgeEnergy[zp] != null) && (mEdgeEnergy[zp].length > sh);
//      }
//   }
//
//   public static final EdgeEnergy DTSA = new DTSAEdgeEnergy();
//
//   /**
//   * A super set to cover the largest range of shells
//   */
//   public static class SuperSetEdgeEnergy
//      extends EdgeEnergy {
//      SuperSetEdgeEnergy() {
//         super("Superset", "Chantler then NIST then DTSA");
//      }
//
//      @Override
//         public double compute(AtomicShell shell) {
//         try {
//            return Chantler2005.compute(shell);
//         }
//         catch (Throwable e0) {
//            try {
//               return NISTxrtdb.compute(shell);
//            }
//            catch (Throwable e1) {
//               try {
//                  return DTSA.compute(shell);
//               }
//               catch (Throwable e2) {
//                  return -1.0;
//               }
//            }
//         }
//      }
//
//      @Override
//         public boolean isSupported(AtomicShell shell) {
//         return Chantler2005.isSupported(shell) || NISTxrtdb.isSupported(shell) || DTSA.isSupported(shell);
//      }
//   }
//
//   public static final EdgeEnergy SuperSet = new SuperSetEdgeEnergy();
//
//   static private final AlgorithmClass[] mAllImplementations = {
//      EdgeEnergy.DTSA,
//      EdgeEnergy.Chantler2005,
//      EdgeEnergy.NISTxrtdb,
//      EdgeEnergy.Wernish84,
//      EdgeEnergy.DHSIonizationEnergy,
//      EdgeEnergy.SuperSet
//   };
//
//};
