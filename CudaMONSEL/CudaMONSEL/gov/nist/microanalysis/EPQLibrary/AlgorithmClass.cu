
#include <hip/hip_runtime.h>
//#include "AlgorithmClass.cuh"
//
///**
//* <p>
//* A simple class that implements various common properties of classes that
//* implement algorithms.
//* </p>
//* <p>
//* AlgorithmClass objects have a general type ('class') name and a specific
//* instance name. They can also contain a reference detailing the source of the
//* algorithm.
//* </p>
//* <p>
//* AlgorithmClass objects also define a list of default AlgorithmClass instances
//* on which they depend. Replacements for these default AlgorithmClass instances
//* can be specified using the Strategy mechanism.
//* <p>
//* Copyright: Pursuant to title 17 Section 105 of the United States Code this
//* software is not subject to copyright protection and is in the public domain
//* </p>
//* <p>
//* Company: National Institute of Standards and Technology
//* </p>
//*
//* @author Nicholas W. M. Ritchie
//* @version 1.0
//*/
//
//namespace AlgorithmClass
//{
//   AlgorithmClass(AlgorithmClass::AlgorithmClassNameT cls, AlgorithmClass::AlgorithmClassNameT name, Reference ref) : mClass(cls), mName(name), mReference(ref)
//   {
//   }
//
//   protected AlgorithmClass(String clss, String name, String ref) {
//      super();
//      mClass = clss;
//      mName = name;
//      mReference = new Reference.CrudeReference(ref);
//   }
//
//   /**
//   * getAllImplementations - Returns a list of all implementations of the
//   * derived algorithm class. Typically this method is implemented by the
//   * abstract base class from which more specific implementations are derived.
//   *
//   * @return List
//   */
//   abstract public List<AlgorithmClass> getAllImplementations();
//
//   /**
//   * compareTo - Sort by name
//   *
//   * @param o UncertainValue
//   * @return int
//   */
//   public int compareTo(AlgorithmClass o) {
//      return toString().compareTo(o.toString());
//   }
//
//   // Overrides Object.toString()
//   @Override
//      public String toString() {
//      return mClass + "[" + mName + "]";
//   }
//
//   /**
//   * getAlgorithmClass - Get the base AlgorithmClass instance of which this
//   * class is an instance.
//   *
//   * @return String
//   */
//   public String getAlgorithmClass() {
//      return mClass;
//   }
//
//   /**
//   * getName - Get the abbreviated name of the algorithm.
//   *
//   * @return String
//   */
//   public String getName() {
//      return mName;
//   }
//
//   /**
//   * getReference - Get the literature reference describing the implementation
//   * of this algorithm.
//   *
//   * @return String
//   */
//   public String getReference() {
//      return mReference.getShortForm();
//   }
//
//   public Reference getReferenceObj() {
//      return mReference;
//   }
//}
