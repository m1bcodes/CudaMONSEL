#include "gov\nist\microanalysis\EPQLibrary\CaveatBase.cuh"

/**
* <p>
* Provides a standardized base set of static items for use in caveats - a
* mechanism for identifying an algorithms limitations.
* </p>
* <p>
* Copyright: Pursuant to title 17 Section 105 of the United States Code this
* software is not subject to copyright protection and is in the public domain
* </p>
* <p>
* Company: National Institute of Standards and Technology
* </p>
*
* @author Nicholas W. M. Ritchie
* @version 1.0
*/

namespace CaveatBase
{
   const char None[] = "No limitations have been identified.";
   const char Broken[] = "The implementation of this algorithm is broken.";
   const char NotImplemented[] = "This algorithm has not been implemented yet.";

   /**
   * isBroken - Is this implementation broken?
   *
   * @param str String - The result from a call to a caveat method
   * @return boolean
   */
   //public static boolean isBroken(String str) {
   //   return (str == Broken) || str.equals(Broken);
   //}

   ///**
   //* isNone - Are there no caveats identified with this algorithm?
   //*
   //* @param str String - The result from a call to a caveat method
   //* @return boolean
   //*/
   //public static boolean isNone(String str) {
   //   return (str == None) || str.equals(None);
   //}

   ///**
   //* isNotImplemented - Is this algorithm not implemented?
   //*
   //* @param str String - The result from a call to a caveat method
   //* @return boolean
   //*/
   //public static boolean isNotImplemented(String str) {
   //   return (str == NotImplemented) || str.equals(NotImplemented);
   //}

   /**
   * appendCaveat - Append a new caveat to a list of existing caveats. If one
   * or both is CaveatBase.None this is handled correctly. If both are
   * CaveatBase.None then CaveatBase.None is returned.
   *
   * @param base String
   * @param str String
   * @return String
   */
   StringT append(StringT base, StringT str)
   {
      if (base == None)
         return str;
      else if (str != None)
         return base + "\n" + str;
      else
         return base;
   }

   ///**
   //* formatCaveat - Format the caveat in such a way that the algorithm is
   //* identified along with the caveat. If the caveat equals CaveatBase.None
   //* then CaveatBase.None is returned.
   //*
   //* @param obj Object
   //* @param str String
   //* @return String
   //*/
   //public static String format(Object obj, String str) {
   //   return str == None ? None : obj.toString() + ": " + str;
   //}

}
