#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\EPQLibrary\MeanIonizationPotential.cuh"
#include "gov\nist\microanalysis\EPQLibrary\CaveatBase.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Composition.cuh"
#include "gov\nist\microanalysis\EPQLibrary\FromSI.cuh"

#include "CudaUtil.h"

namespace MeanIonizationPotential
{
   __host__ __device__ MeanIonizationPotential::MeanIonizationPotential(StringT name, const ReferenceT &reference) : AlgorithmClass("Mean Ionization Potential", name, reference)
   {
   }

   __host__ __device__ void MeanIonizationPotential::initializeDefaultStrategy()
   {
   }

   StringT caveat(const ElementT &el)
   {
      return CaveatBase::None;
   }

   StringT caveat(const CompositionT &comp)
   {
      StringT res(CaveatBase::None);
      for (auto el : comp.getElementSet())
         res = CaveatBase::append(res, caveat(*el));
      return res;
   }

   float MeanIonizationPotential::computeLn(const CompositionT &comp) const
   {
      float m = 0.0f;
      float lnJ = 0.0f;
      for (auto &el : comp.getElementSet()) {
         float cz_a = comp.weightFraction(*el, true) * el->getAtomicNumber() / el->getAtomicWeight();
         m += cz_a;
         lnJ += cz_a * ::logf(FromSI::keV(compute(*el)));
      }
      return ToSI::keV(::expf(lnJ / m));
   }

   Reference::CrudeReference SternheimerCR("Sternheimer quoted in Berger MJ, Seltzer S. NASA Technical Publication SP-4012 (1964)");
   __host__ __device__ Sternheimer64MeanIonizationPotential::Sternheimer64MeanIonizationPotential() :
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      MeanIonizationPotential("Sternheimer 1964", *Reference::d_NullReference)
#else
      MeanIonizationPotential("Sternheimer 1964", SternheimerCR)
#endif
   {
   }

   __host__ __device__ float Sternheimer64MeanIonizationPotential::compute(const ElementT &el) const
   {
      const float z = el.getAtomicNumber();
      return ToSI::eV(9.76f * z + 58.8f * ::powf(z, -0.19f));
   }

   const Sternheimer64MeanIonizationPotential Sternheimer64Ref;
   const MeanIonizationPotential &Sternheimer64 = Sternheimer64Ref;
   __device__ const MeanIonizationPotential *d_Sternheimer64;

   __host__ __device__ float computeSternheimer64(const ElementT &el)
   {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      if (d_Sternheimer64 == nullptr) {
         printf("computeSternheimer64: not initialized on device");
         return NAN;
      }
      return d_Sternheimer64->compute(el);
#else
      return Sternheimer64.compute(el);
#endif
   }

   Reference::CrudeReference BergerSeltzerCR("Berger and Seltzer as implemented by CITZAF 3.06");
   __host__ __device__ BergerAndSeltzerCITZAFMeanIonizationPotential::BergerAndSeltzerCITZAFMeanIonizationPotential() :
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      MeanIonizationPotential("Berger & Seltzer as per JTA", *Reference::d_NullReference)
#else
      MeanIonizationPotential("Berger & Seltzer as per JTA", BergerSeltzerCR)
#endif
   {
   }

   __host__ __device__ float BergerAndSeltzerCITZAFMeanIonizationPotential::compute(const ElementT &el) const
   {
      const float z = el.getAtomicNumber();
      return ToSI::eV(9.76f * z + 58.5f * ::powf(z, -0.19f));
   }

   const BergerAndSeltzerCITZAFMeanIonizationPotential BergerAndSeltzerCITZAFRef;
   const MeanIonizationPotential &BergerAndSeltzerCITZAF = BergerAndSeltzerCITZAFRef;
   __device__ const MeanIonizationPotential *d_BergerAndSeltzerCITZAF;

   Reference::CrudeReference Bloch33CR("Bloch F, F. Z. Phys. 81, 363 (1933)");
   __host__ __device__ Bloch33MeanIonizationPotential::Bloch33MeanIonizationPotential() :
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      MeanIonizationPotential("Bloch 1933", *Reference::d_NullReference)
#else
      MeanIonizationPotential("Bloch 1933", Bloch33CR)
#endif
   {
   }

   __host__ __device__ float Bloch33MeanIonizationPotential::compute(const ElementT &el) const
   {
      const float z = el.getAtomicNumber();
      return ToSI::eV(13.5f * z);
   }

   const Bloch33MeanIonizationPotential Bloch33Ref;
   const MeanIonizationPotential &Bloch33 = Bloch33Ref;
   __device__ const MeanIonizationPotential *d_Bloch33;

   Reference::CrudeReference Wilson41CR("Wilson RR. Phys Rev. 60. 749 (1941)");
   __host__ __device__ Wilson41MeanIonizationPotential::Wilson41MeanIonizationPotential() :
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      MeanIonizationPotential("Wilson 1941", *Reference::d_NullReference)
#else
      MeanIonizationPotential("Wilson 1941", Wilson41CR)
#endif
   {
   }

   __host__ __device__ float Wilson41MeanIonizationPotential::compute(const ElementT &el) const
   {
      const float z = el.getAtomicNumber();
      return ToSI::eV(11.5f * z);
   }

   const Wilson41MeanIonizationPotential Wilson41Ref;
   const MeanIonizationPotential &Wilson41 = Wilson41Ref;
   __device__ const MeanIonizationPotential *d_Wilson41;

   __host__ __device__ float computeWilson41(const ElementT &el)
   {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      if (d_Wilson41 == nullptr) {
         printf("computeWilson41: not initialized on device");
         return NAN;
      }
      return d_Wilson41->compute(el);
#else
      return Wilson41.compute(el);
#endif
   }

   Reference::CrudeReference Springer67CR("Springer G. Meues Jahrbuch Fuer Mineralogie, Monatshefte (1967) 9/10, p. 304");
   __host__ __device__ Springer67MeanIonizationPotential::Springer67MeanIonizationPotential() :
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      MeanIonizationPotential("Springer 1967", *Reference::d_NullReference)
#else
      MeanIonizationPotential("Springer 1967", Springer67CR)
#endif
   {
   }

   __host__ __device__ float Springer67MeanIonizationPotential::compute(const ElementT &el) const
   {
      const float z = el.getAtomicNumber();
      return ToSI::eV(z * (9.0f * (1.0f + ::powf(z, -0.67f)) + 0.03f * z));
   }

   const Springer67MeanIonizationPotential Springer67Ref;
   const MeanIonizationPotential &Springer67 = Springer67Ref;
   __device__ const MeanIonizationPotential *d_Springer67;

   Reference::CrudeReference Heinrich70CR("Heinrich KFJ, Yakowitz H. Mikrochim Acta (1970) p 123");
   __host__ __device__ Heinrich70MeanIonizationPotential::Heinrich70MeanIonizationPotential() :
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      MeanIonizationPotential("Heinrich & Yakowitz 1970", *Reference::d_NullReference)
#else
      MeanIonizationPotential("Heinrich & Yakowitz 1970", Heinrich70CR)
#endif
   {
   }

   __host__ __device__ float Heinrich70MeanIonizationPotential::compute(const ElementT &el) const
   {
      const float z = el.getAtomicNumber();
      return ToSI::eV(z * (12.4f + 0.027f * z));
   }

   const Heinrich70MeanIonizationPotential Heinrich70Ref;
   const MeanIonizationPotential &Heinrich70 = Heinrich70Ref;
   __device__ const MeanIonizationPotential *d_Heinrich70;

   Reference::CrudeReference Duncumb69CR("Duncumb P, Shields-Mason PK, DeCasa C. Proc. 5th Int. Congr. on X-ray Optics and Microanalysis, Springer, Berlin, 1969 p. 146");
   __host__ __device__ Duncumb69MeanIonizationPotential::Duncumb69MeanIonizationPotential() :
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      MeanIonizationPotential("Duncumb & DeCasa 1969", *Reference::d_NullReference)
#else
      MeanIonizationPotential("Duncumb & DeCasa 1969", Duncumb69CR)
#endif
   {
   }

   __host__ __device__ float Duncumb69MeanIonizationPotential::compute(const ElementT &el) const
   {
      const float z = el.getAtomicNumber();
      return ToSI::eV((14.0f * (1.0f - ::expf(-0.1f * z)) + 75.5f / ::powf(z, z / 7.5f) - z / (100.f + z)) * z);
   }

   const Duncumb69MeanIonizationPotential Duncumb69Ref;
   const MeanIonizationPotential &Duncumb69 = Duncumb69Ref;
   __device__ const MeanIonizationPotential *d_Duncumb69;

   Reference::CrudeReference Zeller75CR("Zeller C in Ruste J, Gantois M, J. Phys. D. Appl. Phys 8, 872 (1975)");
   __host__ __device__ Zeller75MeanIonizationPotential::Zeller75MeanIonizationPotential() :
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      MeanIonizationPotential("Zeller 1975", *Reference::d_NullReference)
#else
      MeanIonizationPotential("Zeller 1975", Zeller75CR)
#endif
   {
   }

   __host__ __device__ float Zeller75MeanIonizationPotential::compute(const ElementT &el) const
   {
      const float z = el.getAtomicNumber();
      return ToSI::eV((10.04f + 8.25f * ::expf(-z / 11.22f)) * z);
   }

   const Zeller75MeanIonizationPotential Zeller75Ref;
   const MeanIonizationPotential &Zeller75 = Zeller75Ref;
   __device__ const MeanIonizationPotential *d_Zeller75;

   // https://www.oreilly.com/library/view/c-cookbook/0596007612/ch03s06.html
   static float sciToDub(const std::string& str)
   {
      std::stringstream ss(str);
      float d = 0;
      ss >> d;

      if (ss.fail()) {
         std::string s = "Unable to format ";
         s += str;
         s += " as a number!";
         throw (s);
      }

      return d;
   }

   Reference::CrudeReference Berger64CR("Berger MJ, Seltzer S. NASA Technical Publication SP-4012 (1964)");
   __host__ __device__ Berger64MeanIonizationPotential::Berger64MeanIonizationPotential() :
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      MeanIonizationPotential("Berger & Seltzer 1964", *Reference::d_NullReference)
#else
      MeanIonizationPotential("Berger & Seltzer 1964", Berger64CR)
#endif
   {
   }

   void Berger64MeanIonizationPotential::readTabulatedValues()
   {
      if (!mMeasured.empty()) return;

      std::string name(".\\gov\\nist\\microanalysis\\EPQLibrary\\BergerSeltzer64.csv");
      printf("Reading: %s\n", name.c_str());
      try {
         std::ifstream file(name);
         if (!file.good()) throw 0;
         mMeasured.reserve(92);
         for (CSVIterator loop(file); loop != CSVIterator(); ++loop) {
            if ((*loop)[0][0] != '/')
               mMeasured.push_back(ToSI::eV(sciToDub((*loop)[0])));
         }
         file.close();
      }
      catch (std::exception&) {
         printf("Fatal error while attempting to load the mean ionization potential data file.");
      }
   }

   __host__ __device__ const VectorXf& Berger64MeanIonizationPotential::getData() const
   {
      return mMeasured;
   }

   //__device__ void Berger64MeanIonizationPotential::copyData(const double *data, const unsigned int len)
   //{
   //   mMeasured.resize(len);
   //   memcpy(mMeasured.data(), data, len * sizeof(double));
   //}

   __host__ __device__ float Berger64MeanIonizationPotential::compute(const ElementT &el) const
   {
      return mMeasured[el.getAtomicNumber() - 1];
   }

   Berger64MeanIonizationPotential Berger64Ref;
   Berger64MeanIonizationPotential& Berger64 = Berger64Ref;
   __device__ Berger64MeanIonizationPotential* d_Berger64;

   Reference::CrudeReference Berger83CR("Berger MJ, Seltzer S. NBSIR 82-2550-A - US Dept of Commerce, Washington DC (1983)");
   __host__ __device__ Berger83MeanIonizationPotential::Berger83MeanIonizationPotential() :
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      MeanIonizationPotential("Berger & Seltzer 1983", *Reference::d_NullReference)
#else
      MeanIonizationPotential("Berger & Seltzer 1983", Berger83CR)
#endif
   {
   }

   void Berger83MeanIonizationPotential::readTabulatedValues()
   {
      if (!mMeasured.empty()) return;

      std::string name(".\\gov\\nist\\microanalysis\\EPQLibrary\\BergerSeltzer83.csv");
      printf("Reading: %s\n", name.c_str());
      try {
         std::ifstream file(name);
         if (!file.good()) throw 0;
         mMeasured.reserve(100);
         for (CSVIterator loop(file); loop != CSVIterator(); ++loop) {
            if ((*loop)[0][0] != '/')
               mMeasured.push_back(ToSI::eV(sciToDub((*loop)[0])));
         }
         file.close();
      }
      catch (std::exception&) {
         printf("Fatal error while attempting to load the mean ionization potential data file.");
      }
   }

   __host__ __device__ const VectorXf &Berger83MeanIonizationPotential::getData() const
   {
      return mMeasured;
   }

   //__device__ void Berger83MeanIonizationPotential::copyData(const double *data, const unsigned int len)
   //{
   //   mMeasured.resize(len);
   //   memcpy(mMeasured.data(), data, len * sizeof(double));
   //}

   __host__ __device__ float Berger83MeanIonizationPotential::compute(const ElementT &el) const
   {
      return mMeasured[el.getAtomicNumber() - 1];
   }

   Berger83MeanIonizationPotential Berger83Ref;
   Berger83MeanIonizationPotential& Berger83 = Berger83Ref;
   __device__ Berger83MeanIonizationPotential* d_Berger83;

   const AlgorithmClassT * mAllImplementations[] = {
      &Berger64,
      &Berger83,
      &Bloch33,
      &Duncumb69,
      &BergerAndSeltzerCITZAF,
      &Heinrich70,
      &Springer67,
      &Sternheimer64,
      &Wilson41,
      &Zeller75
   };

   //__device__ const AlgorithmClassT * dAllImplementations[] = {
   //   d_Berger64,
   //   dBerger83,
   //   dBloch33,
   //   dDuncumb69,
   //   dBergerAndSeltzerCITZAF,
   //   dHeinrich70,
   //   dSpringer67,
   //   dSternheimer64,
   //   dWilson41,
   //   dZeller75
   //};

   __global__ void initCuda()
   {
      d_Berger64 = new Berger64MeanIonizationPotential();
      d_Berger83 = new Berger83MeanIonizationPotential();
      d_Bloch33 = new Bloch33MeanIonizationPotential();
      d_Duncumb69 = new Duncumb69MeanIonizationPotential();
      d_BergerAndSeltzerCITZAF = new BergerAndSeltzerCITZAFMeanIonizationPotential();
      d_Heinrich70 = new Heinrich70MeanIonizationPotential();
      d_Springer67 = new Springer67MeanIonizationPotential();
      d_Sternheimer64 = new Sternheimer64MeanIonizationPotential();
      d_Wilson41 = new Wilson41MeanIonizationPotential();
      d_Zeller75 = new Zeller75MeanIonizationPotential();
   }

   template<typename T>
   __global__ void assignDataToBerger64(T *data, int len)
   {
      d_Berger64->assignData<T>(data, len);
   }

   template<typename T>
   __global__ void assignDataToBerger83(T *data, int len)
   {
      d_Berger83->assignData<T>(data, len);
   }

   typedef float data_type;

   void transferDataToCuda()
   {
      data_type* Berger64data = nullptr;
      checkCudaErrors(hipMalloc((void **)&Berger64data, Berger64.getData().size() * sizeof(data_type)));
      checkCudaErrors(hipMemcpy(Berger64data, Berger64.getData().data(), Berger64.getData().size() * sizeof(data_type), hipMemcpyHostToDevice));
      assignDataToBerger64 << <1, 1 >> >(Berger64data, Berger64.getData().size());
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());

      data_type* Berger83data = nullptr;
      checkCudaErrors(hipMalloc((void **)&Berger83data, Berger83.getData().size() * sizeof(data_type)));
      checkCudaErrors(hipMemcpy(Berger83data, Berger83.getData().data(), Berger83.getData().size() * sizeof(data_type), hipMemcpyHostToDevice));
      assignDataToBerger83 << <1, 1 >> >(Berger83data, Berger83.getData().size());
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
   }

   AlgorithmClassT const * const * MeanIonizationPotential::getAllImplementations() const
   {
//#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
//      return dAllImplementations;
//#else
//      return mAllImplementations;
//#endif
      return mAllImplementations;
   }
}
