#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\EPQLibrary\ScreenedRutherfordScatteringAngle.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"

namespace ScreenedRutherfordScatteringAngle
{
   static Reference::CrudeReference REFERENCE(Reference::CrudeReference("NBSMONTE.FOR"));

   ScreenedRutherfordScatteringAngle::ScreenedRutherfordScatteringAngle(const ElementT& elm) : RandomizedScatterT("Screened Rutherford", REFERENCE), mElement(elm)
   {
   }

   ScreenedRutherfordScatteringAngle::ScreenedRutherfordScatteringAngle(int an) : RandomizedScatterT("Screened Rutherford", REFERENCE), mElement(Element::byAtomicNumber(an))
   {
   }
   
   ScreenedRutherfordScatteringAngle::ScreenedRutherfordScatteringAngle(const ScreenedRutherfordScatteringAngle& other) : RandomizedScatterT("Screened Rutherford", REFERENCE), mElement(other.mElement)
   {
   }

   StringT ScreenedRutherfordScatteringAngle::toString() const
   {
      return "CrossSection[Screened-Rutherford," + StringT(mElement.toAbbrev()) + "]";
   }

   
   const ElementT& ScreenedRutherfordScatteringAngle::getElement() const
   {
      return mElement;
   }

   double ScreenedRutherfordScatteringAngle::totalCrossSection(double energy) const
   {
      // Ref: Heinrich 1981 p 459 convert to SI units
      double z = mElement.getAtomicNumber();
      double zp = ::pow(z, 1.0 / 3.0);
      return (7.670843088080456e-38 * zp * (1.0 + z)) / ((energy + ((5.44967975966321e-19 * zp * zp))));
   }

   double ScreenedRutherfordScatteringAngle::randomScatteringAngle(double energy) const
   {
      // This method for calculating the scattering angle is taken from
      // NBSMONTE.FOR
      double alpha = (5.44968e-19 * ::pow(mElement.getAtomicNumber(), 2.0 / 3.0)) / energy;
      double r = Math2::random();
      return ::acos(1 - 2.0 * alpha * r / (1 + alpha - r));
   }

   //const ScreenedRutherfordScatteringAngle SRSA1(Element::H);
   //const ScreenedRutherfordScatteringAngle SRSA2(Element::He);
   //const ScreenedRutherfordScatteringAngle SRSA3(Element::Li);
   //const ScreenedRutherfordScatteringAngle SRSA4(Element::Be);
   //const ScreenedRutherfordScatteringAngle SRSA5(Element::B);
   //const ScreenedRutherfordScatteringAngle SRSA6(Element::C);
   //const ScreenedRutherfordScatteringAngle SRSA7(Element::N);
   //const ScreenedRutherfordScatteringAngle SRSA8(Element::O);
   //const ScreenedRutherfordScatteringAngle SRSA9(Element::F);
   //const ScreenedRutherfordScatteringAngle SRSA10(Element::Ne);
   //const ScreenedRutherfordScatteringAngle SRSA11(Element::Na);
   //const ScreenedRutherfordScatteringAngle SRSA12(Element::Mg);
   //const ScreenedRutherfordScatteringAngle SRSA13(Element::Al);
   //const ScreenedRutherfordScatteringAngle SRSA14(Element::Si);
   //const ScreenedRutherfordScatteringAngle SRSA15(Element::P);
   //const ScreenedRutherfordScatteringAngle SRSA16(Element::S);
   //const ScreenedRutherfordScatteringAngle SRSA17(Element::Cl);
   //const ScreenedRutherfordScatteringAngle SRSA18(Element::Ar);
   //const ScreenedRutherfordScatteringAngle SRSA19(Element::K);
   //const ScreenedRutherfordScatteringAngle SRSA20(Element::Ca);
   //const ScreenedRutherfordScatteringAngle SRSA21(Element::Sc);
   //const ScreenedRutherfordScatteringAngle SRSA22(Element::Ti);
   //const ScreenedRutherfordScatteringAngle SRSA23(Element::V);
   //const ScreenedRutherfordScatteringAngle SRSA24(Element::Cr);
   //const ScreenedRutherfordScatteringAngle SRSA25(Element::Mn);
   //const ScreenedRutherfordScatteringAngle SRSA26(Element::Fe);
   //const ScreenedRutherfordScatteringAngle SRSA27(Element::Co);
   //const ScreenedRutherfordScatteringAngle SRSA28(Element::Ni);
   //const ScreenedRutherfordScatteringAngle SRSA29(Element::Cu);
   //const ScreenedRutherfordScatteringAngle SRSA30(Element::Zn);
   //const ScreenedRutherfordScatteringAngle SRSA31(Element::Ga);
   //const ScreenedRutherfordScatteringAngle SRSA32(Element::Ge);
   //const ScreenedRutherfordScatteringAngle SRSA33(Element::As);
   //const ScreenedRutherfordScatteringAngle SRSA34(Element::Se);
   //const ScreenedRutherfordScatteringAngle SRSA35(Element::Br);
   //const ScreenedRutherfordScatteringAngle SRSA36(Element::Kr);
   //const ScreenedRutherfordScatteringAngle SRSA37(Element::Rb);
   //const ScreenedRutherfordScatteringAngle SRSA38(Element::Sr);
   //const ScreenedRutherfordScatteringAngle SRSA39(Element::Y);
   //const ScreenedRutherfordScatteringAngle SRSA40(Element::Zr);
   //const ScreenedRutherfordScatteringAngle SRSA41(Element::Nb);
   //const ScreenedRutherfordScatteringAngle SRSA42(Element::Mo);
   //const ScreenedRutherfordScatteringAngle SRSA43(Element::Tc);
   //const ScreenedRutherfordScatteringAngle SRSA44(Element::Ru);
   //const ScreenedRutherfordScatteringAngle SRSA45(Element::Rh);
   //const ScreenedRutherfordScatteringAngle SRSA46(Element::Pd);
   //const ScreenedRutherfordScatteringAngle SRSA47(Element::Ag);
   //const ScreenedRutherfordScatteringAngle SRSA48(Element::Cd);
   //const ScreenedRutherfordScatteringAngle SRSA49(Element::In);
   //const ScreenedRutherfordScatteringAngle SRSA50(Element::Sn);
   //const ScreenedRutherfordScatteringAngle SRSA51(Element::Sb);
   //const ScreenedRutherfordScatteringAngle SRSA52(Element::Te);
   //const ScreenedRutherfordScatteringAngle SRSA53(Element::I);
   //const ScreenedRutherfordScatteringAngle SRSA54(Element::Xe);
   //const ScreenedRutherfordScatteringAngle SRSA55(Element::Cs);
   //const ScreenedRutherfordScatteringAngle SRSA56(Element::Ba);
   //const ScreenedRutherfordScatteringAngle SRSA57(Element::La);
   //const ScreenedRutherfordScatteringAngle SRSA58(Element::Ce);
   //const ScreenedRutherfordScatteringAngle SRSA59(Element::Pr);
   //const ScreenedRutherfordScatteringAngle SRSA60(Element::Nd);
   //const ScreenedRutherfordScatteringAngle SRSA61(Element::Pm);
   //const ScreenedRutherfordScatteringAngle SRSA62(Element::Sm);
   //const ScreenedRutherfordScatteringAngle SRSA63(Element::Eu);
   //const ScreenedRutherfordScatteringAngle SRSA64(Element::Gd);
   //const ScreenedRutherfordScatteringAngle SRSA65(Element::Tb);
   //const ScreenedRutherfordScatteringAngle SRSA66(Element::Dy);
   //const ScreenedRutherfordScatteringAngle SRSA67(Element::Ho);
   //const ScreenedRutherfordScatteringAngle SRSA68(Element::Er);
   //const ScreenedRutherfordScatteringAngle SRSA69(Element::Tm);
   //const ScreenedRutherfordScatteringAngle SRSA70(Element::Yb);
   //const ScreenedRutherfordScatteringAngle SRSA71(Element::Lu);
   //const ScreenedRutherfordScatteringAngle SRSA72(Element::Hf);
   //const ScreenedRutherfordScatteringAngle SRSA73(Element::Ta);
   //const ScreenedRutherfordScatteringAngle SRSA74(Element::W);
   //const ScreenedRutherfordScatteringAngle SRSA75(Element::Re);
   //const ScreenedRutherfordScatteringAngle SRSA76(Element::Os);
   //const ScreenedRutherfordScatteringAngle SRSA77(Element::Ir);
   //const ScreenedRutherfordScatteringAngle SRSA78(Element::Pt);
   //const ScreenedRutherfordScatteringAngle SRSA79(Element::Au);
   //const ScreenedRutherfordScatteringAngle SRSA80(Element::Hg);
   //const ScreenedRutherfordScatteringAngle SRSA81(Element::Tl);
   //const ScreenedRutherfordScatteringAngle SRSA82(Element::Pb);
   //const ScreenedRutherfordScatteringAngle SRSA83(Element::Bi);
   //const ScreenedRutherfordScatteringAngle SRSA84(Element::Po);
   //const ScreenedRutherfordScatteringAngle SRSA85(Element::At);
   //const ScreenedRutherfordScatteringAngle SRSA86(Element::Rn);
   //const ScreenedRutherfordScatteringAngle SRSA87(Element::Fr);
   //const ScreenedRutherfordScatteringAngle SRSA88(Element::Ra);
   //const ScreenedRutherfordScatteringAngle SRSA89(Element::Ac);
   //const ScreenedRutherfordScatteringAngle SRSA90(Element::Th);
   //const ScreenedRutherfordScatteringAngle SRSA91(Element::Pa);
   //const ScreenedRutherfordScatteringAngle SRSA92(Element::U);
   //const ScreenedRutherfordScatteringAngle SRSA93(Element::Np);
   //const ScreenedRutherfordScatteringAngle SRSA94(Element::Pu);
   //const ScreenedRutherfordScatteringAngle SRSA95(Element::Am);
   //const ScreenedRutherfordScatteringAngle SRSA96(Element::Cm);

   const ScreenedRutherfordScatteringAngle SRSA1(1);
   const ScreenedRutherfordScatteringAngle SRSA2(2);
   const ScreenedRutherfordScatteringAngle SRSA3(3);
   const ScreenedRutherfordScatteringAngle SRSA4(4);
   const ScreenedRutherfordScatteringAngle SRSA5(5);
   const ScreenedRutherfordScatteringAngle SRSA6(6);
   const ScreenedRutherfordScatteringAngle SRSA7(7);
   const ScreenedRutherfordScatteringAngle SRSA8(8);
   const ScreenedRutherfordScatteringAngle SRSA9(9);
   const ScreenedRutherfordScatteringAngle SRSA10(10);
   const ScreenedRutherfordScatteringAngle SRSA11(11);
   const ScreenedRutherfordScatteringAngle SRSA12(12);
   const ScreenedRutherfordScatteringAngle SRSA13(13);
   const ScreenedRutherfordScatteringAngle SRSA14(14);
   const ScreenedRutherfordScatteringAngle SRSA15(15);
   const ScreenedRutherfordScatteringAngle SRSA16(16);
   const ScreenedRutherfordScatteringAngle SRSA17(17);
   const ScreenedRutherfordScatteringAngle SRSA18(18);
   const ScreenedRutherfordScatteringAngle SRSA19(19);
   const ScreenedRutherfordScatteringAngle SRSA20(20);
   const ScreenedRutherfordScatteringAngle SRSA21(21);
   const ScreenedRutherfordScatteringAngle SRSA22(22);
   const ScreenedRutherfordScatteringAngle SRSA23(23);
   const ScreenedRutherfordScatteringAngle SRSA24(24);
   const ScreenedRutherfordScatteringAngle SRSA25(25);
   const ScreenedRutherfordScatteringAngle SRSA26(26);
   const ScreenedRutherfordScatteringAngle SRSA27(27);
   const ScreenedRutherfordScatteringAngle SRSA28(28);
   const ScreenedRutherfordScatteringAngle SRSA29(29);
   const ScreenedRutherfordScatteringAngle SRSA30(30);
   const ScreenedRutherfordScatteringAngle SRSA31(31);
   const ScreenedRutherfordScatteringAngle SRSA32(32);
   const ScreenedRutherfordScatteringAngle SRSA33(33);
   const ScreenedRutherfordScatteringAngle SRSA34(34);
   const ScreenedRutherfordScatteringAngle SRSA35(35);
   const ScreenedRutherfordScatteringAngle SRSA36(36);
   const ScreenedRutherfordScatteringAngle SRSA37(37);
   const ScreenedRutherfordScatteringAngle SRSA38(38);
   const ScreenedRutherfordScatteringAngle SRSA39(39);
   const ScreenedRutherfordScatteringAngle SRSA40(40);
   const ScreenedRutherfordScatteringAngle SRSA41(41);
   const ScreenedRutherfordScatteringAngle SRSA42(42);
   const ScreenedRutherfordScatteringAngle SRSA43(43);
   const ScreenedRutherfordScatteringAngle SRSA44(44);
   const ScreenedRutherfordScatteringAngle SRSA45(45);
   const ScreenedRutherfordScatteringAngle SRSA46(46);
   const ScreenedRutherfordScatteringAngle SRSA47(47);
   const ScreenedRutherfordScatteringAngle SRSA48(48);
   const ScreenedRutherfordScatteringAngle SRSA49(49);
   const ScreenedRutherfordScatteringAngle SRSA50(50);
   const ScreenedRutherfordScatteringAngle SRSA51(51);
   const ScreenedRutherfordScatteringAngle SRSA52(52);
   const ScreenedRutherfordScatteringAngle SRSA53(53);
   const ScreenedRutherfordScatteringAngle SRSA54(54);
   const ScreenedRutherfordScatteringAngle SRSA55(55);
   const ScreenedRutherfordScatteringAngle SRSA56(56);
   const ScreenedRutherfordScatteringAngle SRSA57(57);
   const ScreenedRutherfordScatteringAngle SRSA58(58);
   const ScreenedRutherfordScatteringAngle SRSA59(59);
   const ScreenedRutherfordScatteringAngle SRSA60(60);
   const ScreenedRutherfordScatteringAngle SRSA61(61);
   const ScreenedRutherfordScatteringAngle SRSA62(62);
   const ScreenedRutherfordScatteringAngle SRSA63(63);
   const ScreenedRutherfordScatteringAngle SRSA64(64);
   const ScreenedRutherfordScatteringAngle SRSA65(65);
   const ScreenedRutherfordScatteringAngle SRSA66(66);
   const ScreenedRutherfordScatteringAngle SRSA67(67);
   const ScreenedRutherfordScatteringAngle SRSA68(68);
   const ScreenedRutherfordScatteringAngle SRSA69(69);
   const ScreenedRutherfordScatteringAngle SRSA70(70);
   const ScreenedRutherfordScatteringAngle SRSA71(71);
   const ScreenedRutherfordScatteringAngle SRSA72(72);
   const ScreenedRutherfordScatteringAngle SRSA73(73);
   const ScreenedRutherfordScatteringAngle SRSA74(74);
   const ScreenedRutherfordScatteringAngle SRSA75(75);
   const ScreenedRutherfordScatteringAngle SRSA76(76);
   const ScreenedRutherfordScatteringAngle SRSA77(77);
   const ScreenedRutherfordScatteringAngle SRSA78(78);
   const ScreenedRutherfordScatteringAngle SRSA79(79);
   const ScreenedRutherfordScatteringAngle SRSA80(80);
   const ScreenedRutherfordScatteringAngle SRSA81(81);
   const ScreenedRutherfordScatteringAngle SRSA82(82);
   const ScreenedRutherfordScatteringAngle SRSA83(83);
   const ScreenedRutherfordScatteringAngle SRSA84(84);
   const ScreenedRutherfordScatteringAngle SRSA85(85);
   const ScreenedRutherfordScatteringAngle SRSA86(86);
   const ScreenedRutherfordScatteringAngle SRSA87(87);
   const ScreenedRutherfordScatteringAngle SRSA88(88);
   const ScreenedRutherfordScatteringAngle SRSA89(89);
   const ScreenedRutherfordScatteringAngle SRSA90(90);
   const ScreenedRutherfordScatteringAngle SRSA91(91);
   const ScreenedRutherfordScatteringAngle SRSA92(92);
   const ScreenedRutherfordScatteringAngle SRSA93(93);
   const ScreenedRutherfordScatteringAngle SRSA94(94);
   const ScreenedRutherfordScatteringAngle SRSA95(95);
   const ScreenedRutherfordScatteringAngle SRSA96(96);

   ScreenedRutherfordScatteringAngle const * mScatter[113] = {
      &SRSA1,
      &SRSA2,
      &SRSA3,
      &SRSA4,
      &SRSA5,
      &SRSA6,
      &SRSA7,
      &SRSA8,
      &SRSA9,
      &SRSA10,
      &SRSA11,
      &SRSA12,
      &SRSA13,
      &SRSA14,
      &SRSA15,
      &SRSA16,
      &SRSA17,
      &SRSA18,
      &SRSA19,
      &SRSA20,
      &SRSA21,
      &SRSA22,
      &SRSA23,
      &SRSA24,
      &SRSA25,
      &SRSA26,
      &SRSA27,
      &SRSA28,
      &SRSA29,
      &SRSA30,
      &SRSA31,
      &SRSA32,
      &SRSA33,
      &SRSA34,
      &SRSA35,
      &SRSA36,
      &SRSA37,
      &SRSA38,
      &SRSA39,
      &SRSA40,
      &SRSA41,
      &SRSA42,
      &SRSA43,
      &SRSA44,
      &SRSA45,
      &SRSA46,
      &SRSA47,
      &SRSA48,
      &SRSA49,
      &SRSA50,
      &SRSA51,
      &SRSA52,
      &SRSA53,
      &SRSA54,
      &SRSA55,
      &SRSA56,
      &SRSA57,
      &SRSA58,
      &SRSA59,
      &SRSA60,
      &SRSA61,
      &SRSA62,
      &SRSA63,
      &SRSA64,
      &SRSA65,
      &SRSA66,
      &SRSA67,
      &SRSA68,
      &SRSA69,
      &SRSA70,
      &SRSA71,
      &SRSA72,
      &SRSA73,
      &SRSA74,
      &SRSA75,
      &SRSA76,
      &SRSA77,
      &SRSA78,
      &SRSA79,
      &SRSA80,
      &SRSA81,
      &SRSA82,
      &SRSA83,
      &SRSA84,
      &SRSA85,
      &SRSA86,
      &SRSA87,
      &SRSA88,
      &SRSA89,
      &SRSA90,
      &SRSA91,
      &SRSA92,
      &SRSA93,
      &SRSA94,
      &SRSA95,
      &SRSA96
   };

   ScreenedRutherfordRandomizedScatterFactory::ScreenedRutherfordRandomizedScatterFactory() : RandomizedScatterFactoryT("Screened Rutherford elastic cross-section", REFERENCE)
   {
   }

   const RandomizedScatterT& ScreenedRutherfordRandomizedScatterFactory::get(const ElementT& elm) const
   {
      return *mScatter[elm.getAtomicNumber()];
   }

   void ScreenedRutherfordRandomizedScatterFactory::initializeDefaultStrategy()
   {
   }

   const ScreenedRutherfordRandomizedScatterFactory FactoryScreenedRutherford;
   const RandomizedScatterFactoryT& Factory = FactoryScreenedRutherford;
}
