#include "hip/hip_runtime.h"
#include "ElementTest.cuh"
#include "..\EPQLibrary\Element.cuh"
#include "..\EPQLibrary\ToSI.cuh"

#include <stdio.h>

extern __device__ double fabs(double);

namespace ElementTest
{
   __device__ void assertEquals(double src, double target, double delta)
   {
      bool b = fabs(src - target) < delta;
      if (!b) {
         printf("values are different: %lf, %lf\n", src, target);
      }
   }

   __device__ void assertEquals(int src, int target)
   {
      if (src != target) {
         printf("values are different: %d, %d\n", src, target);
      }
   }

   __device__ void assertTrue(bool expr)
   {
      if (!expr) {
         printf("expr is not true\n");
      }
   }

   //__device__ void ElementTest::testOne()
   //{
   //   const Element elm = Element.byAtomicNumber(Element.elmTi);
   //   assertEquals(FromSI.eV(elm.meanIonizationPotential()), 247.24, 1.0);
   //   assertEquals(elm.getAtomicWeight(), 47.9, 1.0e-1);
   //   assertEquals(elm.getAtomicNumber(), 22);
   //   assertTrue(elm.compareTo(Element.byAtomicNumber(Element.elmH)) > 0);
   //   assertTrue(elm.compareTo(Element.byAtomicNumber(Element.elmFe)) < 0);
   //   assertTrue(elm.compareTo(Element.byAtomicNumber(Element.elmTi)) == 0);
   //   assertTrue(elm.equals(Element.byName("Ti")));
   //}
}
