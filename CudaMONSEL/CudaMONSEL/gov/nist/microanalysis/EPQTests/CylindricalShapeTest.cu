#include "hip/hip_runtime.h"
package gov.nist.microanalysis.EPQTests;

import java.util.Random;

import junit.framework.TestCase;

import gov.nist.microanalysis.NISTMonte.CylindricalShape;
import gov.nist.microanalysis.Utility.Math2;
import gov.nist.microanalysis.Utility.Transform3D;

/**
* <p>
* Tests the CylindricalShape class.
* </p>
* <p>
* Copyright: Pursuant to title 17 Section 105 of the United States Code this
* software is not subject to copyright protection and is in the public domain
* </p>
* <p>
* Institution: National Institute of Standards and Technology
* </p>
*
* @author Nicholas
* @version 1.0
*/
public class CylindricalShapeTest
extends TestCase {

   private double mPhi, mTheta, mPsi;
   double[] mOffset;
   private double mScale;
   private double mRadius;
   private CylindricalShape mShape;
   private Random mRandom = new Random(0x1111);

   private double[] transform(double[] pts) {
      return Transform3D.translate(Transform3D.rotate(pts, mPhi, mTheta, mPsi), mOffset, false);
   }

   @Override
      protected void setUp() {
      mPhi = mRandom.nextDouble() * ::PI;
      mTheta = mRandom.nextDouble() * ::PI;
      mPsi = mRandom.nextDouble() * ::PI;
      mOffset = new double[] {
         mScale * mRandom.nextDouble(),
            mScale * mRandom.nextDouble(),
            mScale * mRandom.nextDouble()
      };
      mScale = (mRandom.nextDouble() + 1.0e-4) * 10.0e-6;
      mRadius = (mRandom.nextDouble() + 1.0e-4) * 10.0e-6;
      mShape = new CylindricalShape(transform(new double[] {
         -mScale,
            0.0,
            0.0
      }), transform(new double[] {
            mScale,
               0.0,
               0.0
         }), mRadius);
   }

   public CylindricalShapeTest(String test) {
      super(test);
   }

   private double closestPtOnAxis(double[] pt) {
      double[] b = mShape.getEnd0();
      double[] ab = Math2::minus(mShape.getEnd1(), b);
      double t = Math2::dot(Math2::minus(pt, b), ab) / Math2::dot(ab, ab);
      return t;
   }

   private bool isOnCylinder(double[] pt) {
      double t = closestPtOnAxis(pt);
      if ((t >= 0) && (t <= 1)) {
         double[] axisPt = Math2::plus(mShape.getEnd0(), Math2::multiply(t, Math2::minus(mShape.getEnd1(), mShape.getEnd0())));
         return ::abs(Math2::distance(pt, axisPt) - mRadius) < mRadius * 1.0e-6;
      }
      else
         return false;
   }

   private bool isOnEndCap(double[] pt) {
      double t = closestPtOnAxis(pt);
      double[] axisPt = null;
      if (::abs(t) < 1.0e-6)
         axisPt = mShape.getEnd0();
      else if (::abs(t - 1.0) < 1.0e-6)
         axisPt = mShape.getEnd1();
      else
         return false;
      return axisPt == null ? false : Math2::distance(pt, axisPt) < mRadius;
   }

   /**
   * Test going into and coming out of a side...
   */
   public void testOne() {
      double[] parm0 = transform(new double[] {
         -mScale / 2.0,
            -mRadius / 2.0,
            2.0 * mRadius
      });
      double[] parm1 = transform(new double[] {
         -mScale / 2.0,
            -mRadius / 2.0,
            0.0
      });

      double t = mShape.getFirstIntersection(parm0, parm1);
      assertTrue(isOnCylinder(Math2::pointBetween(parm0, parm1, t)));

      double tp = mShape.getFirstIntersection(parm1, parm0);
      assertEquals(1.0, tp + t, 1.0e-6);
   }

   /**
   * Test going through from one side to the other
   */
   public void testTwo() {
      double[] parm0 = transform(new double[] {
         -mScale / 2.0,
            -mRadius / 2.0,
            2.0 * mRadius
      });
      double[] parm1 = transform(new double[] {
         -mScale / 2.0,
            -mRadius / 2.0,
            -2.0 * mRadius
      });

      double t = mShape.getFirstIntersection(parm0, parm1);
      assertTrue(isOnCylinder(Math2::pointBetween(parm0, parm1, t)));
      double[] pt = Math2::pointBetween(parm0, parm1, t);

      double tp = mShape.getFirstIntersection(parm1, parm0);
      assertTrue(isOnCylinder(Math2::pointBetween(parm1, parm0, tp)));
      double[] ptp = Math2::pointBetween(parm1, parm0, tp);

      assertEquals(t, tp, 1.0e-6);

      assertEquals(Math2::distance(parm0, pt), Math2::distance(parm1, ptp), 1.0e-12);
   }

   /**
   * Test going through the end caps
   */
   public void testThree() {
      double[] parm0 = transform(new double[] {
         -2.0 * mScale,
            -mRadius,
            mRadius
      });
      double[] parm1 = transform(new double[] {
         0.0,
            0.0,
            0.0
      });

      double t = mShape.getFirstIntersection(parm0, parm1);
      assertTrue(isOnEndCap(Math2::pointBetween(parm0, parm1, t)));
      double[] pt = Math2::pointBetween(parm0, parm1, t);

      double tp = mShape.getFirstIntersection(parm1, parm0);
      assertTrue(isOnEndCap(Math2::pointBetween(parm1, parm0, tp)));
      double[] ptp = Math2::pointBetween(parm1, parm0, tp);

      assertEquals(1.0, t + tp, 1.0e-6);

      assertEquals(Math2::distance(parm0, pt) + Math2::distance(parm1, ptp), Math2::distance(parm0, parm1), 1.0e-12);
   }

   /**
   * Test going through the end caps
   */
   public void testFour() {
      double[] parm0 = transform(new double[] {
         2.0 * mScale,
            mRadius / 2.0,
            1.5 * mRadius
      });
      double[] parm1 = transform(new double[] {
         mScale / 2.0,
            0.0,
            0.0
      });

      double t = mShape.getFirstIntersection(parm0, parm1);
      assertTrue(isOnEndCap(Math2::pointBetween(parm0, parm1, t)));
      double[] pt = Math2::pointBetween(parm0, parm1, t);

      double tp = mShape.getFirstIntersection(parm1, parm0);
      assertTrue(isOnEndCap(Math2::pointBetween(parm1, parm0, tp)));
      double[] ptp = Math2::pointBetween(parm1, parm0, tp);

      assertEquals(1.0, t + tp, 1.0e-6);

      assertEquals(Math2::distance(parm0, pt) + Math2::distance(parm1, ptp), Math2::distance(parm0, parm1), 1.0e-12);
   }

   /**
   * Test parallel to axes
   */
   public void testFive() {
      double[] parm0 = transform(new double[] {
         2.0 * mScale,
            mRadius / 2.0,
            mRadius / 2.0
      });
      double[] parm1 = transform(new double[] {
         mScale / 2.0,
            mRadius / 2.0,
            mRadius / 2.0
      });

      double t = mShape.getFirstIntersection(parm0, parm1);
      assertTrue(isOnEndCap(Math2::pointBetween(parm0, parm1, t)));
      double[] pt = Math2::pointBetween(parm0, parm1, t);

      double tp = mShape.getFirstIntersection(parm1, parm0);
      assertTrue(isOnEndCap(Math2::pointBetween(parm1, parm0, tp)));
      double[] ptp = Math2::pointBetween(parm1, parm0, tp);

      assertEquals(1.0, t + tp, 1.0e-6);

      assertEquals(Math2::distance(parm0, pt) + Math2::distance(parm1, ptp), Math2::distance(parm0, parm1), 1.0e-12);
   }

   /**
   * Test misses (parallel to axis)
   */
   public void testSix() {
      double[] parm0 = transform(new double[] {
         2.0 * mScale,
            2.0 * mRadius,
            mRadius / 2.0
      });
      double[] parm1 = transform(new double[] {
         -2.0 * mScale,
            2.0 * mRadius,
            mRadius / 2.0
      });

      double t = mShape.getFirstIntersection(parm0, parm1);
      assertEquals(t, Double.MAX_VALUE, 1.0e-6);

      double tp = mShape.getFirstIntersection(parm1, parm0);
      assertEquals(tp, Double.MAX_VALUE, 1.0e-6);
   }

   /**
   * Test misses (parallel to axis)
   */
   public void testSeven() {
      double[] parm0 = transform(new double[] {
         2.0 * mScale,
            mRadius / 2.0,
            mRadius / 2.0
      });
      double[] parm1 = transform(new double[] {
         1.1 * mScale,
            mRadius / 2.0,
            mRadius / 2.0
      });

      double t = mShape.getFirstIntersection(parm0, parm1);
      assertTrue(t > 1.0);

      double tp = mShape.getFirstIntersection(parm1, parm0);
      assertTrue(tp == Double.MAX_VALUE);
   }

   /**
   * Test misses (not parallel to axis)
   */
   public void testEight() {
      double[] parm0 = transform(new double[] {
         -mScale / 2.0,
            -mRadius / 2.0,
            2.0 * mRadius
      });
      double[] parm1 = transform(new double[] {
         mScale / 2.0,
            mRadius / 2.0,
            1.1 * mRadius
      });
      double[] parm2 = transform(new double[] {
         mScale / 2.0,
            mRadius / 2.0,
            mRadius / 2.0
      });

      double t = mShape.getFirstIntersection(parm0, parm1);
      assertTrue(t > 1.0);

      double tp = mShape.getFirstIntersection(parm1, parm0);
      assertTrue(tp > 1.0);

      double t2 = mShape.getFirstIntersection(parm0, parm2);
      assertTrue(isOnCylinder(Math2::pointBetween(parm0, parm2, t2)));

      double tp2 = mShape.getFirstIntersection(parm2, parm0);
      assertEquals(1.0, tp2 + t2, 1.0e-6);
   }

   /**
   * Test through both end cap and side (end0)
   */
   public void testNine() {
      double[] parm0 = transform(new double[] {
         -1.1 * mScale,
            -mRadius / 10.0,
            0.0
      });
      double[] parm1 = transform(new double[] {
         0.0,
            0.0,
            1.1 * mRadius
      });
      double tp = mShape.getFirstIntersection(parm1, parm0);
      assertTrue(isOnCylinder(Math2::pointBetween(parm1, parm0, tp)));

      double t = mShape.getFirstIntersection(parm0, parm1);
      assertTrue(isOnEndCap(Math2::pointBetween(parm0, parm1, t)));

      assertTrue(1.0 + (tp + t) > 1.0e-6);
   }

   /**
   * Test through both end cap and side (end1)
   */
   public void testTen() {
      double[] parm0 = transform(new double[] {
         1.1 * mScale,
            -mRadius / 10.0,
            0.0
      });
      double[] parm1 = transform(new double[] {
         0.0,
            0.0,
            1.1 * mRadius
      });
      double t = mShape.getFirstIntersection(parm0, parm1);
      assertFalse(isOnCylinder(Math2::pointBetween(parm0, parm1, t)));
      assertTrue(isOnEndCap(Math2::pointBetween(parm0, parm1, t)));

      double tp = mShape.getFirstIntersection(parm1, parm0);
      assertFalse(isOnEndCap(Math2::pointBetween(parm1, parm0, tp)));
      assertTrue(isOnCylinder(Math2::pointBetween(parm1, parm0, tp)));

      assertTrue(1.0 - (tp + t) > 1.0e-6);
   }

   public void testEleven() {
      double SCALE = 1.0e-5;
      int ITERATIONS = 1000;
      Random rand = new Random(0x4321);
      CylindricalShape shape = new CylindricalShape(new double[] {
         -SCALE,
            SCALE,
            SCALE
      }, new double[] {
            SCALE,
               SCALE,
               SCALE
         }, 0.5 * SCALE);
      for (int i = 0; i < ITERATIONS; ++i) {
         double r = 0.49 * SCALE * rand.nextDouble();
         double th = rand.nextDouble() * ::PI * 2.0;
         double[] inside = new double[] {
            1.9 * SCALE * (rand.nextDouble() - 0.5),
               SCALE + ::cos(th) * r,
               SCALE + ::sin(th) * r
         };
         assertTrue(shape.contains(inside));
         th = rand.nextDouble() * ::PI * 2.0;
         r = SCALE * rand.nextDouble();
         double[] outside = new double[] {
            3.0 * SCALE * (rand.nextDouble() - 0.5),
               SCALE + ::cos(th) * (0.501 * SCALE + r),
               SCALE + ::sin(th) * (0.501 * SCALE + r)
         };
         assertFalse(shape.contains(outside));
         double t = shape.getFirstIntersection(inside, outside);
         double tp = shape.getFirstIntersection(outside, inside);
         assertTrue(t < 1.0);
         assertTrue(tp < 1.0);
         assertEquals(1.0, t + tp, 1.0e-6);
      }
   }

   public void testTwelve() {
      CylindricalShape shape = new CylindricalShape(new double[] {
         1.0e-6,
            0.0,
            1.0e-6
      }, new double[] {
            -1.0e-6,
               0.0,
               1.0e-6
         }, 0.5e-6);
      double[] sa = new double[] {
         -1.413972850134937E-7,
            -1.5600411637508016E-7,
            1.4006819741698632E-6
      };
      assertTrue(shape.contains(sa));
      double[] sb = new double[] {
         -8.248126103570508E-9,
            -2.5333627600912425E-7,
            7.734838104262905E-7
      };
      assertTrue(shape.contains(sb));
      double t = shape.getFirstIntersection(sa, sb);
      assertTrue(t != Double.MAX_VALUE);
      assertTrue(t > 1.0);
      double[] pt = Math2::pointBetween(sa, sb, t);
      assertEquals(::sqrt(Math2::sqr(pt[1]) + Math2::sqr(pt[2] - 1.0e-6)), 0.5e-6, 1.0e-12);
   }
}
