#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\EPQTests\EdgeEnergyTest.cuh"
#include "gov\nist\microanalysis\EPQLibrary\AtomicShell.cuh"
#include "gov\nist\microanalysis\EPQLibrary\EdgeEnergy.cuh"
#include "gov\nist\microanalysis\EPQLibrary\ToSI.cuh"
#include "gov\nist\microanalysis\EPQLibrary\FromSI.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Composition.cuh"
#include "gov\nist\microanalysis\EPQLibrary\MeanIonizationPotential.cuh"
#include "gov\nist\microanalysis\EPQLibrary\MaterialFactory.cuh"
#include "gov\nist\microanalysis\NISTMonte\Declarations.cuh"

namespace MeanIonizationPotentialTest
{
   static void assertEquals(double a, double b, double err) {
      if (abs(a - b) >= err) printf("wrong: %.10e, %.10e\n", a, b);
   }

   void testOne()
   {
      CompositionT k3189 = MaterialFactory::createMaterial(MaterialFactory::K3189);
      if (false)
         for (auto el : k3189.getElementSet()) {
            printf(el->toString());
            printf("\t");
            printf("%.8e", FromSI::eV(MeanIonizationPotential::Berger64.compute(*el)));
            printf("\t");
            printf("%.10e\n", k3189.weightFraction(*el, true));
         }

      // Berger64
      assertEquals(FromSI::keV(MeanIonizationPotential::Berger64.compute(Element::Si)), 0.172, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Berger64.compute(Element::Al)), 0.163, 0.002);
      assertEquals(FromSI::keV(MeanIonizationPotential::Berger64.computeLn(k3189)), 0.159, 0.001);

      assertEquals(FromSI::keV(MeanIonizationPotential::Berger64.compute(Element::Ca)), 0.228, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Berger64.compute(Element::Ti)), 0.247, 0.001);

      // Duncumb69
      assertEquals(FromSI::keV(MeanIonizationPotential::Duncumb69.compute(Element::Si)), 0.154, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Duncumb69.compute(Element::Al)), 0.142, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Duncumb69.computeLn(k3189)), 0.162, 0.001);

      assertEquals(FromSI::keV(MeanIonizationPotential::Duncumb69.compute(Element::Ca)), 0.239, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Duncumb69.compute(Element::Ti)), 0.270, 0.001);
      // Bloch33
      assertEquals(FromSI::keV(MeanIonizationPotential::Bloch33.compute(Element::Si)), 0.189, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Bloch33.compute(Element::Al)), 0.176, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Bloch33.computeLn(k3189)), 0.165, 0.001);

      assertEquals(FromSI::keV(MeanIonizationPotential::Bloch33.compute(Element::Ca)), 0.270, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Bloch33.compute(Element::Ti)), 0.297, 0.001);

      // Zeller75
      assertEquals(FromSI::keV(MeanIonizationPotential::Zeller75.compute(Element::Si)), 0.174, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Zeller75.compute(Element::Al)), 0.164, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Zeller75.computeLn(k3189)), 0.157, 0.001);

      assertEquals(FromSI::keV(MeanIonizationPotential::Bloch33.compute(Element::Ca)), 0.270, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Bloch33.compute(Element::Ti)), 0.297, 0.001);

      // Springer67
      assertEquals(FromSI::keV(MeanIonizationPotential::Springer67.compute(Element::Si)), 0.154, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Springer67.compute(Element::Al)), 0.143, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Springer67.computeLn(k3189)), 0.137, 0.001);

      assertEquals(FromSI::keV(MeanIonizationPotential::Springer67.compute(Element::Ca)), 0.216, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Springer67.compute(Element::Ti)), 0.238, 0.001);
      // Heinrich70 (disagrees with TryZAF)
      assertEquals(FromSI::keV(MeanIonizationPotential::Heinrich70.compute(Element::Si)), 0.179, 0.001);
      // assertEquals(FromSI::keV(MeanIonizationPotential::Heinrich70.computeLn(k3189)),0.171,0.001);
      // Wilson41
      assertEquals(FromSI::keV(MeanIonizationPotential::Wilson41.compute(Element::Si)), 0.161, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Wilson41.compute(Element::Al)), 0.149, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Wilson41.computeLn(k3189)), 0.141, 0.001);

      assertEquals(FromSI::keV(MeanIonizationPotential::Wilson41.compute(Element::Ca)), 0.230, 0.001);
      assertEquals(FromSI::keV(MeanIonizationPotential::Wilson41.compute(Element::Ti)), 0.253, 0.001);

      // Untested Berger83, Sternheimer64
      assertEquals(MeanIonizationPotential::Berger64.compute(Element::Si), MeanIonizationPotential::Berger83.compute(Element::Si), 0.001);

      printf("MeanIonizationPotentialTest::testOne() completed.\n");
   }
}
