#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\NISTMonte\RegionBase.cuh"
#include "gov\nist\microanalysis\NISTMonte\Electron.cuh"
#include "gov\nist\microanalysis\NISTMonte\MonteCarloSS.cuh"

namespace Electron
{
   static long lastID = 0; // ID of last generated electron

   long getlastIdent()
   {
      return lastID;
   }

   void Electron::Init(double initialPos[], double theta, double phi, double kE)
   {
      mPosition.assign(initialPos, initialPos + 3);
      mPrevPosition.assign(initialPos, initialPos + 3);
      mScatteringElement = (NULL);
      mCurrentRegion = (NULL);
      mPrevRegion = (NULL);
      mEnergy = (kE);
      previousEnergy = (kE);
      mTheta = (theta);
      mPhi = (phi);
      mStepCount = (0);
      mTrajectoryComplete = (false);
      ident = (++lastID);
   }

   Electron::Electron(double initialPos[], double kE)
   {
      Init(initialPos, 0., 0., kE);
   }

   Electron::Electron(double initialPos[], double theta, double phi, double kE)
   {
      Init(initialPos, theta, phi, kE);
   }

   Electron::Electron(const Electron& parent, double theta, double phi, double kE)
   {
      Init(parent.getPosition().data(), theta, phi, kE);
      mCurrentRegion = parent.getCurrentRegion();
      mPrevRegion = mCurrentRegion;
      parentID = parent.getIdent();
   }

   void Electron::setDirection(double theta, double phi)
   {
      mTheta = theta;
      mPhi = phi;
   }

   PositionVecT Electron::getPosition() const
   {
      return mPosition;
   }

   void Electron::setPosition(double newpos[])
   {
      mPosition.assign(newpos, newpos + 3);
   }

   PositionVecT Electron::getPrevPosition() const
   {
      return mPrevPosition;
   }

   const RegionBaseT* Electron::getCurrentRegion() const
   {
      return mCurrentRegion;
   }

   const RegionBaseT* Electron::getPreviousRegion() const
   {
      return mPrevRegion;
   }

   double Electron::getEnergy() const
   {
      return mEnergy;
   }

   double Electron::getPreviousEnergy() const
   {
      return previousEnergy;
   }

   int Electron::getStepCount() const
   {
      return mStepCount;
   }

   double Electron::stepLength() const
   {
      return MonteCarloSS::dist(mPrevPosition.data(), mPosition.data());
   }

   PositionVecT Electron::candidatePoint(double dS) const
   {
      double st = ::sin(mTheta);
      // Calculate the new point as dS distance from mPosition
      return PositionVecT ({
         mPosition[0] + dS * ::cos(mPhi) * st,
            mPosition[1] + dS * ::sin(mPhi) * st,
            mPosition[2] + dS * ::cos(mTheta)
      });
   }

   void Electron::updateDirection(double dTheta, double dPhi)
   {
      // The candidate point is computed by rotating the current trajectory back
      // to the z-axis, deflecting the z-axis by dTheta down from the z-axis and
      // dPhi around the z-axis, then finally rotating back to the original
      // trajectory.

      double ct = ::cos(mTheta), st = ::sin(mTheta);
      double cp = ::cos(mPhi), sp = ::sin(mPhi);
      double ca = ::cos(dTheta), sa = ::sin(dTheta);
      double cb = ::cos(dPhi);

      double xx = cb * ct * sa + ca * st;
      double yy = sa * ::sin(dPhi);
      double dx = cp * xx - sp * yy;
      double dy = cp * yy + sp * xx;
      double dz = ca * ct - cb * sa * st;

      mTheta = ::atan2(::sqrt(dx * dx + dy * dy), dz);
      mPhi = ::atan2(dy, dx);
   }

   void Electron::move(double newPoint[], double dE)
   {
      // Update mPrevPosition and then mPosition
      mPrevPosition = mPosition;
      mPosition.assign(newPoint, newPoint + mPosition.size());

      // Update the energy
      previousEnergy = mEnergy;
      mEnergy += dE;
      ++mStepCount;
   }

   void Electron::setEnergy(double newEnergy)
   {
      mEnergy = newEnergy;
   }

   void Electron::setPreviousEnergy(double newPreviousEnergy)
   {
      previousEnergy = newPreviousEnergy;
   }

   void Electron::setCurrentRegion(const RegionBaseT* reg)
   {
      mPrevRegion = mCurrentRegion;
      mCurrentRegion = reg;
   }

   const ElementT* Electron::getScatteringElement() const
   {
      return mScatteringElement;
   }

   void Electron::setScatteringElement(const ElementT* scatteringElement)
   {
      mScatteringElement = scatteringElement;
   }

   double Electron::getPhi() const
   {
      return mPhi;
   }

   double Electron::getTheta() const
   {
      return mTheta;
   }

   bool Electron::isTrajectoryComplete() const
   {
      return mTrajectoryComplete;
   }

   void Electron::setTrajectoryComplete(bool trajectoryComplete)
   {
      mTrajectoryComplete = trajectoryComplete;
   }

   long Electron::getIdent() const
   {
      return ident;
   }

   long Electron::getParentID() const
   {
      return parentID;
   }

   double DefaultPos[] = { INT_MAX, INT_MAX, INT_MAX };
   Electron Default(DefaultPos, INT_MAX);
}