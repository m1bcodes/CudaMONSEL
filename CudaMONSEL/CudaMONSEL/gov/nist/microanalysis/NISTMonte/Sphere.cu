#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\NISTMonte\Sphere.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"
//import gov.nist.microanalysis.Utility.Transform3D;

namespace Sphere
{
   Sphere::Sphere(const double center[], double radius) : mRadius(radius), mCenter(center, center+3)
   {
   }

   bool Sphere::contains(const double pos[]) const
   {
      return Math2::sqr(pos[0] - mCenter[0]) + Math2::sqr(pos[1] - mCenter[1]) + Math2::sqr(pos[2] - mCenter[2]) <= Math2::sqr(mRadius);
   }

   double Sphere::getRadius() const
   {
      return mRadius;
   }

   double Sphere::getFirstIntersection(const double pos0[], const double pos1[])
   {
      // Compute the intersection of the line between pos0 and pos1 and the
      // shell of the sphere.
      const PositionVecT d = Math2::minus(PositionVecT(pos1, pos1 + 3), PositionVecT(pos0, pos0 + 3));
      const PositionVecT m = Math2::minus(PositionVecT(pos0, pos0 + 3), mCenter);
      const double ma2 = -2.0 * Math2::dot(d, d);
      const double b = 2.0 * Math2::dot(m, d);
      const double c2 = 2.0 * (Math2::dot(m, m) - mRadius * mRadius);
      const double f = b * b + ma2 * c2;
      if (f >= 0) {
         double up = (b + ::sqrt(f)) / ma2;
         double un = (b - ::sqrt(f)) / ma2;
         if (up < 0.0)
            up = INT_MAX;
         if (un < 0.0)
            un = INT_MAX;
         const double res = ::fmin(up, un);
         if (!((res == INT_MAX) || (Math2::magnitude(Math2::plus(m, Math2::multiply(res, d))) - mRadius < ::fmax(1.0e-12, Math2::distance(PositionVecT(pos0, pos0 + 3), PositionVecT(pos1, pos1 + 3)) * 1.0e-9)))) {
            printf("%s\n", std::to_string(Math2::magnitude(Math2::plus(m, Math2::multiply(res, d))) - mRadius).c_str());
         }
         return res;
      }
      return INT_MAX;
   }

   PositionVecT Sphere::getInitialPoint() const
   {
      double res[3];
      res[0] = mCenter[0];
      res[1] = mCenter[1];
      res[2] = mCenter[2] - 0.999 * mRadius; // just inside...
      return PositionVecT(res, res+3);
   }

   PositionVecT Sphere::getPointAt(double phi, double theta, double frac) const
   {
      PositionVecT res(3, 0);
      res[2] = mCenter[2] + mRadius * frac * ::cos(phi);
      res[1] = mCenter[1] + mRadius * frac * ::sin(phi) * ::sin(theta);
      res[0] = mCenter[0] + mRadius * frac * ::sin(phi) * ::cos(theta);
      return res;
   }

   // JavaDoc in ITransform
   void Sphere::rotate(const double pivot[], double phi, double theta, double psi)
   {
      //mCenter = Transform3D.rotate(mCenter, pivot, phi, theta, psi);
   }

   // JavaDoc in ITransform
   void Sphere::translate(const double distance[])
   {
      mCenter[0] += distance[0];
      mCenter[1] += distance[1];
      mCenter[2] += distance[2];
   }

   //void render(TrajectoryVRML.RenderContext vra, Writer wr)
   //   throws IOException
   //{
   //   final NumberFormat nf = NumberFormat.getNumberInstance(Locale.US);
   //   nf.setMaximumFractionDigits(2);
   //   nf.setGroupingUsed(false);
   //   final Color color = vra.getCurrentColor();
   //   final String trStr = nf.format(vra.getTransparency());
   //   final String colorStr = nf.format(color.getRed() / 255.0) + " " + nf.format(color.getGreen() / 255.0) + " "
   //      + nf.format(color.getBlue() / 255.0);
   //   wr.append("Transform {\n");
   //   wr.append(" translation " + nf.format(mCenter[0] / TrajectoryVRML.SCALE) + " "
   //      + nf.format(mCenter[1] / TrajectoryVRML.SCALE) + " " + nf.format(mCenter[2] / TrajectoryVRML.SCALE) + "\n");
   //   wr.append(" children [\n");
   //   wr.append("  Shape {\n");
   //   wr.append("   geometry Sphere { radius " + nf.format(mRadius / TrajectoryVRML.SCALE) + "}\n");
   //   wr.append("   appearance Appearance {\n");
   //   wr.append("    material Material {\n");
   //   wr.append("     emissiveColor " + colorStr + "\n");
   //   wr.append("     transparency " + trStr + "\n");
   //   wr.append("    }\n");
   //   wr.append("   }\n");
   //   wr.append("  }\n");
   //   wr.append(" ]\n");
   //   wr.append("}");
   //   wr.flush();
   //}

   PositionVecT Sphere::getCenter() const
   {
      return mCenter;
   }

   StringT Sphere::toString() const
   {
      return "Sphere[" + std::to_string(mCenter[0]) + std::to_string(mCenter[1]) + std::to_string(mCenter[2]) + ", r=" + std::to_string(mRadius) + "]";
   }
}