#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\Utility\Math2.cuh"
#include "gov\nist\microanalysis\NISTMonte\GaussianBeam.cuh"
#include "gov\nist\microanalysis\NISTMonte\Electron.cuh"
#include "gov\nist\microanalysis\NISTMonte\MonteCarloSS.cuh"

namespace GaussianBeam
{
   GaussianBeam::GaussianBeam(double width)
   {
      mCenter = Math2::multiply(0.99 * MonteCarloSS::ChamberRadius, PositionVecT(Math2::MINUS_Z_AXIS, Math2::MINUS_Z_AXIS + 3));
      mWidth = width;
   }

   void GaussianBeam::setWidth(double width)
   {
      mWidth = width;
   }

   double GaussianBeam::getWidth()
   {
      return mWidth;
   }

   void GaussianBeam::setBeamEnergy(double beamEnergy)
   {
      mBeamEnergy = beamEnergy;
   }

   double GaussianBeam::getBeamEnergy() const
   {
      return mBeamEnergy;
   }

   void GaussianBeam::setCenter(const double center[])
   {
      mCenter.assign(center, center + 3);
   }

   PositionVecT GaussianBeam::getCenter() const
   {
      return mCenter;
   }

   ElectronT GaussianBeam::createElectron() const
   {
      double initialPos[3 * sizeof(double)];
      memcpy(initialPos, mCenter.data(), sizeof(mCenter));
      double r = ::sqrt(-2. * ::log((double)rand() / RAND_MAX)) * mWidth;
      double th = 2.0 * 3.14159265358979323846 * ((double)rand() / RAND_MAX);
      initialPos[0] += r * ::cos(th);
      initialPos[1] += r * ::sin(th);

      return ElectronT(initialPos, mBeamEnergy);
   }
}