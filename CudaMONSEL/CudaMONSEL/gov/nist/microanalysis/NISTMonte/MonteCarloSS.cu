#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\NISTMonte\MonteCarloSS.cuh"
#include "gov\nist\microanalysis\NISTMonte\Electron.cuh"
#include "gov\nist\microanalysis\NISTMonte\IMaterialScatterModel.cuh"
#include "gov\nist\microanalysis\NISTMonte\GaussianBeam.cuh"
#include "gov\nist\microanalysis\NISTMonte\Sphere.cuh"
#include "gov\nist\microanalysis\NISTMonte\RegionBase.cuh"
#include "gov\nist\microanalysis\EPQLibrary\ToSI.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Detector\ElectronProbe.cuh"

#include "gov\nist\microanalysis\Utility\Math2.cuh"

namespace MonteCarloSS
{
   static const int ScatterEvent = 1;
   static const int NonScatterEvent = ScatterEvent + 1;
   static const int BackscatterEvent = ScatterEvent + 2;
   static const int ExitMaterialEvent = ScatterEvent + 3;
   static const int TrajectoryStartEvent = ScatterEvent + 4;
   static const int TrajectoryEndEvent = ScatterEvent + 5;
   static const int LastTrajectoryEvent = ScatterEvent + 6;
   static const int FirstTrajectoryEvent = ScatterEvent + 7;
   static const int StartSecondaryEvent = ScatterEvent + 8;
   static const int EndSecondaryEvent = ScatterEvent + 9;
   static const int PostScatterEvent = ScatterEvent + 10;

   static const int BeamEnergyChanged = 100;
   static const int XAxis = 0;
   static const int YAxis = 1;
   static const int ZAxis = 2;
   const float ChamberRadius = 0.1f;
   const float SMALL_DISP = 1.0e-15f;

   double dist(const double pos0[], const double pos1[])
   {
      return ::sqrt(Math2::sqr(pos1[0] - pos0[0]) + Math2::sqr(pos1[1] - pos0[1]) + Math2::sqr(pos1[2] - pos0[2]));
   }

   MonteCarloSS::MonteCarloSS(ElectronGunT const * gun, RegionT * chamber, ElectronT * electron) : mGun(gun), mChamber(chamber), mElectron(electron)
   {
      //TODO: shift the responsibility to the caller
      //const double center[] = {
      //   0.0,
      //   0.0,
      //   0.0
      //};
      //SphereT sphere(center, ChamberRadius);
      //mGun.setCenter(sphere.getInitialPoint().data());
      //mGun.setBeamEnergy(ToSI::keV(20.0));
      //mChamber = new RegionT(NULL, &NULL_MSM, &sphere);
   }

   //RegionT addSubRegion(const RegionT * parent, const MaterialT& mat, const ShapeT& shape)
   //{
   //   if (parent == NULL) printf("bad");
   //   return new RegionT(parent, new BasicMaterialModel(mat), shape);
   //}

   RegionT* addSubRegion(RegionT& parent, IMaterialScatterModelT& msm, const ShapeT& shape)
   {
      return new RegionT(&parent, &msm, &shape);
   }

   //public Map<Material, Double> getMaterialMap(double[] startPt, double[] endPt) { // used
   //   final HashMap<Material, Double> traj = new HashMap<Material, Double>();
   //   double[] start = startPt;
   //   RegionBase region = mChamber.containingSubRegion(start);
   //   final double eps = 1.0e-7;
   //   while ((region != null) && (distance(start, endPt) > eps)) {
   //      final double[] end = endPt.clone();
   //      final RegionBase nextRegion = region.findEndOfStep(start, end);
   //      double dist = distance(start, end);
   //      if (dist > 0.0) {
   //         if (traj.containsKey(region.getMaterial()))
   //            dist += (traj.get(region.getMaterial())).doubleValue();
   //         traj.put(region.getMaterial(), new Double(dist));
   //      }
   //      start = Math2.plus(end, Math2.multiply(SMALL_DISP, Math2.normalize(Math2.minus(endPt, start))));
   //      region = nextRegion;
   //   }
   //   return traj;
   //}

   //private void fireEvent(int event) {
   //   if (!(mEventListeners.isEmpty() || mDisableEvents)) {
   //      final ActionEvent ae = new ActionEvent(this, event, "MonteCarloSS event");
   //      for (final ActionListener sel : mEventListeners)
   //         sel.actionPerformed(ae);
   //   }
   //}

   const RegionT* MonteCarloSS::getChamber() const
   {
      return mChamber;
   }

   void MonteCarloSS::initializeTrajectory()
   {
      mElectron = mGun->createElectron();
      mElectron->setCurrentRegion(mChamber->containingSubRegion(mElectron->getPosition().data()));
      // Stop when you can't generate any more x-rays
      mElectron->setScatteringElement(NULL);
   }

   void MonteCarloSS::takeStep()
   {
      auto pos0 = mElectron->getPosition();

      auto currentRegion = mElectron->getCurrentRegion();
      if ((currentRegion == NULL) || !(currentRegion->getShape()->contains(pos0.data()))) {
         currentRegion = mChamber->containingSubRegion(pos0.data());
         mElectron->setCurrentRegion(currentRegion);
         if (currentRegion == NULL) {
            mElectron->setTrajectoryComplete(true);
            return;
         }
      }

      auto msm = currentRegion->getScatterModel();
      if (msm == nullptr) printf("MonteCarloSS::takeStep: msm is null\n");

      auto pos1 = mElectron->candidatePoint(msm->randomMeanPathLength(*mElectron));

      auto nextRegion = currentRegion->findEndOfStep(pos0.data(), pos1.data());
      mElectron->move(pos1.data(), msm->calculateEnergyLoss(dist(pos0.data(), pos1.data()), *mElectron));
      bool tc = (mElectron->getEnergy() < msm->getMinEforTracking()) || mElectron->isTrajectoryComplete();
      mElectron->setTrajectoryComplete(tc);
      if (!tc) {
         if (nextRegion == currentRegion) {
            if (mChamber == nullptr) printf("");
            if (mElectron == nullptr);
            if (currentRegion == nullptr);
            //fireEvent(ScatterEvent);
            auto secondary = msm->scatter(*mElectron);
            //fireEvent(PostScatterEvent);
            mElectron->setTrajectoryComplete((mElectron->getEnergy() < msm->getMinEforTracking()) || mElectron->isTrajectoryComplete());
            if (secondary != nullptr) {
               trackSecondaryElectron(secondary);
            }

            if (mElectron->getCurrentRegion() != currentRegion) printf("\n");
         }
         else if (nextRegion != nullptr) {
            //fireEvent(NonScatterEvent);
            auto secondary = msm->barrierScatter(mElectron, nextRegion);
            mElectron->setPosition(mElectron->candidatePoint(SMALL_DISP).data());
            if (!(mElectron->getCurrentRegion()->getShape()->contains(mElectron->getPosition().data())))
               mElectron->setCurrentRegion(mChamber->containingSubRegion(mElectron->getPosition().data()));

            //if (mElectron->getCurrentRegion() != currentRegion)
               //fireEvent(ExitMaterialEvent);
            if (secondary != nullptr) {
               secondary->setPosition(secondary->candidatePoint(SMALL_DISP).data());
               trackSecondaryElectron(secondary);
            }
         }
         else {
            //fireEvent(BackscatterEvent);
            mElectron->setCurrentRegion(nullptr);
            mElectron->setTrajectoryComplete(true);
         }
      }
   }

   void MonteCarloSS::trackSecondaryElectron(ElectronT* newElectron)
   {
      double mMinEnergy = newElectron->getCurrentRegion()->getScatterModel()->getMinEforTracking();
      if (newElectron->getEnergy() > mMinEnergy) {
         // fireEvent(StartSecondaryEvent);
         mElectronStack.push(mElectron);
         mElectron = newElectron;
         //fireEvent(StartSecondaryEvent);
      }
   }

   bool MonteCarloSS::allElectronsComplete()
   {
      bool tc = mElectron->isTrajectoryComplete();
      while (tc && (mElectronStack.size() > 0)) {
         //fireEvent(EndSecondaryEvent);
         mElectron = mElectronStack.top();
         mElectronStack.pop();

         tc = mElectron->isTrajectoryComplete();
      }
      return tc;
   }

   void MonteCarloSS::runTrajectory()
   {
      initializeTrajectory();
      //fireEvent(TrajectoryStartEvent);
      while (!allElectronsComplete())
         takeStep();
      //fireEvent(TrajectoryEndEvent);
   }

   void MonteCarloSS::runMultipleTrajectories(int n)
   {
      //fireEvent(FirstTrajectoryEvent);
      for (int i = 0; i < n; ++i)
         runTrajectory();
      //fireEvent(LastTrajectoryEvent);
   }

   double MonteCarloSS::getBeamEnergy()
   {
      return mGun->getBeamEnergy();
   }

   void MonteCarloSS::setElectronGun(ElectronGunT& gun)
   {
      gun.setBeamEnergy(mGun->getBeamEnergy());
      mGun = &gun;
   }

   //void MonteCarloSS::setBeamEnergy(double beamEnergy)
   //{
   //   mGun->setBeamEnergy(beamEnergy);
   //   //fireEvent(BeamEnergyChanged);
   //}

   PositionVecT MonteCarloSS::computeDetectorPosition(double elevation, double theta)
   {
      double frac = 0.999;
      double r = frac * ChamberRadius;
      //if (mChamber->mShape instanceof Sphere)
      //   r = frac * ((Sphere)mChamber.mShape).getRadius();
      return ElectronProbe::computePosition(0.0, elevation, theta, r);
   }

   //public Set<AtomicShell> getAtomicShellSet()
   //   throws EPQException{ // not used
   //   final Set<AtomicShell> res = new TreeSet<AtomicShell>();
   //   final Set<Element> elements = mChamber.getElements(true);
   //   for (final Object element : elements) {
   //      final Element el = (Element)element;
   //      for (int sh = AtomicShell.K; sh < AtomicShell.NI; ++sh) {
   //         final AtomicShell shell = new AtomicShell(el, sh);
   //         final double ee = shell.getEdgeEnergy();
   //         if ((ee > 0.0) && (ee < mGun.getBeamEnergy()) && (XRayTransition.getStrongestLine(shell) != null))
   //            res.add(shell);
   //      }
   //   }
   //   return res;
   //}

   Element::UnorderedSetT MonteCarloSS::getElementSet() const
   {
      return mChamber->getElements(true);
   }

   //public void estimateTrajectoryVolume(double[] c0, double[] c1) {
   //   c0[0] = (c0[1] = (c0[2] = Double.MAX_VALUE));
   //   c1[0] = (c1[1] = (c1[2] = -Double.MAX_VALUE));
   //   final int nTraj = 100;
   //   mDisableEvents = true;
   //   for (int i = 0; i < nTraj; ++i) {
   //      initializeTrajectory();
   //      while (!mElectron.isTrajectoryComplete()) {
   //         takeStep();
   //         final double[] endPt = mElectron.getPosition();
   //         final RegionBase endRegion = mChamber.containingSubRegion(endPt);
   //         if ((endRegion != null) && (endRegion != mChamber))
   //            for (int j = 0; j < 3; ++j) {
   //               if (endPt[j] < c0[j])
   //                  c0[j] = endPt[j];
   //               if (endPt[j] > c1[j])
   //                  c1[j] = endPt[j];
   //            }
   //      }
   //   }
   //   mDisableEvents = false;
   //}

   //void updateMaterial(const MaterialT& oldMat, const MaterialT& newMat)
   //{
   //   mChamber.updateMaterial(oldMat, new BasicMaterialModel(newMat));
   //}

   void MonteCarloSS::rotate(double pivot[], double phi, double theta, double psi)
   {
      for (const RegionBaseT* r : mChamber->getSubRegions()) {
         //if (r instanceof TransformableRegion) ((TransformableRegionT*)r)->rotate(pivot, phi, theta, psi);
      }
   }

   void MonteCarloSS::translate(double distance[])
   {
      for (const RegionBaseT* r : mChamber->getSubRegions()) {
         //if (r instanceof TransformableRegion) ((TransformableRegionT*)r)->translate(distance);
      }
   }

   const RegionBaseT* MonteCarloSS::findRegionContaining(double point[]) const
   {
      return mChamber->containingSubRegion(point);
   }
}
