#include "gov\nist\microanalysis\NISTMonte\MonteCarloSS.cuh"
#include "gov\nist\microanalysis\NISTMonte\Electron.cuh"
#include "gov\nist\microanalysis\NISTMonte\IMaterialScatterModel.cuh"
#include "gov\nist\microanalysis\NISTMonte\GaussianBeam.cuh"
#include "gov\nist\microanalysis\NISTMonte\Sphere.cuh"
#include "gov\nist\microanalysis\NISTMonte\RegionBase.cuh"
#include "gov\nist\microanalysis\EPQLibrary\ToSI.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Detector\ElectronProbe.cuh"

#include "gov\nist\microanalysis\Utility\ActionListener.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"

#include "Amphibian\Random.cuh"

namespace MonteCarloSS
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
   __constant__ const int ScatterEvent = 1;
   __constant__ const int NonScatterEvent = 2;
   __constant__ const int BackscatterEvent = 3;
   __constant__ const int ExitMaterialEvent = 4;
   __constant__ const int TrajectoryStartEvent = 5;
   __constant__ const int TrajectoryEndEvent = 6;
   __constant__ const int LastTrajectoryEvent = 7;
   __constant__ const int FirstTrajectoryEvent = 8;
   __constant__ const int StartSecondaryEvent = 9;
   __constant__ const int EndSecondaryEvent = 10;
   __constant__ const int PostScatterEvent = 11;

   __constant__ const int BeamEnergyChanged = 100;

   __constant__ static const int XAxis = 0;
   __constant__ static const int YAxis = 1;
   __constant__ static const int ZAxis = 2;
   __constant__ const float ChamberRadius = 0.1f;
   __constant__ const float SMALL_DISP = 1.0e-15f;
#else
   const int ScatterEvent = 1;
   const int NonScatterEvent = ScatterEvent + 1;
   const int BackscatterEvent = ScatterEvent + 2;
   const int ExitMaterialEvent = ScatterEvent + 3;
   const int TrajectoryStartEvent = ScatterEvent + 4;
   const int TrajectoryEndEvent = ScatterEvent + 5;
   const int LastTrajectoryEvent = ScatterEvent + 6;
   const int FirstTrajectoryEvent = ScatterEvent + 7;
   const int StartSecondaryEvent = ScatterEvent + 8;
   const int EndSecondaryEvent = ScatterEvent + 9;
   const int PostScatterEvent = ScatterEvent + 10;

   const int BeamEnergyChanged = 100;

   static const int XAxis = 0;
   static const int YAxis = 1;
   static const int ZAxis = 2;
   const float ChamberRadius = 0.1f;
   const float SMALL_DISP = 1.0e-15f;
#endif

   __host__ __device__ MonteCarloSS::MonteCarloSS(ElectronGunT const * gun, RegionT* chamber, ElectronT* electron) : mGun(gun), mChamber(chamber), mElectron(electron)
   {
      //TODO: shift the responsibility to the caller
      //const double center[] = {
      //   0.0,
      //   0.0,
      //   0.0
      //};
      //SphereT sphere(center, ChamberRadius);
      //mGun.setCenter(sphere.getInitialPoint().data());
      //mGun.setBeamEnergy(ToSI::keV(20.0));
      //mChamber = new RegionT(NULL, &NULL_MSM, &sphere);
   }

   //RegionT addSubRegion(const RegionT * parent, const MaterialT& mat, const ShapeT& shape)
   //{
   //   if (parent == NULL) printf("bad");
   //   return new RegionT(parent, new BasicMaterialModel(mat), shape);
   //}

   //RegionT* addSubRegion(RegionT& parent, IMaterialScatterModelT& msm, ShapeT& shape)
   //{
   //   return new RegionT(&parent, &msm, &shape); // TODO: deal with this, DO NOT USE IT
   //}

   //public Map<Material, Double> getMaterialMap(double[] startPt, double[] endPt) { // used
   //   final HashMap<Material, Double> traj = new HashMap<Material, Double>();
   //   double[] start = startPt;
   //   RegionBase region = mChamber.containingSubRegion(start);
   //   final double eps = 1.0e-7;
   //   while ((region != null) && (distance(start, endPt) > eps)) {
   //      final double[] end = endPt.clone();
   //      final RegionBase nextRegion = region.findEndOfStep(start, end);
   //      double dist = distance(start, end);
   //      if (dist > 0.0) {
   //         if (traj.containsKey(region.getMaterial()))
   //            dist += (traj.get(region.getMaterial())).doubleValue();
   //         traj.put(region.getMaterial(), new Double(dist));
   //      }
   //      start = Math2.plus(end, Math2.multiply(SMALL_DISP, Math2.normalize(Math2.minus(endPt, start))));
   //      region = nextRegion;
   //   }
   //   return traj;
   //}

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
   __constant__ static bool mDisableEvents = false;
#else
   static bool mDisableEvents = false;
#endif
   __host__ __device__ void MonteCarloSS::fireEvent(const int ae)
   {
      if (!(mEventListeners.empty() || mDisableEvents)) {
         for (auto sel : mEventListeners)
            sel->actionPerformed(ae);
      }
   }

   RegionT* MonteCarloSS::getChamber()
   {
      return mChamber;
   }

   __host__ __device__ void MonteCarloSS::initializeTrajectory()
   {
      mElectron = mGun->createElectron();
      auto reg = mChamber->containingSubRegion(mElectron->getPosition());
      mElectron->setCurrentRegion(reg);
      // Stop when you can't generate any more x-rays
      mElectron->setScatteringElement(nullptr);

   }

   __host__ __device__ void MonteCarloSS::takeStep()
   {
      const double *pos0 = mElectron->getPosition();

      const RegionBaseT* currentRegion = mElectron->getCurrentRegion();
      if ((currentRegion == nullptr) || !(currentRegion->getShape()->contains(pos0))) {
         currentRegion = mChamber->containingSubRegion(pos0);
         mElectron->setCurrentRegion(currentRegion);
         if (currentRegion == nullptr) {
            mElectron->setTrajectoryComplete(true);
            return;
         }
      }
      IMaterialScatterModelT* msm = currentRegion->getScatterModel();
      if (msm == nullptr) printf("MonteCarloSS::takeStep: msm is null\n");

      double pos1[3];
      mElectron->candidatePoint(msm->randomMeanPathLength(*mElectron), pos1);
      const RegionBaseT* nextRegion = currentRegion->findEndOfStep(pos0, pos1);
      mElectron->move(pos1, msm->calculateEnergyLoss(Math2::distance3d(pos0, pos1), *mElectron));
      const bool tc = (mElectron->getEnergy() < msm->getMinEforTracking()) || mElectron->isTrajectoryComplete();
      mElectron->setTrajectoryComplete(tc);
      if (!tc) {
         if (nextRegion == currentRegion) {
            if (mChamber == nullptr) printf("MonteCarloSS::takeStep(): mChamber == nullptr");
            if (mElectron == nullptr) printf("MonteCarloSS::takeStep(): mElectron == nullptr");
            if (currentRegion == nullptr) printf("MonteCarloSS::takeStep(): currentRegion == nullptr");
            fireEvent(ScatterEvent);
            ElectronT* secondary = msm->scatter(*mElectron);
            fireEvent(PostScatterEvent);
            mElectron->setTrajectoryComplete((mElectron->getEnergy() < msm->getMinEforTracking()) || mElectron->isTrajectoryComplete());
            if (secondary != nullptr) trackSecondaryElectron(secondary);

            if (mElectron->getCurrentRegion() != currentRegion) printf("MonteCarloSS::takeStep()MonteCarloSS::takeStep(): mElectron->getCurrentRegion() != currentRegion\n");
         }
         else if (nextRegion != nullptr) {
            fireEvent(NonScatterEvent);
            ElectronT* secondary = msm->barrierScatter(mElectron, nextRegion);
            double candpt[3];
            mElectron->candidatePoint(SMALL_DISP, candpt);
            mElectron->setPosition(candpt);
            if (!(mElectron->getCurrentRegion()->getShape()->contains(mElectron->getPosition())))
               mElectron->setCurrentRegion(mChamber->containingSubRegion(mElectron->getPosition()));
            if (mElectron->getCurrentRegion() != currentRegion)
               fireEvent(ExitMaterialEvent);
            if (secondary != nullptr) {
               double secandpt[3];
               secondary->candidatePoint(SMALL_DISP, secandpt);
               secondary->setPosition(secandpt);
               trackSecondaryElectron(secondary);
            }
         }
         else {
            fireEvent(BackscatterEvent);
            mElectron->setCurrentRegion(nullptr);
            mElectron->setTrajectoryComplete(true);
         }
      }
   }

   __host__ __device__ void MonteCarloSS::trackSecondaryElectron(ElectronT* newElectron)
   {
      double mMinEnergy = newElectron->getCurrentRegion()->getScatterModel()->getMinEforTracking();
      if (newElectron->getEnergy() > mMinEnergy) {
         fireEvent(StartSecondaryEvent);
         mElectronStack.push(mElectron);
         mElectron = newElectron;
         fireEvent(StartSecondaryEvent);
      }
   }

   //int MonteCarloSS::getElectronGeneration() const
   //{
   //   return mElectronStack.size();
   //}

   __host__ __device__ void MonteCarloSS::addActionListener(ActionListenerT& sel)
   {
      mEventListeners.push_back(&sel);
   }

   __host__ __device__ void MonteCarloSS::removeActionListener(ActionListenerT& sel)
   {
      auto itr = amp::find(mEventListeners.begin(), mEventListeners.end(), &sel);
      if (itr != mEventListeners.end()) {
         mEventListeners.erase(itr);
      }
   }

   __host__ __device__ bool MonteCarloSS::allElectronsComplete()
   {
      bool tc = mElectron->isTrajectoryComplete();
      while (tc && !mElectronStack.empty()) {
         fireEvent(EndSecondaryEvent);
         delete mElectron;
         mElectron = mElectronStack.top();
         mElectronStack.pop();

         tc = mElectron->isTrajectoryComplete();
      }
      if (tc && mElectronStack.empty() && mElectron) delete mElectron; // deleting the last electron
      return tc;
   }

   __host__ __device__ void MonteCarloSS::runTrajectory()
   {
      initializeTrajectory();
      fireEvent(TrajectoryStartEvent);
      //unsigned int i = 0;
      while (!allElectronsComplete()) {
         takeStep();
         //++i;
      }
      fireEvent(TrajectoryEndEvent);
      //printf("%d steps\n", i);
   }

   __host__ __device__ void MonteCarloSS::runMultipleTrajectories(int n)
   {
      fireEvent(FirstTrajectoryEvent);
      for (int i = 0; i < n; ++i) {
         //printf("itr #%d:\n", i);
         runTrajectory();
      }
      fireEvent(LastTrajectoryEvent);
   }

   __host__ __device__ double MonteCarloSS::getBeamEnergy() const
   {
      return mGun->getBeamEnergy();
   }

   void MonteCarloSS::setElectronGun(ElectronGunT& gun)
   {
      gun.setBeamEnergy(mGun->getBeamEnergy());
      mGun = &gun;
   }

   //void MonteCarloSS::setBeamEnergy(double beamEnergy)
   //{
   //   //mGun->setBeamEnergy(beamEnergy);
   //   //fireEvent(BeamEnergyChanged);
   //}

   void MonteCarloSS::computeDetectorPosition(double elevation, double theta, double res[])
   {
      double frac = 0.999;
      double r = frac * ChamberRadius;
      //if (mChamber->mShape instanceof Sphere)
      //   r = frac * ((Sphere)mChamber.mShape).getRadius();
      ElectronProbe::computePosition(0.0, elevation, theta, r, res);
   }

   __host__ __device__ const ElectronT& MonteCarloSS::getElectron() const
   {
      return *mElectron;
   }

   //public Set<AtomicShell> getAtomicShellSet()
   //   throws EPQException{ // not used
   //   final Set<AtomicShell> res = new TreeSet<AtomicShell>();
   //   final Set<Element> elements = mChamber.getElements(true);
   //   for (final Object element : elements) {
   //      final Element el = (Element)element;
   //      for (int sh = AtomicShell.K; sh < AtomicShell.NI; ++sh) {
   //         final AtomicShell shell = new AtomicShell(el, sh);
   //         final double ee = shell.getEdgeEnergy();
   //         if ((ee > 0.0) && (ee < mGun.getBeamEnergy()) && (XRayTransition.getStrongestLine(shell) != null))
   //            res.add(shell);
   //      }
   //   }
   //   return res;
   //}

   Element::UnorderedSetT MonteCarloSS::getElementSet() const
   {
      return mChamber->getElements(true);
   }

   //public void estimateTrajectoryVolume(double[] c0, double[] c1) {
   //   c0[0] = (c0[1] = (c0[2] = Double.MAX_VALUE));
   //   c1[0] = (c1[1] = (c1[2] = -Double.MAX_VALUE));
   //   final int nTraj = 100;
   //   mDisableEvents = true;
   //   for (int i = 0; i < nTraj; ++i) {
   //      initializeTrajectory();
   //      while (!mElectron.isTrajectoryComplete()) {
   //         takeStep();
   //         final double[] endPt = mElectron.getPosition();
   //         final RegionBase endRegion = mChamber.containingSubRegion(endPt);
   //         if ((endRegion != null) && (endRegion != mChamber))
   //            for (int j = 0; j < 3; ++j) {
   //               if (endPt[j] < c0[j])
   //                  c0[j] = endPt[j];
   //               if (endPt[j] > c1[j])
   //                  c1[j] = endPt[j];
   //            }
   //      }
   //   }
   //   mDisableEvents = false;
   //}

   //void updateMaterial(const MaterialT& oldMat, const MaterialT& newMat)
   //{
   //   mChamber.updateMaterial(oldMat, new BasicMaterialModel(newMat));
   //}

   void MonteCarloSS::rotate(double pivot[], double phi, double theta, double psi)
   {
      for (const RegionBaseT* r : mChamber->getSubRegions()) {
         //if (r instanceof TransformableRegion) ((TransformableRegionT*)r)->rotate(pivot, phi, theta, psi);
      }
   }

   void MonteCarloSS::translate(double distance[])
   {
      for (const RegionBaseT* r : mChamber->getSubRegions()) {
         //if (r instanceof TransformableRegion) ((TransformableRegionT*)r)->translate(distance);
      }
   }

   const RegionBaseT* MonteCarloSS::findRegionContaining(double point[]) const
   {
      return mChamber->containingSubRegion(point);
   }
}
