#include "hip/hip_runtime.h"
//package gov.nist.microanalysis.NISTMonte;

#include "gov\nist\microanalysis\NISTMonte\CylindricalShape.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"
#include "gov\nist\microanalysis\Utility\Transform3D.cuh"

namespace CylindricalShape
{
   static const double EPSILON = 1.0e-40;

   CylindricalShape::CylindricalShape(const double end0[], const double end1[], double radius) : 
      mEnd0(end0, end0 + 3),
      mRadius2(radius * radius)
   {
      mDelta = { end1[0] - end0[0], end1[1] - end0[1], end1[2] - end0[2] };
      if (mRadius2 < 1.0e-30) printf("The cylinder radius is unrealistically small.");
      mLen2 = Math2::sqr(mDelta[0]) + Math2::sqr(mDelta[1]) + Math2::sqr(mDelta[2]);
      if (mLen2 < 1.0e-30) printf("The cylinder length is unrealistically small.");
      mDelta2 = Math2::dot3D(mDelta, mDelta);
   }

   CylindricalShape::CylindricalShape(const CylindricalShape& other) :
      mEnd0(other.mEnd0.begin(), other.mEnd0.end()),
      mDelta(other.mDelta.begin(), other.mDelta.end()),
      mRadius2(other.mRadius2),
      mLen2(other.mLen2),
      mDelta2(other.mDelta2)
   {
   }

   double CylindricalShape::closestPointOnAxis(const double p[]) const
   {
      return (mDelta[0] * (p[0] - mEnd0[0]) + mDelta[1] * (p[1] - mEnd0[1]) + mDelta[2] * (p[2] - mEnd0[2])) / mLen2;
   }

   double CylindricalShape::distanceSqr(const double p[], double u) const
   {
      return Math2::sqr(p[0] - (mEnd0[0] + u * mDelta[0])) + Math2::sqr(p[1] - (mEnd0[1] + u * mDelta[1])) + Math2::sqr(p[2] - (mEnd0[2] + u * mDelta[2]));
   }

   bool CylindricalShape::contains(const double pos[]) const
   {
      // project pos onto the line defined by end0 and end1.
      double u = closestPointOnAxis(pos);
      // Is this point between end0 and end1 and is pos^2 <= mRadius from the
      // line from end0 to end1?
      return (u >= 0) && (u <= 1.0) && (distanceSqr(pos, u) <= mRadius2);
   }

   VectorXd CylindricalShape::getEnd0() const
   {
      return mEnd0;
   }

   VectorXd CylindricalShape::getEnd1() const
   {
      return VectorXd({ mEnd0[0] + mDelta[0], mEnd0[1] + mDelta[1], mEnd0[2] + mDelta[2] });
   }

   static double checkT(double t)
   {
      return t >= 0.0 ? t : INFINITY;
   }

   double CylindricalShape::getFirstIntersection(const double sa[], const double sb[])
   {
      VectorXd saVec(sa, sa + 3), sbVec(sb, sb + 3);
      if (true) {
         double t0 = INFINITY, t1 = INFINITY, tc = INFINITY;
         const VectorXd& n = Math2::minus3d(sb, sa);
         double nd = Math2::dot3D(n, mDelta);
         if (nd != 0.0) {
            // Check end cap 0
            double t = Math2::dot3d(mDelta.data(), Math2::minus3d(mEnd0.data(), sa).data()) / nd;
            if (t > 0.0) {
               const VectorXd& pt = Math2::minus3d(Math2::pointBetween3d(sa, sb, t).data(), mEnd0.data());
               if (Math2::dot3d(pt.data(), pt.data()) < mRadius2)
                  t0 = t;
            }
            // Check end cap 1
            const VectorXd& end1 = Math2::plus(mEnd0, mDelta);
            t = Math2::dot3d(mDelta.data(), Math2::minus3d(end1.data(), sa).data()) / nd;
            if (t > 0.0) {
               const VectorXd& pt = Math2::minus3d(Math2::pointBetween3d(sa, sb, t).data(), end1.data());
               if (Math2::dot3d(pt.data(), pt.data()) < mRadius2)
                  t1 = t;
            }
         }
         double a = mDelta2 * Math2::dot3d(n.data(), n.data()) - nd * nd;
         if (::abs(a) > EPSILON) {
            const VectorXd& m = Math2::minus3d(sa, mEnd0.data());
            double mn = Math2::dot3d(m.data(), n.data());
            double b = mDelta2 * mn - nd * Math2::dot3d(m.data(), mDelta.data());
            double md = Math2::dot3d(m.data(), mDelta.data());
            // Consider the side of the cylinder
            double c = mDelta2 * (Math2::dot3d(m.data(), m.data()) - mRadius2) - md * md;
            double discr = b * b - a * c;
            if (discr >= 0.0) {
               double tm = (-b - ::sqrt(discr)) / a;
               double tp = (-b + ::sqrt(discr)) / a;
               double t = ::fmin(tm > 0.0 ? tm : INFINITY, tp > 0.0 ? tp : INFINITY);
               if ((t != INFINITY) && (md + t * nd >= 0.0) && (md + t * nd <= mDelta2))
                  tc = t;
            }
         }
         return ::fmin(t0, ::fmin(t1, tc));
      }
      else {
         auto m = Math2::minus3d(sa, mEnd0.data()), n = Math2::minus3d(sb, sa);
         double md = Math2::dot3d(m.data(), mDelta.data()), nd = Math2::dot3d(n.data(), mDelta.data()), dd = Math2::dot3d(mDelta.data(), mDelta.data());
         // Segment fully outside end caps...
         if ((md < 0.0) && (md + nd < 0.0))
            return INFINITY;
         if ((md > dd) && (md + nd > dd))
            return INFINITY;
         double nn = Math2::dot3d(n.data(), n.data()), mn = Math2::dot3d(m.data(), n.data());
         double a = dd * nn - nd * nd;
         double k = Math2::dot3d(m.data(), m.data()) - mRadius2;
         double c = dd * k - md * md;
         if (::abs(a) < EPSILON) {
            if (md < 0.0)
               return checkT(-mn / nn);
            else if (md > dd)
               return checkT((nd - mn) / nn);
            else
               return 0.0;
         }
         double b = dd * mn - nd * md;
         double disc = b * b - a * c;
         if (disc < 0.0)
            return INFINITY;
         double t = (-b - ::sqrt(disc)) / a; // Always a >= 0.0
         if (t < 0.0)
            t = (-b + ::sqrt(disc)) / a;
         if (!(::abs(distanceSqr(Math2::plus3d(sa, Math2::multiply3d(t, n.data()).data()).data(), closestPointOnAxis(Math2::plus3d(sa, Math2::multiply3d(t, n.data()).data()).data()) - mRadius2)) < 1.0e-10 * mRadius2)) printf("CylindricalShape::getFirstIntersection: < 1.0e-10 * mRadius2 (%.10e)\n", mRadius2);
         // Check end caps
         if (md + t * nd < 0.0) {
            t = -md / nd;
            return k + 2.0 * t * (mn + t * nn) <= 0.0 ? checkT(t) : INFINITY;
         }
         else if (md + t * nd > dd) {
            t = (dd - md) / nd;
            return k + dd - 2.0 * md + t * (2.0 * (mn - nd) + t * nn) <= 0.0 ? checkT(t) : INFINITY;
         }
         return checkT(t);
      }
   }

   void CylindricalShape::rotate(const double pivot[], double phi, double theta, double psi)
   {
      mEnd0 = Transform3D::rotate(mEnd0.data(), pivot, phi, theta, psi);
      mDelta = Transform3D::rotate(mDelta.data(), phi, theta, psi);
   }

   void CylindricalShape::translate(const double distance[])
   {
      mEnd0[0] += distance[0];
      mEnd0[1] += distance[1];
      mEnd0[2] += distance[2];
   }

   double CylindricalShape::getRadius() const
   {
      return ::sqrt(mRadius2);
   }

   double CylindricalShape::getLength() const
   {
      return ::sqrt(mDelta[0] * mDelta[0] + mDelta[1] * mDelta[1] + mDelta[2] * mDelta[2]);
   }

   StringT CylindricalShape::toString() const
   {
      StringT res = "Cylinder([";
      double* end1 = getEnd1().data();
      res += std::to_string(getEnd0()[0]) + "," + std::to_string(getEnd0()[1]) + "," + std::to_string(getEnd0()[2]) + "],[";
      res += std::to_string(end1[0]) + "," + std::to_string(end1[1]) + "," + std::to_string(end1[2]) + "],";
      res += std::to_string(getRadius()) + ")";
      return res.c_str();
   }

   ///**
   //* @see gov.nist.microanalysis.NISTMonte.TrajectoryVRML.IRender#render(gov.nist.microanalysis.NISTMonte.TrajectoryVRML.RenderContext,
   //*      java.io.Writer)
   //*/
   //public void render(TrajectoryVRML.RenderContext vra, Writer wr)
   //   throws IOException{
   //   final NumberFormat nf = NumberFormat.getNumberInstance(Locale.US);
   //   nf.setMaximumFractionDigits(3);
   //   nf.setGroupingUsed(false);
   //   final Color color = vra.getCurrentColor();
   //   final String trStr = nf.format(vra.getTransparency());
   //   final String colorStr = nf.format(color.getRed() / 255.0) + " " + nf.format(color.getGreen() / 255.0) + " "
   //      + nf.format(color.getBlue() / 255.0);
   //   wr.append("\nTransform {\n");
   //   // r is the cross product (1,0,0) x norm(mDelta)
   //   {
   //      final double dm = Math2::magnitude(mDelta);
   //      assert(dm > 0.0);
   //      final double[] r = {
   //         mDelta[2] / dm,
   //         0.0,
   //         -mDelta[0] / dm,
   //      };
   //      final double rm = Math2::magnitude(r);
   //      if (rm > 0.0) { // if rotation required...
   //         nf.setMaximumFractionDigits(5);
   //         double th = ::asin(rm);
   //         if (mDelta[1] < 0.0)
   //            th = ::PI - th;
   //         wr.append(" rotation " + nf.format(r[0] / rm) + " " + nf.format(r[1] / rm) + " " + nf.format(r[2] / rm) + " "
   //            + nf.format(th) + "\n");
   //         nf.setMaximumFractionDigits(3);
   //      }
   //   }
   //   wr.append(" translation " + nf.format((mEnd0[0] + mDelta[0] / 2.0) / TrajectoryVRML.SCALE) + " "
   //      + nf.format((mEnd0[1] + mDelta[1] / 2.0) / TrajectoryVRML.SCALE) + " "
   //      + nf.format((mEnd0[2] + mDelta[2] / 2.0) / TrajectoryVRML.SCALE) + "\n");
   //   wr.append(" children [\n");
   //   wr.append("  Shape {\n");
   //   wr.append("   geometry Cylinder {\n");
   //   wr.append("    radius " + nf.format(getRadius() / TrajectoryVRML.SCALE) + "\n");
   //   wr.append("    height " + nf.format(getLength() / TrajectoryVRML.SCALE) + "\n");
   //   wr.append("    bottom TRUE\n");
   //   wr.append("    side TRUE\n");
   //   wr.append("    top TRUE\n");
   //   wr.append("   }\n");
   //   wr.append("   appearance Appearance {\n");
   //   wr.append("    material Material {\n");
   //   wr.append("     emissiveColor " + colorStr + "\n");
   //   wr.append("     transparency " + trStr + "\n");
   //   wr.append("    }\n");
   //   wr.append("   }\n");
   //   wr.append("  }\n");
   //   wr.append(" ]\n");
   //   wr.append("}");
   //   wr.flush();
   //}
}
