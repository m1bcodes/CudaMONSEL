#include "hip/hip_runtime.h"
//package gov.nist.microanalysis.NISTMonte;

#include "gov\nist\microanalysis\NISTMonte\CylindricalShape.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"
#include "gov\nist\microanalysis\Utility\Transform3D.cuh"

namespace CylindricalShape
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
   __constant__ static const double EPSILON = 1.0e-40;
#else
   static const double EPSILON = 1.0e-40;
#endif

   __host__ __device__ CylindricalShape::CylindricalShape(const double end0[], const double end1[], double radius)
   {
      memcpy(mEnd0, end0, sizeof(double) * 3);
      memcpy(mEnd1, end1, sizeof(double) * 3);
      mRadius2 = radius * radius;
      mDelta[0] = end1[0] - end0[0];
      mDelta[1] = end1[1] - end0[1];
      mDelta[2] = end1[2] - end0[2];

      if (mRadius2 < 1.0e-30) printf("The cylinder radius is unrealistically small.");
      mLen2 = Math2::sqr(mDelta[0]) + Math2::sqr(mDelta[1]) + Math2::sqr(mDelta[2]);
      if (mLen2 < 1.0e-30) printf("The cylinder length is unrealistically small.");
      mDelta2 = Math2::dot3d(mDelta, mDelta);
   }

   __host__ __device__ CylindricalShape::CylindricalShape(const CylindricalShape& other) :
      mRadius2(other.mRadius2),
      mLen2(other.mLen2),
      mDelta2(other.mDelta2)
   {
      memcpy(mEnd0, other.mEnd0, sizeof(double) * 3);
      memcpy(mEnd1, other.mEnd1, sizeof(double) * 3);
      memcpy(mDelta, other.mDelta, sizeof(double) * 3);
   }

   __host__ __device__ double CylindricalShape::closestPointOnAxis(const double p[]) const
   {
      return (mDelta[0] * (p[0] - mEnd0[0]) + mDelta[1] * (p[1] - mEnd0[1]) + mDelta[2] * (p[2] - mEnd0[2])) / mLen2;
   }

   __host__ __device__ double CylindricalShape::distanceSqr(const double p[], double u) const
   {
      return Math2::sqr(p[0] - (mEnd0[0] + u * mDelta[0])) + Math2::sqr(p[1] - (mEnd0[1] + u * mDelta[1])) + Math2::sqr(p[2] - (mEnd0[2] + u * mDelta[2]));
   }

   __host__ __device__ bool CylindricalShape::contains(const double pos[]) const
   {
      // project pos onto the line defined by end0 and end1.
      const double u = closestPointOnAxis(pos);
      // Is this point between end0 and end1 and is pos^2 <= mRadius from the
      // line from end0 to end1?
      return (u >= 0) && (u <= 1.0) && (distanceSqr(pos, u) <= mRadius2);
   }

   const double* CylindricalShape::getEnd0() const
   {
      return mEnd0;
   }

   const double* CylindricalShape::getEnd1() const
   {
      return mEnd1;
   }

   __host__ __device__ static double checkT(double t)
   {
      return t >= 0.0 ? t : INFINITY;
   }

   __host__ __device__ double CylindricalShape::getFirstIntersection(const double sa[], const double sb[])
   {
      if (true) {
         double t0 = INFINITY, t1 = INFINITY, tc = INFINITY;
         double n[3];
         Math2::minus3d(sb, sa, n);
         const double nd = Math2::dot3d(n, mDelta);
         if (nd != 0.0) {
            // Check end cap 0
            double m1[3];
            Math2::minus3d(mEnd0, sa, m1);
            double t = Math2::dot3d(mDelta, m1) / nd;
            if (t > 0.0) {
               double pt[3];
               double ptbtw[3];
               Math2::pointBetween3d(sa, sb, t, ptbtw);
               Math2::minus3d(ptbtw, mEnd0, pt);
               if (Math2::dot3d(pt, pt) < mRadius2)
                  t0 = t;
            }
            // Check end cap 1
            double end1[3];
            Math2::plus3d(mEnd0, mDelta, end1);
            double m2[3];
            Math2::minus3d(end1, sa, m2);
            t = Math2::dot3d(mDelta, m2) / nd;
            if (t > 0.0) {
               double pt[3];
               double ptbtw[3];
               Math2::pointBetween3d(sa, sb, t, ptbtw);
               Math2::minus3d(ptbtw, end1, pt);
               if (Math2::dot3d(pt, pt) < mRadius2)
                  t1 = t;
            }
         }
         const double a = mDelta2 * Math2::dot3d(n, n) - nd * nd;
         if (::fabs(a) > EPSILON) {
            double m[3];
            Math2::minus3d(sa, mEnd0, m);
            const double mn = Math2::dot3d(m, n);
            const double b = mDelta2 * mn - nd * Math2::dot3d(m, mDelta);
            const double md = Math2::dot3d(m, mDelta);
            // Consider the side of the cylinder
            const double c = mDelta2 * (Math2::dot3d(m, m) - mRadius2) - md * md;
            const double discr = b * b - a * c;
            if (discr >= 0.0) {
               const double tm = (-b - ::sqrt(discr)) / a;
               const double tp = (-b + ::sqrt(discr)) / a;
               const double t = ::fmin(tm > 0.0 ? tm : INFINITY, tp > 0.0 ? tp : INFINITY);
               if ((t != INFINITY) && (md + t * nd >= 0.0) && (md + t * nd <= mDelta2))
                  tc = t;
            }
         }
         return ::fmin(t0, ::fmin(t1, tc));
      }
      else {
         double m[3];
         Math2::minus3d(sa, mEnd0, m);
         double n[3];
         Math2::minus3d(sb, sa, n);
         const double md = Math2::dot3d(m, mDelta), nd = Math2::dot3d(n, mDelta), dd = Math2::dot3d(mDelta, mDelta);
         // Segment fully outside end caps...
         if ((md < 0.0) && (md + nd < 0.0))
            return INFINITY;
         if ((md > dd) && (md + nd > dd))
            return INFINITY;
         const double nn = Math2::dot3d(n, n), mn = Math2::dot3d(m, n);
         const double a = dd * nn - nd * nd;
         const double k = Math2::dot3d(m, m) - mRadius2;
         const double c = dd * k - md * md;
         if (::abs(a) < EPSILON) {
            if (md < 0.0)
               return checkT(-mn / nn);
            else if (md > dd)
               return checkT((nd - mn) / nn);
            else
               return 0.0;
         }
         const double b = dd * mn - nd * md;
         const double disc = b * b - a * c;
         if (disc < 0.0)
            return INFINITY;
         double t = (-b - ::sqrt(disc)) / a; // Always a >= 0.0
         if (t < 0.0)
            t = (-b + ::sqrt(disc)) / a;
         double mult0[3];
         Math2::multiply3d(t, n, mult0);
         double p0[3];
         Math2::plus3d(sa, mult0, p0);
         double mult1[3];
         Math2::multiply3d(t, n, mult1);
         double p1[3];
         Math2::plus3d(sa, mult1, p1);

         if (!(::fabs(distanceSqr(p0, closestPointOnAxis(p1) - mRadius2)) < 1.0e-10 * mRadius2)) printf("CylindricalShape::getFirstIntersection: < 1.0e-10 * mRadius2 (%.10e)\n", mRadius2);
         // Check end caps
         if (md + t * nd < 0.0) {
            t = -md / nd;
            return k + 2.0 * t * (mn + t * nn) <= 0.0 ? checkT(t) : INFINITY;
         }
         else if (md + t * nd > dd) {
            t = (dd - md) / nd;
            return k + dd - 2.0 * md + t * (2.0 * (mn - nd) + t * nn) <= 0.0 ? checkT(t) : INFINITY;
         }
         return checkT(t);
      }
   }

   void CylindricalShape::rotate(const double pivot[], double phi, double theta, double psi)
   {
      Transform3D::rotate3d(mEnd0, pivot, phi, theta, psi, mEnd0);
      Transform3D::rotate3d(mDelta, phi, theta, psi, mDelta);
   }

   __host__ __device__ void CylindricalShape::translate(const double distance[])
   {
      mEnd0[0] += distance[0];
      mEnd0[1] += distance[1];
      mEnd0[2] += distance[2];
   }

   __host__ __device__ double CylindricalShape::getRadius() const
   {
      return ::sqrt(mRadius2);
   }

   double CylindricalShape::getLength() const
   {
      return ::sqrt(mDelta[0] * mDelta[0] + mDelta[1] * mDelta[1] + mDelta[2] * mDelta[2]);
   }

   __host__ __device__ StringT CylindricalShape::toString() const
   {
      StringT res = "CylindricalShape([";
      res += amp::to_string(mEnd0[0]) + "," + amp::to_string(mEnd0[1]) + "," + amp::to_string(mEnd0[2]) + "],[";
      res += amp::to_string(mEnd1[0]) + "," + amp::to_string(mEnd1[1]) + "," + amp::to_string(mEnd1[2]) + "],";
      res += amp::to_string(getRadius()) + ")";
      return res.c_str();
   }

   //public void render(TrajectoryVRML.RenderContext vra, Writer wr)
   //   throws IOException{
   //   final NumberFormat nf = NumberFormat.getNumberInstance(Locale.US);
   //   nf.setMaximumFractionDigits(3);
   //   nf.setGroupingUsed(false);
   //   final Color color = vra.getCurrentColor();
   //   final String trStr = nf.format(vra.getTransparency());
   //   final String colorStr = nf.format(color.getRed() / 255.0) + " " + nf.format(color.getGreen() / 255.0) + " "
   //      + nf.format(color.getBlue() / 255.0);
   //   wr.append("\nTransform {\n");
   //   // r is the cross product (1,0,0) x norm(mDelta)
   //   {
   //      final double dm = Math2::magnitude(mDelta);
   //      assert(dm > 0.0);
   //      final double[] r = {
   //         mDelta[2] / dm,
   //         0.0,
   //         -mDelta[0] / dm,
   //      };
   //      final double rm = Math2::magnitude(r);
   //      if (rm > 0.0) { // if rotation required...
   //         nf.setMaximumFractionDigits(5);
   //         double th = ::asin(rm);
   //         if (mDelta[1] < 0.0)
   //            th = ::PI - th;
   //         wr.append(" rotation " + nf.format(r[0] / rm) + " " + nf.format(r[1] / rm) + " " + nf.format(r[2] / rm) + " "
   //            + nf.format(th) + "\n");
   //         nf.setMaximumFractionDigits(3);
   //      }
   //   }
   //   wr.append(" translation " + nf.format((mEnd0[0] + mDelta[0] / 2.0) / TrajectoryVRML.SCALE) + " "
   //      + nf.format((mEnd0[1] + mDelta[1] / 2.0) / TrajectoryVRML.SCALE) + " "
   //      + nf.format((mEnd0[2] + mDelta[2] / 2.0) / TrajectoryVRML.SCALE) + "\n");
   //   wr.append(" children [\n");
   //   wr.append("  Shape {\n");
   //   wr.append("   geometry Cylinder {\n");
   //   wr.append("    radius " + nf.format(getRadius() / TrajectoryVRML.SCALE) + "\n");
   //   wr.append("    height " + nf.format(getLength() / TrajectoryVRML.SCALE) + "\n");
   //   wr.append("    bottom TRUE\n");
   //   wr.append("    side TRUE\n");
   //   wr.append("    top TRUE\n");
   //   wr.append("   }\n");
   //   wr.append("   appearance Appearance {\n");
   //   wr.append("    material Material {\n");
   //   wr.append("     emissiveColor " + colorStr + "\n");
   //   wr.append("     transparency " + trStr + "\n");
   //   wr.append("    }\n");
   //   wr.append("   }\n");
   //   wr.append("  }\n");
   //   wr.append(" ]\n");
   //   wr.append("}");
   //   wr.flush();
   //}
}
