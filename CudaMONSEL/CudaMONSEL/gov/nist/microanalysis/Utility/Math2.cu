#include "hip/hip_runtime.h"
#include "Math2.cuh"

#include <math.h>

namespace Math2
{
   const double PI = 3.14159265358979323846;

   const double ORIGIN_3D[] = {
      0.0,
      0.0,
      0.0
   };
   const double ONE[] = {
      1.0,
      1.0,
      1.0
   };
   const double X_AXIS[] = {
      1.0,
      0.0,
      0.0
   };
   const double Y_AXIS[] = {
      0.0,
      1.0,
      0.0
   };
   const double Z_AXIS[] = {
      0.0,
      0.0,
      1.0
   };
   const double MINUS_X_AXIS[] = {
      -1.0,
      0.0,
      0.0
   };
   const double MINUS_Y_AXIS[] = {
      0.0,
      -1.0,
      0.0
   };
   const double MINUS_Z_AXIS[] = {
      0.0,
      0.0,
      -1.0
   };

   //   /**
   //   * SQRT_PI - The square root of Pi ~ 1.772
   //   */
   //   const double SQRT_PI = Math.sqrt(Math.PI);
   //
   //   /**
   //   * A random number generator. It is invoked by, e.g., rgen.nextDouble(). By
   //   * default it functions similarly to Math.double(), with an initial seed
   //   * produced by a procedure unlikely to give the same value under repeated
   //   * invocations.
   //   */
   //   public Random rgen = new Random();
   //
   //   /**
   //   * initializeRandom - When called with an argument of type long, it
   //   * initializes the Math2.rgen random number generator with that seed value.
   //   * When called without an argument it sets the seed using a procedure
   //   * unlikely to give the same value under repeated invocations.
   //   */
   //   public void initializeRandom(long seed) {
   //      rgen = new Random(seed);
   //   }
   //
   //   public void initializeRandom() {
   //      rgen = new Random();
   //   }
   //
   //   /**
   //   * sqr - returns x*x
   //   *
   //   * @param x double
   //   * @return double
   //   */
   double sqr(double x)
   {
      return x * x;
   }
   //
   //   /**
   //   * erf - The error function (2/sqrt(pi))*Integrate[Exp[-t^2],{t,0,x}] <br>
   //   * Source: Numerical Recipes in C - 2nd Edition
   //   *
   //   * @param x double
   //   * @return double
   //   */
   //   const double erf(double x) {
   //      return x < 0.0 ? -gammap(0.5, x * x) : gammap(0.5, x * x);
   //   }
   //
   //   /**
   //   * erfc - The complementary error function (1-erf(x)) <br>
   //   * Source: Numerical Recipes in C - 2nd Edition
   //   *
   //   * @param x double
   //   * @return double
   //   */
   //   const double erfc(double x) {
   //      return x < 0.0 ? 1.0 + gammap(0.5, x * x) : gammq(0.5, x * x);
   //   }
   //
   //   /**
   //   * gammaq - The incomplete gamma function Q(a,x) = 1 - P(a,x) <br>
   //   * Source: Numerical Recipes in C - 2nd Edition
   //   *
   //   * @param a double
   //   * @param x double
   //   * @return double
   //   */
   //   const double gammq(double a, double x) {
   //      assert(x >= 0.0);
   //      assert(a > 0.0);
   //      if (x < (a + 1.0))
   //         return 1.0 - gser(a, x);
   //      else
   //         return gcf(a, x);
   //   }
   //
   //   /**
   //   * gammap - Computes the incomplete gamma function P(a,x) by selecting
   //   * between series and continued fraction representations based on the size of
   //   * the arguments. <br>
   //   * Source: Numerical Recipes in C - 2nd Edition
   //   *
   //   * @param a double
   //   * @param x double
   //   * @return double
   //   */
   //   const double gammap(double a, double x) {
   //      assert(x >= 0.0);
   //      assert(a > 0.0);
   //      if (x < (a + 1.0))
   //         return gser(a, x);
   //      else
   //         return 1.0 - gcf(a, x);
   //   }
   //
   //   /**
   //   * chiSquaredConfidenceLevel - Computes the table in Press et al 15.6 which
   //   * displays dChiSq as a function of confidence level and degrees of freedom.
   //   *
   //   * @param confidence double on (0,1)
   //   * @param degreesOfFreedom int > 0
   //   * @return double
   //   */
   //   public double chiSquaredConfidenceLevel(double confidence, int degreesOfFreedom) {
   //      assert(confidence > 0.0) && (confidence < 1.0) : "Confidence must be in the range (0, 1).";
   //      assert degreesOfFreedom > 0 : "Degrees of freedom must be 1 or larger.";
   //      if ((confidence < 0.0) || (confidence >= 1.0) || (degreesOfFreedom <= 0))
   //         return Double.NaN;
   //      final FindRoot fr = new FindRoot(){
   //         double mDegreesOfFreedom;
   //         double mConfidenceLimit;
   //
   //         @Override
   //            public void initialize(double[] dd) {
   //            mDegreesOfFreedom = dd[0];
   //            mConfidenceLimit = dd[1];
   //         }
   //
   //         @Override
   //            public double function(double x0) {
   //            return gammap(0.5 * mDegreesOfFreedom, 0.5 * x0) - mConfidenceLimit;
   //         }
   //      };
   //      fr.initialize(new double[] {
   //         degreesOfFreedom,
   //            confidence
   //      });
   //      return fr.perform(1.0, 2.0 * degreesOfFreedom + 50.0, 1.0e-3, 100);
   //   }
   //
   //   /**
   //   * gser - Calculates the incomplete gamma function P(a,x) by evaluating the
   //   * series representation. <br>
   //   * Source: Numerical Recipes in C - 2nd Edition
   //   *
   //   * @param a double
   //   * @param x double
   //   * @return double
   //   */
   //   private final double gser(double a, double x) {
   //      assert(x >= 0.0);
   //      final double ITMAX = 100;
   //      final double EPS = 3.0e-7;
   //      if (x == 0.0)
   //         return 0.0;
   //      else {
   //         double ap = a;
   //         double sum = 1.0 / a;
   //         double del = sum;
   //         for (int n = 1; n <= ITMAX; ++n) {
   //            ++ap;
   //            del *= x / ap;
   //            sum += del;
   //            if (Math.abs(del) < Math.abs(sum) * EPS)
   //               return sum * Math.exp(-x + a * Math.log(x) - gammaln(a));
   //         }
   //         assert false : "a too large, ITMAX too small in routine gser";
   //         return sum * Math.exp(-x + a * Math.log(x) - gammaln(a));
   //      }
   //   }
   //
   //   /**
   //   * gcf - Implements the incomplete gamma function Q(a,x) evaluated as a
   //   * continued fraction. <br>
   //   * Source: Numerical Recipes in C - 2nd Edition
   //   *
   //   * @param a double
   //   * @param x double
   //   * @return double
   //   */
   //   private final double gcf(double a, double x) {
   //      final double ITMAX = 100;
   //      final double EPS = 3.0e-7;
   //      final double FPMIN = 1.0e-30;
   //      double b = x + 1.0 - a;
   //      double c = 1.0 / FPMIN;
   //      double d = 1.0 / b;
   //      double h = d;
   //      for (int i = 1; i <= ITMAX; ++i) {
   //         final double an = -i * (i - a);
   //         b += 2.0;
   //         d = an * d + b;
   //         if (Math.abs(d) < FPMIN)
   //            d = FPMIN;
   //         c = b + an / c;
   //         if (Math.abs(c) < FPMIN)
   //            c = FPMIN;
   //         d = 1.0 / d;
   //         final double del = d * c;
   //         h *= del;
   //         if (Math.abs(del - 1.0) < EPS)
   //            break;
   //         assert i != ITMAX : "a too large, ITMAX too small in gcf";
   //      }
   //      return Math.exp(-x + a * Math.log(x) - gammaln(a)) * h;
   //   }
   //
   //   /**
   //   * gammaln - The natural log of the gamma function. <br>
   //   * Source: Numerical Recipes in C - 2nd Edition
   //   *
   //   * @param xx double
   //   * @return double
   //   */
   //   const double gammaln(double xx) {
   //      // Coefficients used by gammaln
   //      final double[] coeff = {
   //         76.18009172947146,
   //         -86.50532032941677,
   //         24.01409824083091,
   //         -1.231739572450155,
   //         0.1208650973866179e-2,
   //         -0.5395239384953e-5
   //      };
   //      double y = xx;
   //      double tmp = xx + 5.5;
   //      tmp -= (xx + 0.5) * Math.log(tmp);
   //      double ser = 1.000000000190015;
   //      for (int j = 0; j <= 5; ++j) {
   //         y += 1.0;
   //         ser += coeff[j] / y;
   //      }
   //      return -tmp + Math.log(2.5066282746310005 * ser / xx);
   //   }
   //
   //   /**
   //   * expRand - Selects a random value from an exponential distibution. The mean
   //   * value returned is 1.0.
   //   *
   //   * @return double - Returns a random variable in the range [0,infinity)
   //   */
   //   final public double expRand() {
   //      return -Math.log(rgen.nextDouble());
   //   }
   //
   //   /**
   //   * Computes a random 3-vector uniform in solid angle using the algorithm of
   //   * Robert Knop in Commun. ACM, ACM, 1970, 13, 326
   //   *
   //   * @return double[3]
   //   */
   //   final public double[] randomDir() {
   //      double x, y, s;
   //      do {
   //         x = 2.0 * (Math.random() - 0.5);
   //         y = 2.0 * (Math.random() - 0.5);
   //         s = x * x + y * y;
   //      } while (s > 1.0);
   //      final double z = 2.0 * s - 1.0;
   //      s = Math.sqrt((1 - z * z) / s);
   //      x *= s;
   //      y *= s;
   //      return new double[] {
   //         x,
   //            y,
   //            z
   //      };
   //   }
   
   /**
   * distance - What is the Euclidean distance between p1 and p2.
   *
   * @param p1
   * @param p2
   * @return double
   */
   double distance(const VectorXd& p1, const VectorXd& p2)
   {
      if (p1.size() != p2.size()) return -1;
      double sum2 = 0.0;
      for (int i = 0; i < p1.size(); ++i) {
         sum2 += Math2::sqr(p2[i] - p1[i]);
      }
      return ::sqrt(sum2);
   }

   double distance3d(const VectorXd& p1, const VectorXd& p2)
   {
      double sum2 = 0.0;
      for (int i = 0; i < 3; ++i)
         sum2 += Math2::sqr(p2[i] - p1[i]);
      return ::sqrt(sum2);
   }
   
   //   /**
   //   * distanceSqr - What is the square of the Euclidean distance between p1 and
   //   * p2.
   //   *
   //   * @param p1
   //   * @param p2
   //   * @return double
   //   */
   //   final public double distanceSqr(double[] p1, double[] p2) {
   //      assert(p1.length == p2.length);
   //      double sum2 = 0.0;
   //      for (int i = 0; i < p1.length; ++i)
   //         sum2 += Math2.sqr(p2[i] - p1[i]);
   //      return sum2;
   //   }
   //
   //   /**
   //   * magnitude - What is the length of the specified vector?
   //   *
   //   * @param p
   //   * @return double
   //   */
   //   final public double magnitude(double[] p) {
   //      double sum2 = 0.0;
   //      for (int i = 0; i < p.length; ++i)
   //         sum2 += p[i] * p[i];
   //      return Math.sqrt(sum2);
   //   }

   double magnitude(const VectorXd& p)
   {
      double sum2 = 0.0;
      for (int i = 0; i < p.size(); ++i)
         sum2 += p[i] * p[i];
      return ::sqrt(sum2);
   }

   //   /**
   //   * normalize - Returns a copy of the input vector normalized to 1 length of
   //   * 1.0.
   //   *
   //   * @param p The vector to normalize
   //   * @return A new vector containing the normalized result.
   //   */
   //double[] normalize(double[] p) {
   //   return divide(p, Math2.magnitude(p));
   //}

   VectorXd normalize(const VectorXd& p)
   {
      return Math2::divide(p, Math2::magnitude(p));
   }
   
   //   /**
   //   * sum - Returns the sum of the specified array
   //   *
   //   * @param da
   //   * @return The sum of the specified array
   //   */
   //   final public double sum(double[] da) {
   //      double res = 0.0;
   //      for (int i = 0; i < da.length; ++i)
   //         res += da[i];
   //      return res;
   //   }
   //
   //   /**
   //   * sum - Returns the sum of the specified array
   //   *
   //   * @param da
   //   * @return The sum of the specified array
   //   */
   //   final public int sum(int[] da) {
   //      int res = 0;
   //      for (int i = 0; i < da.length; ++i)
   //         res += da[i];
   //      return res;
   //   }
   //
   //   /**
   //   * plus - Returns the vector sum a + b
   //   *
   //   * @param a
   //   * @param b
   //   * @return The vector a+b
   //   */
   //double* plus(double a[], int alen, double b[], int blen)
   //{
   //   if (alen != blen) {
   //      printf("Both arguments to the plus operator must be the same length.");
   //      return;
   //   }
   //   double* res = new double[alen];
   //   for (int i = 0; i < alen; ++i)
   //      res[i] = a[i] + b[i];
   //   return res;
   //}

   VectorXd plus(const VectorXd& a, const VectorXd& b)
   {
      VectorXd res(a.size(), 0);
      if (a.size() != b.size()) {
         printf("Math2::plus bad len: %d, %d", a.size(), b.size());
         return res;
      }
      for (int i = 0; i < a.size(); ++i)
         res[i] = a[i] + b[i];
      return res;
   }
   //
   //   /**
   //   * plus - Returns the vector sum a + b
   //   *
   //   * @param a
   //   * @param b
   //   * @return The vector a[i]+b
   //   */
   //   final public double[] plus(double[] a, double b) {
   //      final double[] res = new double[a.length];
   //      for (int i = 0; i < a.length; ++i)
   //         res[i] = a[i] + b;
   //      return res;
   //   }
   //
   //   /**
   //   * Replaces a with the sum a+b and returns a+b in a.
   //   *
   //   * @param a
   //   * @param b
   //   * @return The vector a+b
   //   */
   //   final public double[] plusEquals(double[] a, double[] b) {
   //      if (a.length != b.length)
   //         throw new IllegalArgumentException("Both arguments to the plus operator must be the same length.");
   //      for (int i = 0; i < a.length; ++i)
   //         a[i] += b[i];
   //      return a;
   //   }
   //
   //   /**
   //   * minus - Returns the vector difference a - b
   //   *
   //   * @param a
   //   * @param b
   //   * @return The vector a+b
   //   */
   //double* minus(double a[], int alen, double b[], int blen)
   //{
   //   if (alen != blen) {
   //      printf("Both arguments to the plus operator must be the same length.");
   //      return NULL;
   //   }
   //   double* res = new double[alen];
   //   for (int i = 0; i < alen; ++i)
   //      res[i] = a[i] - b[i];
   //   return res;
   //}
   //
   VectorXd minus(const VectorXd& a, const VectorXd& b)
   {
      VectorXd res(a.size(), 0);
      if (a.size() != b.size()) {
         printf("Math2::plus bad len: %d, %d", a.size(), b.size());
         return res;
      }
      for (int i = 0; i < a.size(); ++i)
         res[i] = a[i] - b[i];
      return res;
   }
   //
   //   /**
   //   * minus - Returns the vector sum a + b
   //   *
   //   * @param a
   //   * @param b
   //   * @return The vector a[i]+b
   //   */
   //   final public double[] minus(double[] a, double b) {
   //      final double[] res = new double[a.length];
   //      for (int i = 0; i < a.length; ++i)
   //         res[i] = a[i] - b;
   //      return res;
   //   }
   
      /**
      * dot - Compute the dot product of two equal lengthed vectors
      *
      * @param a
      * @param b
      * @return The dot product of a and b
      */
   double dot(const VectorXd& a, const VectorXd& b)
   {
      if (a.size() != b.size()) printf("Both arguments to the dot product must be the same length.");
      double res = 0.0;
      for (int i = 0; i < a.size(); ++i) {
         res += a[i] * b[i];
      }
      return res;
   }
   
   //   /**
   //   * Returns a vector in which each of the coordinates equals -1 times the
   //   * coordinates in a.
   //   *
   //   * @param a
   //   * @return double [] = -a
   //   */
   //   final public double[] negative(double[] a) {
   //      final double[] na = new double[a.length];
   //      for (int i = 0; i < a.length; ++i)
   //         na[i] = -a[i];
   //      return na;
   //   }
   //
      /**
      * cross - Computes the cross product of two three-vectors a and b
      *
      * @param a
      * @param b
      * @return A three-vector perpendicular to both a and b and of length |a||b|
      *         sin(th) where th is the angle between a and b
      */
   VectorXd cross(const VectorXd& a, const VectorXd& b)
   {
      if ((a.size() != 3) | (b.size()!= 3)) printf("Both arguments to the cross product must be the three-vectors.");
      return VectorXd({ a[1] * b[2] - a[2] * b[1], a[2] * b[0] - a[0] * b[2], a[0] * b[1] - a[1] * b[0] });
   }
   //
   //   /**
   //   * multiply - Returns a vector containing the product of a, a scalar, times
   //   * b, a vector.
   //   *
   //   * @param a
   //   * @param b
   //   * @return A vector containing a*b
   //   */
   //   final public double[] multiply(double a, double[] b) {
   //      final double[] res = new double[b.length];
   //      for (int i = 0; i < b.length; ++i)
   //         res[i] = a * b[i];
   //      return res;
   //   }
   //
   VectorXd multiply(double a, const VectorXd& b)
   {
      VectorXd res(b.size(), 0);
      for (int i = 0; i < 3; ++i)
         res[i] = a * b[i];
      return res;
   }
   //
   //   /**
   //   * multiply - Returns a vector containing the product of a, a vector, times
   //   * b, a vector where the multiplication is done piecewise.
   //   *
   //   * @param a
   //   * @param b
   //   * @return A vector containing a[i]*b[i] for all i
   //   */
   //   final public double[] multiply(double[] a, double[] b) {
   //      final double[] res = new double[Math.min(a.length, b.length)];
   //      for (int i = 0; i < res.length; ++i)
   //         res[i] = a[i] * b[i];
   //      return res;
   //   }
   //
   //   final public double[] timesEquals(double a, double[] b) {
   //      for (int i = 0; i < b.length; ++i)
   //         b[i] = a * b[i];
   //      return b;
   //   }
   //
   //   final public double[] abs(double[] data) {
   //      final double[] res = new double[data.length];
   //      for (int i = 0; i < res.length; ++i)
   //         res[i] = (data[i] > 0.0 ? data[i] : 0.0);
   //      return res;
   //   }
   //
   /**
   * Returns a point that is a fraction <code>f</code> of the distance between
   * <code>a</code> and <code>b</code>. If f=0, then the result is
   * <code>a</code>. If f=1, then the result is <code>b</code>. This function
   * works in an arbitrary number of dimensions but <code>a</code> and
   * <code>b</code> must have the same number of dimensions.
   *
   * @param a
   * @param b
   * @param f
   * @return double[] - a point
   */
   VectorXd pointBetween(const VectorXd& a, const VectorXd& b, double f)
   {
      VectorXd res(a.size());
      for (int i = 0; i < res.size(); ++i)
         res[i] = a[i] + (b[i] - a[i]) * f;
      return res;
   }
   //
   //   /**
   //   * Returns true if the argument vector has magnitude of 1.0.
   //   *
   //   * @param a
   //   * @return boolean
   //   */
   //   final boolean isUnitVector(double[] a) {
   //      return Math.abs(Math2.magnitude(a) - 1.0) < a.length * Double.MIN_VALUE;
   //   }
   //
   //   /**
   //   * angleBetween - Returns the angle between the vector a and vector b in
   //   * radians.
   //   *
   //   * @param a
   //   * @param b
   //   * @return The angle 0.0 (parallel) to Pi (antiparallel)
   //   */
   //   final public double angleBetween(double[] a, double[] b) {
   //      return Math.acos(dot(a, b) / (magnitude(a) * magnitude(b)));
   //   }
   //
   //   /**
   //   * divide - Returns a vector containing a, a vector, divided by b, a scalar.
   //   *
   //   * @param a
   //   * @param b
   //   * @return A vector containing a/b
   //   */
   //   final public double[] divide(double[] a, double b) {
   //      final double[] res = new double[a.length];
   //      for (int i = 0; i < a.length; ++i)
   //         res[i] = a[i] / b;
   //      return res;
   //   }
   //
   VectorXd divide(const VectorXd& a, double b)
   {
      VectorXd res(a.size(), 0);
      for (int i = 0; i < 3; ++i)
         res[i] = a[i] / b;
      return res;
   }
   //
   //   final public double[] divideEquals(double[] a, double b) {
   //      for (int i = 0; i < a.length; ++i)
   //         a[i] = a[i] / b;
   //      return a;
   //   }
   //
   //   /**
   //   * A numerically stable method for solving the quadratic equation a*x^2 + b*x
   //   * + c = 0.
   //   *
   //   * @param a
   //   * @param b
   //   * @param c
   //   * @return double[2] containing the solutions or null if there is no real
   //   *         solution.
   //   */
   //   final public double[] quadraticSolver(double a, double b, double c) {
   //      double r = b * b - 4.0 * a * c;
   //      if (r < 0.0)
   //         return null;
   //      double q = -0.5 * (b + Math.signum(b) * Math.sqrt(r));
   //      return new double[] {
   //         q / a,
   //            c / q
   //      };
   //   }
   //
   //   const double cubeRoot(double x) {
   //      return x < 0.0 ? -Math.pow(-x, 1.0 / 3.0) : Math.pow(x, 1.0 / 3.0);
   //   }
   //
   //   public double[] cubicSolver(double a, double b, double c, double d) {
   //      // find the discriminant
   //      final double f = (3.0 * c / a - (b * b) / (a * a)) / 3.0;
   //      final double g = (2.0 * Math.pow(b / a, 3.0) - 9 * b * c / (a * a) + 27.0 * d / a) / 27.0;
   //      final double h = (g * g) / 4.0 + Math.pow(f, 3.0) / 27;
   //      // evaluate discriminant
   //      if (f == 0.0 && g == 0.0 && h == 0.0) {
   //         // 3 equal roots
   //         final double x = -cubeRoot(d / a);
   //         return new double[] {
   //            x,
   //               x,
   //               x
   //         };
   //      }
   //      else if (h <= 0) {
   //         // 3 real roots
   //         final double i = Math.sqrt((g * g) / 4.0 - h);
   //         final double j = cubeRoot(i);
   //         final double k = Math.acos(-(g / (2.0 * i)));
   //         final double m = Math.cos(k / 3.0);
   //         final double n = Math.sqrt(3.0) * Math.sin(k / 3.0);
   //         final double p = -(b / (3.0 * a));
   //         return new double[] {
   //            2 * j * m + p,
   //               -j * (m + n) + p,
   //               -j * (m - n) + p
   //         };
   //      }
   //      else {
   //         // 1 real root and 2 complex roots
   //         final double r = -0.5 * g + Math.sqrt(h);
   //         final double s = cubeRoot(r);
   //         final double t = -0.5 * g - Math.sqrt(h);
   //         final double u = cubeRoot(t);
   //         final double p = -(b / (3.0 * a));
   //         return new double[] {
   //            (s + u) + p
   //         };
   //      }
   //   }
   //
   //   /**
   //   * Compute the polynomial <code>coeff[0]+coeff[1]*x+coeff[2]*x^2+...</code>
   //   *
   //   * @param coeff
   //   * @param x
   //   * @return The result as a double
   //   */
   //   final public double polynomial(double[] coeff, double x) {
   //      double res = coeff[coeff.length - 1];
   //      for (int i = coeff.length - 2; i >= 0; --i)
   //         res = res * x + coeff[i];
   //      return res;
   //   }
   //
   //   public double closestTo(double[] vals, double val) {
   //      double res = vals[0];
   //      for (int i = 1; i < vals.length; ++i)
   //         if (Math.abs(vals[i] - val) < Math.abs(res - val))
   //            res = vals[i];
   //      return res;
   //   }
   //
   //   /**
   //   * Solve the polynomial equation
   //   * <code>coeff[0]+coeff[1]*x+coeff[2]*x^2+..=0</code> for x. Polynomials up
   //   * to order cubic are supported. Only real roots will be returned.
   //   *
   //   * @param coeff
   //   * @return An array containing all the real roots
   //   * @throws EPQException
   //   */
   //   final public double[] solvePoly(double[] coeff)
   //      throws EPQException{
   //      switch (coeff.length) {
   //      case 2:
   //         return new double[] {
   //            -coeff[0] / coeff[1]
   //         };
   //      case 3:
   //         return quadraticSolver(coeff[2], coeff[1], coeff[0]);
   //      case 4:
   //         return cubicSolver(coeff[3], coeff[2], coeff[1], coeff[0]);
   //      default:
   //         throw new EPQException("Solution not available");
   //      }
   //   }
   //
   //      final public double[] solvePoly(double[] coeff, double y)
   //      throws EPQException{
   //      double[] tmp = coeff.clone();
   //      tmp[0] -= y;
   //      return solvePoly(tmp);
   //   }
   //
   //      /**
   //      * li - A naive implementation of the Logarithmic Integral
   //      *
   //      * @param x
   //      * @return double
   //      */
   //      final public double li(double x) {
   //      if (x <= 1.0)
   //         throw new IllegalArgumentException("x>1.0 :" + Double.toString(x));
   //      final double lx = Math.log(x);
   //      double res = Math.log(lx) + 0.577215664901532860;
   //      double ff = 1.0;
   //      double lxp = 1.0;
   //      for (double f = 1.0; f < 20.0; ++f) {
   //         ff *= f;
   //         lxp *= lx;
   //         res += lxp / (ff * f);
   //      }
   //      return res;
   //   }
   //
   //   /**
   //   * Returns x trimmed such that if x is less than x0 then x0 is returned and
   //   * if x is greater than x1 then x1 is returned, otherwise x is returned.
   //   *
   //   * @param x
   //   * @param x0
   //   * @param x1
   //   * @return return x < x0 ? x0 : (x > x1 ? x1 : x);
   //   */
   //   final public double bound(double x, double x0, double x1) {
   //      if (x0 > x1) {
   //         final double t = x0;
   //         x0 = x1;
   //         x1 = t;
   //      }
   //      return Double.isNaN(x) ? x : (x < x0 ? x0 : (x > x1 ? x1 : x));
   //   }
   //
   //   /**
   //   * Returns x trimmed such that if x is less than x0 then x0 is returned and
   //   * if x is greater or equal to x1 then x1-1 is returned, otherwise x is
   //   * returned.
   //   *
   //   * @param x
   //   * @param x0 where x0 &lt; x1
   //   * @param x1 where x1 &gt; x0
   //   * @return return x < x0 ? x0 : (x >= x1 ? x1-1 : x);
   //   */
   //   final public int bound(int x, int x0, int x1) {
   //      assert(x0 < x1);
   //      return x < x0 ? x0 : (x >= x1 ? x1 - 1 : x);
   //   }
   //
   //   /**
   //   * Returns x if x>0, 0 if x<=0
   //   *
   //   * @param x
   //   * @return x if x>0, 0 if x<=0
   //   */
   //   const double positive(double x) {
   //      return x > 0.0 ? x : 0.0;
   //   }
   //
   //   /**
   //   * Returns x if x<0, 0 otherwise
   //   *
   //   * @param x
   //   * @return x if x<0, 0 otherwise
   //   */
   //   const double negative(double x) {
   //      return x < 0.0 ? x : 0.0;
   //   }
   //
   //   /**
   //   * Compute the number of permutions of N items chosen M at a time.
   //   *
   //   * @param n
   //   * @param m
   //   * @return double
   //   */
   //   final public int binomialCoefficient(int n, int m) {
   //      if ((n >= m) && (m > 0)) {
   //         double res = 1.0;
   //         for (int i = m + 1; i <= n; ++i)
   //            res *= i;
   //         for (int i = n - m; i > 0; --i)
   //            res /= i;
   //         assert(int) res == Math.round(res) : Double.toString(res);
   //         return (int)Math.round(res);
   //      }
   //      else
   //         return 0;
   //   }
   //
   //   final public double max(double[] da) {
   //      double res = -Double.MAX_VALUE;
   //      for (final double d : da)
   //         if (d > res)
   //            res = d;
   //      return res;
   //   }
   //
   //   final public double max(double[][] m) {
   //      double max = Math2.max(m[m.length - 1]);
   //      for (int h = m.length - 2; h >= 0; --h) {
   //         final double tmp = Math2.max(m[h]);
   //         if (tmp > max)
   //            max = tmp;
   //      }
   //      return max;
   //   }
   //
   //   final public int max(int[] da) {
   //      int res = da[0];
   //      for (final int d : da)
   //         if (d > res)
   //            res = d;
   //      return res;
   //   }
   //
   //   final public double min(double[] da) {
   //      double res = Double.MAX_VALUE;
   //      for (final double d : da)
   //         if (d < res)
   //            res = d;
   //      return res;
   //   }
   //
   //   final public int min(int[] da) {
   //      int res = da[0];
   //      for (final int d : da)
   //         if (d < res)
   //            res = d;
   //      return res;
   //   }
   //
   //   final public double min(double[][] m) {
   //      double min = Math2.min(m[m.length - 1]);
   //      for (int h = m.length - 2; h >= 0; --h) {
   //         final double tmp = Math2.min(m[h]);
   //         if (tmp < min)
   //            min = tmp;
   //      }
   //      return min;
   //   }
   //
   //   /**
   //   * Extract a slice of data from the array <code>data</code> starting with the
   //   * element indexed by <code>start</code> and of length <code>len</code>. This
   //   * function will fail with an IndexOutOfBoundsException exception if
   //   * <code>st+len>data.length</code>.
   //   *
   //   * @param data
   //   * @param st
   //   * @param len
   //   * @return double[] of length len
   //   */
   //   public double[] slice(double[] data, int st, int len) {
   //      final double[] res = new double[len];
   //      System.arraycopy(data, st, res, 0, len);
   //      return res;
   //   }
   //
   //   public double pNorm(double[] data, double p) {
   //      double res = 0.0;
   //      for (int i = 0; i < data.length; ++i)
   //         res += Math.pow(Math.abs(data[i]), p);
   //      return Math.pow(res, 1.0 / p);
   //   }
   //
   //   public double infinityNorm(double[] data) {
   //      double res = 0.0;
   //      for (int i = 0; i < data.length; ++i)
   //         if (res < Math.abs(data[i]))
   //            res = Math.abs(data[i]);
   //      return res;
   //   }
   //
   //   /**
   //   * Computes the Legendre polynomials up to n=10.
   //   *
   //   * @param x
   //   * @param n In range 0 to 10
   //   * @return double
   //   */
   //   public double Legendre(double x, int n) {
   //      switch (n) {
   //      case 0:
   //         return 1.0;
   //      case 1:
   //         return x;
   //      case 2:
   //         return 0.5 * (-1.0 + 3.0 * x * x);
   //      case 3:
   //         return 0.5 * x * (-3.0 + 5.0 * x * x);
   //      case 4: {
   //         final double xx = x * x;
   //         return 0.125 * (3.0 + xx * (-30.0 + xx * 35.0));
   //      }
   //      case 5: {
   //         final double xx = x * x;
   //         return 0.125 * x * (15.0 + xx * (-70.0 + xx * 63.0));
   //      }
   //      case 6: {
   //         final double xx = x * x;
   //         return 0.0625 * (-5.0 + xx * (105.0 + xx * (-315.0 + xx * 231.0)));
   //      }
   //      case 7: {
   //         final double xx = x * x;
   //         return 0.0625 * x * (-35.0 + xx * (315.0 + xx * (-693.0 + 429.0 * xx)));
   //      }
   //      case 8: {
   //         final double xx = x * x;
   //         return 0.0078125 * (35.0 + xx * (-1260.0 + xx * (6930.0 + xx * (-12012.0 + xx * 6435.0))));
   //      }
   //      case 9: {
   //         final double xx = x * x;
   //         return 0.0078125 * x * (315.0 + xx * (-4620.0 + xx * (18018.0 + xx * (-25740.0 + xx * 12155.0))));
   //      }
   //      case 10: {
   //         final double xx = x * x;
   //         return 0.00390625 * (-63.0 + xx * (3465.0 + xx * (-30030.0 + xx * (90090.0 + xx * (-109395.0 + xx * 46189.0)))));
   //      }
   //      default:
   //         throw new IllegalArgumentException("Legendre order out of range [0,10].");
   //      }
   //   }
   //
   //   /**
   //   * Are <code>a</code> and <code>b</code> approximately equal to within
   //   * <code>frac</code> of the average of a+b. Use with care on numbers
   //   * <code>a</code> and <code>b</code> which are both strictly positive or
   //   * strictly negative.
   //   *
   //   * @param a
   //   * @param b
   //   * @param frac
   //   * @return approxEquals
   //   */
   //   public boolean approxEquals(double a, double b, double frac) {
   //      assert frac > 0.0;
   //      assert frac < 1.0;
   //      assert Math.abs(a + b) > Math.abs(a);
   //      return Math.abs(a - b) < 0.5 * Math.abs(a + b) * frac;
   //   }
   //
   //   /**
   //   * Convolves the specified kernel with the specified vector. The end members
   //   * of <code>v</code> are reused when the kernel extends past the ends.
   //   *
   //   * @param v
   //   * @param kernel
   //   * @return double[]
   //   */
   //   public double[] convolve(double[] v, double[] kernel) {
   //      assert kernel.length % 2 == 1;
   //      final double[] res = new double[v.length];
   //      final int mid = kernel.length / 2;
   //      for (int i = 0; i < res.length; ++i)
   //         for (int j = 0; j < kernel.length; ++j)
   //            res[i] += kernel[j] * v[bound(i + j - mid, 0, v.length)];
   //      return res;
   //   }
   //
   //   public String toString(double[] vec) {
   //      return toString(vec, NumberFormat.getInstance());
   //   }
   //
   //   public String toString(double[] vec, NumberFormat nf) {
   //      final StringBuffer sb = new StringBuffer();
   //      if (vec.length > 0) {
   //         sb.append(nf.format(vec[0]));
   //         for (int i = 1; i < vec.length; ++i) {
   //            sb.append(",");
   //            sb.append(nf.format(vec[i]));
   //         }
   //      }
   //      return sb.toString();
   //   }
   //
   //   public boolean isNaN(double[] arr) {
   //      for (final double d : arr)
   //         if (Double.isNaN(d))
   //            return true;
   //      return false;
   //   }
   //
   //   /**
   //   * Converts a double into a continued fraction to within the specified
   //   * tolerance.
   //   *
   //   * @param val
   //   * @param tol
   //   * @return long[]
   //   */
   //   public long[] toContinuedFraction(double val, double tol) {
   //      final long[] res = new long[10];
   //      final double[] num = new double[res.length + 2];
   //      final double[] den = new double[res.length + 2];
   //      num[1] = 1.0;
   //      den[0] = 1.0;
   //      final double sign = Math.signum(val);
   //      double rem = Math.abs(val);
   //      for (int i = 0; i < res.length; ++i) {
   //         res[i] = (long)Math.floor(rem);
   //         num[i + 2] = res[i] * num[i + 1] + num[i];
   //         den[i + 2] = res[i] * den[i + 1] + den[i];
   //         System.out.println(num[i + 2] / den[i + 2]);
   //         rem -= res[i];
   //         if (Math.abs(num[i + 2] / den[i + 2] - Math.abs(val)) < tol) {
   //            res[0] = (long)(sign * res[0]);
   //            return Arrays.copyOfRange(res, 0, i + 1);
   //         }
   //         rem = 1.0 / rem;
   //      }
   //      res[0] = (long)(sign * res[0]);
   //      return res;
   //   }
   //
   //   /**
   //   * Convert a continued fraction into a double
   //   *
   //   * @param cf
   //   * @return double
   //   */
   //   public double toDecimal(long[] cf) {
   //      double x = cf[cf.length - 1], y = 1.0;
   //      for (int i = cf.length - 2; i >= 1; --i) {
   //         double oldX = x;
   //         x = cf[i] * x + y;
   //         y = oldX;
   //      }
   //      return cf[0] > 0 ? cf[0] + y / x : cf[0] - y / x;
   //   }
   //
   //   /**
   //   * Convert a continued fraction into a a long[2] containing the numerator [0]
   //   * and denominator [1] of a fraction.
   //   *
   //   * @param cf
   //   * @return long[2]
   //   */
   //   public long[] toFraction(long[] cf) {
   //      long x = cf[cf.length - 1], y = 1;
   //      for (int i = cf.length - 2; i >= 1; --i) {
   //         long oldX = x;
   //         x = cf[i] * x + y;
   //         y = oldX;
   //      }
   //      if (cf[0] > 0)
   //         return new long[] {
   //         cf[0] * x + y,
   //            x
   //      };
   //      else
   //         return new long[] {
   //         cf[0] * x - y,
   //            x
   //      };
   //   }
   //
   //   public Matrix createRowMatrix(double[] vals) {
   //      return new Matrix(vals, vals.length);
   //   }
   //
   //   /**
   //   * Computes the greatest common divisor of a and b. The result is always
   //   * positive.
   //   *
   //   * @param a
   //   * @param b
   //   * @return gcd(a,b) -> a % gcd(a,b)==0 and b % gcd(a,b)==0
   //   */
   //   public long gcd(long a, long b) {
   //      if (b == 0)
   //         return Math.abs(a);
   //      return gcd(b, a - b * (a / b));
   //   }
   //
   //   /**
   //   * Computes the quadratic equation (ax^2+bx+c=0) in a numerically stable
   //   * manner.
   //   *
   //   * @param a Quadratic coefficient
   //   * @param b Linear coefficient
   //   * @param c Offset
   //   * @return The two roots or null if there are no real roots.
   //   */
   //   public double[] solveQuadratic(double a, double b, double c)
   //      throws EPQException{
   //      return solvePoly(new double[] {
   //         c,
   //            b,
   //            a
   //      });
   //   }
   //
   //      /**
   //      * Solves the cubic equation (x^3+ax^2+bx+c=0) in a numerically stable
   //      * manner.
   //      *
   //      * @param a Quadratic coefficient
   //      * @param b Linear coefficient
   //      * @param c Offset
   //      * @return The two roots or null if there are no real roots.
   //      */
   //      public double[] solveCubic(double a, double b, double c) {
   //      final double q = (a * a - 3.0 * b) / 9.0;
   //      final double r = (2.0 * a * a * a - 9.0 * a * b + 27.0 * c) / 54.0;
   //      if (r * r < q * q * q) {
   //         // Three real roots
   //         final double th = Math.acos(r / Math.pow(q, 1.5));
   //         return new double[] {
   //            -2.0 * q * Math.cos(th / 3.0) - a / 3.0,
   //               -2.0 * q * Math.cos((th + 2.0 * Math.PI) / 3.0) - a / 3.0,
   //               -2.0 * q * Math.cos((th - 2.0 * Math.PI) / 3.0) - a / 3.0
   //         };
   //      }
   //      else {
   //         // One real root
   //         final double A = -Math.signum(r) * Math.pow((Math.abs(r) + Math.sqrt(r * r - q * q * q)), 1.0 / 3.0);
   //         final double B = (a == 0.0 ? 0.0 : q / a);
   //         return new double[] {
   //            (A + B) - a / 3.0
   //         };
   //      }
   //   }
   //
   //   public double findRoot(double[] coeffs, double x1, double x2, double xacc)
   //      throws EPQException{
   //      final int MAXIT = 100;
   //      double[] deriv = new double[coeffs.length - 1];
   //      for (int i = 0; i < deriv.length; ++i)
   //         deriv[i] = coeffs[i + 1] * (i + 1);
   //      final double fl = polynomial(coeffs, x1);
   //      final double fh = polynomial(coeffs, x2);
   //      if (Math.signum(fl) == Math.signum(fh))
   //         throw new EPQException("End points must bracket the root in Math2.findRoot.");
   //      double xl = (fl < 0.0 ? x1 : x2);
   //      double xh = (fl < 0.0 ? x2 : x1);
   //      double rts = 0.5 * (x1 + x2);
   //      double dxold = Math.abs(x2 - x1);
   //      double dx = dxold;
   //      double f = polynomial(coeffs, rts);
   //      double df = polynomial(deriv, rts);
   //      for (int j = 0; j < MAXIT; ++j) {
   //         if ((((rts - xh) * df - f) * ((rts - xl) * df - f) >= 0.0) || (Math.abs(2.0 * f) > Math.abs(dxold * df))) {
   //            dxold = dx;
   //            dx = 0.5 * (xh - xl);
   //            rts = xl + dx;
   //            if (xl == rts)
   //               return rts;
   //         }
   //         else {
   //            dxold = dx;
   //            dx = f / df;
   //            final double temp = rts;
   //            rts -= dx;
   //            if (temp == rts)
   //               return rts;
   //         }
   //         if (Math.abs(dx) < xacc)
   //            return rts;
   //         f = polynomial(coeffs, rts);
   //         df = polynomial(deriv, rts);
   //         if (f < 0.0)
   //            xl = rts;
   //         else
   //            xh = rts;
   //      }
   //      throw new EPQException("Maximum iteration count exceeded in Math2.rootFind");
   //   }

   double random()
   {
      return (double)rand() / RAND_MAX;
   }

   int randomInt(int mod)
   {
      return rand() % mod;
   }

   double expRand()
   {
      return -::log(random());
   }

   double toRadians(double deg)
   {
      return deg * PI / 180.0;
   }

   double generateGaussianNoise(const double mean, const double stdDev)
   {
      static bool hasSpare = false;
      static double spare;

      if (hasSpare) {
         hasSpare = false;
         return mean + stdDev * spare;
      }

      hasSpare = true;
      static double u, v, s;
      do {
         u = (rand() / ((double)RAND_MAX)) * 2.0 - 1.0;
         v = (rand() / ((double)RAND_MAX)) * 2.0 - 1.0;
         s = u * u + v * v;
      }
      while ((s >= 1.0) || (s == 0.0));
      s = sqrt(-2.0 * log(s) / s);
      spare = v * s;
      return mean + stdDev * u * s;
   }
}