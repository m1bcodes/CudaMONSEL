#ifndef LINKED_LIST_CU
#define LINKED_LIST_CU

#include "hip/hip_runtime.h"

template<typename Key, typename Value>
class Node
{
public:
   __host__ __device__ static void InsertHead(Node** head, Key key, Value val);
   __host__ __device__ static void RemoveHead(Node** head);

   __host__ __device__ static void RemoveAll(Node** head);

   __host__ __device__ static void PrintList(Node* head);

private:
   Node* next;
   Key key;
   Value val;
};

#endif