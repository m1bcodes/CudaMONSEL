#include "hip/hip_runtime.h"
#include "UncertainValue2.cuh"
#include "..\..\..\..\Amphibian\Math.cuh"

const char UncertainValue2::DEFAULT[] = "Default";

//int UncertainValue2::sDefIndex = 0;
////const long UncertainValue2::serialVersionUID = 119495064970078787L;

//const UncertainValue2 UncertainValue2::ONE(1.0);
//const UncertainValue2 UncertainValue2::ZERO(0.0);
//const UncertainValue2 UncertainValue2::NaN(Double.NaN);
//const UncertainValue2 UncertainValue2::POSITIVE_INFINITY(Double.POSITIVE_INFINITY);
//const UncertainValue2 UncertainValue2::NEGATIVE_INFINITY(Double.NEGATIVE_INFINITY);

//__host__ __device__ UncertainValue2::UncertainValue2(double v, double dv) : mValue(v)
//{
//   char tmpName[MAX_LEN];
//   String::IToA(tmpName, ++sDefIndex);
//   UncertainValue2::UncertainValue2(v, tmpName, dv);
//}

//__host__ __device__ UncertainValue2::UncertainValue2(double v) : mValue(v)
//{
//   UncertainValue2::UncertainValue2(v, 0.0);
//}

__host__ __device__ UncertainValue2::UncertainValue2(double v, char source[], double dv) : mValue(v)
{
   assignComponent(source, dv);
}

__host__ __device__ UncertainValue2::UncertainValue2(double v, Node<String, double>* sigmas) : mValue(v)
{
   while (sigmas != NULL) {
      assignComponent(sigmas->GetKey(), sigmas->GetValue());
      sigmas = sigmas->GetNext();
   }
}

__host__ __device__ void UncertainValue2::assignComponent(String name, double sigma)
{
   if (sigma != 0.0) {
      Node<String, double>::InsertHead(&mSigmas, name, sigma);
   }
   else {
      Node<String, double>::Remove(&mSigmas, name, String::AreEqual);
   }
}
//
//double UncertainValue2::doubleValue() {
//   return mValue;
//}
//
//bool UncertainValue2::isUncertain() {
//   return mSigmas != NULL;
//}
//
//double UncertainValue2::uncertainty() {
//   double varnc = variance();
//   return Math::sqrt(varnc, varnc / 1000000);
//}
//
//double UncertainValue2::variance() {
//   double sigma2 = 0.0;
//   Node<String, double>* head = mSigmas;
//   while(head != NULL) {
//      double v = head->GetValue();
//      sigma2 += v * v;
//      head = head->GetNext();
//   }
//   return sigma2;
//}
//
//double UncertainValue2::fractionalUncertainty() {
//   return (mValue == 0 || isNaN()) ? 0 : Math::abs(uncertainty() / mValue);
//}
//
//bool UncertainValue2::isNaN() {
//   return mNotANumber;
//}
//
////int hashCode() {
////   return Objects.hash(mValue, mSigmas);
////}
//
//bool UncertainValue2::equals(UncertainValue2 * const obj) {
//   if (this == obj) {
//      return true;
//   }
//   if (obj == NULL) {
//      return false;
//   }
//   UncertainValue2 other = (UncertainValue2)*obj;
//   return Node<String, double>::AreEquivalentSets(mSigmas, other.mSigmas, String::AreEqual, [](double a, double b) { return a == b; }) && (mValue == other.mValue);
//}
