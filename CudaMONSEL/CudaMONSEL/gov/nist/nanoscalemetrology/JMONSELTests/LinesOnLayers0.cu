#include "hip/hip_runtime.h"
#include "gov\nist\nanoscalemetrology\JMONSELTests\LinesOnLayers0.cuh"

#include "gov\nist\microanalysis\EPQLibrary\ToSI.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\NISTMonte\NullMaterialScatterModel.cuh"
#include "gov\nist\microanalysis\NISTMonte\MonteCarloSS.cuh"
#include "gov\nist\microanalysis\NISTMonte\GaussianBeam.cuh"
#include "gov\nist\microanalysis\NISTMonte\RegionBase.cuh"
#include "gov\nist\microanalysis\NISTMonte\Sphere.cuh"
#include "gov\nist\microanalysis\NISTMonte\BackscatterStats.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"
#include "gov\nist\microanalysis\Utility\Histogram.cuh"

#include "gov\nist\nanoscalemetrology\JMONSEL\SEmaterial.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\ExpQMBarrierSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\MONSEL_MaterialScatterModel.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\SelectableElasticSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NISTMottRS.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\JoyLuoNieminenCSD.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\FittedInelSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\GanachaudMokraniPolaronTrapSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\TabulatedInelasticSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\GanachaudMokraniPhononInelasticSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NormalMultiPlaneShape.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NormalIntersectionShape.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NShapes.cuh"

#include "gov\nist\nanoscalemetrology\JMONSEL\NUTableInterpolation.cuh"

#include "Amphibian\random.cuh"

#include <fstream>

#include <chrono>

#include "CudaUtil.h"

namespace LinesOnLayers
{
   __global__ void verifyNUTable1d(const char* fn)
   {
      const NUTableInterpolationT* table = NUTableInterpolation::getInstance(fn);
      const VectorXd& data = table->gettable1d();
      printf("GPU %s\n", fn);
      for (auto v : data) {
         printf("%.5e ", v);
      }
      printf("\n");
   }

   __global__ void verifyNUTable2d(const char* fn, const int r)
   {
      const NUTableInterpolationT* table = NUTableInterpolation::getInstance(fn);
      const MatrixXd& data = table->gettable2d();
      printf("GPU %s: row %d\n", fn, r);
      for (auto v : data[r]) {
         printf("%.5e ", v);
      }
      printf("\n");
   }

   __global__ void verifyNUTable3d(const char* fn, const int r, const int c)
   {
      const NUTableInterpolationT* table = NUTableInterpolation::getInstance(fn);
      const Matrix3DXf& data = table->gettable3d();
      printf("GPU %s: row %d, col %d\n", fn, r, c);
      for (auto v : data[r][c]) {
         printf("%.5e ", v);
      }
      printf("\n");
   }

   void transferDataToCuda()
   {
      NUTableInterpolation::initFactory << <1, 1 >> >();
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());

      StringT tablePath = "C:\\Program Files\\NIST\\JMONSEL\\ScatteringTables\\glassyCTables\\";
      const StringT IIMFPPennInterpglassy = tablePath + "IIMFPPennInterpglassyCSI.csv";
      const StringT SimReducedDeltaEglassy = tablePath + "interpNUSimReducedDeltaEglassyCSI.csv";
      const StringT simTableThetaNUglassy = tablePath + "interpsimTableThetaNUglassyCSI.csv";
      const StringT SimESE0NUglassy = tablePath + "interpSimESE0NUglassyCSI.csv";
      NUTableInterpolation::transferDataToCuda(IIMFPPennInterpglassy.c_str());
      NUTableInterpolation::transferDataToCuda(SimReducedDeltaEglassy.c_str());
      NUTableInterpolation::transferDataToCuda(simTableThetaNUglassy.c_str());
      NUTableInterpolation::transferDataToCuda(SimESE0NUglassy.c_str());

      //tablePath = "C:\\Program Files\\NIST\\JMONSEL\\ScatteringTables\\SiTables\\";
      //const StringT IIMFPFullPennInterpSiSI = tablePath + "IIMFPFullPennInterpSiSI.csv";
      //const StringT interpNUSimReducedDeltaEFullPennSiSI = tablePath + "interpNUSimReducedDeltaEFullPennSiSI.csv";
      //const StringT interpNUThetaFullPennSiBGSI = tablePath + "interpNUThetaFullPennSiBGSI.csv";
      //const StringT interpSimESE0NUSiBGSI = tablePath + "interpSimESE0NUSiBGSI.csv";
      //NUTableInterpolation::copyDataToCuda(IIMFPFullPennInterpSiSI.c_str());
      //NUTableInterpolation::copyDataToCuda(interpNUSimReducedDeltaEFullPennSiSI.c_str());
      //NUTableInterpolation::copyDataToCuda(interpNUThetaFullPennSiBGSI.c_str());
      //NUTableInterpolation::copyDataToCuda(interpSimESE0NUSiBGSI.c_str());

      const char* fn = IIMFPPennInterpglassy.c_str();
      char* d_fn = nullptr;

      checkCudaErrors(hipMalloc((void **)&d_fn, (IIMFPPennInterpglassy.size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemset(d_fn, 0, (IIMFPPennInterpglassy.size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemcpy(d_fn, fn, (IIMFPPennInterpglassy.size() + 1) * sizeof(char), hipMemcpyHostToDevice));
      verifyNUTable1d << <1, 1 >> >(d_fn);
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipFree(d_fn));
      const NUTableInterpolationT* table0 = NUTableInterpolation::getInstance(fn);
      const VectorXd& data0 = table0->gettable1d();
      printf("CPU %s\n", fn);
      for (auto v : data0) {
         printf("%.5e ", v);
      }
      printf("\n");

      fn = SimReducedDeltaEglassy.c_str();
      checkCudaErrors(hipMalloc((void **)&d_fn, (SimReducedDeltaEglassy.size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemset(d_fn, 0, (SimReducedDeltaEglassy.size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemcpy(d_fn, fn, (SimReducedDeltaEglassy.size() + 1) * sizeof(char), hipMemcpyHostToDevice));
      verifyNUTable2d << <1, 1 >> >(d_fn, 0);
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipFree(d_fn));
      const NUTableInterpolationT* table1 = NUTableInterpolation::getInstance(fn);
      const MatrixXd& data1 = table1->gettable2d();
      printf("CPU %s: row %d\n", fn, 0);
      for (auto v : data1[0]) {
         printf("%.5e ", v);
      }
      printf("\n");

      fn = simTableThetaNUglassy.c_str();
      checkCudaErrors(hipMalloc((void **)&d_fn, (simTableThetaNUglassy.size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemset(d_fn, 0, (simTableThetaNUglassy.size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemcpy(d_fn, fn, (simTableThetaNUglassy.size() + 1) * sizeof(char), hipMemcpyHostToDevice));
      verifyNUTable3d << <1, 1 >> >(d_fn, 50, 50);
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipFree(d_fn));
      const NUTableInterpolationT* table2 = NUTableInterpolation::getInstance(fn);
      const Matrix3DXf& data2 = table2->gettable3d();
      printf("CPU %s: row %d, col %d\n", fn, 50, 50);
      for (auto v : data2[50][50]) {
         printf("%.5e ", v);
      }
      printf("\n");

      tablePath = "C:\\Program Files\\NIST\\JMONSEL\\ScatteringTables\\SiTables\\";
      const StringT IIMFPFullPennInterpSiSI = tablePath + "IIMFPFullPennInterpSiSI.csv";
      const StringT interpNUSimReducedDeltaEFullPennSiSI = tablePath + "interpNUSimReducedDeltaEFullPennSiSI.csv";
      const StringT interpNUThetaFullPennSiBGSI = tablePath + "interpNUThetaFullPennSiBGSI.csv";
      const StringT interpSimESE0NUSiBGSI = tablePath + "interpSimESE0NUSiBGSI.csv";
      NUTableInterpolation::transferDataToCuda(IIMFPFullPennInterpSiSI.c_str());
      NUTableInterpolation::transferDataToCuda(interpNUSimReducedDeltaEFullPennSiSI.c_str());
      NUTableInterpolation::transferDataToCuda(interpNUThetaFullPennSiBGSI.c_str());
      NUTableInterpolation::transferDataToCuda(interpSimESE0NUSiBGSI.c_str());
   }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
   __constant__ const int nTrajectories = 100;

   __constant__ const double pitchnm = 180;
   __constant__ const int nlines = 3;
   __constant__ const double hnm = 120;
   __constant__ const double wnm = 80;
   __constant__ const double linelengthnm = 1000;
   __constant__ const double thetardeg = 3;
   __constant__ const double thetaldeg = 3;
   __constant__ const double radrnm = 20;
   __constant__ const double radlnm = 20;
   __constant__ const double layer1thicknessnm = 80;
   __constant__ const double layer2thicknessnm = 200;

   __constant__ const double beamEeVvals[] = { 500. };
   __constant__ const int beamEeVvalsLen = 1;
   __constant__ const double beamsizenm = 0.5;
   __constant__ const double deepnm = 15;

   __constant__ const bool trajImg = true;
   __constant__ const int trajImgMaxTraj = 50;
   __constant__ const double trajImgSize = 200e-9;

   __constant__ const bool VRML = false;
   __constant__ const int VRMLImgMaxTraj = 0;

   //__device__ SEmaterialT* vacuum = nullptr;
   //__device__ ExpQMBarrierSMT* vacuumBarrier = nullptr;
   //__device__ ZeroCSDT* sZeroCSD = nullptr;

   //__device__ MONSEL_MaterialScatterModelT* vacuumMSM = nullptr;

   __constant__ const double PMMAbreakE = 1.60217653e-19 * 45.;
   __constant__ const double PMMAdensity = 1190.;
   __constant__ const double PMMAworkfun = 5.5;
   __constant__ const double PMMAbandgap = 5.;
   __constant__ const double PMMAEFermi = -5.;//-PMMAbandgap;
   __constant__ const double PMMApotU = -5.5 - (-5.);

   //__device__ SEmaterialT* PMMA = nullptr;

   //__device__ SelectableElasticSMT* PMMANISTMott = nullptr;

   //__device__ JoyLuoNieminenCSDT* PMMACSD = nullptr;
   //__device__ FittedInelSMT* PMMAfittedInel = nullptr;
   //__device__ GanachaudMokraniPolaronTrapSMT* PMMApolaron = nullptr;

   //__device__ ExpQMBarrierSMT* pmmaeqmbsm = nullptr;

   //__device__ MONSEL_MaterialScatterModelT* PMMAMSM = nullptr;

   //__device__ MONSEL_MaterialScatterModelT* PMMAMSMDeep = nullptr;

   //__device__ MONSEL_MaterialScatterModelT* ARCMSM = nullptr;

   __constant__ const double glCdensity = 1800.;
   __constant__ const double glCworkfun = 5.0;
   __constant__ const double glCbandgap = 0.;
   __constant__ const double glCEFermi = 20.4;
   __constant__ const double glCpotU = -5. - 20.4;

   //__device__ SEmaterialT* glC = nullptr;

   //__device__ SelectableElasticSMT* glCNISTMott = nullptr;

   //__device__ TabulatedInelasticSMT* glCDS = nullptr;

   //__device__ ExpQMBarrierSMT* glceqmbsm = nullptr;

   //__device__ MONSEL_MaterialScatterModelT* glCMSM = nullptr;

   //__device__ MONSEL_MaterialScatterModelT* glCMSMDeep = nullptr;

   __constant__ const double phononE = 0.063;
   __constant__ const double phononStrength = 3.;

   __constant__ const double Sidensity = 2330.;
   __constant__ const double Siworkfun = 4.85;
   __constant__ const double Sibandgap = 1.1;
   __constant__ const double SiEFermi = -1.1;//-Sibandgap;
   __constant__ const double SipotU = -44.85 - (-1.1);//-Siworkfun - SiEFermi;

   //__device__ SEmaterialT* Si = nullptr;

   //__device__ SelectableElasticSMT* SiNISTMott = nullptr;

   //__device__ TabulatedInelasticSMT* SiDS = nullptr;

   //__device__ GanachaudMokraniPhononInelasticSMT* Siphonon = nullptr;

   //__device__ ExpQMBarrierSMT* sieqmbsm = nullptr;

   //__device__ MONSEL_MaterialScatterModelT* SiMSM = nullptr;

   //__device__ MONSEL_MaterialScatterModelT* SiMSMDeep = nullptr;

   //__device__ SphereT* sphere = nullptr;

   __constant__ const double pitch = 180 * 1.e-9;
   __constant__ const double h = 120 * 1.e-9;
   __constant__ const double w = 80 * 1.e-9;
   __constant__ const double linelength = 1000 * 1.e-9;

   __constant__ const double radperdeg = 3.14159265358979323846 / 180.;
   __constant__ const double thetar = 3 * 3.14159265358979323846 / 180.;
   __constant__ const double thetal = 3 * 3.14159265358979323846 / 180.;
   __constant__ const double radr = 20 * 1.e-9;
   __constant__ const double radl = 20 * 1.e-9;
   __constant__ const double layer1thickness = 80 * 1.e-9;
   __constant__ const double layer2thickness = 200 * 1.e-9;
   __constant__ const double beamsize = 0.5 * 1.e-9;
   __constant__ const double deep = 15 * 1.e-9;

   __constant__ const double center[] = {
      0.0,
      0.0,
      0.0
   };

   __constant__ const double beamEeV = 500.;
   __constant__ const double beamE = 1.60217653e-19 * 500;
   __constant__ const double binSizeEV = 10.;

   //__device__ NullMaterialScatterModelT* NULL_MSM = nullptr;

   //__device__ RegionT* chamber = nullptr;

   __constant__ const double normalvector[] = { 0., 0., -1. };
   __constant__ const double layer1Pos[] = { 0., 0., 0. };

   //__device__ NormalMultiPlaneShapeT* layer1 = nullptr;
   //__device__ PlaneT* pl1 = nullptr;
   //__device__ RegionT* layer1Region = nullptr;

   __constant__ const double layer2Pos[] = { 0., 0., 80 * 1.e-9 };
   //__device__ NormalMultiPlaneShapeT* layer2 = nullptr;
   //__device__ PlaneT* pl2 = nullptr;
   //__device__ RegionT* layer2Region = nullptr;

   __constant__ const double layer3Pos[] = { 0., 0., 80 * 1.e-9 + 200 * 1.e-9 };
   //__device__ NormalMultiPlaneShapeT* layer3 = nullptr;
   //__device__ PlaneT* pl3 = nullptr;
   //__device__ RegionT* layer3Region = nullptr;

   __constant__ const double layer4Pos[] = { 0., 0., 80 * 1.e-9 + 200 * 1.e-9 + 15 * 1.e-9 };
   //__device__ NormalMultiPlaneShapeT* layer4 = nullptr;
   //__device__ PlaneT* pl4 = nullptr;
   //__device__ RegionT* layer4Region = nullptr;

   //__device__ RegionT* deepRegion = nullptr;

   __constant__ const double leftmostLineCenterx = -180. * 1.e-9 * (3. / 2.);
   __constant__ const double xcenter = -180. * 1.e-9 * (3. / 2.) + 0 * 180 * 1.e-9;

   //__device__ NormalIntersectionShapeT* line = nullptr;
   //__device__ RegionT* lineRegion = nullptr;

   __device__ double* yvals = nullptr;
   __device__ double* xvals = nullptr;
   __device__ unsigned int yvalsSize = 0;
   __device__ unsigned int xvalsSize = 0;
#else
   const int nTrajectories = 100;

   const double pitchnm = 180;
   const int nlines = 3;
   const double hnm = 120;
   const double wnm = 80;
   const double linelengthnm = 1000;
   const double thetardeg = 3;
   const double thetaldeg = 3;
   const double radrnm = 20;
   const double radlnm = 20;
   const double layer1thicknessnm = 80;
   const double layer2thicknessnm = 200;

   const double beamEeVvals[] = { 500. };
   const int beamEeVvalsLen = 1;
   const double beamsizenm = 0.5;
   const double deepnm = 15;

   const bool trajImg = true;
   const int trajImgMaxTraj = 50;
   const double trajImgSize = 200e-9;

   const bool VRML = false;
   const int VRMLImgMaxTraj = 0;

   //SEmaterialT* vacuum = nullptr;
   //ExpQMBarrierSMT* vacuumBarrier = nullptr;
   //ZeroCSDT* sZeroCSD = nullptr;

   //MONSEL_MaterialScatterModelT* vacuumMSM = nullptr;

   const double PMMAbreakE = 1.60217653e-19 * 45.;
   const double PMMAdensity = 1190.;
   const double PMMAworkfun = 5.5;
   const double PMMAbandgap = 5.;
   const double PMMAEFermi = -5.;//-PMMAbandgap;
   const double PMMApotU = -5.5 - (-5.);

   //SEmaterialT* PMMA = nullptr;

   //SelectableElasticSMT* PMMANISTMott = nullptr;

   //JoyLuoNieminenCSDT* PMMACSD = nullptr;
   //FittedInelSMT* PMMAfittedInel = nullptr;
   //GanachaudMokraniPolaronTrapSMT* PMMApolaron = nullptr;

   //ExpQMBarrierSMT* pmmaeqmbsm = nullptr;

   //MONSEL_MaterialScatterModelT* PMMAMSM = nullptr;

   //MONSEL_MaterialScatterModelT* PMMAMSMDeep = nullptr;

   //MONSEL_MaterialScatterModelT* ARCMSM = nullptr;

   const double glCdensity = 1800.;
   const double glCworkfun = 5.0;
   const double glCbandgap = 0.;
   const double glCEFermi = 20.4;
   const double glCpotU = -5. - 20.4;

   //SEmaterialT* glC = nullptr;

   //SelectableElasticSMT* glCNISTMott = nullptr;

   //TabulatedInelasticSMT* glCDS = nullptr;

   //ExpQMBarrierSMT* glceqmbsm = nullptr;

   //MONSEL_MaterialScatterModelT* glCMSM = nullptr;

   //MONSEL_MaterialScatterModelT* glCMSMDeep = nullptr;

   const double phononE = 0.063;
   const double phononStrength = 3.;

   const double Sidensity = 2330.;
   const double Siworkfun = 4.85;
   const double Sibandgap = 1.1;
   const double SiEFermi = -1.1;//-Sibandgap;
   const double SipotU = -44.85 - (-1.1);//-Siworkfun - SiEFermi;

   //SEmaterialT* Si = nullptr;

   //SelectableElasticSMT* SiNISTMott = nullptr;

   //TabulatedInelasticSMT* SiDS = nullptr;

   //GanachaudMokraniPhononInelasticSMT* Siphonon = nullptr;

   //ExpQMBarrierSMT* sieqmbsm = nullptr;

   //MONSEL_MaterialScatterModelT* SiMSM = nullptr;

   //MONSEL_MaterialScatterModelT* SiMSMDeep = nullptr;

   //SphereT* sphere = nullptr;

   const double pitch = 180 * 1.e-9;
   const double h = 120 * 1.e-9;
   const double w = 80 * 1.e-9;
   const double linelength = 1000 * 1.e-9;

   const double radperdeg = 3.14159265358979323846 / 180.;
   const double thetar = 3 * 3.14159265358979323846 / 180.;
   const double thetal = 3 * 3.14159265358979323846 / 180.;
   const double radr = 20 * 1.e-9;
   const double radl = 20 * 1.e-9;
   const double layer1thickness = 80 * 1.e-9;
   const double layer2thickness = 200 * 1.e-9;
   const double beamsize = 0.5 * 1.e-9;
   const double deep = 15 * 1.e-9;

   const double center[] = {
      0.0,
      0.0,
      0.0
   };

   const double beamEeV = 500.;
   const double beamE = 1.60217653e-19 * 500;
   const double binSizeEV = 10.;

   //NullMaterialScatterModelT* NULL_MSM = nullptr;

   //RegionT* chamber = nullptr;

   const double normalvector[] = { 0., 0., -1. };
   const double layer1Pos[] = { 0., 0., 0. };

   //NormalMultiPlaneShapeT* layer1 = nullptr;
   //PlaneT* pl1 = nullptr;
   //RegionT* layer1Region = nullptr;

   const double layer2Pos[] = { 0., 0., 80 * 1.e-9 };
   //NormalMultiPlaneShapeT* layer2 = nullptr;
   //PlaneT* pl2 = nullptr;
   //RegionT* layer2Region = nullptr;

   const double layer3Pos[] = { 0., 0., 80 * 1.e-9 + 200 * 1.e-9 };
   //NormalMultiPlaneShapeT* layer3 = nullptr;
   //PlaneT* pl3 = nullptr;
   //RegionT* layer3Region = nullptr;

   const double layer4Pos[] = { 0., 0., 80 * 1.e-9 + 200 * 1.e-9 + 15 * 1.e-9 };
   //NormalMultiPlaneShapeT* layer4 = nullptr;
   //PlaneT* pl4 = nullptr;
   //RegionT* layer4Region = nullptr;

   //RegionT* deepRegion = nullptr;

   const double leftmostLineCenterx = -180. * 1.e-9 * (3. / 2.);
   const double xcenter = -180. * 1.e-9 * (3. / 2.) + 0 * 180 * 1.e-9;

   //NormalIntersectionShapeT* line = nullptr;
   //RegionT* lineRegion = nullptr;

   double* yvals = nullptr;
   double* xvals = nullptr;
   unsigned int yvalsSize = 0;
   unsigned int xvalsSize = 0;
#endif

   __global__ void initCuda()
   //void initCuda()
   {
      printf("LinesOnLayers: initCuda\n");
      for (int i = 0; i < 10; ++i) {
         printf("%.10e\n", Random::random());
      }

      //vacuum = new SEmaterialT();
      //vacuum->setName("SE vacuum");
      //vacuumBarrier = new ExpQMBarrierSMT(vacuum);
      //sZeroCSD = new ZeroCSDT();

      //vacuumMSM = new MONSEL_MaterialScatterModelT(vacuum, vacuumBarrier, sZeroCSD);

//#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
//      const ElementT* componentsCOH[] = { Element::dC, Element::dO, Element::dH };
//#else
//      const ElementT* componentsCOH[] = { &Element::C, &Element::O, &Element::H };
//#endif
      //CompositionT* PMMAcomp = new CompositionT();
      //const double compositionCOH[] = { 5, 2, 8 };
      //PMMAcomp->defineByMoleFraction(componentsCOH, 3, compositionCOH, 3);
      //SEmaterialT* PMMA = new SEmaterialT(*PMMAcomp, PMMAdensity);
      //PMMA->setName("PMMA");
      //PMMA->setWorkfunction(ToSI::eV(PMMAworkfun));
      //PMMA->setBandgap(ToSI::eV(PMMAbandgap));
      //PMMA->setEnergyCBbottom(ToSI::eV(PMMApotU));

//#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
//      PMMANISTMott = new SelectableElasticSMT(*PMMA, *NISTMottRS::d_Factory);
//#else
//      PMMANISTMott = new SelectableElasticSMT(*PMMA, NISTMottRS::Factory);
//#endif

      //PMMACSD = new JoyLuoNieminenCSDT(*PMMA, PMMAbreakE);
      //PMMAfittedInel = new FittedInelSMT(*PMMA, ToSI::eV(65.4), *PMMACSD);
      //PMMApolaron = new GanachaudMokraniPolaronTrapSMT(2.e7, 1. / ToSI::eV(4.));

      //pmmaeqmbsm = new ExpQMBarrierSMT(PMMA);

      //PMMAMSM = new MONSEL_MaterialScatterModelT(PMMA, pmmaeqmbsm, sZeroCSD);
      //PMMAMSM->addScatterMechanism(PMMANISTMott);
      //PMMAMSM->addScatterMechanism(PMMAfittedInel);
      //PMMAMSM->addScatterMechanism(PMMApolaron);

      //PMMAMSM->setCSD(PMMACSD);

      //PMMAMSMDeep = new MONSEL_MaterialScatterModelT(PMMA, pmmaeqmbsm, sZeroCSD);
      //PMMAMSMDeep->addScatterMechanism(PMMANISTMott);
      //PMMAMSMDeep->addScatterMechanism(PMMAfittedInel);
      //PMMAMSMDeep->addScatterMechanism(PMMApolaron);

      //PMMAMSMDeep->setCSD(PMMACSD);
      //PMMAMSMDeep->setMinEforTracking(ToSI::eV(50.));

      //ARCMSM = PMMAMSM;

//#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
//      const ElementT* glCComponents[] = { Element::dC };
//#else
//      const ElementT* glCComponents[] = { &Element::C };
//#endif
//      const double glCComposition[] = { 1. };
//      glC = new SEmaterialT(glCComponents, 1, glCComposition, 1, glCdensity, "glassy Carbon");
//      glC->setWorkfunction(ToSI::eV(glCworkfun));
//      glC->setEnergyCBbottom(ToSI::eV(glCpotU));
//      glC->setBandgap(ToSI::eV(glCbandgap));
//      const double glCCoreEnergy[] = { ToSI::eV(284.2) };
//      glC->setCoreEnergy(glCCoreEnergy, 1);
//
//#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
//      glCNISTMott = new SelectableElasticSMT(*glC, *NISTMottRS::d_Factory);
//#else
//      glCNISTMott = new SelectableElasticSMT(*glC, NISTMottRS::Factory);
//#endif

      //StringT tablePath = "C:\\Program Files\\NIST\\JMONSEL\\ScatteringTables\\glassyCTables\\";
      //const StringT IIMFPPennInterpglassy = tablePath + "IIMFPPennInterpglassyCSI.csv";
      //const StringT SimReducedDeltaEglassy = tablePath + "interpNUSimReducedDeltaEglassyCSI.csv";
      //const StringT simTableThetaNUglassy = tablePath + "interpsimTableThetaNUglassyCSI.csv";
      //const StringT SimESE0NUglassy = tablePath + "interpSimESE0NUglassyCSI.csv";
      //const char* glCTables[] = {
      //   IIMFPPennInterpglassy.c_str(),
      //   SimReducedDeltaEglassy.c_str(),
      //   simTableThetaNUglassy.c_str(),
      //   SimESE0NUglassy.c_str()
      //};

      //glCDS = new TabulatedInelasticSMT(*glC, 3, glCTables);

      //glceqmbsm = new ExpQMBarrierSMT(glC);

      //glCMSM = new MONSEL_MaterialScatterModelT(glC, glceqmbsm, sZeroCSD);
      //glCMSM->addScatterMechanism(glCNISTMott);
      //glCMSM->addScatterMechanism(glCDS);

      //glCMSMDeep = new MONSEL_MaterialScatterModelT(glC, glceqmbsm, sZeroCSD);
      //glCMSMDeep->addScatterMechanism(glCNISTMott);
      //glCMSMDeep->addScatterMechanism(glCDS);

      //glCMSMDeep->setMinEforTracking(ToSI::eV(50.));

//#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
//      const ElementT* SiComponent[] = { Element::dSi };
//#else
//      const ElementT* SiComponent[] = { &Element::Si };
//#endif
      //const double SiComposition[] = { 1. };
      //Si = new SEmaterialT(SiComponent, 1, SiComposition, 1, Sidensity, "Silicon");
      //Si->setWorkfunction(ToSI::eV(Siworkfun));
      //Si->setEnergyCBbottom(ToSI::eV(SipotU));
      //Si->setBandgap(ToSI::eV(Sibandgap));
      //const double SiCoreEnergy[] = { ToSI::eV(99.2), ToSI::eV(99.8), ToSI::eV(149.7), ToSI::eV(1839.) };
      //Si->setCoreEnergy(SiCoreEnergy, 4);

//#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
//      SiNISTMott = new SelectableElasticSMT(*Si, *NISTMottRS::d_Factory);
//#else
//      SiNISTMott = new SelectableElasticSMT(*Si, NISTMottRS::Factory);
//#endif

      //tablePath = "C:\\Program Files\\NIST\\JMONSEL\\ScatteringTables\\SiTables\\";
      //const StringT IIMFPFullPennInterpSiSI = tablePath + "IIMFPFullPennInterpSiSI.csv";
      //const StringT interpNUSimReducedDeltaEFullPennSiSI = tablePath + "interpNUSimReducedDeltaEFullPennSiSI.csv";
      //const StringT interpNUThetaFullPennSiBGSI = tablePath + "interpNUThetaFullPennSiBGSI.csv";
      //const StringT interpSimESE0NUSiBGSI = tablePath + "interpSimESE0NUSiBGSI.csv";
      //const char* SiTables[] = {
      //   IIMFPFullPennInterpSiSI.c_str(),
      //   interpNUSimReducedDeltaEFullPennSiSI.c_str(),
      //   interpNUThetaFullPennSiBGSI.c_str(),
      //   interpSimESE0NUSiBGSI.c_str()
      //};

      //SiDS = new TabulatedInelasticSMT(*Si, 3, SiTables, ToSI::eV(13.54));

      //Siphonon = new GanachaudMokraniPhononInelasticSMT(phononStrength, ToSI::eV(phononE), 300., 11.7, 1.);

      //sieqmbsm = new ExpQMBarrierSMT(Si);

      //SiMSM = new MONSEL_MaterialScatterModelT(Si, sieqmbsm, sZeroCSD);
      //SiMSM->addScatterMechanism(SiNISTMott);
      //SiMSM->addScatterMechanism(SiDS);
      //SiMSM->addScatterMechanism(Siphonon);

      //SiMSMDeep = new MONSEL_MaterialScatterModelT(Si, sieqmbsm, sZeroCSD);
      //SiMSMDeep->addScatterMechanism(SiNISTMott);
      //SiMSMDeep->addScatterMechanism(SiDS);
      //SiMSMDeep->addScatterMechanism(Siphonon);

      //SiMSMDeep->setMinEforTracking(ToSI::eV(50.));

      //sphere = new SphereT(center, MonteCarloSS::ChamberRadius);

      //NULL_MSM = new NullMaterialScatterModelT();
      //chamber = new RegionT(nullptr, NULL_MSM, sphere);
      //chamber->updateMaterial(*(chamber->getScatterModel()), *vacuumMSM);

      //layer1 = new NormalMultiPlaneShapeT();
      //pl1 = new PlaneT(normalvector, layer1Pos);
      //layer1->addPlane(pl1);
      //layer1Region = new RegionT(chamber, ARCMSM, (NormalShapeT*)layer1);

      //layer2 = new NormalMultiPlaneShapeT();
      //pl2 = new PlaneT(normalvector, layer2Pos);
      //layer2->addPlane(pl2);
      //layer2Region = new RegionT(layer1Region, glCMSM, (NormalShapeT*)layer2);

      //layer3 = new NormalMultiPlaneShapeT();
      //pl3 = new PlaneT(normalvector, layer2Pos);
      //layer3->addPlane(pl3);
      //layer3Region = new RegionT(layer2Region, SiMSM, (NormalShapeT*)layer3);

      //layer4 = new NormalMultiPlaneShapeT();
      //pl4 = new PlaneT(normalvector, layer4Pos);
      //layer4Region = new RegionT(layer3Region, SiMSM, (NormalShapeT*)layer4);

      //deepRegion = new RegionT(layer3Region, SiMSMDeep, (NormalShapeT*)layer4);

      //for (int i = 0; i < nlines; ++i) {
      //   double xcenter = leftmostLineCenterx + i*pitch;
      //   NormalIntersectionShapeT* line = (NormalIntersectionShapeT*)NShapes::createLine(-h, w, linelength, thetal, thetar, radl, radr);
      //   const double newLinePos[] = { xcenter, 0., 0. };
      //   line->translate(newLinePos);
      //   RegionT lineRegion(&chamber, &PMMAMSM, line);
      //}

      //line = (NormalIntersectionShapeT*)NShapes::createLine(-h, w, linelength, thetal, thetar, radl, radr);
      //lineRegion = new RegionT(chamber, PMMAMSM, line);

      VectorXd yvalstmp(128);
      for (int i = -64; i < 64; i += 1) {
         yvalstmp.push_back(i);
      }

      const double xbottom = wnm / 2.;
      const double xtop = wnm / 2. - hnm * ::tan(thetar);
      const double xstart = xbottom - 100.5;
      const double xstop = xbottom + 100.5;
      const double xfinestart = xtop - 20.5;
      double xfinestop;
      if (thetar < 0.) xfinestop = xtop + 20.5;
      else xfinestop = wnm / 2. + 20.5;

      VectorXd xvalstmp(128);
      double deltax = 5.;
      double x = xstart;
      while (x < xfinestart) {
         xvalstmp.push_back(x);
         x += deltax;
      }
      x = xfinestart;
      deltax = 1;
      while (x < xfinestop) {
         xvalstmp.push_back(x);
         x += deltax;
      }
      x = xfinestop;
      deltax = 5.;
      while (x < xstop) {
         xvalstmp.push_back(x);
         x += deltax;
      }
      xvalstmp.push_back(xstop);

      yvalsSize = yvalstmp.size();
      xvalsSize = xvalstmp.size();

      yvals = new double[yvalsSize];
      xvals = new double[xvalsSize];

      memcpy(yvals, yvalstmp.data(), yvalsSize * sizeof(double));
      memcpy(xvals, xvalstmp.data(), xvalsSize * sizeof(double));
   }

   __global__ void runCudaSinglePixel()
   {
      int r = blockIdx.y*blockDim.y + threadIdx.y;
      int c = blockIdx.x*blockDim.x + threadIdx.x;

      int blockId = blockIdx.x + blockIdx.y * gridDim.x;
      int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
      printf("%d, %d (%d) began\n", r, c, threadId);

      const double ynm = yvals[r];
      const double y = ynm * 1.e-9;
      const double xnm = xvals[c];
      const double x = xnm * 1.e-9;

      SEmaterialT vacuum;
      vacuum.setName("SE vacuum");
      ExpQMBarrierSMT vacuumBarrier(&vacuum);
      ZeroCSDT sZeroCSD;

      MONSEL_MaterialScatterModelT vacuumMSM(&vacuum, &vacuumBarrier, &sZeroCSD);

      const ElementT* componentsCOH[] = { Element::dC, Element::dO, Element::dH };

      CompositionT PMMAcomp;
      const double compositionCOH[] = { 5, 2, 8 };
      PMMAcomp.defineByMoleFraction(componentsCOH, 3, compositionCOH, 3);
      SEmaterialT PMMA(PMMAcomp, PMMAdensity);
      PMMA.setName("PMMA");
      PMMA.setWorkfunction(ToSI::eV(PMMAworkfun));
      PMMA.setBandgap(ToSI::eV(PMMAbandgap));
      PMMA.setEnergyCBbottom(ToSI::eV(PMMApotU));

      SelectableElasticSMT PMMANISTMott(PMMA, *NISTMottRS::d_Factory);

      JoyLuoNieminenCSDT PMMACSD(PMMA, PMMAbreakE);
      FittedInelSMT PMMAfittedInel(PMMA, ToSI::eV(65.4), PMMACSD);
      GanachaudMokraniPolaronTrapSMT PMMApolaron(2.e7, 1. / ToSI::eV(4.));

      ExpQMBarrierSMT pmmaeqmbsm(&PMMA);

      MONSEL_MaterialScatterModelT PMMAMSM(&PMMA, &pmmaeqmbsm, &sZeroCSD);
      PMMAMSM.addScatterMechanism(&PMMANISTMott);
      PMMAMSM.addScatterMechanism(&PMMAfittedInel);
      PMMAMSM.addScatterMechanism(&PMMApolaron);

      PMMAMSM.setCSD(&PMMACSD);

      MONSEL_MaterialScatterModelT PMMAMSMDeep(&PMMA, &pmmaeqmbsm, &sZeroCSD);
      PMMAMSMDeep.addScatterMechanism(&PMMANISTMott);
      PMMAMSMDeep.addScatterMechanism(&PMMAfittedInel);
      PMMAMSMDeep.addScatterMechanism(&PMMApolaron);

      PMMAMSMDeep.setCSD(&PMMACSD);
      PMMAMSMDeep.setMinEforTracking(ToSI::eV(50.));

      MONSEL_MaterialScatterModelT& ARCMSM = PMMAMSM;

      const ElementT* glCComponents[] = { Element::dC };
      const double glCComposition[] = { 1. };
      SEmaterialT glC(glCComponents, 1, glCComposition, 1, glCdensity, "glassy Carbon");
      glC.setWorkfunction(ToSI::eV(glCworkfun));
      glC.setEnergyCBbottom(ToSI::eV(glCpotU));
      glC.setBandgap(ToSI::eV(glCbandgap));
      const double glCCoreEnergy[] = { ToSI::eV(284.2) };
      glC.setCoreEnergy(glCCoreEnergy, 1);

      SelectableElasticSMT glCNISTMott(glC, *NISTMottRS::d_Factory);

      StringT tablePath = "C:\\Program Files\\NIST\\JMONSEL\\ScatteringTables\\glassyCTables\\";
      const StringT IIMFPPennInterpglassy = tablePath + "IIMFPPennInterpglassyCSI.csv";
      const StringT SimReducedDeltaEglassy = tablePath + "interpNUSimReducedDeltaEglassyCSI.csv";
      const StringT simTableThetaNUglassy = tablePath + "interpsimTableThetaNUglassyCSI.csv";
      const StringT SimESE0NUglassy = tablePath + "interpSimESE0NUglassyCSI.csv";
      const char* glCTables[] = {
         IIMFPPennInterpglassy.c_str(),
         SimReducedDeltaEglassy.c_str(),
         simTableThetaNUglassy.c_str(),
         SimESE0NUglassy.c_str()
      };
      TabulatedInelasticSMT glCDS(glC, 3, glCTables);

      ExpQMBarrierSMT glceqmbsm(&glC);

      MONSEL_MaterialScatterModelT glCMSM(&glC, &glceqmbsm, &sZeroCSD);
      glCMSM.addScatterMechanism(&glCNISTMott);
      glCMSM.addScatterMechanism(&glCDS);

      MONSEL_MaterialScatterModelT glCMSMDeep(&glC, &glceqmbsm, &sZeroCSD);
      glCMSMDeep.addScatterMechanism(&glCNISTMott);
      glCMSMDeep.addScatterMechanism(&glCDS);

      glCMSMDeep.setMinEforTracking(ToSI::eV(50.));

      const ElementT* SiComponent[] = { Element::dSi };
      const double SiComposition[] = { 1. };
      SEmaterialT Si(SiComponent, 1, SiComposition, 1, Sidensity, "Silicon");
      Si.setWorkfunction(ToSI::eV(Siworkfun));
      Si.setEnergyCBbottom(ToSI::eV(SipotU));
      Si.setBandgap(ToSI::eV(Sibandgap));
      const double SiCoreEnergy[] = { ToSI::eV(99.2), ToSI::eV(99.8), ToSI::eV(149.7), ToSI::eV(1839.) };
      Si.setCoreEnergy(SiCoreEnergy, 4);

      SelectableElasticSMT SiNISTMott(Si, *NISTMottRS::d_Factory);

      tablePath = "C:\\Program Files\\NIST\\JMONSEL\\ScatteringTables\\SiTables\\";
      const StringT IIMFPFullPennInterpSiSI = tablePath + "IIMFPFullPennInterpSiSI.csv";
      const StringT interpNUSimReducedDeltaEFullPennSiSI = tablePath + "interpNUSimReducedDeltaEFullPennSiSI.csv";
      const StringT interpNUThetaFullPennSiBGSI = tablePath + "interpNUThetaFullPennSiBGSI.csv";
      const StringT interpSimESE0NUSiBGSI = tablePath + "interpSimESE0NUSiBGSI.csv";
      const char* SiTables[] = {
         IIMFPFullPennInterpSiSI.c_str(),
         interpNUSimReducedDeltaEFullPennSiSI.c_str(),
         interpNUThetaFullPennSiBGSI.c_str(),
         interpSimESE0NUSiBGSI.c_str()
      };
      
      TabulatedInelasticSMT SiDS(Si, 3, SiTables, ToSI::eV(13.54));

      GanachaudMokraniPhononInelasticSMT Siphonon(phononStrength, ToSI::eV(phononE), 300., 11.7, 1.);

      ExpQMBarrierSMT sieqmbsm(&Si);

      MONSEL_MaterialScatterModelT SiMSM(&Si, &sieqmbsm, &sZeroCSD);
      SiMSM.addScatterMechanism(&SiNISTMott);
      SiMSM.addScatterMechanism(&SiDS);
      SiMSM.addScatterMechanism(&Siphonon);

      MONSEL_MaterialScatterModelT SiMSMDeep(&Si, &sieqmbsm, &sZeroCSD);
      SiMSMDeep.addScatterMechanism(&SiNISTMott);
      SiMSMDeep.addScatterMechanism(&SiDS);
      SiMSMDeep.addScatterMechanism(&Siphonon);

      SiMSMDeep.setMinEforTracking(ToSI::eV(50.));

      SphereT sphere(center, MonteCarloSS::ChamberRadius);

      NullMaterialScatterModelT NULL_MSM;

      RegionT chamber(nullptr, &NULL_MSM, &sphere);
      chamber.updateMaterial(*(chamber.getScatterModel()), vacuumMSM);

      NormalMultiPlaneShapeT layer1;
      PlaneT pl1(normalvector, layer1Pos);
      layer1.addPlane(&pl1);
      RegionT layer1Region(&chamber, &ARCMSM, (NormalShapeT*)&layer1);

      NormalMultiPlaneShapeT layer2;
      PlaneT pl2(normalvector, layer2Pos);
      layer2.addPlane(&pl2);
      RegionT layer2Region(&layer1Region, &glCMSM, (NormalShapeT*)&layer2);

      NormalMultiPlaneShapeT layer3;
      PlaneT pl3(normalvector, layer2Pos);
      layer3.addPlane(&pl3);
      RegionT layer3Region(&layer2Region, &SiMSM, (NormalShapeT*)&layer3);

      NormalMultiPlaneShapeT layer4;
      PlaneT pl4(normalvector, layer4Pos);
      RegionT layer4Region(&layer3Region, &SiMSM, (NormalShapeT*)&layer4);

      RegionT deepRegion(&layer3Region, &SiMSMDeep, (NormalShapeT*)&layer4);

      NormalIntersectionShapeT* line = (NormalIntersectionShapeT*)NShapes::createLine(-h, w, linelength, thetal, thetar, radl, radr);
      RegionT lineRegion(&chamber, &PMMAMSM, line);

      GaussianBeamT eg(beamsize, beamE, center);
      const double egCenter[] = { x, y, -h - 20. * 1.e-9 };
      eg.setCenter(egCenter);
      MonteCarloSST monte(&eg, &chamber, nullptr);

      const int nbins = (int)(beamEeV / binSizeEV);
      BackscatterStatsT back(monte, nbins); //printf("48\n");
      monte.addActionListener(back);

      monte.runMultipleTrajectories(nTrajectories);

      const HistogramT& hist = back.backscatterEnergyHistogram(); //printf("49\n");

      const double energyperbineV = beamEeV / hist.binCount();
      const double maxSEbin = 50. / energyperbineV;
      int totalSE = 0;
      for (int j = 0; j < (int)maxSEbin; ++j) {
         totalSE = totalSE + hist.counts(j);
      }

      const double SEf = (float)totalSE / nTrajectories;
      const double bsf = back.backscatterFraction() - SEf;
      printf("%lf %lf %lf %lf %lf\n", beamEeV, xnm, ynm, bsf, SEf);
      monte.removeActionListener(back);

      printf("%d, %d (%d) ended\n", r, c, threadId);
   }

   //__global__ void runCudaSinglePixel(int r, int c)
   //{
   //   int blockId = blockIdx.x + blockIdx.y * gridDim.x;
   //   int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
   //   printf("%d, %d (%d) began\n", r, c, threadId);

   //   const double ynm = yvals[r];
   //   const double y = ynm * 1.e-9;
   //   const double xnm = xvals[c];
   //   const double x = xnm * 1.e-9;

   //   GaussianBeamT eg(beamsize, beamE, center);
   //   const double egCenter[] = { x, y, -h - 20. * 1.e-9 };
   //   eg.setCenter(egCenter);
   //   MonteCarloSST monte(&eg, chamber, nullptr);

   //   const int nbins = (int)(beamEeV / binSizeEV);
   //   BackscatterStatsT back(monte, nbins); //printf("48\n");
   //   monte.addActionListener(back);

   //   monte.runMultipleTrajectories(nTrajectories);

   //   const HistogramT& hist = back.backscatterEnergyHistogram(); //printf("49\n");

   //   const double energyperbineV = beamEeV / hist.binCount();
   //   const double maxSEbin = 50. / energyperbineV;
   //   int totalSE = 0;
   //   for (int j = 0; j < (int)maxSEbin; ++j) {
   //      totalSE = totalSE + hist.counts(j);
   //   }

   //   const double SEf = (float)totalSE / nTrajectories;
   //   const double bsf = back.backscatterFraction() - SEf;
   //   printf("%lf %lf %lf %lf %lf\n", beamEeV, xnm, ynm, bsf, SEf);
   //   monte.removeActionListener(back);

   //   printf("%d, %d (%d) ended\n", r, c, threadId);
   //}
//
//   __global__ void runCuda()
//   //void runCuda()
//   {
//      printf("LinesOnLayers: runCuda (%d, %d)\n", yvalsSize, xvalsSize);
//
//      //printf("\n# Trajectories at each landing position: %d", nTrajectories);
//      //printf("\n# Pitch of lines (nm): %.10e", pitchnm);
//      //printf("\n# lines: %d", nlines);
//      //printf("\nLine height (nm): %.10e", hnm);
//      //printf("\nLine bottom width (nm): %.10e", wnm);
//      //printf("\nLine length (nm): %.10e", linelengthnm);
//      //printf("\nLeft and right sidewall angles (deg): %.10e %.10e", thetaldeg, thetardeg);
//      //printf("\nLeft and right top corner radii (nm): %.10e %.10e", radlnm, radrnm);
//      //printf("\nThicknesses of 1st and second layers (nm): %.10e %.10e", layer1thicknessnm, layer2thicknessnm);
//      //printf("\nBeam landing energies (eV): ");
//
//      //for (int i = 0; i < beamEeVvalsLen; i++) {
//      //   printf("\n%.10e", beamEeVvals[i]);
//      //}
//      //printf("\nBeam size (standard deviation, in nm): %.10e", beamsizenm);
//
//      //printf("\n");
//      //printf("\nbeamE (eV)\t x(nm)\t y (nm)\t BSE yield\t SE yield");
//      GaussianBeamT* eg = new GaussianBeamT(beamsize, beamE, center);
//      MonteCarloSST* monte = new MonteCarloSST(eg, chamber, nullptr);
//
//#if (!(defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
//      std::string output;
//      auto start = std::chrono::system_clock::now();
//#endif
//
//      for (int r = 0; r < yvalsSize; ++r) {
//         const double ynm = yvals[r];
//         const double y = ynm * 1.e-9;
//         for (int c = 0; c < xvalsSize; ++c) {
//            double xnm = xvals[c];
//            const double x = xnm * 1.e-9;
//            const double egCenter[] = { x, y, -h - 20. * 1.e-9 };
//            eg->setCenter(egCenter);
//
//            const int nbins = (int)(beamEeV / binSizeEV);
//            BackscatterStatsT* back = new BackscatterStatsT(*monte, nbins); //printf("48\n");
//            monte->addActionListener(*back);
//
//            monte->runMultipleTrajectories(nTrajectories);
//
//            const HistogramT& hist = back->backscatterEnergyHistogram(); //printf("49\n");
//
//            const double energyperbineV = beamEeV / hist.binCount();
//            const double maxSEbin = 50. / energyperbineV;
//            int totalSE = 0;
//            for (int j = 0; j < (int)maxSEbin; ++j) {
//               totalSE = totalSE + hist.counts(j);
//            }
//
//            const double SEf = (float)totalSE / nTrajectories;
//            const double bsf = back->backscatterFraction() - SEf;
//            printf("%lf %lf %lf %lf %lf\n", beamEeV, xnm, ynm, bsf, SEf);
//#if (!(defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
//            output += std::to_string(beamEeV) + " " + std::to_string(xnm) + " " + std::to_string(ynm) + " " + std::to_string(bsf) + " " + std::to_string(SEf) + "\n";
//#endif
//            monte->removeActionListener(*back);
//            delete back;
//         }
//      }
//#if (!(defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
//      auto end = std::chrono::system_clock::now();
//      std::chrono::duration<double> elapsed_seconds = end - start;
//      std::time_t end_time = std::chrono::system_clock::to_time_t(end);
//      std::cout << std::endl << "finished computation at " << std::ctime(&end_time) << "elapsed time: " << elapsed_seconds.count() << "s" << std::endl;
//      output += std::to_string(elapsed_seconds.count());
//      
//      std::ofstream myfile;
//      myfile.open("output.txt");
//      myfile << output.c_str();
//      myfile.close();
//#endif
//   }
}