#include "hip/hip_runtime.h"
#include "gov\nist\nanoscalemetrology\JMONSELTests\LinesOnLayers0.cuh"

#include "gov\nist\microanalysis\EPQLibrary\ToSI.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\NISTMonte\NullMaterialScatterModel.cuh"
#include "gov\nist\microanalysis\NISTMonte\MonteCarloSS.cuh"
#include "gov\nist\microanalysis\NISTMonte\GaussianBeam.cuh"
#include "gov\nist\microanalysis\NISTMonte\RegionBase.cuh"
#include "gov\nist\microanalysis\NISTMonte\Sphere.cuh"
#include "gov\nist\microanalysis\NISTMonte\BackscatterStats.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"
#include "gov\nist\microanalysis\Utility\Histogram.cuh"

#include "gov\nist\nanoscalemetrology\JMONSEL\SEmaterial.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\ExpQMBarrierSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\MONSEL_MaterialScatterModel.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\SelectableElasticSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NISTMottRS.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\JoyLuoNieminenCSD.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\FittedInelSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\GanachaudMokraniPolaronTrapSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\TabulatedInelasticSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\GanachaudMokraniPhononInelasticSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NormalMultiPlaneShape.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NormalIntersectionShape.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NShapes.cuh"

#include "gov\nist\nanoscalemetrology\JMONSEL\NUTableInterpolation.cuh"

#include "Amphibian\random.cuh"

#include <fstream>

#include <chrono>

#include "CudaUtil.h"

namespace LinesOnLayers
{
   __global__ void verifyNUTable1d(const char* fn)
   {
      const NUTableInterpolationT* table = NUTableInterpolation::getInstance(fn);
      const VectorXd& data = table->gettable1d();
      printf("GPU %s\n", fn);
      for (auto v : data) {
         printf("%.5e ", v);
      }
      printf("\n");
   }

   __global__ void verifyNUTable2d(const char* fn, const int r)
   {
      const NUTableInterpolationT* table = NUTableInterpolation::getInstance(fn);
      const MatrixXd& data = table->gettable2d();
      printf("GPU %s: row %d\n", fn, r);
      for (auto v : data[r]) {
         printf("%.5e ", v);
      }
      printf("\n");
   }

   __global__ void verifyNUTable3d(const char* fn, const int r, const int c)
   {
      const NUTableInterpolationT* table = NUTableInterpolation::getInstance(fn);
      const Matrix3DXf& data = table->gettable3d();
      printf("GPU %s: row %d, col %d\n", fn, r, c);
      for (auto v : data[r][c]) {
         printf("%.5e ", v);
      }
      printf("\n");
   }

   void copyDataToCuda()
   {
      NUTableInterpolation::initFactory << <1, 1 >> >();
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());

      StringT tablePath = "C:\\Program Files\\NIST\\JMONSEL\\ScatteringTables\\glassyCTables\\";
      const StringT IIMFPPennInterpglassy = tablePath + "IIMFPPennInterpglassyCSI.csv";
      const StringT SimReducedDeltaEglassy = tablePath + "interpNUSimReducedDeltaEglassyCSI.csv";
      const StringT simTableThetaNUglassy = tablePath + "interpsimTableThetaNUglassyCSI.csv";
      const StringT SimESE0NUglassy = tablePath + "interpSimESE0NUglassyCSI.csv";
      NUTableInterpolation::copyDataToCuda(IIMFPPennInterpglassy.c_str());
      NUTableInterpolation::copyDataToCuda(SimReducedDeltaEglassy.c_str());
      NUTableInterpolation::copyDataToCuda(simTableThetaNUglassy.c_str());
      NUTableInterpolation::copyDataToCuda(SimESE0NUglassy.c_str());

      //tablePath = "C:\\Program Files\\NIST\\JMONSEL\\ScatteringTables\\SiTables\\";
      //const StringT IIMFPFullPennInterpSiSI = tablePath + "IIMFPFullPennInterpSiSI.csv";
      //const StringT interpNUSimReducedDeltaEFullPennSiSI = tablePath + "interpNUSimReducedDeltaEFullPennSiSI.csv";
      //const StringT interpNUThetaFullPennSiBGSI = tablePath + "interpNUThetaFullPennSiBGSI.csv";
      //const StringT interpSimESE0NUSiBGSI = tablePath + "interpSimESE0NUSiBGSI.csv";
      //NUTableInterpolation::copyDataToCuda(IIMFPFullPennInterpSiSI.c_str());
      //NUTableInterpolation::copyDataToCuda(interpNUSimReducedDeltaEFullPennSiSI.c_str());
      //NUTableInterpolation::copyDataToCuda(interpNUThetaFullPennSiBGSI.c_str());
      //NUTableInterpolation::copyDataToCuda(interpSimESE0NUSiBGSI.c_str());

      const char* fn = IIMFPPennInterpglassy.c_str();
      char* d_fn = nullptr;

      checkCudaErrors(hipMalloc((void **)&d_fn, (IIMFPPennInterpglassy.size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemset(d_fn, 0, (IIMFPPennInterpglassy.size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemcpy(d_fn, fn, (IIMFPPennInterpglassy.size() + 1) * sizeof(char), hipMemcpyHostToDevice));
      verifyNUTable1d << <1, 1 >> >(d_fn);
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipFree(d_fn));
      const NUTableInterpolationT* table0 = NUTableInterpolation::getInstance(fn);
      const VectorXd& data0 = table0->gettable1d();
      printf("CPU %s\n", fn);
      for (auto v : data0) {
         printf("%.5e ", v);
      }
      printf("\n");

      fn = SimReducedDeltaEglassy.c_str();
      checkCudaErrors(hipMalloc((void **)&d_fn, (SimReducedDeltaEglassy.size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemset(d_fn, 0, (SimReducedDeltaEglassy.size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemcpy(d_fn, fn, (SimReducedDeltaEglassy.size() + 1) * sizeof(char), hipMemcpyHostToDevice));
      verifyNUTable2d << <1, 1 >> >(d_fn, 0);
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipFree(d_fn));
      const NUTableInterpolationT* table1 = NUTableInterpolation::getInstance(fn);
      const MatrixXd& data1 = table1->gettable2d();
      printf("CPU %s: row %d\n", fn, 0);
      for (auto v : data1[0]) {
         printf("%.5e ", v);
      }
      printf("\n");

      fn = simTableThetaNUglassy.c_str();
      checkCudaErrors(hipMalloc((void **)&d_fn, (simTableThetaNUglassy.size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemset(d_fn, 0, (simTableThetaNUglassy.size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemcpy(d_fn, fn, (simTableThetaNUglassy.size() + 1) * sizeof(char), hipMemcpyHostToDevice));
      verifyNUTable3d << <1, 1 >> >(d_fn, 50, 50);
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipFree(d_fn));
      const NUTableInterpolationT* table2 = NUTableInterpolation::getInstance(fn);
      const Matrix3DXf& data2 = table2->gettable3d();
      printf("CPU %s: row %d, col %d\n", fn, 50, 50);
      for (auto v : data2[50][50]) {
         printf("%.5e ", v);
      }
      printf("\n");

      tablePath = "C:\\Program Files\\NIST\\JMONSEL\\ScatteringTables\\SiTables\\";
      const StringT IIMFPFullPennInterpSiSI = tablePath + "IIMFPFullPennInterpSiSI.csv";
      const StringT interpNUSimReducedDeltaEFullPennSiSI = tablePath + "interpNUSimReducedDeltaEFullPennSiSI.csv";
      const StringT interpNUThetaFullPennSiBGSI = tablePath + "interpNUThetaFullPennSiBGSI.csv";
      const StringT interpSimESE0NUSiBGSI = tablePath + "interpSimESE0NUSiBGSI.csv";
      NUTableInterpolation::copyDataToCuda(IIMFPFullPennInterpSiSI.c_str());
      NUTableInterpolation::copyDataToCuda(interpNUSimReducedDeltaEFullPennSiSI.c_str());
      NUTableInterpolation::copyDataToCuda(interpNUThetaFullPennSiBGSI.c_str());
      NUTableInterpolation::copyDataToCuda(interpSimESE0NUSiBGSI.c_str());
   }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
   __constant__ const int nTrajectories = 100;

   __constant__ const double pitchnm = 180;
   __constant__ const int nlines = 3;
   __constant__ const double hnm = 120;
   __constant__ const double wnm = 80;
   __constant__ const double linelengthnm = 1000;
   __constant__ const double thetardeg = 3;
   __constant__ const double thetaldeg = 3;
   __constant__ const double radrnm = 20;
   __constant__ const double radlnm = 20;
   __constant__ const double layer1thicknessnm = 80;
   __constant__ const double layer2thicknessnm = 200;

   __constant__ const double beamEeVvals[] = { 500. };
   __constant__ const int beamEeVvalsLen = 1;
   __constant__ const double beamsizenm = 0.5;
   __constant__ const double deepnm = 15;

   __constant__ const bool trajImg = true;
   __constant__ const int trajImgMaxTraj = 50;
   __constant__ const double trajImgSize = 200e-9;

   __constant__ const bool VRML = false;
   __constant__ const int VRMLImgMaxTraj = 0;

   __device__ SEmaterialT* vacuum = nullptr;
   __device__ ExpQMBarrierSMT* vacuumBarrier = nullptr;
   __device__ ZeroCSDT* sZeroCSD = nullptr;

   __device__ MONSEL_MaterialScatterModelT* vacuumMSM = nullptr;

   __constant__ const double PMMAbreakE = 1.60217653e-19 * 45.;
   __constant__ const double PMMAdensity = 1190.;
   __constant__ const double PMMAworkfun = 5.5;
   __constant__ const double PMMAbandgap = 5.;
   __constant__ const double PMMAEFermi = -5.;//-PMMAbandgap;
   __constant__ const double PMMApotU = -5.5 - (-5.);

   __device__ SEmaterialT* PMMA = nullptr;

   __device__ SelectableElasticSMT* PMMANISTMott = nullptr;

   __device__ JoyLuoNieminenCSDT* PMMACSD = nullptr;
   __device__ FittedInelSMT* PMMAfittedInel = nullptr;
   __device__ GanachaudMokraniPolaronTrapSMT* PMMApolaron = nullptr;

   __device__ ExpQMBarrierSMT* pmmaeqmbsm = nullptr;

   __device__ MONSEL_MaterialScatterModelT* PMMAMSM = nullptr;

   __device__ MONSEL_MaterialScatterModelT* PMMAMSMDeep = nullptr;

   __device__ MONSEL_MaterialScatterModelT* ARCMSM = nullptr;

   __constant__ const double glCdensity = 1800.;
   __constant__ const double glCworkfun = 5.0;
   __constant__ const double glCbandgap = 0.;
   __constant__ const double glCEFermi = 20.4;
   __constant__ const double glCpotU = -5. - 20.4;

   __device__ SEmaterialT* glC = nullptr;

   __device__ SelectableElasticSMT* glCNISTMott = nullptr;

   __device__ TabulatedInelasticSMT* glCDS = nullptr;

   __device__ ExpQMBarrierSMT* glceqmbsm = nullptr;

   __device__ MONSEL_MaterialScatterModelT* glCMSM = nullptr;

   __device__ MONSEL_MaterialScatterModelT* glCMSMDeep = nullptr;

   __constant__ const double phononE = 0.063;
   __constant__ const double phononStrength = 3.;

   __constant__ const double Sidensity = 2330.;
   __constant__ const double Siworkfun = 4.85;
   __constant__ const double Sibandgap = 1.1;
   __constant__ const double SiEFermi = -1.1;//-Sibandgap;
   __constant__ const double SipotU = -44.85 - (-1.1);//-Siworkfun - SiEFermi;

   __device__ SEmaterialT* Si = nullptr;

   __device__ SelectableElasticSMT* SiNISTMott = nullptr;

   __device__ TabulatedInelasticSMT* SiDS = nullptr;

   __device__ GanachaudMokraniPhononInelasticSMT* Siphonon = nullptr;

   __device__ ExpQMBarrierSMT* sieqmbsm = nullptr;

   __device__ MONSEL_MaterialScatterModelT* SiMSM = nullptr;

   __device__ MONSEL_MaterialScatterModelT* SiMSMDeep = nullptr;

   __device__ SphereT* sphere = nullptr;
   __device__ GaussianBeamT* eg = nullptr;

   __constant__ const double pitch = 180 * 1.e-9;
   __constant__ const double h = 120 * 1.e-9;
   __constant__ const double w = 80 * 1.e-9;
   __constant__ const double linelength = 1000 * 1.e-9;

   __constant__ const double radperdeg = 3.14159265358979323846 / 180.;
   __constant__ const double thetar = 3 * 3.14159265358979323846 / 180.;
   __constant__ const double thetal = 3 * 3.14159265358979323846 / 180.;
   __constant__ const double radr = 20 * 1.e-9;
   __constant__ const double radl = 20 * 1.e-9;
   __constant__ const double layer1thickness = 80 * 1.e-9;
   __constant__ const double layer2thickness = 200 * 1.e-9;
   __constant__ const double beamsize = 0.5 * 1.e-9;
   __constant__ const double deep = 15 * 1.e-9;

   __constant__ const double center[] = {
      0.0,
      0.0,
      0.0
   };

   __constant__ const double beamEeV = 500.;
   __constant__ const double beamE = 1.60217653e-19 * 500;
   __constant__ const double binSizeEV = 10.;

   __device__ NullMaterialScatterModelT* NULL_MSM = nullptr;

   __device__ RegionT* chamber = nullptr;

   __constant__ const double normalvector[] = { 0., 0., -1. };
   __constant__ const double layer1Pos[] = { 0., 0., 0. };

   __device__ NormalMultiPlaneShapeT* layer1 = nullptr;
   __device__ PlaneT* pl1 = nullptr;
   __device__ RegionT* layer1Region = nullptr;

   __constant__ const double layer2Pos[] = { 0., 0., 80 * 1.e-9 };
   __device__ NormalMultiPlaneShapeT* layer2 = nullptr;
   __device__ PlaneT* pl2 = nullptr;
   __device__ RegionT* layer2Region = nullptr;

   __constant__ const double layer3Pos[] = { 0., 0., 80 * 1.e-9 + 200 * 1.e-9 };
   __device__ NormalMultiPlaneShapeT* layer3 = nullptr;
   __device__ PlaneT* pl3 = nullptr;
   __device__ RegionT* layer3Region = nullptr;

   __constant__ const double layer4Pos[] = { 0., 0., 80 * 1.e-9 + 200 * 1.e-9 + 15 * 1.e-9 };
   __device__ NormalMultiPlaneShapeT* layer4 = nullptr;
   __device__ PlaneT* pl4 = nullptr;
   __device__ RegionT* layer4Region = nullptr;

   __device__ RegionT* deepRegion = nullptr;

   __constant__ const double leftmostLineCenterx = -180. * 1.e-9 * (3. / 2.);
   __constant__ const double xcenter = -180. * 1.e-9 * (3. / 2.) + 0 * 180 * 1.e-9;

   __device__ NormalIntersectionShapeT* line = nullptr;
   __device__ RegionT* lineRegion = nullptr;

   __device__ MonteCarloSST* monte = nullptr;
#else
   const int nTrajectories = 100;

   const double pitchnm = 180;
   const int nlines = 3;
   const double hnm = 120;
   const double wnm = 80;
   const double linelengthnm = 1000;
   const double thetardeg = 3;
   const double thetaldeg = 3;
   const double radrnm = 20;
   const double radlnm = 20;
   const double layer1thicknessnm = 80;
   const double layer2thicknessnm = 200;

   const double beamEeVvals[] = { 500. };
   const int beamEeVvalsLen = 1;
   const double beamsizenm = 0.5;
   const double deepnm = 15;

   const bool trajImg = true;
   const int trajImgMaxTraj = 50;
   const double trajImgSize = 200e-9;

   const bool VRML = false;
   const int VRMLImgMaxTraj = 0;

   SEmaterialT* vacuum = nullptr;
   ExpQMBarrierSMT* vacuumBarrier = nullptr;
   ZeroCSDT* sZeroCSD = nullptr;

   MONSEL_MaterialScatterModelT* vacuumMSM = nullptr;

   const double PMMAbreakE = 1.60217653e-19 * 45.;
   const double PMMAdensity = 1190.;
   const double PMMAworkfun = 5.5;
   const double PMMAbandgap = 5.;
   const double PMMAEFermi = -5.;//-PMMAbandgap;
   const double PMMApotU = -5.5 - (-5.);

   SEmaterialT* PMMA = nullptr;

   SelectableElasticSMT* PMMANISTMott = nullptr;

   JoyLuoNieminenCSDT* PMMACSD = nullptr;
   FittedInelSMT* PMMAfittedInel = nullptr;
   GanachaudMokraniPolaronTrapSMT* PMMApolaron = nullptr;

   ExpQMBarrierSMT* pmmaeqmbsm = nullptr;

   MONSEL_MaterialScatterModelT* PMMAMSM = nullptr;

   MONSEL_MaterialScatterModelT* PMMAMSMDeep = nullptr;

   MONSEL_MaterialScatterModelT* ARCMSM = nullptr;

   const double glCdensity = 1800.;
   const double glCworkfun = 5.0;
   const double glCbandgap = 0.;
   const double glCEFermi = 20.4;
   const double glCpotU = -5. - 20.4;

   SEmaterialT* glC = nullptr;

   SelectableElasticSMT* glCNISTMott = nullptr;

   TabulatedInelasticSMT* glCDS = nullptr;

   ExpQMBarrierSMT* glceqmbsm = nullptr;

   MONSEL_MaterialScatterModelT* glCMSM = nullptr;

   MONSEL_MaterialScatterModelT* glCMSMDeep = nullptr;

   const double phononE = 0.063;
   const double phononStrength = 3.;

   const double Sidensity = 2330.;
   const double Siworkfun = 4.85;
   const double Sibandgap = 1.1;
   const double SiEFermi = -1.1;//-Sibandgap;
   const double SipotU = -44.85 - (-1.1);//-Siworkfun - SiEFermi;

   SEmaterialT* Si = nullptr;

   SelectableElasticSMT* SiNISTMott = nullptr;

   TabulatedInelasticSMT* SiDS = nullptr;

   GanachaudMokraniPhononInelasticSMT* Siphonon = nullptr;

   ExpQMBarrierSMT* sieqmbsm = nullptr;

   MONSEL_MaterialScatterModelT* SiMSM = nullptr;

   MONSEL_MaterialScatterModelT* SiMSMDeep = nullptr;

   SphereT* sphere = nullptr;
   GaussianBeamT* eg = nullptr;

   const double pitch = 180 * 1.e-9;
   const double h = 120 * 1.e-9;
   const double w = 80 * 1.e-9;
   const double linelength = 1000 * 1.e-9;

   const double radperdeg = 3.14159265358979323846 / 180.;
   const double thetar = 3 * 3.14159265358979323846 / 180.;
   const double thetal = 3 * 3.14159265358979323846 / 180.;
   const double radr = 20 * 1.e-9;
   const double radl = 20 * 1.e-9;
   const double layer1thickness = 80 * 1.e-9;
   const double layer2thickness = 200 * 1.e-9;
   const double beamsize = 0.5 * 1.e-9;
   const double deep = 15 * 1.e-9;

   const double center[] = {
      0.0,
      0.0,
      0.0
   };

   const double beamEeV = 500.;
   const double beamE = 1.60217653e-19 * 500;
   const double binSizeEV = 10.;

   NullMaterialScatterModelT* NULL_MSM = nullptr;

   RegionT* chamber = nullptr;

   const double normalvector[] = { 0., 0., -1. };
   const double layer1Pos[] = { 0., 0., 0. };

   NormalMultiPlaneShapeT* layer1 = nullptr;
   PlaneT* pl1 = nullptr;
   RegionT* layer1Region = nullptr;

   const double layer2Pos[] = { 0., 0., 80 * 1.e-9 };
   NormalMultiPlaneShapeT* layer2 = nullptr;
   PlaneT* pl2 = nullptr;
   RegionT* layer2Region = nullptr;

   const double layer3Pos[] = { 0., 0., 80 * 1.e-9 + 200 * 1.e-9 };
   NormalMultiPlaneShapeT* layer3 = nullptr;
   PlaneT* pl3 = nullptr;
   RegionT* layer3Region = nullptr;

   const double layer4Pos[] = { 0., 0., 80 * 1.e-9 + 200 * 1.e-9 + 15 * 1.e-9 };
   NormalMultiPlaneShapeT* layer4 = nullptr;
   PlaneT* pl4 = nullptr;
   RegionT* layer4Region = nullptr;

   RegionT* deepRegion = nullptr;

   const double leftmostLineCenterx = -180. * 1.e-9 * (3. / 2.);
   const double xcenter = -180. * 1.e-9 * (3. / 2.) + 0 * 180 * 1.e-9;

   NormalIntersectionShapeT* line = nullptr;
   RegionT* lineRegion = nullptr;

   MonteCarloSST* monte = nullptr;
#endif

   __global__ void initCuda()
   //void initCuda()
   {
      printf("LinesOnLayers: initCuda\n");      for (int i = 0; i < 10; ++i) {
         printf("%.10e\n", Random::random());
      }

      vacuum = new SEmaterialT(); printf("0");
      vacuum->setName("SE vacuum");
      vacuumBarrier = new ExpQMBarrierSMT(vacuum); printf("1");
      sZeroCSD = new ZeroCSDT(); printf("2");

      vacuumMSM = new MONSEL_MaterialScatterModelT(vacuum, vacuumBarrier, sZeroCSD); printf("3");

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      const ElementT* componentsCOH[] = { Element::dC, Element::dO, Element::dH };
#else
      const ElementT* componentsCOH[] = { &Element::C, &Element::O, &Element::H };
#endif
      CompositionT* PMMAcomp = new CompositionT(); printf("4");
      const double compositionCOH[] = { 5, 2, 8 };
      PMMAcomp->defineByMoleFraction(componentsCOH, 3, compositionCOH, 3);
      SEmaterialT* PMMA = new SEmaterialT(*PMMAcomp, PMMAdensity); printf("4.5");
      PMMA->setName("PMMA");
      PMMA->setWorkfunction(ToSI::eV(PMMAworkfun));
      PMMA->setBandgap(ToSI::eV(PMMAbandgap));
      PMMA->setEnergyCBbottom(ToSI::eV(PMMApotU));

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      PMMANISTMott = new SelectableElasticSMT(*PMMA, *NISTMottRS::d_Factory); printf("5");
#else
      PMMANISTMott = new SelectableElasticSMT(*PMMA, NISTMottRS::Factory); printf("5");
#endif

      PMMACSD = new JoyLuoNieminenCSDT(*PMMA, PMMAbreakE); printf("6");
      PMMAfittedInel = new FittedInelSMT(*PMMA, ToSI::eV(65.4), *PMMACSD); printf("7");
      PMMApolaron = new GanachaudMokraniPolaronTrapSMT(2.e7, 1. / ToSI::eV(4.)); printf("8");

      pmmaeqmbsm = new ExpQMBarrierSMT(PMMA); printf("9");

      PMMAMSM = new MONSEL_MaterialScatterModelT(PMMA, pmmaeqmbsm, sZeroCSD); printf("10");
      PMMAMSM->addScatterMechanism(PMMANISTMott);
      PMMAMSM->addScatterMechanism(PMMAfittedInel);
      PMMAMSM->addScatterMechanism(PMMApolaron);

      PMMAMSM->setCSD(PMMACSD);

      PMMAMSMDeep = new MONSEL_MaterialScatterModelT(PMMA, pmmaeqmbsm, sZeroCSD); printf("11");
      PMMAMSMDeep->addScatterMechanism(PMMANISTMott);
      PMMAMSMDeep->addScatterMechanism(PMMAfittedInel);
      PMMAMSMDeep->addScatterMechanism(PMMApolaron);

      PMMAMSMDeep->setCSD(PMMACSD);
      PMMAMSMDeep->setMinEforTracking(ToSI::eV(50.));

      ARCMSM = PMMAMSM; printf("12");

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      const ElementT* glCComponents[] = { Element::dC };
#else
      const ElementT* glCComponents[] = { &Element::C };
#endif
      const double glCComposition[] = { 1. };
      glC = new SEmaterialT(glCComponents, 1, glCComposition, 1, glCdensity, "glassy Carbon"); printf("13");
      glC->setWorkfunction(ToSI::eV(glCworkfun));
      glC->setEnergyCBbottom(ToSI::eV(glCpotU));
      glC->setBandgap(ToSI::eV(glCbandgap));
      const double glCCoreEnergy[] = { ToSI::eV(284.2) };
      glC->setCoreEnergy(glCCoreEnergy, 1);

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      glCNISTMott = new SelectableElasticSMT(*glC, *NISTMottRS::d_Factory); printf("14");
#else
      glCNISTMott = new SelectableElasticSMT(*glC, NISTMottRS::Factory); printf("14");
#endif

      StringT tablePath = "C:\\Program Files\\NIST\\JMONSEL\\ScatteringTables\\glassyCTables\\";
      const StringT IIMFPPennInterpglassy = tablePath + "IIMFPPennInterpglassyCSI.csv";
      const StringT SimReducedDeltaEglassy = tablePath + "interpNUSimReducedDeltaEglassyCSI.csv";
      const StringT simTableThetaNUglassy = tablePath + "interpsimTableThetaNUglassyCSI.csv";
      const StringT SimESE0NUglassy = tablePath + "interpSimESE0NUglassyCSI.csv";
      const char* glCTables[] = {
         IIMFPPennInterpglassy.c_str(),
         SimReducedDeltaEglassy.c_str(),
         simTableThetaNUglassy.c_str(),
         SimESE0NUglassy.c_str()
      };

      glCDS = new TabulatedInelasticSMT(*glC, 3, glCTables); printf("15");

      glceqmbsm = new ExpQMBarrierSMT(glC); printf("16");

      glCMSM = new MONSEL_MaterialScatterModelT(glC, glceqmbsm, sZeroCSD); printf("17");
      glCMSM->addScatterMechanism(glCNISTMott);
      glCMSM->addScatterMechanism(glCDS);

      glCMSMDeep = new MONSEL_MaterialScatterModelT(glC, glceqmbsm, sZeroCSD); printf("18");
      glCMSMDeep->addScatterMechanism(glCNISTMott);
      glCMSMDeep->addScatterMechanism(glCDS);

      glCMSMDeep->setMinEforTracking(ToSI::eV(50.));

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      const ElementT* SiComponent[] = { Element::dSi };
#else
      const ElementT* SiComponent[] = { &Element::Si };
#endif
      const double SiComposition[] = { 1. };
      Si = new SEmaterialT(SiComponent, 1, SiComposition, 1, Sidensity, "Silicon"); printf("19");
      Si->setWorkfunction(ToSI::eV(Siworkfun));
      Si->setEnergyCBbottom(ToSI::eV(SipotU));
      Si->setBandgap(ToSI::eV(Sibandgap));
      const double SiCoreEnergy[] = { ToSI::eV(99.2), ToSI::eV(99.8), ToSI::eV(149.7), ToSI::eV(1839.) };
      Si->setCoreEnergy(SiCoreEnergy, 4);

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      SiNISTMott = new SelectableElasticSMT(*Si, *NISTMottRS::d_Factory); printf("20");
#else
      SiNISTMott = new SelectableElasticSMT(*Si, NISTMottRS::Factory); printf("20");
#endif

      tablePath = "C:\\Program Files\\NIST\\JMONSEL\\ScatteringTables\\SiTables\\";
      const StringT IIMFPFullPennInterpSiSI = tablePath + "IIMFPFullPennInterpSiSI.csv";
      const StringT interpNUSimReducedDeltaEFullPennSiSI = tablePath + "interpNUSimReducedDeltaEFullPennSiSI.csv";
      const StringT interpNUThetaFullPennSiBGSI = tablePath + "interpNUThetaFullPennSiBGSI.csv";
      const StringT interpSimESE0NUSiBGSI = tablePath + "interpSimESE0NUSiBGSI.csv";
      const char* SiTables[] = {
         IIMFPFullPennInterpSiSI.c_str(),
         interpNUSimReducedDeltaEFullPennSiSI.c_str(),
         interpNUThetaFullPennSiBGSI.c_str(),
         interpSimESE0NUSiBGSI.c_str()
      };

      SiDS = new TabulatedInelasticSMT(*Si, 3, SiTables, ToSI::eV(13.54)); printf("21");

      Siphonon = new GanachaudMokraniPhononInelasticSMT(phononStrength, ToSI::eV(phononE), 300., 11.7, 1.); printf("22");

      sieqmbsm = new ExpQMBarrierSMT(Si); printf("23");

      SiMSM = new MONSEL_MaterialScatterModelT(Si, sieqmbsm, sZeroCSD); printf("24");
      SiMSM->addScatterMechanism(SiNISTMott);
      SiMSM->addScatterMechanism(SiDS);
      SiMSM->addScatterMechanism(Siphonon);

      SiMSMDeep = new MONSEL_MaterialScatterModelT(Si, sieqmbsm, sZeroCSD); printf("25");
      SiMSMDeep->addScatterMechanism(SiNISTMott);
      SiMSMDeep->addScatterMechanism(SiDS);
      SiMSMDeep->addScatterMechanism(Siphonon);

      SiMSMDeep->setMinEforTracking(ToSI::eV(50.));

      sphere = new SphereT(center, MonteCarloSS::ChamberRadius); printf("26");
      eg = new GaussianBeamT(beamsize, beamE, center); printf("27");

      NULL_MSM = new NullMaterialScatterModelT(); printf("28");
      chamber = new RegionT(nullptr, NULL_MSM, sphere); printf("29");
      chamber->updateMaterial(*(chamber->getScatterModel()), *vacuumMSM);

      layer1 = new NormalMultiPlaneShapeT(); printf("30");
      pl1 = new PlaneT(normalvector, layer1Pos); printf("31");
      layer1->addPlane(pl1);
      layer1Region = new RegionT(chamber, ARCMSM, (NormalShapeT*)layer1); printf("32");

      layer2 = new NormalMultiPlaneShapeT(); printf("33");
      pl2 = new PlaneT(normalvector, layer2Pos); printf("34");
      layer2->addPlane(pl2);
      layer2Region = new RegionT(layer1Region, glCMSM, (NormalShapeT*)layer2); printf("35");

      layer3 = new NormalMultiPlaneShapeT(); printf("36");
      pl3 = new PlaneT(normalvector, layer2Pos); printf("37");
      layer3->addPlane(pl3);
      layer3Region = new RegionT(layer2Region, SiMSM, (NormalShapeT*)layer3); printf("38");

      layer4 = new NormalMultiPlaneShapeT(); printf("39");
      pl4 = new PlaneT(normalvector, layer4Pos); printf("40");
      layer4Region = new RegionT(layer3Region, SiMSM, (NormalShapeT*)layer4); printf("41");

      deepRegion = new RegionT(layer3Region, SiMSMDeep, (NormalShapeT*)layer4); printf("42");

      //for (int i = 0; i < nlines; ++i) {
      //   double xcenter = leftmostLineCenterx + i*pitch;
      //   NormalIntersectionShapeT* line = (NormalIntersectionShapeT*)NShapes::createLine(-h, w, linelength, thetal, thetar, radl, radr);
      //   const double newLinePos[] = { xcenter, 0., 0. };
      //   line->translate(newLinePos);
      //   RegionT lineRegion(&chamber, &PMMAMSM, line);
      //}

      line = (NormalIntersectionShapeT*)NShapes::createLine(-h, w, linelength, thetal, thetar, radl, radr); printf("43");
      lineRegion = new RegionT(chamber, PMMAMSM, line); printf("44");
   }

   __global__ void runCuda()
   //void runCuda()
   {
      printf("LinesOnLayers: runCuda\n");

      //VectorXd yvals(128); printf("45");
      //for (int i = -64; i < 64; i += 1) {
      //   yvals.push_back(i);
      //}
      VectorXd yvals(1, -64); printf("45");

      const double xbottom = wnm / 2.;
      const double xtop = wnm / 2. - hnm * ::tan(thetar);
      const double xstart = xbottom - 100.5;
      const double xstop = xbottom + 100.5;
      const double xfinestart = xtop - 20.5;
      double xfinestop;
      if (thetar < 0.) xfinestop = xtop + 20.5;
      else xfinestop = wnm / 2. + 20.5;

      //VectorXd xvals(128); printf("46");
      //double deltax = 5.;
      //double x = xstart;
      //while (x < xfinestart) {
      //   xvals.push_back(x);
      //   x += deltax;
      //}
      //x = xfinestart;
      //deltax = 1;
      //while (x < xfinestop) {
      //   xvals.push_back(x);
      //   x += deltax;
      //}
      //x = xfinestop;
      //deltax = 5.;
      //while (x < xstop) {
      //   xvals.push_back(x);
      //   x += deltax;
      //}
      //xvals.push_back(xstop);
      VectorXd xvals(1, xstart); printf("46");

      monte = new MonteCarloSST(eg, chamber, eg->createElectron()); printf("47");

      //printf("\n# Trajectories at each landing position: %d", nTrajectories);
      //printf("\n# Pitch of lines (nm): %.10e", pitchnm);
      //printf("\n# lines: %d", nlines);
      //printf("\nLine height (nm): %.10e", hnm);
      //printf("\nLine bottom width (nm): %.10e", wnm);
      //printf("\nLine length (nm): %.10e", linelengthnm);
      //printf("\nLeft and right sidewall angles (deg): %.10e %.10e", thetaldeg, thetardeg);
      //printf("\nLeft and right top corner radii (nm): %.10e %.10e", radlnm, radrnm);
      //printf("\nThicknesses of 1st and second layers (nm): %.10e %.10e", layer1thicknessnm, layer2thicknessnm);
      //printf("\nBeam landing energies (eV): ");

      //for (int i = 0; i < beamEeVvalsLen; i++) {
      //   printf("\n%.10e", beamEeVvals[i]);
      //}
      //printf("\nBeam size (standard deviation, in nm): %.10e", beamsizenm);

      //printf("\n");
      //printf("\nbeamE (eV)\t x(nm)\t y (nm)\t BSE yield\t SE yield");

#if (!(defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
      std::string output;
      auto start = std::chrono::system_clock::now();
#endif
      for (auto ynm : yvals) {
         //double ynm = yvals[0];
         const double y = ynm * 1.e-9;
         for (auto xnm : xvals) {
            double x = xnm*1.e-9;
            const double egCenter[] = { x, y, -h - 20.*1.e-9 };
            eg->setCenter(egCenter);

            const int nbins = (int)(beamEeV / binSizeEV);
            BackscatterStatsT* back = new BackscatterStatsT(*monte, nbins); //printf("48\n");
            monte->addActionListener(*back);

            monte->runMultipleTrajectories(nTrajectories);

            const HistogramT& hist = back->backscatterEnergyHistogram(); //printf("49\n");

            const double energyperbineV = beamEeV / hist.binCount();
            const double maxSEbin = 50. / energyperbineV;
            int totalSE = 0;
            for (int j = 0; j < (int)maxSEbin; ++j) {
               totalSE = totalSE + hist.counts(j);
            }

            const double SEf = (float)totalSE / nTrajectories;
            const double bsf = back->backscatterFraction() - SEf;
            printf("%lf %lf %lf %lf %lf\n", beamEeV, xnm, ynm, bsf, SEf);

#if (!(defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
            output += std::to_string(beamEeV) + " " + std::to_string(xnm) + " " + std::to_string(ynm) + " " + std::to_string(bsf) + " " + std::to_string(SEf) + "\n";
#endif

            monte->removeActionListener(*back);
            delete back;
         }
      }
#if (!(defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
      auto end = std::chrono::system_clock::now();
      std::chrono::duration<double> elapsed_seconds = end - start;
      std::time_t end_time = std::chrono::system_clock::to_time_t(end);
      std::cout << std::endl << "finished computation at " << std::ctime(&end_time) << "elapsed time: " << elapsed_seconds.count() << "s" << std::endl;
      output += std::to_string(elapsed_seconds.count());
      
      std::ofstream myfile;
      myfile.open("output.txt");
      myfile << output.c_str();
      myfile.close();
#endif
   }
}