#include "hip/hip_runtime.h"
#include "gov\nist\nanoscalemetrology\JMONSEL\NShapes.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NormalMultiPlaneShape.cuh"
#include "gov\nist\microanalysis\NISTMonte\MultiPlaneShape.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NormalIntersectionShape.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NormalCylindricalShape.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NormalUnionShape.cuh"

namespace NShapes
{
   static void normalize(double vec[], double res[])
   {
      double norm = ::sqrt((vec[0] * vec[0]) + (vec[1] * vec[1]) + (vec[2] * vec[2]));
      res[0] = vec[0] / norm;
      res[1] = vec[1] / norm;
      res[2] = vec[2] / norm;
   }

   // returns a vector pointing in the opposite direction of v
   static void invert(double v[], double res[])
   {
      res[0] = -v[0];
      res[1] = -v[1];
      res[2] = -v[2];
   }

   // Add a plane offset by dist*normal from pt
   //static void addOffsetPlane(NormalMultiPlaneShapeT& shape, double normal[], double pt[], double dist)
   //{
   //   normal = normalize(normal).data();
   //   shape.addPlane(normal, new double[] {
   //      pt[0] + (normal[0] * dist),
   //         pt[1] + (normal[1] * dist),
   //         pt[2] + (normal[2] * dist)
   //   });
   //}

   static void addPlane(NormalMultiPlaneShapeT& shape, PlaneT& plane)
   {
      shape.addPlane(plane);
   }

   //NormalMultiPlaneShapeT createNormalFilm(double normal[], double pt1[], double thickness)
   //{
   //   NormalMultiPlaneShapeT mp;
   //   mp.addPlane(normal, pt1);
   //   addOffsetPlane(mp, invert(normal), pt1, thickness);
   //   return mp;
   //}

   // TODO: write a destructor
   NormalShapeT* createLine(
      double topz, // z of the top face
      double width, // line width
      double length, // length of line
      double thetal, // angle of right sidewall
      double thetar, // angle of left sidewall
      double radl, // radius of top right corner
      double radr // radius of top left corner
      ) {
      // Parameter checks
      if (radr < 0.)
         radr = 0.;
      if (radl < 0.)
         radl = 0.;

      /* First, construct the enclosure */
      NormalMultiPlaneShapeT* enclosure = new NormalMultiPlaneShapeT();
      // Add top plane
      double signz = !topz ? 0 : (topz > 0 ? 1 : -1);
      if (signz == 0.) signz = 1.; // For rare case of 0-height specification
      const double n0[] = { 0., 0., signz }, p0[] = { 0., 0., topz };
      PlaneT* pl0 = new PlaneT(n0, p0);
      enclosure->addPlane(*pl0);
      // Add bottom plane
      double n1[] = { 0., 0., -signz }, p1[] = { 0., 0., 0. };
      PlaneT* pl1 = new PlaneT(n1, p1);
      enclosure->addPlane(*pl1);
      // Add end caps
      const double n2[] = { 0., 1., 0. }, p2[] = { 0., length / 2., 0. }; // Right end 
      PlaneT* pl2 = new PlaneT(n2, p2);
      enclosure->addPlane(*pl2);
      const double n3[] = { 0., -1., 0. }, p3[] = { 0., -length / 2., 0. }; // Left end
      PlaneT* pl3 = new PlaneT(n3, p3);
      enclosure->addPlane(*pl3);

      NormalMultiPlaneShapeT* rightNMPS = new NormalMultiPlaneShapeT();
      NormalShapeT* rightSide = NULL;

      // Add right sidewall
      const double costhetar = ::cos(thetar);
      const double sinthetar = ::sin(thetar);

      const double n4[] = { costhetar, 0., signz * sinthetar }, p4[] = { width / 2, 0., 0. };
      PlaneT* pl4 = new PlaneT(n4, p4);
      rightNMPS->addPlane(*pl4);
      // If radr>0 add a clipping plane and the cylinder
      double root2 = ::sqrt(2.);
      double absz = signz * topz;
      if (radr > 0) {
         const double rad = ::sqrt(1 - sinthetar);
         const double nr[] = { rad / root2, 0., (signz * costhetar) / root2 / rad }, pr[] = { ((width / 2.) - (radr / costhetar)) + (((radr - absz) * sinthetar) / costhetar), 0., topz };
         PlaneT* plr = new PlaneT(nr, pr);
         rightNMPS->addPlane(*plr);
         // Construct cylinder for right corner
         const double xc = ((width / 2.) - (radr / ::cos(thetar))) + ((radr - absz) * ::tan(thetar));
         const double zc = topz - (signz * radr);
         const double end0r[] = { xc, -length / 2., zc }, end1r[] = { xc, length / 2., zc };
         NormalCylindricalShapeT* rcylinder = new NormalCylindricalShapeT(end0r, end1r, radr);
         rightSide = new NormalUnionShapeT(*rightNMPS, *rcylinder);
      }
      else
         rightSide = rightNMPS;

      NormalMultiPlaneShapeT* leftNMPS = new NormalMultiPlaneShapeT();
      NormalShapeT* leftSide = NULL;

      // Add left sidewall
      const double costhetal = ::cos(thetal);
      const double sinthetal = ::sin(thetal);
      const double n6[] = { -costhetal, 0., signz * sinthetal }, p6[] = { -width / 2, 0., 0. };
      PlaneT* pl6 = new PlaneT(n6, p6);
      leftNMPS->addPlane(*pl6);
      // If radl>0 add a clipping plane and the cylinder
      if (radl > 0.) {
         const double rad = ::sqrt(1 - sinthetal);
         const double n8[] = { -rad / root2, 0., (signz * costhetal) / root2 / rad }, p8[] = { ((-width / 2.) + (radl / costhetal)) - (((radl - absz) * sinthetal) / costhetal), 0., topz };
         PlaneT* pl = new PlaneT(n8, p8);
         leftNMPS->addPlane(*pl);
         const double xc = ((width / 2.) - (radl / ::cos(thetal))) + ((radl - absz) * ::tan(thetal));
         const double zc = topz - (signz * radl);
         // Construct cylinder for left corner
         const double end0[] = { -xc, -length / 2., zc }, end1[] = { -xc, length / 2., zc };
         NormalCylindricalShapeT* lcylinder = new NormalCylindricalShapeT(end0, end1, radl);
         leftSide = new NormalUnionShapeT(*leftNMPS, *lcylinder);
      }
      else
         leftSide = leftNMPS;

      NormalIntersectionShapeT* nts = new NormalIntersectionShapeT(*leftSide, *rightSide);
      NormalIntersectionShapeT* nis = new NormalIntersectionShapeT(*nts, *enclosure);
      return nis;
   }
}