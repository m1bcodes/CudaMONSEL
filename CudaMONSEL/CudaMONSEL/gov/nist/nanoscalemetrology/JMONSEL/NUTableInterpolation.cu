#include "hip/hip_runtime.h"
#include "gov\nist\nanoscalemetrology\JMONSEL\NUTableInterpolation.cuh"

#include "gov\nist\nanoscalemetrology\JMONSELutils\NULagrangeInterpolation.cuh"

#include <fstream>

#include "CudaUtil.h"

namespace NUTableInterpolation
{
   /**
   * RegularTableInterpolation - Create an interpolation table from the named
   * resource. The table is assumed to be stored in the resource as numbers (in
   * character format) separated by white space. The numbers are in this order:
   * Number of input variables for this table (N), # of values taken by the 1st
   * input variable, the monotonic list of these values, ... (repeated for 2nd,
   * 3rd, up to Nth input variable), then a list of the tabulated values in
   * order with the Nth input variable varying most rapidly, the N-1st next,
   * and so on, with the 1st varying most slowly.
   *
   * @param tableFileName - A String providing the name of the resource (data
   *           file) that stores the table to be interpolated.
   */
   __host__ __device__ NUTableInterpolation::NUTableInterpolation(char const * tableFileName) :
      table1d(0, 0),
      table2d(0, VectorXd(0, 0)),
      table3d(0, MatrixXf(0, VectorXf(0, 0))),
      table4d(0, Matrix3DXd(0, MatrixXd(0, VectorXd(0, 0)))),
      x(0, VectorXd(0, 0)),
      domain(0, VectorXd(0, 0)),
      range(2, 0),
      tableFileName(tableFileName)
   {
      range[0] = INFINITY;
      range[1] = -INFINITY;

#if (!(defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0)))
      ReadTable(tableFileName);
#endif
   }

   /**
   * interpolate - Interpolates this object's table to determine the value at
   * the supplied input coordinate. If the supplied coordinate lies outside the
   * domain of the table, this method extrapolates. This can very quickly lead
   * to very poor estimates. The calling routine is responsible for checking
   * the input against the domain if extrapolation is to be avoided.
   *
   * @param xval - double[] of length in principle equal to the dimension of
   *           the table. For convenience it is allowed to be greater, in which
   *           case the unnecessary values at the end of the array are ignored.
   * @param order - int The interpolation order, 1 for linear, 3 for cubic,
   *           etc.
   * @return double - The estimated value of the tabulated function at the
   *         supplied coordinate.
   */
   __host__ __device__ double NUTableInterpolation::interpolate(double xval[], int xvallen, int order) const
   {
      if (xvallen < dim)
         printf("Attempt to interpolate %s at x with %d dimensions", tableFileName.c_str(), dim);

      switch (dim) {
      case 1:
         return NULagrangeInterpolation::d1(table1d.data(), table1d.size(), x[0].data(), x[0].size(), order, xval[0])[0];
      case 2:
         return NULagrangeInterpolation::d2(table2d, x, order, xval, xvallen)[0];
      case 3:
         return NULagrangeInterpolation::d3(table3d, x, order, xval, xvallen)[0];
      case 4:
         return NULagrangeInterpolation::d4(table4d, x, order, xval, xvallen)[0];
      default:
         printf("NUTableInterpolation::interpolate: Table dimensions must be 1<=dim<=4");
         return NAN;
      }
   }

   void NUTableInterpolation::ReadTable(char const * tableFileName)
   {
      printf("void NUTableInterpolation::ReadTable: Reading %s\n", tableFileName);
      try {
         std::fstream myfile(tableFileName, std::ios_base::in);
         if (!myfile.good()) throw 0;

         int a;
         while (myfile >> a) {
            dim = (int)a;
            if ((dim < 1) || (dim > 4))
               printf("NUTableInterpolation::ReadTable: Table dimensions must be 1<=dim<=4");
            /*
            * Note: I think I could write a general N-dimension interpolation
            * using techniques similar to Mick Flanagan's PolyCubicSpline
            * algorithm.
            */
            VectorXi nPoints(dim, 0);
            x.resize(dim);

            domain.resize(dim, VectorXd(2, 0));

            for (int i = 0; i < dim; i++) {
               myfile >> a;
               nPoints[i] = a;
               x[i].resize(nPoints[i]);

               for (int j = 0; j < nPoints[i]; j++) {
                  myfile >> x[i][j];
               }

               if (x[i][0] < x[i][nPoints[i] - 1]) {
                  domain[i][0] = x[i][0];
                  domain[i][1] = x[i][nPoints[i] - 1];
               }
               else {
                  domain[i][1] = x[i][0];
                  domain[i][0] = x[i][nPoints[i] - 1];
               }
            }

            switch (dim) {
            case 1:
               table1d.resize(nPoints[0]);
               for (int i = 0; i < nPoints[0]; i++) {
                  double tmp;
                  myfile >> tmp;
                  table1d[i] = tmp;
                  if (table1d[i] < range[0])
                     range[0] = table1d[i];
                  else if (table1d[i] > range[1])
                     range[1] = table1d[i];
               }
               break;
            case 2:
               table2d.resize(nPoints[0], VectorXd(nPoints[1], 0));
               for (int i = 0; i < nPoints[0]; i++)
                  for (int j = 0; j < nPoints[1]; j++) {
                     double tmp;
                     myfile >> tmp;
                     table2d[i][j] = tmp;
                     if (table2d[i][j] < range[0])
                        range[0] = table2d[i][j];
                     else if (table2d[i][j] > range[1])
                        range[1] = table2d[i][j];
                  }
               break;
            case 3:
               //table3d.resize(nPoints[0], MatrixXd(nPoints[1], VectorXd(nPoints[2], 0)));
               table3d.resize(nPoints[0], MatrixXf(nPoints[1], VectorXf(nPoints[2], 0)));
               for (int i = 0; i < nPoints[0]; i++)
                  for (int j = 0; j < nPoints[1]; j++)
                     for (int k = 0; k < nPoints[2]; k++) {
                        double tmp;
                        myfile >> tmp;
                        table3d[i][j][k] = tmp;
                        if (table3d[i][j][k] < range[0])
                           range[0] = table3d[i][j][k];
                        else if (table3d[i][j][k] > range[1])
                           range[1] = table3d[i][j][k];
                     }
               break;
            case 4:
               table4d.resize(nPoints[0], Matrix3DXd(nPoints[1], MatrixXd(nPoints[1], VectorXd(nPoints[2], 0))));
               for (int i = 0; i < nPoints[0]; i++) {
                  for (int j = 0; j < nPoints[1]; j++) {
                     for (int k = 0; k < nPoints[2]; k++) {
                        for (int m = 0; m < nPoints[3]; m++) {
                           double tmp;
                           myfile >> tmp;
                           table4d[i][j][k][m] = tmp;
                           if (table4d[i][j][k][m] < range[0])
                              range[0] = table4d[i][j][k][m];
                           else if (table4d[i][j][k][m] > range[1])
                              range[1] = table4d[i][j][k][m];
                        }
                     }
                  }
               }
               break;
            }
         }

         myfile.close();
      }
      catch (std::exception&) {
         printf("NUTableInterpolation::ReadTable: failed reading file %s\n", tableFileName);
      }
   }

   __host__ __device__ const VectorXd& NUTableInterpolation::gettable1d() const
   {
      return table1d;
   }

   __host__ __device__ const MatrixXd& NUTableInterpolation::gettable2d() const
   {
      return table2d;
   }

   __host__ __device__ const Matrix3DXf& NUTableInterpolation::gettable3d() const
   {
      return table3d;
   }

   __host__ __device__ const Matrix4DXd& NUTableInterpolation::gettable4d() const
   {
      return table4d;
   }

   __host__ __device__ const MatrixXd& NUTableInterpolation::getx() const
   {
      return x;
   }

   __host__ __device__ const MatrixXd& NUTableInterpolation::getdomain() const
   {
      return domain;
   }

   __host__ __device__ const VectorXd& NUTableInterpolation::getrange() const
   {
      return range;
   }

   __host__ __device__ int NUTableInterpolation::getdim() const
   {
      return dim;
   }

   __host__ __device__ StringT NUTableInterpolation::gettableFileName() const
   {
      return tableFileName;
   }

   __device__ void NUTableInterpolation::copytable1d(const double* data, const unsigned int len)
   {
      table1d.assign(data, data + len);
   }

   __device__ void NUTableInterpolation::copytable2d(const unsigned int i0, const double* data, const unsigned int len)
   {
      table2d[i0].assign(data, data + len);
   }

   __device__ void NUTableInterpolation::copytable3d(const unsigned int i0, const unsigned int i1, const float* data, const unsigned int len)
   {
      table3d[i0][i1].assign(data, data + len);
   }

   __device__ void NUTableInterpolation::copytable4d(const unsigned int i0, const unsigned int i1, const unsigned int d2, const double* data, const unsigned int len)
   {
      table4d[i0][i1][d2].assign(data, data + len);
   }

   __device__ void NUTableInterpolation::copyx(const unsigned int i0, const double* data, const unsigned int len)
   {
      x[i0].assign(data, data + len);
   }

   __device__ void NUTableInterpolation::copydomain(const unsigned int i0, const double* data, const unsigned int len)
   {
      domain[i0].assign(data, data + len);
   }

   __device__ void NUTableInterpolation::copyrange(const double* data, const unsigned int len)
   {
      range.assign(data, data + len);
   }

   __device__ void NUTableInterpolation::copydim(int d)
   {
      dim = d;
   }

   __device__ void NUTableInterpolation::copytableFileName(const char* data)
   {
      tableFileName = data;
   }

   __device__ void NUTableInterpolation::resizetable2d(const unsigned int d0)
   {
      table2d.resize(d0);
   }

   __device__ void NUTableInterpolation::resizetable3d_0(const unsigned int d0)
   {
      table3d.resize(d0);
   }

   __device__ void NUTableInterpolation::resizetable3d_1(const unsigned int i0, const unsigned int d1)
   {
      table3d[i0].resize(d1);
   }

   __device__ void NUTableInterpolation::resizetable4d_0(const unsigned int d0)
   {
      table4d.resize(d0);
   }

   __device__ void NUTableInterpolation::resizetable4d_1(const unsigned int i0, const unsigned int d1)
   {
      table4d[i0].resize(d1);
   }

   __device__ void NUTableInterpolation::resizetable4d_2(const unsigned int i0, const unsigned int i1, const unsigned int d2)
   {
      table4d[i0][i1].resize(d2);
   }

   __device__ void NUTableInterpolation::resizex(const unsigned int d0)
   {
      x.resize(d0);
   }

   __device__ void NUTableInterpolation::resizedomain(const unsigned int d0)
   {
      domain.resize(d0);
   }

   __host__ __device__ NUTableInterpolationFactory::NUTableInterpolationFactory()
   {
   }

   /**
   * getInstance - Returns an instance of a RegularTableInterpolation object
   * for the table contained in the named resource.
   *
   * @param tableFileName - A String providing the full path name of the data
   *           file that stores the table to be interpolated.
   */
   __host__ __device__ const NUTableInterpolation* NUTableInterpolationFactory::getInstance(char const * tableFileName)
   {
      printf("NUTableInterpolation* getInstance: %s\n", tableFileName);
      const NUTableInterpolation* uniqueInstance = nullptr;
      StringT key(tableFileName);

      if (!instanceMap.ContainsKey(key)) {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
         printf("NUTableInterpolationFactory::getInstance: need to copy %s on to device\n", tableFileName);
#endif
         uniqueInstance = new NUTableInterpolation(tableFileName);
         instanceMap.Put(key, uniqueInstance);
      }
      uniqueInstance = instanceMap[key];

      if (!uniqueInstance) printf("const NUTableInterpolation* NUTableInterpolationFactory::getInstance: failed\n");
      return uniqueInstance;
   }

   __device__ void NUTableInterpolationFactory::setInstance(StringT k, const NUTableInterpolation* v)
   {
      instanceMap.Put(k, v);
   }

   static NUTableInterpolationFactory Factory;
   __device__ static NUTableInterpolationFactory* d_Factory = nullptr;

   __global__ void initFactory()
   {
      d_Factory = new NUTableInterpolationFactory();
   }

   __host__ __device__ const NUTableInterpolation* getInstance(char const * tableFileName)
   {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
      return d_Factory->getInstance(tableFileName);
#else
      return Factory.getInstance(tableFileName);
#endif
   }

   __device__ NUTableInterpolation* newOne = nullptr;

   __global__ void initNewNUTableOnDevice(char const * fn)
   {
      newOne = new NUTableInterpolation(fn);
   }

   __global__ void copytable1d(const double* data, const unsigned int len)
   {
      newOne->copytable1d(data, len);
   }

   __global__ void copytable2d(const unsigned int i0, const double* data, const unsigned int len)
   {
      newOne->copytable2d(i0, data, len);
   }

   __global__ void copytable3d(const unsigned int i0, const unsigned int i1, const float* data, const unsigned int len)
   {
      newOne->copytable3d(i0, i1, data, len);
   }

   __global__ void copytable4d(const unsigned int i0, const unsigned int i1, const unsigned int d2, const double* data, const unsigned int len)
   {
      newOne->copytable4d(i0, i1, d2, data, len);
   }

   __global__ void copyx(const unsigned int i0, const double* data, const unsigned int len)
   {
      newOne->copyx(i0, data, len);
   }

   __global__ void copydomain(const unsigned int i0, const double* data, const unsigned int len)
   {
      newOne->copydomain(i0, data, len);
   }

   __global__ void copyrange(const double* data, const unsigned int len)
   {
      newOne->copyrange(data, len);
   }

   __global__ void copydim(int d)
   {
      newOne->copydim(d);
   }

   __global__ void copytableFileName(const char* data)
   {
      newOne->copytableFileName(data);
   }

   __global__ void setInstance()
   {
      d_Factory->setInstance(newOne->gettableFileName(), newOne);
   }

   __global__ void resizetable2d(const unsigned int n)
   {
      newOne->resizetable2d(n);
   }

   __global__ void resizetable3d_0(const unsigned int d0)
   {
      newOne->resizetable3d_0(d0);
   }

   __global__ void resizetable3d_1(const unsigned int i0, const unsigned int d1)
   {
      newOne->resizetable3d_1(i0, d1);
   }

   __global__ void resizetable4d_0(const unsigned int d0)
   {
      newOne->resizetable4d_0(d0);
   }

   __global__ void resizetable4d_1(const unsigned int i0, const unsigned int d1)
   {
      newOne->resizetable4d_1(i0, d1);
   }

   __global__ void resizetable4d_2(const unsigned int i0, const unsigned int i1, const unsigned int d2)
   {
      newOne->resizetable4d_2(i0, i1, d2);
   }

   __global__ void resizex(const unsigned int n)
   {
      newOne->resizex(n);
   }

   __global__ void resizedomain(const unsigned int n)
   {
      newOne->resizedomain(n);
   }

   void copyDataToCuda(char const * tableFileName)
   {
      NUTableInterpolation const * ptr = getInstance(tableFileName);
      
      char* d_tableFileName = nullptr;
      checkCudaErrors(hipMalloc((void **)&d_tableFileName, (ptr->gettableFileName().size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemset(d_tableFileName, NULL, (ptr->gettableFileName().size() + 1) * sizeof(char)));
      checkCudaErrors(hipMemcpy(d_tableFileName, ptr->gettableFileName().c_str(), (ptr->gettableFileName().size() + 1) * sizeof(char), hipMemcpyHostToDevice));
      initNewNUTableOnDevice << <1, 1 >> >(d_tableFileName);
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipFree(d_tableFileName));

      double* d_table1d = nullptr;
      checkCudaErrors(hipMalloc((void **)&d_table1d, ptr->gettable1d().size() * sizeof(double)));
      checkCudaErrors(hipMemcpy(d_table1d, ptr->gettable1d().data(), ptr->gettable1d().size() * sizeof(double), hipMemcpyHostToDevice));
      copytable1d << <1, 1 >> >(d_table1d, ptr->gettable1d().size());
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipFree(d_table1d));

      double* d_table2d = nullptr;
      resizetable2d << <1, 1 >> >(ptr->gettable2d().size());
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      for (int i = 0; i < ptr->gettable2d().size(); ++i) {
         checkCudaErrors(hipMalloc((void **)&d_table2d, ptr->gettable2d()[i].size() * sizeof(double)));
         checkCudaErrors(hipMemcpy(d_table2d, ptr->gettable2d()[i].data(), ptr->gettable2d()[i].size() * sizeof(double), hipMemcpyHostToDevice));
         copytable2d << <1, 1 >> >(i, d_table2d, ptr->gettable2d()[i].size());
         checkCudaErrors(hipDeviceSynchronize());
         checkCudaErrors(hipGetLastError());
         checkCudaErrors(hipFree(d_table2d));
      }

      float* d_table3d = nullptr;
      resizetable3d_0 << <1, 1 >> >(ptr->gettable3d().size());
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      for (int i = 0; i < ptr->gettable3d().size(); ++i) {
         resizetable3d_1 << <1, 1 >> >(i, ptr->gettable3d()[i].size());
         checkCudaErrors(hipDeviceSynchronize());
         checkCudaErrors(hipGetLastError());
         for (int j = 0; j < ptr->gettable3d()[i].size(); ++j) {
            checkCudaErrors(hipMalloc((void **)&d_table3d, ptr->gettable3d()[i][j].size() * sizeof(float)));
            checkCudaErrors(hipMemcpy(d_table3d, ptr->gettable3d()[i][j].data(), ptr->gettable3d()[i][j].size() * sizeof(float), hipMemcpyHostToDevice));
            copytable3d << <1, 1 >> >(i, j, d_table3d, ptr->gettable3d()[i][j].size());
            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipFree(d_table3d));
         }
      }

      double* d_table4d = nullptr;
      resizetable4d_0 << <1, 1 >> >(ptr->gettable4d().size());
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      for (int i = 0; i < ptr->gettable4d().size(); ++i) {
         resizetable4d_1 << <1, 1 >> >(i, ptr->gettable4d()[i].size());
         checkCudaErrors(hipDeviceSynchronize());
         checkCudaErrors(hipGetLastError());
         for (int j = 0; j < ptr->gettable4d()[i].size(); ++j) {
            resizetable4d_2 << <1, 1 >> >(i, j, ptr->gettable4d()[i][j].size());
            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipGetLastError());
            for (int k = 0; k < ptr->gettable4d()[i][j].size(); ++k) {
               checkCudaErrors(hipMalloc((void **)&d_table4d, ptr->gettable4d()[i][j][k].size() * sizeof(double)));
               checkCudaErrors(hipMemcpy(d_table4d, ptr->gettable4d()[i][j][k].data(), ptr->gettable4d()[i][j][k].size() * sizeof(double), hipMemcpyHostToDevice));
               copytable4d << <1, 1 >> >(i, j, k, d_table4d, ptr->gettable4d()[i][j][k].size());
               checkCudaErrors(hipDeviceSynchronize());
               checkCudaErrors(hipGetLastError());
               checkCudaErrors(hipFree(d_table4d));
            }
         }
      }

      double* d_x = nullptr;
      resizex << <1, 1 >> >(ptr->getx().size());
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      for (int i = 0; i < ptr->getx().size(); ++i) {
         checkCudaErrors(hipMalloc((void **)&d_x, ptr->getx()[i].size() * sizeof(double)));
         checkCudaErrors(hipMemcpy(d_x, ptr->getx()[i].data(), ptr->getx()[i].size() * sizeof(double), hipMemcpyHostToDevice));
         copyx << <1, 1 >> >(i, d_x, ptr->getx()[i].size());
         checkCudaErrors(hipDeviceSynchronize());
         checkCudaErrors(hipGetLastError());
         checkCudaErrors(hipFree(d_x));
      }

      double* d_domain = nullptr;
      resizedomain << <1, 1 >> >(ptr->getdomain().size());
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      for (int i = 0; i < ptr->getdomain().size(); ++i) {
         checkCudaErrors(hipMalloc((void **)&d_domain, ptr->getdomain()[i].size() * sizeof(double)));
         checkCudaErrors(hipMemcpy(d_domain, ptr->getdomain()[i].data(), ptr->getdomain()[i].size() * sizeof(double), hipMemcpyHostToDevice));
         copydomain << <1, 1 >> >(i, d_domain, ptr->getdomain()[i].size());
         checkCudaErrors(hipDeviceSynchronize());
         checkCudaErrors(hipGetLastError());
         checkCudaErrors(hipFree(d_domain));
      }

      double* d_range = nullptr;
      checkCudaErrors(hipMalloc((void **)&d_range, ptr->getrange().size() * sizeof(double)));
      checkCudaErrors(hipMemcpy(d_range, ptr->getrange().data(), ptr->getrange().size() * sizeof(double), hipMemcpyHostToDevice));
      copyrange << <1, 1 >> >(d_range, ptr->getrange().size());
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipFree(d_range));

      copydim << <1, 1 >> >(ptr->getdim());
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());

      setInstance << <1, 1 >> >();
      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipGetLastError());
   }
}
