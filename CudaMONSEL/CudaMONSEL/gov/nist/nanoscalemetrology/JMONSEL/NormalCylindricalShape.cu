#include "hip/hip_runtime.h"
#include "gov\nist\nanoscalemetrology\JMONSEL\NormalCylindricalShape.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"
#include "gov\nist\microanalysis\Utility\Transform3D.cuh"

namespace NormalCylindricalShape
{
   __host__ __device__ NormalCylindricalShape::NormalCylindricalShape(const double end0[], const double end1[], double radius) :
      CylindricalShapeT(end0, end1, radius),
      radius2(radius * radius)
   {
      memcpy(this->end0, end0, sizeof(double) * 3);
      Math2::minus3d(end1, end0, axis);
      mLen2 = Math2::dot3d(axis, axis);
      mLen = ::sqrt(mLen2);
      Math2::divide3d(axis, mLen, normalizedaxis);
   }

   __host__ __device__ bool NormalCylindricalShape::contains(const double pos[]) const
   {
      return CylindricalShapeT::contains(pos);
   }

   __host__ __device__ bool NormalCylindricalShape::contains(const double pos0[], const double pos1[]) const
   {
      const double p0c[] = { pos0[0] - end0[0], pos0[1] - end0[1], pos0[2] - end0[2] }; // pos0 - end0
      const double p0cdotn = (p0c[0] * normalizedaxis[0]) + (p0c[1] * normalizedaxis[1]) + (p0c[2] * normalizedaxis[2]);
      const double p0cSquared = (p0c[0] * p0c[0]) + (p0c[1] * p0c[1]) + (p0c[2] * p0c[2]);
      const double r2 = p0cSquared - (p0cdotn * p0cdotn); // Distance of pos0
      // from
      // axis
      const bool on_end0 = (p0cdotn == 0) && (r2 <= radius2);
      const bool on_end1 = (p0cdotn == mLen) && (r2 <= radius2);
      const bool on_cylinder = (r2 == radius2) && (p0cdotn >= 0) && (p0cdotn <= mLen);
      if (on_end0) {
         const double delta[] = {
            pos1[0] - pos0[0],
            pos1[1] - pos0[1],
            pos1[2] - pos0[2]
         };
         // The normal vector on this endcap is -normalizedaxis
         const double deltadotnormalizedaxis = (delta[0] * normalizedaxis[0]) + (delta[1] * normalizedaxis[1]) + (delta[2] * normalizedaxis[2]);
         if (deltadotnormalizedaxis == 0)
            // endcap
            /*
            * If the trajectory lies wholly within the endcap we must resort to
            * an arbitrary assignment.
            */
            if (normalizedaxis[0] == 0) {
               if (normalizedaxis[1] == 0)
                  return normalizedaxis[2] > 0;
               else
                  return normalizedaxis[1] > 0;
            }
            else
               return normalizedaxis[0] > 0;
         return deltadotnormalizedaxis > 0;
      }
      else if (on_end1) {
         const double delta[] = {
            pos1[0] - pos0[0],
            pos1[1] - pos0[1],
            pos1[2] - pos0[2]
         };
         // The normal vector on this endcap is normalizedaxis
         const double deltadotnormalizedaxis = (delta[0] * normalizedaxis[0]) + (delta[1] * normalizedaxis[1]) + (delta[2] * normalizedaxis[2]);
         if (deltadotnormalizedaxis == 0.)
            // endcap
            if (normalizedaxis[0] == 0) {
               if (normalizedaxis[1] == 0)
                  return normalizedaxis[2] > 0;
               else
                  return normalizedaxis[1] > 0;
            }
            else
               return normalizedaxis[0] > 0;
         return deltadotnormalizedaxis < 0;
      }
      else if (on_cylinder) {
         const double delta[] = {
            pos1[0] - pos0[0],
            pos1[1] - pos0[1],
            pos1[2] - pos0[2]
         }; // Vector pointing in direction of
         // trajectory
         const double nvtmp[] = {
            p0c[0] - (p0cdotn * normalizedaxis[0]),
            p0c[1] - (p0cdotn * normalizedaxis[1]),
            p0c[2] - (p0cdotn * normalizedaxis[2])
         }; // An outward
         // pointing vector
         // at pos0
         return ((nvtmp[0] * delta[0]) + (nvtmp[1] * delta[1]) + (nvtmp[2] * delta[2])) < 0;
      }
      // Here if pos0 is not on the boundary. This is the usual case.
      return (r2 < radius2) && (p0cdotn > 0) && (p0cdotn < mLen);
   }

   __host__ __device__ double NormalCylindricalShape::getFirstIntersection(const double pos0[], const double pos1[])
   {
      return getFirstNormal(pos0, pos1)[3];
   }

   __host__ __device__ bool NormalCylindricalShape::isNormalShape() const
   {
      return true;
   }

   __host__ __device__ const double* NormalCylindricalShape::getFirstNormal(const double pos0[], const double pos1[])
   {
      nv[0] = 0.;
      nv[1] = 0.;
      nv[2] = 0.;
      nv[3] = INFINITY;

      // Various differences and dot products that we will need:
      /*
      * double[] p0c = Math2.minus(pos0, end0); //pos0 - end0 double p0cSquared
      * = Math2.dot(p0c, p0c); double[] delta = Math2.minus(pos1, pos0); //
      * pos1 - pos0 double p0cdotn = Math2.dot(p0c, normalizedaxis); double
      * deltadotn = Math2.dot(delta, normalizedaxis); double deltadotnSquared =
      * deltadotn * deltadotn; double p0cdotdelta = Math2.dot(p0c, delta);
      * double deltaSquared = Math2.dot(delta, delta); // delta^2
      */

      // Following version avoids function call overhead & shaves about 34%
      // off
      // time!
      const double p0c[] = {
         pos0[0] - end0[0],
         pos0[1] - end0[1],
         pos0[2] - end0[2]
      }; // pos0 - end0
      const double p0cSquared = (p0c[0] * p0c[0]) + (p0c[1] * p0c[1]) + (p0c[2] * p0c[2]);
      const double delta[] = {
         pos1[0] - pos0[0],
         pos1[1] - pos0[1],
         pos1[2] - pos0[2]
      }; // pos1 - pos0
      const double p0cdotn = (p0c[0] * normalizedaxis[0]) + (p0c[1] * normalizedaxis[1]) + (p0c[2] * normalizedaxis[2]);
      const double deltadotn = (delta[0] * normalizedaxis[0]) + (delta[1] * normalizedaxis[1]) + (delta[2] * normalizedaxis[2]);
      const double deltadotnSquared = deltadotn * deltadotn;
      const double p0cdotdelta = (delta[0] * p0c[0]) + (delta[1] * p0c[1]) + (delta[2] * p0c[2]);
      const double deltaSquared = (delta[0] * delta[0]) + (delta[1] * delta[1]) + (delta[2] * delta[2]); // delta^2

      /*
      * There are 4 possible intersections, two through the end caps and two
      * through the cylinder body. We'll number these 1, 2, 3, and 4 to
      * remember which one is the nearest.
      */
      int intersectionnumber = 0;

      double u = 0.;
      double u1 = 0.;
      double u2 = 0.;
      double projection = 0.;// Stores projection of intersection onto
      // cylinder
      // axis
      double savedprojection = 0.;

      // Check end caps
      /*
      * Ignore deltadotaxis=0 as this means trajectory is parallel to the end
      * caps. Such a trajectory either misses the end caps, or if it hits them
      * represents a grazing collision that we can ignore.
      */
      if (deltadotn != 0.) {
         // 1st end cap (end0)
         u1 = -p0cdotn / deltadotn; // Distance to plane containing endcap
         /*
         * rSquared = p0cSquared+2*u1*p0cdotdelta+u1*u1*deltaSquared; is
         * in-plane distance to end0, squared. Condition for a valid endcap
         * intersection is we strike the plane of the endcap within the step
         * and the position of the intersection is within r of the center.
         */
         if ((u1 > 0.) && (u1 <= 1.) && ((p0cSquared + (2 * u1 * p0cdotdelta) + (u1 * u1 * deltaSquared)) <= radius2)) {
            nv[3] = u1;
            intersectionnumber = 1;
         }

         // 2nd end cap (end0+axis)
         u2 = u1 + (mLen / deltadotn); // Distance to plane containing other
         // endcap

         if ((u2 > 0.) && (u2 <= 1.) && (u2 < nv[3])
            && (((p0cSquared + (2 * u2 * p0cdotdelta) + (u2 * u2 * deltaSquared) + mLen2)
            - (2 * mLen * (p0cdotn + (u2 * deltadotn)))) <= radius2)) {
            nv[3] = u2;
            intersectionnumber = 2;
         }
      }

      /*
      * Here we check for intersections in the cylindrical wall. This involves
      * solution of a quadratic equation: soln = (-b/2 +/- sqrt((b/2)^2-a*c))/a
      * In the following lines we construct this solution
      */
      if (!(((u1 < 0) && (u2 < 0)) || ((u1 > 1) && (u2 > 1)))) {
         /*
         * The above "if" statement may or may not improve speed. The logic of
         * the test is this: If u1<0 and u2<0 then pos0 and pos1 are "above"
         * both end caps. If u1>1 && u2>1 they are "below" both end caps. In
         * either case there can be no meaningful intersection with the
         * cylindrical wall either, since any such intersection must be outside
         * the endcaps. Thus, this test may allow us to avoid the following
         * time-consuming calculation, but at the cost of the additional time
         * to conduct the test even in those cases where we must go ahead
         * anyway. Thus, whether the test is a net benefit depends upon the
         * relative number of times we find ourselves in each of these
         * situations. If the cylinder is "encapsulated" in a layer with planes
         * defined by the endcaps (or perhaps even better, inside of a
         * rectangular block), then trajectories outside of this region check
         * for plane intersections (fast) instead of cylindrical ones (slow).
         * They never check the cylinder intersections because the cylinder is
         * not a subregion of the region they occupy. Those inside the
         * subregion always check, as they should, and so spare themselves the
         * cost of this test. With proper sample description the case in which
         * the test is beneficial never arises. Still, the cost of the test
         * appears to be only ~5% in timing trials, so it is probably worth it
         * in case the user doesn't encapsulate his cylinders.
         */

         /*
         * If at least one of the points is outside the cylinder we have to
         * check for intersections. Otherwise we skip it. Probably this test is
         * a net benefit, since for well-designed sample descriptions our
         * trajectory should mainly be called when we are inside the cylinder
         * or at least near it.
         */

         const double r0Squared = p0cSquared - (p0cdotn * p0cdotn); // radius of
         // pos0
         const double r1Squared = (r0Squared + (2. * (p0cdotdelta - (p0cdotn * deltadotn))) + deltaSquared) - deltadotnSquared; // radius
         // of
         // pos1

         if ((r0Squared > radius2) || (r1Squared > radius2)) {
            const double a = deltaSquared - deltadotnSquared;
            const double minusbover2 = (p0cdotn * deltadotn) - p0cdotdelta;
            double term = (minusbover2 * minusbover2) - (a * (r0Squared - radius2));
            if (term >= 0) { // The quadratic has real solutions
               term = ::sqrt(term); // term is now the square root

               u = (minusbover2 + term) / a; // 1st quadratic solution
               if ((u > 0) && (u <= 1) && (u < nv[3])) { // Solution falls
                  // within
                  // step.
                  /*
                  * We check whether the projection of the intersection point
                  * onto the cylinder axis falls between the end caps.
                  */
                  projection = p0cdotn + (u * deltadotn);
                  if ((projection >= 0) && (projection <= mLen)) {
                     nv[3] = u;
                     intersectionnumber = 3;
                     savedprojection = projection; // We'll need this
                     // for normal
                     // vector
                  }
               }

               u = (minusbover2 - term) / a; // 2nd quadratic solution
               if ((u > 0) && (u <= 1) && (u < nv[3])) { // Solution falls
                  // within
                  // step.
                  /*
                  * We check whether the projection of the intersection point
                  * onto the cylinder axis falls between the end caps.
                  */
                  projection = p0cdotn + (u * deltadotn);
                  if ((projection >= 0) && (projection <= mLen)) {
                     nv[3] = u;
                     intersectionnumber = 4;
                     savedprojection = projection; // We'll need this
                     // for normal
                     // vector
                  }
               }
            }
         } // End of "is this a net savings?" if statement
      }

      // Decide which intersection we chose
      switch (intersectionnumber) {
      case 1:
         nv[0] = -normalizedaxis[0];
         nv[1] = -normalizedaxis[1];
         nv[2] = -normalizedaxis[2];
         return nv;
      case 2:
         nv[0] = normalizedaxis[0];
         nv[1] = normalizedaxis[1];
         nv[2] = normalizedaxis[2];
         return nv;
      case 3:
      case 4:
         /*
         * Method 1, by subracting axial component to leave perpendicular
         * one
         */
         const double normalv[] = {
            (p0c[0] + (nv[3] * delta[0])) - (savedprojection * normalizedaxis[0]),
            (p0c[1] + (nv[3] * delta[1])) - (savedprojection * normalizedaxis[1]),
            (p0c[2] + (nv[3] * delta[2])) - (savedprojection * normalizedaxis[2])
         };
         const double normalvmag = ::sqrt((normalv[0] * normalv[0]) + (normalv[1] * normalv[1]) + (normalv[2] * normalv[2]));
         nv[0] = normalv[0] / normalvmag;
         nv[1] = normalv[1] / normalvmag;
         nv[2] = normalv[2] / normalvmag;
         return nv;

         /*
         * Method 2 using cross products Temporarily store p0c+u delta double[]
         * normalv = {p0c[0]+u*delta[0], p0c[1]+u*delta[1],p0c[2]+u*delta[2]};
         * normalv = Math2.normalize(Math2.cross(
         * Math2.cross(normalizedaxis,normalv),normalizedaxis)); result[0] =
         * normalv[0]; result[1] = normalv[1]; result[2] = normalv[2]; return
         * result;
         */
      }
      return nv; // None of the above. There was no intersection.
   }

   void NormalCylindricalShape::rotate(const double pivot[], double phi, double theta, double psi)
   {
      Transform3D::rotate3d(end0, pivot, phi, theta, psi, end0);
      Transform3D::rotate3d(axis, phi, theta, psi, axis);
      Math2::divide3d(axis, ::sqrt(mLen2), normalizedaxis);
      CylindricalShapeT::rotate(pivot, phi, theta, psi);
   }

   void NormalCylindricalShape::translate(const double distance[])
   {
      end0[0] += distance[0];
      end0[1] += distance[1];
      end0[2] += distance[2];
      CylindricalShapeT::translate(distance);
   }

   __host__ __device__ const double* NormalCylindricalShape::getPreviousNormal() const
   {
      return nv;
   }

   __host__ __device__ StringT NormalCylindricalShape::toString() const
   {
      return "NormalCylindricalShape";
   }
}