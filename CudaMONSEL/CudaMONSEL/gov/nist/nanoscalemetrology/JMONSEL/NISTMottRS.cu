#include "hip/hip_runtime.h"
#include "gov\nist\nanoscalemetrology\JMONSEL\NISTMottRS.cuh"

#include "gov\nist\microanalysis\EPQLibrary\Reference.cuh"
#include "gov\nist\microanalysis\EPQLibrary\ToSI.cuh"
#include "gov\nist\microanalysis\EPQLibrary\Element.cuh"
#include "gov\nist\microanalysis\EPQLibrary\PhysicalConstants.cuh"
#include "gov\nist\microanalysis\EPQLibrary\ScreenedRutherfordScatteringAngle.cuh"
#include "gov\nist\microanalysis\EPQLibrary\BrowningEmpiricalCrossSection.cuh"
#include "gov\nist\microanalysis\EPQLibrary\NISTMottScatteringAngle.cuh"
#include "gov\nist\microanalysis\Utility\Math2.cuh"

#include "gov\nist\nanoscalemetrology\JMONSELutils\ULagrangeInterpolation.cuh"

namespace NISTMottRS
{
   static const double MAX_NISTMOTT = ToSI::keV(20.0);
   static const double MIN_NISTMOTT = ToSI::keV(0.050);

   static const int qINTERPOLATIONORDER = 3;
   static const int sigmaINTERPOLATIONORDER = 3;
   static const double scale = PhysicalConstants::BohrRadius * PhysicalConstants::BohrRadius;

   static const int SPWEM_LEN = 61;
   static const int X1_LEN = 201;
   static const double DL50 = ::log(MIN_NISTMOTT);
   static const double PARAM = (::log(MAX_NISTMOTT) - DL50) / 60.0;

   static const Reference::Author* al[] = {
      &Reference::FSalvat,
      &Reference::AJablonski,
      &Reference::CPowell
   };
   static const Reference::WebSite mReferenceWebsite("http://www.nist.gov/srd/nist64.htm", "NIST Electron Elastic-Scattering Cross-Section Database version 3.1", "AUGUST 24, 2007", al, 3);

   //static double sciToDub(const std::string& str)
   //{
   //   std::string tmp = str.substr(str.find_first_not_of(" "));
   //   std::stringstream ss(tmp);
   //   double d = 0;
   //   ss >> d;

   //   if (ss.fail()) {
   //      std::string s = "Unable to format ";
   //      s += tmp;
   //      s += " as a number!";
   //      throw s;
   //   }

   //   return d;
   //}

   //void NISTMottRS::loadData(int an)
   //{
   //   const std::string name(an < 10 ? ".\\gov\\nist\\microanalysis\\EPQLibrary\\NistXSec/E0" + std::to_string(an) + ".D64" : ".\\gov\\nist\\microanalysis\\EPQLibrary\\NistXSec/E" + std::to_string(an) + ".D64");
   //   printf("Reading: %s\n", name.c_str());
   //   try {
   //      std::ifstream t(name);
   //      if (!t.good()) throw 0;
   //      std::string line;
   //      std::getline(t, line);
   //      for (int j = 0; j < SPWEM_LEN; ++j) {
   //         std::getline(t, line);
   //         mSpwem[j] = sciToDub(line);
   //         for (int i = 0; i < X1_LEN; ++i){
   //            std::getline(t, line);
   //            mX1[j][i] = sciToDub(line);
   //         }
   //      }
   //      t.close();
   //   }
   //   catch (std::exception& ex) {
   //      printf("Unable to construct NISTMottRS: %s\n", name.c_str());
   //   }
   //}

   NISTMottRS::NISTMottRS(const ElementT& elm, int method) :
      RandomizedScatterT("NIST Elastic cross-section", mReferenceWebsite),
      mElement(elm),
      method(method),
      mRutherford(ScreenedRutherfordScatteringAngle::getSRSA(elm.getAtomicNumber())),
      mBrowning(BrowningEmpiricalCrossSection::getBECS(elm.getAtomicNumber())),
      extrapolateBelowEnergy(method == 1 ? ToSI::eV(50.) : ToSI::eV(100.)),
      mSpwem(NISTMottScatteringAngle::getNISTMSA(elm.getAtomicNumber()).getSpwem()),
      mX1(NISTMottScatteringAngle::getNISTMSA(elm.getAtomicNumber()).getX1()),
      MottXSatMinEnergy(totalCrossSection(extrapolateBelowEnergy)),
      sfBrowning(MottXSatMinEnergy / mBrowning.totalCrossSection(extrapolateBelowEnergy))
   {
      //loadData(elm.getAtomicNumber());
      if (!(method >= 1 && method <= 3))
         printf("NISTMottRS::setMethod: Invalid NISTMottRS method: method must = 1, 2, or 3.");
      sprintf(name, "CrossSection[NIST-Mott, %s]", mElement.toAbbrev());
   }

   const char* NISTMottRS::toString()
   {
      return name;
   }

   const ElementT& NISTMottRS::getElement() const
   {
      return mElement;
   }

   double NISTMottRS::totalCrossSection(double energy) const
   {
      if (energy < extrapolateBelowEnergy) {
         if (method == 3) { // linear interpolation
            return MottXSatMinEnergy * energy / extrapolateBelowEnergy;
         }
         else { // Browning interpolation
            return sfBrowning * mBrowning.totalCrossSection(energy);
         }
      }
      else if (energy < MAX_NISTMOTT) {
         return scale * ULagrangeInterpolation::d1(mSpwem, DL50, PARAM, sigmaINTERPOLATIONORDER, ::log(energy))[0];
      }
      else {
         return mRutherford.totalCrossSection(energy);
      }
   }

   double NISTMottRS::randomScatteringAngle(double energy) const
   {
      /*
      * Even in method 3 (linear interpolation) we use Browning for the angular
      * distribution.
      */
      if (energy < extrapolateBelowEnergy) {
         //if (mBrowning == null) // never happens
         //   mBrowning = new BrowningEmpiricalCrossSection(mElement);
         // sfBrowning = this.totalCrossSection(MIN_NISTMOTT) / mBrowning.totalCrossSection(MIN_NISTMOTT);
         return mBrowning.randomScatteringAngle(energy);

      }
      else if (energy < MAX_NISTMOTT) {
         const double x0[] =  {
            DL50,
               0.
         };
         const double xinc[] = {
            PARAM,
            0.005
         };
         const double x[] = {
            ::log(energy),
            Math2::random()
         };
         const double q = ULagrangeInterpolation::d2(mX1, x0, 2, xinc, 2, qINTERPOLATIONORDER, x, 2)[0];
         const double com = 1.0 - (2.0 * q * q);
         return com > -1.0 ? (com < 1.0 ? ::acos(com) : 0.0) : Math2::PI;
      }
      else {
         return mRutherford.randomScatteringAngle(energy);
      }
   }

   int NISTMottRS::getMethod() const
   {
      return method;
   }

   //void NISTMottRS::setMethod(int method)
   //{
   //   if (method >= 1 && method <= 3)
   //      this->method = method;
   //   else
   //      printf("NISTMottRS::setMethod: Invalid NISTMottRS method: method must = 1, 2, or 3.");
   //   if (method == 2 || method == 3)
   //      extrapolateBelowEnergy = ToSI::eV(100.);
   //   MottXSatMinEnergy = totalCrossSection(extrapolateBelowEnergy);
   //}

   //const NISTMottRS NMRS1_1(Element::H, 1);
   //const NISTMottRS NMRS2_1(Element::He, 1);
   //const NISTMottRS NMRS3_1(Element::Li, 1);
   //const NISTMottRS NMRS4_1(Element::Be, 1);
   //const NISTMottRS NMRS5_1(Element::B, 1);
   //const NISTMottRS NMRS6_1(Element::C, 1);
   //const NISTMottRS NMRS7_1(Element::N, 1);
   //const NISTMottRS NMRS8_1(Element::O, 1);
   //const NISTMottRS NMRS9_1(Element::F, 1);
   //const NISTMottRS NMRS10_1(Element::Ne, 1);
   //const NISTMottRS NMRS11_1(Element::Na, 1);
   //const NISTMottRS NMRS12_1(Element::Mg, 1);
   //const NISTMottRS NMRS13_1(Element::Al, 1);
   //const NISTMottRS NMRS14_1(Element::Si, 1);
   //const NISTMottRS NMRS15_1(Element::P, 1);
   //const NISTMottRS NMRS16_1(Element::S, 1);
   //const NISTMottRS NMRS17_1(Element::Cl, 1);
   //const NISTMottRS NMRS18_1(Element::Ar, 1);
   //const NISTMottRS NMRS19_1(Element::K, 1);
   //const NISTMottRS NMRS20_1(Element::Ca, 1);
   //const NISTMottRS NMRS21_1(Element::Sc, 1);
   //const NISTMottRS NMRS22_1(Element::Ti, 1);
   //const NISTMottRS NMRS23_1(Element::V, 1);
   //const NISTMottRS NMRS24_1(Element::Cr, 1);
   //const NISTMottRS NMRS25_1(Element::Mn, 1);
   //const NISTMottRS NMRS26_1(Element::Fe, 1);
   //const NISTMottRS NMRS27_1(Element::Co, 1);
   //const NISTMottRS NMRS28_1(Element::Ni, 1);
   //const NISTMottRS NMRS29_1(Element::Cu, 1);
   //const NISTMottRS NMRS30_1(Element::Zn, 1);
   //const NISTMottRS NMRS31_1(Element::Ga, 1);
   //const NISTMottRS NMRS32_1(Element::Ge, 1);
   //const NISTMottRS NMRS33_1(Element::As, 1);
   //const NISTMottRS NMRS34_1(Element::Se, 1);
   //const NISTMottRS NMRS35_1(Element::Br, 1);
   //const NISTMottRS NMRS36_1(Element::Kr, 1);
   //const NISTMottRS NMRS37_1(Element::Rb, 1);
   //const NISTMottRS NMRS38_1(Element::Sr, 1);
   //const NISTMottRS NMRS39_1(Element::Y, 1);
   //const NISTMottRS NMRS40_1(Element::Zr, 1);
   //const NISTMottRS NMRS41_1(Element::Nb, 1);
   //const NISTMottRS NMRS42_1(Element::Mo, 1);
   //const NISTMottRS NMRS43_1(Element::Tc, 1);
   //const NISTMottRS NMRS44_1(Element::Ru, 1);
   //const NISTMottRS NMRS45_1(Element::Rh, 1);
   //const NISTMottRS NMRS46_1(Element::Pd, 1);
   //const NISTMottRS NMRS47_1(Element::Ag, 1);
   //const NISTMottRS NMRS48_1(Element::Cd, 1);
   //const NISTMottRS NMRS49_1(Element::In, 1);
   //const NISTMottRS NMRS50_1(Element::Sn, 1);
   //const NISTMottRS NMRS51_1(Element::Sb, 1);
   //const NISTMottRS NMRS52_1(Element::Te, 1);
   //const NISTMottRS NMRS53_1(Element::I, 1);
   //const NISTMottRS NMRS54_1(Element::Xe, 1);
   //const NISTMottRS NMRS55_1(Element::Cs, 1);
   //const NISTMottRS NMRS56_1(Element::Ba, 1);
   //const NISTMottRS NMRS57_1(Element::La, 1);
   //const NISTMottRS NMRS58_1(Element::Ce, 1);
   //const NISTMottRS NMRS59_1(Element::Pr, 1);
   //const NISTMottRS NMRS60_1(Element::Nd, 1);
   //const NISTMottRS NMRS61_1(Element::Pm, 1);
   //const NISTMottRS NMRS62_1(Element::Sm, 1);
   //const NISTMottRS NMRS63_1(Element::Eu, 1);
   //const NISTMottRS NMRS64_1(Element::Gd, 1);
   //const NISTMottRS NMRS65_1(Element::Tb, 1);
   //const NISTMottRS NMRS66_1(Element::Dy, 1);
   //const NISTMottRS NMRS67_1(Element::Ho, 1);
   //const NISTMottRS NMRS68_1(Element::Er, 1);
   //const NISTMottRS NMRS69_1(Element::Tm, 1);
   //const NISTMottRS NMRS70_1(Element::Yb, 1);
   //const NISTMottRS NMRS71_1(Element::Lu, 1);
   //const NISTMottRS NMRS72_1(Element::Hf, 1);
   //const NISTMottRS NMRS73_1(Element::Ta, 1);
   //const NISTMottRS NMRS74_1(Element::W, 1);
   //const NISTMottRS NMRS75_1(Element::Re, 1);
   //const NISTMottRS NMRS76_1(Element::Os, 1);
   //const NISTMottRS NMRS77_1(Element::Ir, 1);
   //const NISTMottRS NMRS78_1(Element::Pt, 1);
   //const NISTMottRS NMRS79_1(Element::Au, 1);
   //const NISTMottRS NMRS80_1(Element::Hg, 1);
   //const NISTMottRS NMRS81_1(Element::Tl, 1);
   //const NISTMottRS NMRS82_1(Element::Pb, 1);
   //const NISTMottRS NMRS83_1(Element::Bi, 1);
   //const NISTMottRS NMRS84_1(Element::Po, 1);
   //const NISTMottRS NMRS85_1(Element::At, 1);
   //const NISTMottRS NMRS86_1(Element::Rn, 1);
   //const NISTMottRS NMRS87_1(Element::Fr, 1);
   //const NISTMottRS NMRS88_1(Element::Ra, 1);
   //const NISTMottRS NMRS89_1(Element::Ac, 1);
   //const NISTMottRS NMRS90_1(Element::Th, 1);
   //const NISTMottRS NMRS91_1(Element::Pa, 1);
   //const NISTMottRS NMRS92_1(Element::U, 1);
   //const NISTMottRS NMRS93_1(Element::Np, 1);
   //const NISTMottRS NMRS94_1(Element::Pu, 1);
   //const NISTMottRS NMRS95_1(Element::Am, 1);
   //const NISTMottRS NMRS96_1(Element::Cm, 1);

   const NISTMottRS* mScatter1[113];

   //const NISTMottRS NMRS1_2(Element::H, 2);
   //const NISTMottRS NMRS2_2(Element::He, 2);
   //const NISTMottRS NMRS3_2(Element::Li, 2);
   //const NISTMottRS NMRS4_2(Element::Be, 2);
   //const NISTMottRS NMRS5_2(Element::B, 2);
   //const NISTMottRS NMRS6_2(Element::C, 2);
   //const NISTMottRS NMRS7_2(Element::N, 2);
   //const NISTMottRS NMRS8_2(Element::O, 2);
   //const NISTMottRS NMRS9_2(Element::F, 2);
   //const NISTMottRS NMRS10_2(Element::Ne, 2);
   //const NISTMottRS NMRS11_2(Element::Na, 2);
   //const NISTMottRS NMRS12_2(Element::Mg, 2);
   //const NISTMottRS NMRS13_2(Element::Al, 2);
   //const NISTMottRS NMRS14_2(Element::Si, 2);
   //const NISTMottRS NMRS15_2(Element::P, 2);
   //const NISTMottRS NMRS16_2(Element::S, 2);
   //const NISTMottRS NMRS17_2(Element::Cl, 2);
   //const NISTMottRS NMRS18_2(Element::Ar, 2);
   //const NISTMottRS NMRS19_2(Element::K, 2);
   //const NISTMottRS NMRS20_2(Element::Ca, 2);
   //const NISTMottRS NMRS21_2(Element::Sc, 2);
   //const NISTMottRS NMRS22_2(Element::Ti, 2);
   //const NISTMottRS NMRS23_2(Element::V, 2);
   //const NISTMottRS NMRS24_2(Element::Cr, 2);
   //const NISTMottRS NMRS25_2(Element::Mn, 2);
   //const NISTMottRS NMRS26_2(Element::Fe, 2);
   //const NISTMottRS NMRS27_2(Element::Co, 2);
   //const NISTMottRS NMRS28_2(Element::Ni, 2);
   //const NISTMottRS NMRS29_2(Element::Cu, 2);
   //const NISTMottRS NMRS30_2(Element::Zn, 2);
   //const NISTMottRS NMRS31_2(Element::Ga, 2);
   //const NISTMottRS NMRS32_2(Element::Ge, 2);
   //const NISTMottRS NMRS33_2(Element::As, 2);
   //const NISTMottRS NMRS34_2(Element::Se, 2);
   //const NISTMottRS NMRS35_2(Element::Br, 2);
   //const NISTMottRS NMRS36_2(Element::Kr, 2);
   //const NISTMottRS NMRS37_2(Element::Rb, 2);
   //const NISTMottRS NMRS38_2(Element::Sr, 2);
   //const NISTMottRS NMRS39_2(Element::Y, 2);
   //const NISTMottRS NMRS40_2(Element::Zr, 2);
   //const NISTMottRS NMRS41_2(Element::Nb, 2);
   //const NISTMottRS NMRS42_2(Element::Mo, 2);
   //const NISTMottRS NMRS43_2(Element::Tc, 2);
   //const NISTMottRS NMRS44_2(Element::Ru, 2);
   //const NISTMottRS NMRS45_2(Element::Rh, 2);
   //const NISTMottRS NMRS46_2(Element::Pd, 2);
   //const NISTMottRS NMRS47_2(Element::Ag, 2);
   //const NISTMottRS NMRS48_2(Element::Cd, 2);
   //const NISTMottRS NMRS49_2(Element::In, 2);
   //const NISTMottRS NMRS50_2(Element::Sn, 2);
   //const NISTMottRS NMRS51_2(Element::Sb, 2);
   //const NISTMottRS NMRS52_2(Element::Te, 2);
   //const NISTMottRS NMRS53_2(Element::I, 2);
   //const NISTMottRS NMRS54_2(Element::Xe, 2);
   //const NISTMottRS NMRS55_2(Element::Cs, 2);
   //const NISTMottRS NMRS56_2(Element::Ba, 2);
   //const NISTMottRS NMRS57_2(Element::La, 2);
   //const NISTMottRS NMRS58_2(Element::Ce, 2);
   //const NISTMottRS NMRS59_2(Element::Pr, 2);
   //const NISTMottRS NMRS60_2(Element::Nd, 2);
   //const NISTMottRS NMRS61_2(Element::Pm, 2);
   //const NISTMottRS NMRS62_2(Element::Sm, 2);
   //const NISTMottRS NMRS63_2(Element::Eu, 2);
   //const NISTMottRS NMRS64_2(Element::Gd, 2);
   //const NISTMottRS NMRS65_2(Element::Tb, 2);
   //const NISTMottRS NMRS66_2(Element::Dy, 2);
   //const NISTMottRS NMRS67_2(Element::Ho, 2);
   //const NISTMottRS NMRS68_2(Element::Er, 2);
   //const NISTMottRS NMRS69_2(Element::Tm, 2);
   //const NISTMottRS NMRS70_2(Element::Yb, 2);
   //const NISTMottRS NMRS71_2(Element::Lu, 2);
   //const NISTMottRS NMRS72_2(Element::Hf, 2);
   //const NISTMottRS NMRS73_2(Element::Ta, 2);
   //const NISTMottRS NMRS74_2(Element::W, 2);
   //const NISTMottRS NMRS75_2(Element::Re, 2);
   //const NISTMottRS NMRS76_2(Element::Os, 2);
   //const NISTMottRS NMRS77_2(Element::Ir, 2);
   //const NISTMottRS NMRS78_2(Element::Pt, 2);
   //const NISTMottRS NMRS79_2(Element::Au, 2);
   //const NISTMottRS NMRS80_2(Element::Hg, 2);
   //const NISTMottRS NMRS81_2(Element::Tl, 2);
   //const NISTMottRS NMRS82_2(Element::Pb, 2);
   //const NISTMottRS NMRS83_2(Element::Bi, 2);
   //const NISTMottRS NMRS84_2(Element::Po, 2);
   //const NISTMottRS NMRS85_2(Element::At, 2);
   //const NISTMottRS NMRS86_2(Element::Rn, 2);
   //const NISTMottRS NMRS87_2(Element::Fr, 2);
   //const NISTMottRS NMRS88_2(Element::Ra, 2);
   //const NISTMottRS NMRS89_2(Element::Ac, 2);
   //const NISTMottRS NMRS90_2(Element::Th, 2);
   //const NISTMottRS NMRS91_2(Element::Pa, 2);
   //const NISTMottRS NMRS92_2(Element::U, 2);
   //const NISTMottRS NMRS93_2(Element::Np, 2);
   //const NISTMottRS NMRS94_2(Element::Pu, 2);
   //const NISTMottRS NMRS95_2(Element::Am, 2);
   //const NISTMottRS NMRS96_2(Element::Cm, 2);

   const NISTMottRS* mScatter2[113];

   //const NISTMottRS NMRS1_3(Element::H, 3);
   //const NISTMottRS NMRS2_3(Element::He, 3);
   //const NISTMottRS NMRS3_3(Element::Li, 3);
   //const NISTMottRS NMRS4_3(Element::Be, 3);
   //const NISTMottRS NMRS5_3(Element::B, 3);
   //const NISTMottRS NMRS6_3(Element::C, 3);
   //const NISTMottRS NMRS7_3(Element::N, 3);
   //const NISTMottRS NMRS8_3(Element::O, 3);
   //const NISTMottRS NMRS9_3(Element::F, 3);
   //const NISTMottRS NMRS10_3(Element::Ne, 3);
   //const NISTMottRS NMRS11_3(Element::Na, 3);
   //const NISTMottRS NMRS12_3(Element::Mg, 3);
   //const NISTMottRS NMRS13_3(Element::Al, 3);
   //const NISTMottRS NMRS14_3(Element::Si, 3);
   //const NISTMottRS NMRS15_3(Element::P, 3);
   //const NISTMottRS NMRS16_3(Element::S, 3);
   //const NISTMottRS NMRS17_3(Element::Cl, 3);
   //const NISTMottRS NMRS18_3(Element::Ar, 3);
   //const NISTMottRS NMRS19_3(Element::K, 3);
   //const NISTMottRS NMRS20_3(Element::Ca, 3);
   //const NISTMottRS NMRS21_3(Element::Sc, 3);
   //const NISTMottRS NMRS22_3(Element::Ti, 3);
   //const NISTMottRS NMRS23_3(Element::V, 3);
   //const NISTMottRS NMRS24_3(Element::Cr, 3);
   //const NISTMottRS NMRS25_3(Element::Mn, 3);
   //const NISTMottRS NMRS26_3(Element::Fe, 3);
   //const NISTMottRS NMRS27_3(Element::Co, 3);
   //const NISTMottRS NMRS28_3(Element::Ni, 3);
   //const NISTMottRS NMRS29_3(Element::Cu, 3);
   //const NISTMottRS NMRS30_3(Element::Zn, 3);
   //const NISTMottRS NMRS31_3(Element::Ga, 3);
   //const NISTMottRS NMRS32_3(Element::Ge, 3);
   //const NISTMottRS NMRS33_3(Element::As, 3);
   //const NISTMottRS NMRS34_3(Element::Se, 3);
   //const NISTMottRS NMRS35_3(Element::Br, 3);
   //const NISTMottRS NMRS36_3(Element::Kr, 3);
   //const NISTMottRS NMRS37_3(Element::Rb, 3);
   //const NISTMottRS NMRS38_3(Element::Sr, 3);
   //const NISTMottRS NMRS39_3(Element::Y, 3);
   //const NISTMottRS NMRS40_3(Element::Zr, 3);
   //const NISTMottRS NMRS41_3(Element::Nb, 3);
   //const NISTMottRS NMRS42_3(Element::Mo, 3);
   //const NISTMottRS NMRS43_3(Element::Tc, 3);
   //const NISTMottRS NMRS44_3(Element::Ru, 3);
   //const NISTMottRS NMRS45_3(Element::Rh, 3);
   //const NISTMottRS NMRS46_3(Element::Pd, 3);
   //const NISTMottRS NMRS47_3(Element::Ag, 3);
   //const NISTMottRS NMRS48_3(Element::Cd, 3);
   //const NISTMottRS NMRS49_3(Element::In, 3);
   //const NISTMottRS NMRS50_3(Element::Sn, 3);
   //const NISTMottRS NMRS51_3(Element::Sb, 3);
   //const NISTMottRS NMRS52_3(Element::Te, 3);
   //const NISTMottRS NMRS53_3(Element::I, 3);
   //const NISTMottRS NMRS54_3(Element::Xe, 3);
   //const NISTMottRS NMRS55_3(Element::Cs, 3);
   //const NISTMottRS NMRS56_3(Element::Ba, 3);
   //const NISTMottRS NMRS57_3(Element::La, 3);
   //const NISTMottRS NMRS58_3(Element::Ce, 3);
   //const NISTMottRS NMRS59_3(Element::Pr, 3);
   //const NISTMottRS NMRS60_3(Element::Nd, 3);
   //const NISTMottRS NMRS61_3(Element::Pm, 3);
   //const NISTMottRS NMRS62_3(Element::Sm, 3);
   //const NISTMottRS NMRS63_3(Element::Eu, 3);
   //const NISTMottRS NMRS64_3(Element::Gd, 3);
   //const NISTMottRS NMRS65_3(Element::Tb, 3);
   //const NISTMottRS NMRS66_3(Element::Dy, 3);
   //const NISTMottRS NMRS67_3(Element::Ho, 3);
   //const NISTMottRS NMRS68_3(Element::Er, 3);
   //const NISTMottRS NMRS69_3(Element::Tm, 3);
   //const NISTMottRS NMRS70_3(Element::Yb, 3);
   //const NISTMottRS NMRS71_3(Element::Lu, 3);
   //const NISTMottRS NMRS72_3(Element::Hf, 3);
   //const NISTMottRS NMRS73_3(Element::Ta, 3);
   //const NISTMottRS NMRS74_3(Element::W, 3);
   //const NISTMottRS NMRS75_3(Element::Re, 3);
   //const NISTMottRS NMRS76_3(Element::Os, 3);
   //const NISTMottRS NMRS77_3(Element::Ir, 3);
   //const NISTMottRS NMRS78_3(Element::Pt, 3);
   //const NISTMottRS NMRS79_3(Element::Au, 3);
   //const NISTMottRS NMRS80_3(Element::Hg, 3);
   //const NISTMottRS NMRS81_3(Element::Tl, 3);
   //const NISTMottRS NMRS82_3(Element::Pb, 3);
   //const NISTMottRS NMRS83_3(Element::Bi, 3);
   //const NISTMottRS NMRS84_3(Element::Po, 3);
   //const NISTMottRS NMRS85_3(Element::At, 3);
   //const NISTMottRS NMRS86_3(Element::Rn, 3);
   //const NISTMottRS NMRS87_3(Element::Fr, 3);
   //const NISTMottRS NMRS88_3(Element::Ra, 3);
   //const NISTMottRS NMRS89_3(Element::Ac, 3);
   //const NISTMottRS NMRS90_3(Element::Th, 3);
   //const NISTMottRS NMRS91_3(Element::Pa, 3);
   //const NISTMottRS NMRS92_3(Element::U, 3);
   //const NISTMottRS NMRS93_3(Element::Np, 3);
   //const NISTMottRS NMRS94_3(Element::Pu, 3);
   //const NISTMottRS NMRS95_3(Element::Am, 3);
   //const NISTMottRS NMRS96_3(Element::Cm, 3);

   const NISTMottRS* mScatter3[113];

   const NISTMottRS& getNMRS1(int an)
   {
      return *mScatter1[an];
   }

   const NISTMottRS& getNMRS2(int an)
   {
      return *mScatter2[an];
   }

   const NISTMottRS& getNMRS3(int an)
   {
      return *mScatter3[an];
   }

   NISTMottRSFactory::NISTMottRSFactory(int method) : RandomizedScatterFactoryT("NIST Mott Inelastic Cross-Section", mReferenceWebsite), method(method >= 1 && method <= 3 ? method : 1)
   {
   }

   const RandomizedScatterT& NISTMottRSFactory::get(const ElementT& elm) const
   {
      switch (method) {
      case 1:
         return getNMRS1(elm.getAtomicNumber());
      case 2:
         return getNMRS2(elm.getAtomicNumber());
      case 3:
         return getNMRS3(elm.getAtomicNumber());
      default:
         return getNMRS1(elm.getAtomicNumber());
      }
   }

   void NISTMottRSFactory::initializeDefaultStrategy()
   {
   }

   const NISTMottRSFactory FactoryRef = NISTMottRSFactory(1);
   const NISTMottRSFactory Factory100Ref = NISTMottRSFactory(2);
   const NISTMottRSFactory Factory100LinRef = NISTMottRSFactory(3);

   const RandomizedScatterFactoryT& Factory = FactoryRef;
   const RandomizedScatterFactoryT& Factory100 = Factory100Ref;
   const RandomizedScatterFactoryT& Factory100Lin = Factory100LinRef;

   void init()
   {
      mScatter1[1] = new NISTMottRS(Element::H, 1);
      mScatter1[2] = new NISTMottRS(Element::He, 1);
      mScatter1[3] = new NISTMottRS(Element::Li, 1);
      mScatter1[4] = new NISTMottRS(Element::Be, 1);
      mScatter1[5] = new NISTMottRS(Element::B, 1);
      mScatter1[6] = new NISTMottRS(Element::C, 1);
      mScatter1[7] = new NISTMottRS(Element::N, 1);
      mScatter1[8] = new NISTMottRS(Element::O, 1);
      mScatter1[9] = new NISTMottRS(Element::F, 1);
      mScatter1[10] = new NISTMottRS(Element::Ne, 1);
      mScatter1[11] = new NISTMottRS(Element::Na, 1);
      mScatter1[12] = new NISTMottRS(Element::Mg, 1);
      mScatter1[13] = new NISTMottRS(Element::Al, 1);
      mScatter1[14] = new NISTMottRS(Element::Si, 1);
      mScatter1[15] = new NISTMottRS(Element::P, 1);
      mScatter1[16] = new NISTMottRS(Element::S, 1);
      mScatter1[17] = new NISTMottRS(Element::Cl, 1);
      mScatter1[18] = new NISTMottRS(Element::Ar, 1);
      mScatter1[19] = new NISTMottRS(Element::K, 1);
      mScatter1[20] = new NISTMottRS(Element::Ca, 1);
      mScatter1[21] = new NISTMottRS(Element::Sc, 1);
      mScatter1[22] = new NISTMottRS(Element::Ti, 1);
      mScatter1[23] = new NISTMottRS(Element::V, 1);
      mScatter1[24] = new NISTMottRS(Element::Cr, 1);
      mScatter1[25] = new NISTMottRS(Element::Mn, 1);
      mScatter1[26] = new NISTMottRS(Element::Fe, 1);
      mScatter1[27] = new NISTMottRS(Element::Co, 1);
      mScatter1[28] = new NISTMottRS(Element::Ni, 1);
      mScatter1[29] = new NISTMottRS(Element::Cu, 1);
      mScatter1[30] = new NISTMottRS(Element::Zn, 1);
      mScatter1[31] = new NISTMottRS(Element::Ga, 1);
      mScatter1[32] = new NISTMottRS(Element::Ge, 1);
      mScatter1[33] = new NISTMottRS(Element::As, 1);
      mScatter1[34] = new NISTMottRS(Element::Se, 1);
      mScatter1[35] = new NISTMottRS(Element::Br, 1);
      mScatter1[36] = new NISTMottRS(Element::Kr, 1);
      mScatter1[37] = new NISTMottRS(Element::Rb, 1);
      mScatter1[38] = new NISTMottRS(Element::Sr, 1);
      mScatter1[39] = new NISTMottRS(Element::Y, 1);
      mScatter1[40] = new NISTMottRS(Element::Zr, 1);
      mScatter1[41] = new NISTMottRS(Element::Nb, 1);
      mScatter1[42] = new NISTMottRS(Element::Mo, 1);
      mScatter1[43] = new NISTMottRS(Element::Tc, 1);
      mScatter1[44] = new NISTMottRS(Element::Ru, 1);
      mScatter1[45] = new NISTMottRS(Element::Rh, 1);
      mScatter1[46] = new NISTMottRS(Element::Pd, 1);
      mScatter1[47] = new NISTMottRS(Element::Ag, 1);
      mScatter1[48] = new NISTMottRS(Element::Cd, 1);
      mScatter1[49] = new NISTMottRS(Element::In, 1);
      mScatter1[50] = new NISTMottRS(Element::Sn, 1);
      mScatter1[51] = new NISTMottRS(Element::Sb, 1);
      mScatter1[52] = new NISTMottRS(Element::Te, 1);
      mScatter1[53] = new NISTMottRS(Element::I, 1);
      mScatter1[54] = new NISTMottRS(Element::Xe, 1);
      mScatter1[55] = new NISTMottRS(Element::Cs, 1);
      mScatter1[56] = new NISTMottRS(Element::Ba, 1);
      mScatter1[57] = new NISTMottRS(Element::La, 1);
      mScatter1[58] = new NISTMottRS(Element::Ce, 1);
      mScatter1[59] = new NISTMottRS(Element::Pr, 1);
      mScatter1[60] = new NISTMottRS(Element::Nd, 1);
      mScatter1[61] = new NISTMottRS(Element::Pm, 1);
      mScatter1[62] = new NISTMottRS(Element::Sm, 1);
      mScatter1[63] = new NISTMottRS(Element::Eu, 1);
      mScatter1[64] = new NISTMottRS(Element::Gd, 1);
      mScatter1[65] = new NISTMottRS(Element::Tb, 1);
      mScatter1[66] = new NISTMottRS(Element::Dy, 1);
      mScatter1[67] = new NISTMottRS(Element::Ho, 1);
      mScatter1[68] = new NISTMottRS(Element::Er, 1);
      mScatter1[69] = new NISTMottRS(Element::Tm, 1);
      mScatter1[70] = new NISTMottRS(Element::Yb, 1);
      mScatter1[71] = new NISTMottRS(Element::Lu, 1);
      mScatter1[72] = new NISTMottRS(Element::Hf, 1);
      mScatter1[73] = new NISTMottRS(Element::Ta, 1);
      mScatter1[74] = new NISTMottRS(Element::W, 1);
      mScatter1[75] = new NISTMottRS(Element::Re, 1);
      mScatter1[76] = new NISTMottRS(Element::Os, 1);
      mScatter1[77] = new NISTMottRS(Element::Ir, 1);
      mScatter1[78] = new NISTMottRS(Element::Pt, 1);
      mScatter1[79] = new NISTMottRS(Element::Au, 1);
      mScatter1[80] = new NISTMottRS(Element::Hg, 1);
      mScatter1[81] = new NISTMottRS(Element::Tl, 1);
      mScatter1[82] = new NISTMottRS(Element::Pb, 1);
      mScatter1[83] = new NISTMottRS(Element::Bi, 1);
      mScatter1[84] = new NISTMottRS(Element::Po, 1);
      mScatter1[85] = new NISTMottRS(Element::At, 1);
      mScatter1[86] = new NISTMottRS(Element::Rn, 1);
      mScatter1[87] = new NISTMottRS(Element::Fr, 1);
      mScatter1[88] = new NISTMottRS(Element::Ra, 1);
      mScatter1[89] = new NISTMottRS(Element::Ac, 1);
      mScatter1[90] = new NISTMottRS(Element::Th, 1);
      mScatter1[91] = new NISTMottRS(Element::Pa, 1);
      mScatter1[92] = new NISTMottRS(Element::U, 1);
      mScatter1[93] = new NISTMottRS(Element::Np, 1);
      mScatter1[94] = new NISTMottRS(Element::Pu, 1);
      mScatter1[95] = new NISTMottRS(Element::Am, 1);
      mScatter1[96] = new NISTMottRS(Element::Cm, 1);

      mScatter2[1] = new NISTMottRS(Element::H, 2);
      mScatter2[2] = new NISTMottRS(Element::He, 2);
      mScatter2[3] = new NISTMottRS(Element::Li, 2);
      mScatter2[4] = new NISTMottRS(Element::Be, 2);
      mScatter2[5] = new NISTMottRS(Element::B, 2);
      mScatter2[6] = new NISTMottRS(Element::C, 2);
      mScatter2[7] = new NISTMottRS(Element::N, 2);
      mScatter2[8] = new NISTMottRS(Element::O, 2);
      mScatter2[9] = new NISTMottRS(Element::F, 2);
      mScatter2[10] = new NISTMottRS(Element::Ne, 2);
      mScatter2[11] = new NISTMottRS(Element::Na, 2);
      mScatter2[12] = new NISTMottRS(Element::Mg, 2);
      mScatter2[13] = new NISTMottRS(Element::Al, 2);
      mScatter2[14] = new NISTMottRS(Element::Si, 2);
      mScatter2[15] = new NISTMottRS(Element::P, 2);
      mScatter2[16] = new NISTMottRS(Element::S, 2);
      mScatter2[17] = new NISTMottRS(Element::Cl, 2);
      mScatter2[18] = new NISTMottRS(Element::Ar, 2);
      mScatter2[19] = new NISTMottRS(Element::K, 2);
      mScatter2[20] = new NISTMottRS(Element::Ca, 2);
      mScatter2[21] = new NISTMottRS(Element::Sc, 2);
      mScatter2[22] = new NISTMottRS(Element::Ti, 2);
      mScatter2[23] = new NISTMottRS(Element::V, 2);
      mScatter2[24] = new NISTMottRS(Element::Cr, 2);
      mScatter2[25] = new NISTMottRS(Element::Mn, 2);
      mScatter2[26] = new NISTMottRS(Element::Fe, 2);
      mScatter2[27] = new NISTMottRS(Element::Co, 2);
      mScatter2[28] = new NISTMottRS(Element::Ni, 2);
      mScatter2[29] = new NISTMottRS(Element::Cu, 2);
      mScatter2[30] = new NISTMottRS(Element::Zn, 2);
      mScatter2[31] = new NISTMottRS(Element::Ga, 2);
      mScatter2[32] = new NISTMottRS(Element::Ge, 2);
      mScatter2[33] = new NISTMottRS(Element::As, 2);
      mScatter2[34] = new NISTMottRS(Element::Se, 2);
      mScatter2[35] = new NISTMottRS(Element::Br, 2);
      mScatter2[36] = new NISTMottRS(Element::Kr, 2);
      mScatter2[37] = new NISTMottRS(Element::Rb, 2);
      mScatter2[38] = new NISTMottRS(Element::Sr, 2);
      mScatter2[39] = new NISTMottRS(Element::Y, 2);
      mScatter2[40] = new NISTMottRS(Element::Zr, 2);
      mScatter2[41] = new NISTMottRS(Element::Nb, 2);
      mScatter2[42] = new NISTMottRS(Element::Mo, 2);
      mScatter2[43] = new NISTMottRS(Element::Tc, 2);
      mScatter2[44] = new NISTMottRS(Element::Ru, 2);
      mScatter2[45] = new NISTMottRS(Element::Rh, 2);
      mScatter2[46] = new NISTMottRS(Element::Pd, 2);
      mScatter2[47] = new NISTMottRS(Element::Ag, 2);
      mScatter2[48] = new NISTMottRS(Element::Cd, 2);
      mScatter2[49] = new NISTMottRS(Element::In, 2);
      mScatter2[50] = new NISTMottRS(Element::Sn, 2);
      mScatter2[51] = new NISTMottRS(Element::Sb, 2);
      mScatter2[52] = new NISTMottRS(Element::Te, 2);
      mScatter2[53] = new NISTMottRS(Element::I, 2);
      mScatter2[54] = new NISTMottRS(Element::Xe, 2);
      mScatter2[55] = new NISTMottRS(Element::Cs, 2);
      mScatter2[56] = new NISTMottRS(Element::Ba, 2);
      mScatter2[57] = new NISTMottRS(Element::La, 2);
      mScatter2[58] = new NISTMottRS(Element::Ce, 2);
      mScatter2[59] = new NISTMottRS(Element::Pr, 2);
      mScatter2[60] = new NISTMottRS(Element::Nd, 2);
      mScatter2[61] = new NISTMottRS(Element::Pm, 2);
      mScatter2[62] = new NISTMottRS(Element::Sm, 2);
      mScatter2[63] = new NISTMottRS(Element::Eu, 2);
      mScatter2[64] = new NISTMottRS(Element::Gd, 2);
      mScatter2[65] = new NISTMottRS(Element::Tb, 2);
      mScatter2[66] = new NISTMottRS(Element::Dy, 2);
      mScatter2[67] = new NISTMottRS(Element::Ho, 2);
      mScatter2[68] = new NISTMottRS(Element::Er, 2);
      mScatter2[69] = new NISTMottRS(Element::Tm, 2);
      mScatter2[70] = new NISTMottRS(Element::Yb, 2);
      mScatter2[71] = new NISTMottRS(Element::Lu, 2);
      mScatter2[72] = new NISTMottRS(Element::Hf, 2);
      mScatter2[73] = new NISTMottRS(Element::Ta, 2);
      mScatter2[74] = new NISTMottRS(Element::W, 2);
      mScatter2[75] = new NISTMottRS(Element::Re, 2);
      mScatter2[76] = new NISTMottRS(Element::Os, 2);
      mScatter2[77] = new NISTMottRS(Element::Ir, 2);
      mScatter2[78] = new NISTMottRS(Element::Pt, 2);
      mScatter2[79] = new NISTMottRS(Element::Au, 2);
      mScatter2[80] = new NISTMottRS(Element::Hg, 2);
      mScatter2[81] = new NISTMottRS(Element::Tl, 2);
      mScatter2[82] = new NISTMottRS(Element::Pb, 2);
      mScatter2[83] = new NISTMottRS(Element::Bi, 2);
      mScatter2[84] = new NISTMottRS(Element::Po, 2);
      mScatter2[85] = new NISTMottRS(Element::At, 2);
      mScatter2[86] = new NISTMottRS(Element::Rn, 2);
      mScatter2[87] = new NISTMottRS(Element::Fr, 2);
      mScatter2[88] = new NISTMottRS(Element::Ra, 2);
      mScatter2[89] = new NISTMottRS(Element::Ac, 2);
      mScatter2[90] = new NISTMottRS(Element::Th, 2);
      mScatter2[91] = new NISTMottRS(Element::Pa, 2);
      mScatter2[92] = new NISTMottRS(Element::U, 2);
      mScatter2[93] = new NISTMottRS(Element::Np, 2);
      mScatter2[94] = new NISTMottRS(Element::Pu, 2);
      mScatter2[95] = new NISTMottRS(Element::Am, 2);
      mScatter2[96] = new NISTMottRS(Element::Cm, 2);

      mScatter3[1] = new NISTMottRS(Element::H, 3);
      mScatter3[2] = new NISTMottRS(Element::He, 3);
      mScatter3[3] = new NISTMottRS(Element::Li, 3);
      mScatter3[4] = new NISTMottRS(Element::Be, 3);
      mScatter3[5] = new NISTMottRS(Element::B, 3);
      mScatter3[6] = new NISTMottRS(Element::C, 3);
      mScatter3[7] = new NISTMottRS(Element::N, 3);
      mScatter3[8] = new NISTMottRS(Element::O, 3);
      mScatter3[9] = new NISTMottRS(Element::F, 3);
      mScatter3[10] = new NISTMottRS(Element::Ne, 3);
      mScatter3[11] = new NISTMottRS(Element::Na, 3);
      mScatter3[12] = new NISTMottRS(Element::Mg, 3);
      mScatter3[13] = new NISTMottRS(Element::Al, 3);
      mScatter3[14] = new NISTMottRS(Element::Si, 3);
      mScatter3[15] = new NISTMottRS(Element::P, 3);
      mScatter3[16] = new NISTMottRS(Element::S, 3);
      mScatter3[17] = new NISTMottRS(Element::Cl, 3);
      mScatter3[18] = new NISTMottRS(Element::Ar, 3);
      mScatter3[19] = new NISTMottRS(Element::K, 3);
      mScatter3[20] = new NISTMottRS(Element::Ca, 3);
      mScatter3[21] = new NISTMottRS(Element::Sc, 3);
      mScatter3[22] = new NISTMottRS(Element::Ti, 3);
      mScatter3[23] = new NISTMottRS(Element::V, 3);
      mScatter3[24] = new NISTMottRS(Element::Cr, 3);
      mScatter3[25] = new NISTMottRS(Element::Mn, 3);
      mScatter3[26] = new NISTMottRS(Element::Fe, 3);
      mScatter3[27] = new NISTMottRS(Element::Co, 3);
      mScatter3[28] = new NISTMottRS(Element::Ni, 3);
      mScatter3[29] = new NISTMottRS(Element::Cu, 3);
      mScatter3[30] = new NISTMottRS(Element::Zn, 3);
      mScatter3[31] = new NISTMottRS(Element::Ga, 3);
      mScatter3[32] = new NISTMottRS(Element::Ge, 3);
      mScatter3[33] = new NISTMottRS(Element::As, 3);
      mScatter3[34] = new NISTMottRS(Element::Se, 3);
      mScatter3[35] = new NISTMottRS(Element::Br, 3);
      mScatter3[36] = new NISTMottRS(Element::Kr, 3);
      mScatter3[37] = new NISTMottRS(Element::Rb, 3);
      mScatter3[38] = new NISTMottRS(Element::Sr, 3);
      mScatter3[39] = new NISTMottRS(Element::Y, 3);
      mScatter3[40] = new NISTMottRS(Element::Zr, 3);
      mScatter3[41] = new NISTMottRS(Element::Nb, 3);
      mScatter3[42] = new NISTMottRS(Element::Mo, 3);
      mScatter3[43] = new NISTMottRS(Element::Tc, 3);
      mScatter3[44] = new NISTMottRS(Element::Ru, 3);
      mScatter3[45] = new NISTMottRS(Element::Rh, 3);
      mScatter3[46] = new NISTMottRS(Element::Pd, 3);
      mScatter3[47] = new NISTMottRS(Element::Ag, 3);
      mScatter3[48] = new NISTMottRS(Element::Cd, 3);
      mScatter3[49] = new NISTMottRS(Element::In, 3);
      mScatter3[50] = new NISTMottRS(Element::Sn, 3);
      mScatter3[51] = new NISTMottRS(Element::Sb, 3);
      mScatter3[52] = new NISTMottRS(Element::Te, 3);
      mScatter3[53] = new NISTMottRS(Element::I, 3);
      mScatter3[54] = new NISTMottRS(Element::Xe, 3);
      mScatter3[55] = new NISTMottRS(Element::Cs, 3);
      mScatter3[56] = new NISTMottRS(Element::Ba, 3);
      mScatter3[57] = new NISTMottRS(Element::La, 3);
      mScatter3[58] = new NISTMottRS(Element::Ce, 3);
      mScatter3[59] = new NISTMottRS(Element::Pr, 3);
      mScatter3[60] = new NISTMottRS(Element::Nd, 3);
      mScatter3[61] = new NISTMottRS(Element::Pm, 3);
      mScatter3[62] = new NISTMottRS(Element::Sm, 3);
      mScatter3[63] = new NISTMottRS(Element::Eu, 3);
      mScatter3[64] = new NISTMottRS(Element::Gd, 3);
      mScatter3[65] = new NISTMottRS(Element::Tb, 3);
      mScatter3[66] = new NISTMottRS(Element::Dy, 3);
      mScatter3[67] = new NISTMottRS(Element::Ho, 3);
      mScatter3[68] = new NISTMottRS(Element::Er, 3);
      mScatter3[69] = new NISTMottRS(Element::Tm, 3);
      mScatter3[70] = new NISTMottRS(Element::Yb, 3);
      mScatter3[71] = new NISTMottRS(Element::Lu, 3);
      mScatter3[72] = new NISTMottRS(Element::Hf, 3);
      mScatter3[73] = new NISTMottRS(Element::Ta, 3);
      mScatter3[74] = new NISTMottRS(Element::W, 3);
      mScatter3[75] = new NISTMottRS(Element::Re, 3);
      mScatter3[76] = new NISTMottRS(Element::Os, 3);
      mScatter3[77] = new NISTMottRS(Element::Ir, 3);
      mScatter3[78] = new NISTMottRS(Element::Pt, 3);
      mScatter3[79] = new NISTMottRS(Element::Au, 3);
      mScatter3[80] = new NISTMottRS(Element::Hg, 3);
      mScatter3[81] = new NISTMottRS(Element::Tl, 3);
      mScatter3[82] = new NISTMottRS(Element::Pb, 3);
      mScatter3[83] = new NISTMottRS(Element::Bi, 3);
      mScatter3[84] = new NISTMottRS(Element::Po, 3);
      mScatter3[85] = new NISTMottRS(Element::At, 3);
      mScatter3[86] = new NISTMottRS(Element::Rn, 3);
      mScatter3[87] = new NISTMottRS(Element::Fr, 3);
      mScatter3[88] = new NISTMottRS(Element::Ra, 3);
      mScatter3[89] = new NISTMottRS(Element::Ac, 3);
      mScatter3[90] = new NISTMottRS(Element::Th, 3);
      mScatter3[91] = new NISTMottRS(Element::Pa, 3);
      mScatter3[92] = new NISTMottRS(Element::U, 3);
      mScatter3[93] = new NISTMottRS(Element::Np, 3);
      mScatter3[94] = new NISTMottRS(Element::Pu, 3);
      mScatter3[95] = new NISTMottRS(Element::Am, 3);
      mScatter3[96] = new NISTMottRS(Element::Cm, 3);
   }
}