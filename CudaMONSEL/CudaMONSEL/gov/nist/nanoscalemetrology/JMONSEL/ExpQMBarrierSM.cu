#include "hip/hip_runtime.h"
#include "gov\nist\microanalysis\EPQLibrary\PhysicalConstants.cuh"
#include "gov\nist\microanalysis\NISTMonte\RegionBase.cuh"
#include "gov\nist\microanalysis\NISTMonte\Electron.cuh"
#include "gov\nist\microanalysis\NISTMonte\Shape.cuh"
#include "gov\nist\microanalysis\NISTMonte\MonteCarloSS.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\ExpQMBarrierSM.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\SEMaterial.cuh"
#include "gov\nist\nanoscalemetrology\JMONSEL\NormalShape.cuh"

namespace ExpQMBarrierSM
{
   ExpQMBarrierSM::ExpQMBarrierSM(const MaterialT* mat) :
      u0(mat->isSEmaterial() ? ((SEmaterialT*)mat)->getEnergyCBbottom() : 0),
      classical(true)
   {
   }

   ExpQMBarrierSM::ExpQMBarrierSM(const MaterialT* mat, double lambda) : 
      u0(mat->isSEmaterial() ? ((SEmaterialT*)mat)->getEnergyCBbottom() : 0),
      classical(false),
      lambda(lambda),
      lambdaFactor((PhysicalConstants::PI * lambda * ::sqrt(PhysicalConstants::ElectronMass / 2.)) / PhysicalConstants::PlanckReduced)
   {
   }

   static double sharpBarrierT(double rootPerpE, double rootDiff)
   {
      double ksum = rootPerpE + rootDiff;
      return (4. * rootPerpE * rootDiff) / (ksum * ksum);
   }

   double ExpQMBarrierSM::generalBarrierT(double rootPerpE, double rootDiff) const
   {
      double k1 = lambdaFactor * rootPerpE;
      if (k1 > 50.)
         return 1.;
      double k2 = lambdaFactor * rootDiff;
      double kplus = k1 + k2;
      double kminus = k1 - k2;
      double sinhPlus = ::sinh(kplus);
      double sinhMinus = ::sinh(kminus);
      double ratio = sinhMinus / sinhPlus;
      return 1. - (ratio * ratio);
   }

   ElectronT* ExpQMBarrierSM::barrierScatter(ElectronT* pe, const RegionBaseT* nextRegion) const
   {
      const MaterialT& nextmaterial = nextRegion->getMaterial();
      const RegionBaseT* currentRegion = pe->getCurrentRegion();

      if (!(currentRegion != nextRegion)) {
         printf("ExpQMBarrierSM::barrierScatter: currentRegion == nextRegion");
         return NULL;
      }

      double deltaU;
      const MaterialT& currentMaterial = currentRegion->getMaterial();
      if (currentMaterial.isSEmaterial())
         deltaU = -((SEmaterialT)currentMaterial).getEnergyCBbottom();
      else
         deltaU = 0.;

      if (!(deltaU == -u0)) {
         printf("ExpQMBarrierSM::barrierScatter: deltaU != -u0 (%.10e, %.10e)", deltaU, -u0);
         return NULL;
      }
      if (nextmaterial.isSEmaterial())
         deltaU += ((SEmaterialT)nextmaterial).getEnergyCBbottom();

      /* FIND THE OUTWARD POINTING NORMAL AT THE BOUNDARY */
      PositionVecT nb; // We'll store it here

      if (currentRegion->isContainingRegion(*nextRegion)) {
         const RegionBaseT* struckRegion = nextRegion; // usually this is true
         /*
         * Sometimes we cross multiple boundaries at once. The while loop
         * checks and corrects for this.
         */
         while (struckRegion->getParent() != currentRegion)
            struckRegion = struckRegion->getParent();
         const ShapeT* intersectedshape = struckRegion->getShape();
         if (intersectedshape->isNormalShape()) {
            nb = ((NormalShapeT*)intersectedshape)->getPreviousNormal();
            for (int i = 0; i < 3; i++)
               nb[i] *= -1;
         }
      }
      else {
         const ShapeT* intersectedshape = currentRegion->getShape();
         if (intersectedshape->isNormalShape())
            nb = ((NormalShapeT*)intersectedshape)->getPreviousNormal();
      }

      // GET THE VECTOR IN THE ELECTRON'S DIRECTION OF MOTION
      double theta0 = pe->getTheta();
      double phi0 = pe->getPhi();
      double sintheta0 = ::sin(theta0);
      PositionVecT n0({ sintheta0 * ::cos(phi0), sintheta0 * ::sin(phi0), ::cos(theta0) });

      /*
      * If the intersected shape is not a NormalShape, we still haven't
      * initialized nb. We have no data in this case, so we must make do with
      * an arbitrary assignment: Let nb be the same as the electron direction.
      * This choice gives maximum transmission probability and no deflection of
      * the electron's path.
      */
      if (!nb.empty())
         nb = n0;

      /*
      * Let the angle of incidence be called alpha. Cos(alpha) is given by the
      * dot product
      */
      double cosalpha = (n0[0] * nb[0]) + (n0[1] * nb[1]) + (n0[2] * nb[2]);

      if (cosalpha <= 0.) {
         /*
         * This case corresponds to the electron "hitting" the barrier while
         * moving away from it. I.e., it didn't really hit the barrier. This
         * can happen, e.g., if electric field alters the electron's direction
         * of motion. We give it a nudge away from the barrier towards the
         * inside
         */
         PositionVecT pos0 = pe->getPosition();
         double tmppos[] = { pos0[0] - (MonteCarloSS::SMALL_DISP * nb[0]), pos0[1] - (MonteCarloSS::SMALL_DISP * nb[1]), pos0[2] - (MonteCarloSS::SMALL_DISP * nb[2]) };
         pe->setPosition(tmppos);

         return NULL;
      }

      if (deltaU == 0.) {
         /*
         * This corresponds to no barrier. This is usually due to a
         * mathematical boundary with the same material on both sides. It
         * transmits, so we give it a nudge off of the barrier toward the
         * outside, update the electron's region, and return.
         */
         auto pos0 = pe->getPosition();
         double tmppos[] = { pos0[0] + (MonteCarloSS::SMALL_DISP * nb[0]), pos0[1] + (MonteCarloSS::SMALL_DISP * nb[1]), pos0[2] + (MonteCarloSS::SMALL_DISP * nb[2]) };
         pe->setPosition(tmppos);
         pe->setCurrentRegion(nextRegion);

         return NULL;
      }

      double kE0 = pe->getEnergy();
      double perpE;
      if (kE0 <= 0.)
         perpE = 0.;
      else
         perpE = cosalpha * cosalpha * kE0;
      double rootPerpE = 0.;
      double rootDiff = 0.;

      /* DECIDE WHETHER IT TRANSMITS OR NOT */
      bool transmits;
      if ((perpE == 0.) || (perpE <= deltaU))
         /*
         * Even if deltaU<0 (the electron is stepping downhill) the quantum
         * mechanical formula gives transmission = 0 when perpE = 0.
         */
         transmits = false;
      else {
         rootPerpE = ::sqrt(perpE);
         rootDiff = ::sqrt(perpE - deltaU);
         if (classical)
            transmits = true; // Since we already know perpE>deltaU
         else {
            double transmissionProb;
            if (lambda == 0.)
               transmissionProb = sharpBarrierT(rootPerpE, rootDiff);
            else
               transmissionProb = generalBarrierT(rootPerpE, rootDiff);

            double r = Math2::random();
            transmits = r < transmissionProb;
         }
      }

      /*
      * COMPUTE DIRECTION AND ENERGY FOR EACH OF THE CASES: TRANSMISSION
      * OR REFLECTION
      */
      PositionVecT nf(3, 0); // Direction vector after scattering
      if (transmits) { // Transmission
         double factor = cosalpha * ((rootDiff / rootPerpE) - 1.);
         for (int i = 0; i < 3; i++)
            nf[i] = n0[i] + (factor * nb[i]);
         /* Normalize the z component to use later computing theta. */
         // nf[2] /= Math.sqrt(1. + (2. * cosalpha + factor) * factor);
         nf[2] /= ::sqrt((nf[0] * nf[0]) + (nf[1] * nf[1]) + (nf[2] * nf[2]));

         pe->setEnergy(kE0 - deltaU);
         pe->setCurrentRegion(nextRegion);
         auto pos0 = pe->getPosition();

         double tmppos[] = { pos0[0] + (MonteCarloSS::SMALL_DISP * nb[0]), pos0[1] + (MonteCarloSS::SMALL_DISP * nb[1]), pos0[2] + (MonteCarloSS::SMALL_DISP * nb[2]) };
         pe->setPosition(tmppos);
      }
      else { // Total internal reflection
         double twocosalpha = 2. * cosalpha;
         for (int i = 0; i < 3; i++)
            nf[i] = n0[i] - (nb[i] * twocosalpha);

         auto pos0 = pe->getPosition();
         double tmppos[] = { pos0[0] - (MonteCarloSS::SMALL_DISP * nb[0]), pos0[1] - (MonteCarloSS::SMALL_DISP * nb[1]), pos0[2] - (MonteCarloSS::SMALL_DISP * nb[2]) };
         pe->setPosition(tmppos);
      }

      double thetaf = ::acos(nf[2]);
      double phif = ::atan2(nf[1], nf[0]);

      pe->setDirection(thetaf, phif);
      return NULL;
   }

   StringT ExpQMBarrierSM::toString() const
   {
      return "ExpQMBarrierSM(" + mat->toString() + "," + std::to_string(u0) + "," + std::to_string(lambda) + ")";
   }
}